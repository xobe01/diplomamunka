#include "hip/hip_runtime.h"
﻿#include "kernel_p.h"

std::vector<Plane*> allPlanes;
std::vector<Point*> fitAddedPoints;
std::vector<Edge*> fitEdges;
std::vector<std::vector<Point*>> points;
std::vector<std::vector<Point*>> addedPoints;
std::vector<std::vector<Edge*>> edges;
std::vector<std::vector<Plane*>> planes;

size_t horizontalCount = 0;
size_t verticalCount = 0;
std::vector<int> currentPlaneId;
std::vector<int> currentOutlineId;
const double objectPointDistance = 5;
const double planeDistanceTreshold = 0.05;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData(size_t pointCloudIndex)
{
	points.push_back({});
	addedPoints.push_back({});
	edges.push_back({});
	planes.push_back({});
	currentPlaneId.push_back(1);
	currentOutlineId.push_back(1);
	std::string myText;
	std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw_" +
		(pointCloudTestIndex == -1 ? (pointCloudCount == 0 ? "test" : std::to_string(pointCloudIndex)) : std::to_string(pointCloudTestIndex)) + ".txt");
	getline(MyReadFile, myText);
	std::replace(myText.begin(), myText.end(), ',', '.');
	std::stringstream ss(myText);
	std::string _x, _y, _z;
	std::getline(ss, _x, ';');
	std::getline(ss, _y, ';');
	std::getline(ss, _z, ';');
	egoCarPos.push_back({ std::stof(_x),  std::stof(_y),  std::stof(_z) });
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	std::vector<std::string> stringContainer(std::stoi(myText));
	points[pointCloudIndex].resize(verticalCount * horizontalCount);
	while (getline(MyReadFile, myText))
	{
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x != myText) {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[pointCloudIndex][getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}
	}
	MyReadFile.close();
}

void groundSegmentation(size_t pointCloudIndex)
{ //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points[pointCloudIndex].size(); i++) {
		if (points[pointCloudIndex][i] && points[pointCloudIndex][i]->position.y < groundLevel) groundLevel = points[pointCloudIndex][i]->position.y;
	}

	for (size_t i = 0; i < points[pointCloudIndex].size(); i++) {
		if (points[pointCloudIndex][i] && points[pointCloudIndex][i]->position.y <= groundLevel + 0.1) {
			points[pointCloudIndex][i] = nullptr;
		}
	}
}

void setPointsMarked(std::vector<Point*> points, bool isMarked, bool isMarked2)
{
	for (size_t i = 0; i < points.size(); i++) {
		points[i]->isMarked = isMarked;
		points[i]->isMarked2 = isMarked2;
	}
}

int spikeType(Point* p, int arriveDirection, bool onlyMarkedNeighbours, size_t pointCloudIndex)
{
	//arriveDirection
	// 0 - from left
	// 1 - from up
	// 2 - from rigth
	// 3 - from down
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	size_t neighbourCount = 0;
	bool isNeighbour[4] = { false, false, false, false };
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x, y - 1)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
				points[pointCloudIndex][getOffset(x + 1, y)] };

	Point* diagNeighbourPoints[4] = { points[pointCloudIndex][getOffset(x - 1, y - 1)], points[pointCloudIndex][getOffset(x + 1, y - 1)], points[pointCloudIndex][getOffset(x + 1, y + 1)],
		points[pointCloudIndex][getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j != 0 || y > 0) && (j != 1 || y < verticalCount - 1) && neighbourPoints[j]->plane == p->plane &&
			(!onlyMarkedNeighbours || neighbourPoints[j]->isMarked)) {
			neighbourCount++;
			isNeighbour[j] = true;
		}
	}
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarkedNeighbours || diagNeighbourPoints[j]->isMarked)) {
			diagIsNeighbour[j] = true;
		}
	}
	if (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[2] && !diagIsNeighbour[0]) || (isNeighbour[0] && isNeighbour[3] && !diagIsNeighbour[1])
		|| (isNeighbour[1] && isNeighbour[2] && !diagIsNeighbour[3]) || (isNeighbour[1] && isNeighbour[3] && !diagIsNeighbour[2])))
		return -1;
	if ((neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3]))) || (neighbourCount > 1 &&
		((arriveDirection == 0 && !isNeighbour[1]) || (arriveDirection == 1 && !isNeighbour[2]) || (arriveDirection == 2 && !isNeighbour[0])
			|| (arriveDirection == 3 && !isNeighbour[3]))))
		return 0;
	if (neighbourCount == 1)
		return 1;
	if (neighbourCount > 1)
		return 2;
	return 3;
}

bool has2NonSpikeNeighbours(Point* p, bool onlyMarked, size_t pointCloudIndex)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x, y - 1)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
				points[pointCloudIndex][getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && neighbourPoints[j]->plane == p->plane && (j != 0 || y > 0) && (j != 1 || y < verticalCount - 1) &&
			spikeType(neighbourPoints[j], -1, onlyMarked, pointCloudIndex) > 1 && (!onlyMarked || neighbourPoints[j]->isMarked))
			neighbourCount++;
	}
	return neighbourCount > 1;
}

bool checkIfBridge(Point* p, bool onlyMarked, size_t pointCloudIndex)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x, y - 1)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
				points[pointCloudIndex][getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && neighbourPoints[j]->plane == p->plane && (j != 0 || y > 0) && (j != 1 || y < verticalCount - 1) &&
			spikeType(neighbourPoints[j], -1, onlyMarked, pointCloudIndex) > 1 && (!onlyMarked || neighbourPoints[j]->isMarked) && has2NonSpikeNeighbours(neighbourPoints[j], onlyMarked,
				pointCloudIndex))
			neighbourCount++;
	}
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* diagNeighbourPoints[4] = { points[pointCloudIndex][getOffset(x - 1, y - 1)], points[pointCloudIndex][getOffset(x + 1, y - 1)], points[pointCloudIndex][getOffset(x + 1, y + 1)],
		points[pointCloudIndex][getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarked || diagNeighbourPoints[j]->isMarked)) {
			diagIsNeighbour[j] = true;
		}
	}
	if (((!diagIsNeighbour[0] && !diagIsNeighbour[2]) || (!diagIsNeighbour[1] && !diagIsNeighbour[3])) && neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints, size_t pointCloudIndex)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p, false, pointCloudIndex)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(p1->position - center->position, p2->position - center->position);
}

void calculateNormal(Point* point, size_t pointCloudIndex)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[pointCloudIndex][getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[pointCloudIndex][getOffset(x + 1, y)];
	Point* neighbourPoint3 = points[pointCloudIndex][getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[pointCloudIndex][getOffset(x - 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

int areNeighbours(Point* p1, Point* p2, size_t pointCloudIndex)
{
	if (points[pointCloudIndex][getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[pointCloudIndex][getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 2;
	if (points[pointCloudIndex][getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 3;
	if (points[pointCloudIndex][getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

void choosePoints(const Vec3<Point*> planePoints, Plane* basePlane, /*out*/ Plane* plane, size_t pointCloudIndex)
{
	if (basePlane) {
		plane->pointDirections[0] = basePlane->pointDirections[0];
		plane->pointDirections[1] = basePlane->pointDirections[1];
		plane->normal = basePlane->normal;
	}
	else {
		auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.y->position -
			planePoints.x->position, planePoints.z->position - planePoints.x->position));
		Vec3<double> horizontalDirection = { 0,0,0 };
		Vec3<double> verticalDirection = { 0,0,0 };
		Point* neighbours[2] = { planePoints.y, planePoints.z };
		for each (auto neighbour in neighbours) {
			switch (areNeighbours(planePoints.x, neighbour, pointCloudIndex)) {
			case 1:
			horizontalDirection = neighbour->position - planePoints.x->position;
			break;
			case 2:
			verticalDirection = neighbour->position - planePoints.x->position;
			break;
			case 3:
			horizontalDirection = planePoints.x->position - neighbour->position;
			break;
			case 4:
			verticalDirection = planePoints.x->position - neighbour->position;
			break;
			default:
			break;
			}
		}
		plane->pointDirections[0] = horizontalDirection;
		plane->pointDirections[1] = verticalDirection;
		plane->normal = normal;
	}
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId[pointCloudIndex];
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);	
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x, y - 1)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
				points[pointCloudIndex][getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j != 0 || y > 0) && (j != 1 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(plane->normal, neighbourPoints[j]->position - plane->planePointPos));
					if (dist <= planeDistanceTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
						if ((neighbourPoints[j]->normal - plane->normal).length() < 0.05) {
							plane->normal = Vec3<double>::normalize(plane->normal * (plane->points.size() - 1) + neighbourPoints[j]->normal);
						}
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}	
}

void findPlanes(size_t pointCloudIndex)
{
	size_t minPointCount = 10;
	size_t counter = 1;
	double normalTreshold = 0.01;
	
	for (size_t i = 0; i < points[pointCloudIndex].size(); i++)
		if (points[pointCloudIndex][i]) {
			points[pointCloudIndex][i]->isMarked = true;
			points[pointCloudIndex][i]->isMarked2 = true;
			calculateNormal(points[pointCloudIndex][i], pointCloudIndex);
		}
	
	
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points[pointCloudIndex].size(); j++) {
		if (points[pointCloudIndex][j] && points[pointCloudIndex][j]->isMarked) 
		{
			auto start = std::chrono::steady_clock::now();
			nextStepPoints.push_back(points[pointCloudIndex][j]);
			while (nextStepPoints.size() > 0) {
				
				Plane* plane = new Plane();
				bool planeMerged;
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x, y - 1)], points[pointCloudIndex][getOffset(x + 1, y)], points[pointCloudIndex][getOffset(x, y + 1)],
							points[pointCloudIndex][getOffset(x - 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k != 0 || y > 0) && (k != 2 || y < verticalCount - 1) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if (normals[k].length() > 0 && normals[(k + 1) % 4].length() > 0 &&
								(normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) 
							{
								
								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, nullptr,
									plane, pointCloudIndex);
								
								break;
							}
						}
						if (plane->points.size() > 0)
							break;
					}
				}
				if (plane->points.size() > 0) {
					planes[pointCloudIndex].push_back(plane);
					currentPlaneId[pointCloudIndex]++;
				}
				else
					delete plane;
				nextStepPoints = tempNextStepPoints;
				
			}		
			auto end = std::chrono::steady_clock::now();
			dbgTimeCounter += (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000;
		}		
	}
	
	for (size_t i = 0; i < points[pointCloudIndex].size(); i++) if (points[pointCloudIndex][i]) points[pointCloudIndex][i]->isMarked = false;
	for (size_t i = 0; i < points[pointCloudIndex].size(); i++) if (points[pointCloudIndex][i]) points[pointCloudIndex][i]->isMarked2 = false;
	
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		
		int originalSize = planes[pointCloudIndex][i]->points.size();
		
		while (isThereBridge(planes[pointCloudIndex][i]->points, pointCloudIndex)) {}
		
		if (originalSize != planes[pointCloudIndex][i]->points.size() && planes[pointCloudIndex][i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) planes[pointCloudIndex][i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[pointCloudIndex][i]->points[0], nullptr, nullptr }, planes[pointCloudIndex][i], plane, pointCloudIndex);
				if (plane->points.size() < planes[pointCloudIndex][i]->points.size()) {
					for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) {
						if (planes[pointCloudIndex][i]->points[j]->plane != planes[pointCloudIndex][i]) {
							planes[pointCloudIndex][i]->points.erase(planes[pointCloudIndex][i]->points.begin() + j);
							j--;
						}
					}
					planes[pointCloudIndex].push_back(plane);
					currentPlaneId[pointCloudIndex]++;
				}
				else {
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[pointCloudIndex][i];
					}
					delete plane;
					break;
				}
			}
		}		
	}	
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++)
		planes[pointCloudIndex][i]->calculateAvaragePointPos();
}

bool hasNonSpykeNeighbour(size_t x, size_t y, size_t pointCloudIndex)
{
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 4; i++) {
		switch (i) {
		case 0: //to right
		neighbourPoint = points[pointCloudIndex][getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[pointCloudIndex][getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[pointCloudIndex][getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[pointCloudIndex][getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && spikeType(neighbourPoint, -1, false, pointCloudIndex) == 2)
			return true;
	}
	return false;
}

void findNextPoint(Point*& startPoint, size_t direction, size_t pointCloudIndex, /*out*/
	std::vector<std::pair<Point*, int>>& currentEdge, std::vector<Point*>& spikePoints, size_t dbgPlaneIndex, std::vector<Plane*> dbgPlanes)
{
	//isMarked -- turned off after the edge is complete
	//isMarked2 -- turned off when point added to edge

	Point* currentPoint = nullptr;
	std::pair<Point*, size_t> previousSavedPoint = { nullptr, 0 };
	bool isFirstPoint = true;
	bool comeFromDeadEnd = false;
	bool isPreviousSpike = false;
	bool wasThereNonSpike = false;
	bool isHole = direction == 1;
	while (currentPoint != startPoint || comeFromDeadEnd) {

		if (!currentPoint)
			currentPoint = startPoint;
		if (currentPoint->horizontalIndex == 109 && currentPoint->verticalIndex == 63) {
			std::cout << "asd";
		}
		Point* neighbourPoint = nullptr;
		size_t x = currentPoint->horizontalIndex;
		size_t y = currentPoint->verticalIndex;
		isPreviousSpike = currentPoint != previousSavedPoint.first && spikeType(currentPoint, (!wasThereNonSpike || currentPoint == startPoint) ? -1 : ((direction + 1) % 4), !isHole,
			pointCloudIndex) == 0;
		if (!isPreviousSpike || currentEdge.size() == 0) {
			if (currentPoint->isMarked2) {
				currentEdge.push_back({ currentPoint, direction });
			}
		}
		else spikePoints.push_back(currentPoint);
		if (isHole && isPreviousSpike) {
			currentPoint = previousSavedPoint.first;
			direction = (previousSavedPoint.second + 1) % 4;
			continue;
		}
		if (true || !isHole) currentPoint->isMarked2 = false;
		for (size_t i = 0; i < 4; i++) {
			/*if (checkIfBridge(currentPoint)) {
				i += 2;
				direction = (direction + 2) % 4;
			}*/
			switch (direction) {
			case 0: //to right
			neighbourPoint = points[pointCloudIndex][getOffset(x + 1, y)];
			break;
			case 1: //to down
			neighbourPoint = y == verticalCount - 1 ? nullptr : points[pointCloudIndex][getOffset(x, y + 1)];
			break;
			case 2: //to left
			neighbourPoint = points[pointCloudIndex][getOffset(x - 1, y)];
			break;
			case 3: //to up
			neighbourPoint = y == 0 ? nullptr : points[pointCloudIndex][getOffset(x, y - 1)];
			break;
			default:
			break;
			}
			if (neighbourPoint == startPoint) {
				currentPoint = startPoint;
				currentEdge[0].second = (direction + 3) % 4;
				comeFromDeadEnd = false;
				break;
			}
			if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr && neighbourPoint->isMarked2
				&& (isPreviousSpike || spikeType(neighbourPoint, -1, !isHole, pointCloudIndex) <= 1 || spikeType(currentPoint, direction, !isHole, pointCloudIndex) > 0)) {
				if (isFirstPoint) {
					currentEdge[0].second = (direction + 1) % 4;
					isFirstPoint = false;
				}
				if (!wasThereNonSpike && spikeType(currentPoint, -1, !isHole, pointCloudIndex) > 1) {
					if (currentEdge.size() > 1) {
						auto helper = currentEdge[0];
						currentEdge[0] = currentEdge[1];
						currentEdge[1] = helper;
						startPoint = currentPoint;
						if (!hasNonSpykeNeighbour(startPoint->horizontalIndex, startPoint->verticalIndex, pointCloudIndex)) {
							for (size_t j = 0; j < currentEdge.size(); j++) {
								spikePoints.push_back(currentEdge[j].first);
								currentEdge[j].first->plane = nullptr;
							}
							currentEdge.clear();
							return;
						}
					}
					isPreviousSpike = false;
					wasThereNonSpike = true;
				}
				if (!isPreviousSpike)
					previousSavedPoint = { currentPoint, (direction + (4 - i)) % 4 };
				auto neighbourSpikeType = spikeType(neighbourPoint, -1, !isHole, pointCloudIndex);
				if ((!isHole && checkIfBridge(neighbourPoint, true, pointCloudIndex)) || neighbourSpikeType == -1 || neighbourSpikeType == 1 || (wasThereNonSpike &&
					currentEdge.size() > 1 && isPreviousSpike && neighbourSpikeType == 2 && neighbourPoint->isMarked)) {
					auto savedPoint = neighbourSpikeType == 1 ? neighbourPoint : currentPoint;
					if (savedPoint->isMarked2 || isPreviousSpike)
						currentEdge.push_back({ savedPoint, (direction + 3) % 4 });
					if (neighbourSpikeType == -1 || (!isHole && checkIfBridge(neighbourPoint, true, pointCloudIndex))) {
						neighbourPoint->isMarked = false;
						neighbourPoint->isMarked2 = false;
						neighbourPoint->plane = nullptr;
					}
					savedPoint->isMarked2 = false;
					currentPoint = previousSavedPoint.first;
					direction = previousSavedPoint.second;
					isPreviousSpike = false;
					comeFromDeadEnd = true;
					break;
				}
				else {
					comeFromDeadEnd = false;
					currentPoint = neighbourPoint;
				}
				direction = (direction + 3) % 4;
				break;
			}
			direction += direction == 3 ? -3 : 1;
			if (i == 3) {
				if (!wasThereNonSpike) {
					for (size_t j = 0; j < currentEdge.size(); j++) {
						currentEdge[j].first->plane = nullptr;
						spikePoints.push_back(currentEdge[j].first);
					}
					currentEdge.clear();
					return;
				}
				if (currentPoint == startPoint)
					return;
				currentPoint = previousSavedPoint.first;
				direction = previousSavedPoint.second;
				std::cout << "INVALID EDGE SEARCH" << std::endl;;
			}
		}
	}
}

bool isEdgePoint(Point* point, size_t pointCloudIndex)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[pointCloudIndex][getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[pointCloudIndex][getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[pointCloudIndex][getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[pointCloudIndex][getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

Vec3<double> getRay(size_t hoizontalIndex, size_t verticalIndex)
{
	hoizontalIndex = (hoizontalIndex + horizontalCount) % horizontalCount;
	hoizontalIndex = (hoizontalIndex + horizontalCount) % horizontalCount;
	return  Vec3<double>::normalize({ sin(2 * PI * ((double)hoizontalIndex / horizontalCount)),
				rayAngles.second / 45 + (rayAngles.first - rayAngles.second) / 45 / verticalCount * (double)verticalIndex,
				cos(2 * PI * ((double)hoizontalIndex / horizontalCount)) });
}

void saveEdgeNeighbours(Edge* edge, size_t pointCloudIndex)
{
	const double backgroundPlaneDistTreshold = 0.5;
	auto normal = Vec3<double>::crossProduct(edge->pointsWithDir[0].first->plane->pointDirections[0],
		edge->pointsWithDir[0].first->plane->pointDirections[1]);//edge->pointsWithDir[0].first->plane->normal;
	auto planePointPos = edge->pointsWithDir[0].first->plane->planePointPos;
	for (size_t i = 0; i < edge->pointsWithDir.size(); i++) {
		auto point = edge->pointsWithDir[i].first;
		size_t x = point->horizontalIndex;
		size_t y = point->verticalIndex;
		Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x + 1, y)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
			points[pointCloudIndex][getOffset(x, y - 1)] };
		Vec3<double> neighbourRays[4] = { getRay(x + 1, y), getRay(x, y + 1), getRay(x - 1, y), getRay(x, y - 1) };
		for (size_t j = 0; j < 4; j++) {
			if ((y > 0 || j != 3) && (y < verticalCount - 1 || j != 1) && (!neighbourPoints[j] || neighbourPoints[j]->plane !=
				edge->pointsWithDir[i].first->plane)) {
				double d = Vec3<double>::dot_product(normal, planePointPos);
				if (Vec3<double>::dot_product(normal, neighbourRays[j]) == 0) { // No intersection, the line is parallel to the plane
					continue;
				}
				float x = (Vec3<double>::dot_product(normal, planePointPos) - Vec3<double>::dot_product(normal, egoCarPos[pointCloudIndex])) /
					Vec3<double>::dot_product(normal, neighbourRays[j]);
				auto intersection = egoCarPos[pointCloudIndex] + neighbourRays[j] * x;
				if (!neighbourPoints[j] || ((intersection - egoCarPos[pointCloudIndex]).length() + backgroundPlaneDistTreshold) <
					(neighbourPoints[j]->position - egoCarPos[pointCloudIndex]).length()) {
					edge->closestNeighbourPoints.push_back({ intersection, {0,0,0} });
				}
			}
		}
	}
}

void findEdgePoints(size_t pointCloudIndex)
{
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		std::vector<Point*> outerConnectedEdgePoints;
		std::vector<Point*> holeConnectedEdgePoints;
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) planes[pointCloudIndex][i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) planes[pointCloudIndex][i]->points[j]->isMarked2 = true;
		for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) if (isEdgePoint(planes[pointCloudIndex][i]->points[j], pointCloudIndex))
			edgePointsInPlane.push_back(planes[pointCloudIndex][i]->points[j]);
		while (edgePointsInPlane.size() > 0) {
			Edge* currentEdge = new Edge();
			edges[pointCloudIndex].push_back(currentEdge);
			std::vector<Point*> tempEdgePointsInPlane;
			Point* startPoint = edgePointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
				if ((edgePointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - edgePointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| edgePointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = edgePointsInPlane[j]->horizontalIndex;
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
				else if (edgePointsInPlane[j]->horizontalIndex == minHorizontalCoord && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
			}
			size_t direction = 2;
			if (spikeType(startPoint, -1, true, pointCloudIndex) == -1) {
				size_t x = startPoint->horizontalIndex;
				size_t y = startPoint->verticalIndex;
				startPoint->plane = nullptr;
				startPoint->isMarked = false;
				startPoint->isMarked2 = false;
				for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) {
					if (planes[pointCloudIndex][i]->points[j] == startPoint) {
						planes[pointCloudIndex][i]->points[j]->plane = nullptr;
						planes[pointCloudIndex][i]->points.erase(planes[pointCloudIndex][i]->points.begin() + j);
						break;
					}
				}
				startPoint = points[pointCloudIndex][getOffset(x + 1, y)];
				minVerticalCoord = y;
				for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
					if (edgePointsInPlane[j]->horizontalIndex == x + 1 && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
						minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
						startPoint = edgePointsInPlane[j];
					}
				}
			}
			else if (startPoint->verticalIndex > 0 && points[pointCloudIndex][getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)] &&
				points[pointCloudIndex][getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)]->plane == startPoint->plane &&
				points[pointCloudIndex][getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex - 1)] &&
				points[pointCloudIndex][getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex - 1)]->plane == startPoint->plane &&
				points[pointCloudIndex][getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex)] &&
				points[pointCloudIndex][getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex)]->plane == startPoint->plane) {
				direction = 1;
				currentEdge->isHole = true;
			}

			if (i == 19 && planes[pointCloudIndex][i]->edges.size() == 0) {
				std::cout << "asd";
				//return;
			}
			std::vector<Point*> spikePoints;
			setPointsMarked(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints, false, false);
			setPointsMarked(currentEdge->isHole ? outerConnectedEdgePoints : holeConnectedEdgePoints, true, false);
			findNextPoint(startPoint, direction, pointCloudIndex, currentEdge->pointsWithDir, spikePoints, i, planes[pointCloudIndex]);
			setPointsMarked(outerConnectedEdgePoints, false, false);
			setPointsMarked(holeConnectedEdgePoints, false, false);
			for (size_t j = 0; j < currentEdge->pointsWithDir.size(); j++) {
				currentEdge->pointsWithDir[j].first->isMarked = false;
				(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints).push_back(currentEdge->pointsWithDir[j].first);
			}
			for (size_t j = 0; j < spikePoints.size(); j++) {
				spikePoints[j]->isMarked = false;
				(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints).push_back(spikePoints[j]);
			}
			for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
				if (edgePointsInPlane[j]->isMarked) tempEdgePointsInPlane.push_back(edgePointsInPlane[j]);
			}
			edgePointsInPlane = tempEdgePointsInPlane;
			if (currentEdge->pointsWithDir.size() > 3) {
				if (direction == 1) {
					currentEdge->pointsWithDir.insert(currentEdge->pointsWithDir.begin(),
						currentEdge->pointsWithDir[currentEdge->pointsWithDir.size() - 1]);
					currentEdge->pointsWithDir.pop_back();
				}
				for (size_t k = 0; k < currentEdge->pointsWithDir.size(); k++) {
					currentEdge->pointsWithDir[k].first->outlineId = currentOutlineId[pointCloudIndex];
				}
				currentOutlineId[pointCloudIndex]++;
				currentEdge->startPoint = startPoint;
				planes[pointCloudIndex][i]->edges.push_back(currentEdge);
				saveEdgeNeighbours(currentEdge, pointCloudIndex);
			}
		}
		for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) planes[pointCloudIndex][i]->points[j]->isMarked = false;
		for (size_t j = 0; j < planes[pointCloudIndex][i]->points.size(); j++) planes[pointCloudIndex][i]->points[j]->isMarked2 = false;
	}
}

const double newPointAcceptTreshold = 0.95;
const double inf = 1000000;

bool isStraightPoint(size_t pointIndex, Edge* edge, size_t& previousNeighbourCount,
	Vec3<double>& straigthDir, size_t pointCloudIndex)
{
	const double newDirTreshold = 0.1;
	Point* point = edge->pointsWithDir[pointIndex].first;
	if (point->horizontalIndex == 666 && point->verticalIndex == 29) {
		//std::cout << "asd";
	}
	size_t previousNeighbourCountStore = previousNeighbourCount;
	size_t neighbourCount = 0;
	size_t neighbourEdgeCount = 0;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Plane* plane = point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x, y - 1)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
		points[pointCloudIndex][getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i != 0) && (y < verticalCount - 1 || i != 1) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane) {
			neighbourCount++;
			isNeighbour[i] = true;
			if (neighbourPoints[i]->outlineId > 0)
				neighbourEdgeCount++;
		}
	}
	previousNeighbourCount = neighbourCount;
	if (pointIndex > 0 && areNeighbours(pointIndex < (edge->pointsWithDir.size() - 1) ? edge->pointsWithDir[pointIndex + 1].first : edge->startPoint,
		point, pointCloudIndex) == 0 && neighbourCount < 3) { //deadend
		auto previousPoint = edge->pointsWithDir[pointIndex - 1];
		auto previousPreviousPoint = pointIndex > 1 ? edge->pointsWithDir[pointIndex - 2] : edge->pointsWithDir[edge->pointsWithDir.size() - 2
			+ pointIndex];
		straigthDir = { 0,0,0 };
		if (areNeighbours(previousPoint.first, previousPreviousPoint.first, pointCloudIndex) > 0 && ((previousPoint.first->verticalIndex ==
			previousPreviousPoint.first->verticalIndex && previousPoint.first->horizontalIndex == point->horizontalIndex &&
			previousPoint.first->verticalIndex != point->verticalIndex) || (previousPoint.first->horizontalIndex ==
				previousPreviousPoint.first->horizontalIndex && previousPoint.first->verticalIndex == point->verticalIndex &&
				previousPoint.first->horizontalIndex != point->horizontalIndex))) //if curve swap with previous
		{
			point->isCorner = true;
			edge->pointsWithDir[pointIndex - 1] = edge->pointsWithDir[pointIndex];
			edge->pointsWithDir[pointIndex] = previousPoint;
			if (pointIndex > 1) edge->pointsWithDir[pointIndex - 2].first->isCorner = true;
			return true;
		}
		return false;
	}
	if (neighbourCount == 3 && (neighbourEdgeCount == 2 && (pointIndex == edge->pointsWithDir.size() - 1 ||
		areNeighbours(point, edge->pointsWithDir[pointIndex + 1].first, pointCloudIndex)
				> 0))) {
		if (straigthDir.length() == 0) {
			if (pointIndex > 0)
				straigthDir = Vec3<double>::normalize(point->position - edge->pointsWithDir[pointIndex - 1].first->position);
		}
		else {
			auto newDir = Vec3<double>::normalize(point->position - edge->pointsWithDir[pointIndex - 1].first->position);
			if ((straigthDir - newDir).length() > newDirTreshold) {
				straigthDir = newDir;
				return false;
			}
		}
		return true;
	}
	straigthDir = { 0,0,0 };
	if (neighbourCount == 4 && (pointIndex == 0 ? edge->pointsWithDir[edge->pointsWithDir.size() - 1].first :
		edge->pointsWithDir[pointIndex - 1].first)->isCorner &&
		previousNeighbourCountStore == 2)
		return true;
	if (previousNeighbourCountStore == 4) {
		edge->pointsWithDir[pointIndex - 1].first->isCorner = false;
	}
	return false;
}

void findCorners(size_t pointCloudIndex)
{
	for (size_t k = 0; k < planes[pointCloudIndex].size(); k++) {
		for (size_t i = 0; i < planes[pointCloudIndex][k]->edges.size(); i++) {
			size_t previousNeighbourCount = 0;
			Vec3<double> straightDir = { 0,0,0 };
			for (size_t j = 0; j < planes[pointCloudIndex][k]->edges[i]->pointsWithDir.size(); j++) {
				if (!isStraightPoint(j, planes[pointCloudIndex][k]->edges[i], previousNeighbourCount, straightDir, pointCloudIndex)) {
					planes[pointCloudIndex][k]->edges[i]->pointsWithDir[j].first->isCorner = true;
				}
			}
		}
	}
}

Point* createNewPoint(Vec3<double> newPointPos, Point* point, std::vector<Point*> neighbours, size_t addedCount, size_t pointCloudIndex, bool createBeforePoint = false,
	bool isCornerPoint = false)
{
	double deleteDurroundingCornersTreshold = 0.1;
	Point* newPoint = new Point(newPointPos, neighbours[0]->horizontalIndex, isCornerPoint ? verticalCount + 1 : verticalCount, point->plane);
	addedPoints[pointCloudIndex].push_back(newPoint);
	newPoint->isCorner = true;
	newPoint->outlineId = point->outlineId;
	for (size_t j = 0; j < point->plane->edges.size(); j++) {
		if (point->plane->edges[j]->pointsWithDir[0].first->outlineId == point->outlineId) {
			for (size_t k = 0; k < point->plane->edges[j]->pointsWithDir.size(); k++) {
				if (point->plane->edges[j]->pointsWithDir[k].first == point) {
					auto index = k + (createBeforePoint ? 0 : 1) + addedCount;
					point->plane->edges[j]->pointsWithDir.insert(point->plane->edges[j]->pointsWithDir.begin() + index, { newPoint, -1 });
					break;
				}
			}
			break;
		}
	}
	for (size_t i = 0; i < neighbours.size(); i++) {
		newPoint->neighbourPlaneNeighbours[i] = neighbours[i];
	}
	return newPoint;
}

const double twoPointDifferenceTreshold = 0.5;

Point* addNewPoint(Point* point, Point*& neighbour, Plane* plane, size_t addedCount, size_t neighbourIndex, size_t pointCloudIndex)
{
	if (point->createdNeighbourPoints[neighbourIndex] != nullptr) { //created by other plane
		auto createdNeighbour = point->createdNeighbourPoints[neighbourIndex];
		createdNeighbour->isCorner = true;
		for (size_t j = 0; j < point->plane->edges.size(); j++) {
			if (point->plane->edges[j]->pointsWithDir[0].first->outlineId == point->outlineId) {
				for (size_t k = 0; k < point->plane->edges[j]->pointsWithDir.size(); k++) {
					if (point->plane->edges[j]->pointsWithDir[k].first == point) {
						for (size_t l = 0; l < 4; l++) {
							if (point->plane->edges[j]->pointsWithDir[k + l + 1].first == createdNeighbour) {
								point->plane->edges[j]->pointsWithDir.insert(point->plane->edges[j]->pointsWithDir.begin() + k + 1 + addedCount,
									{ createdNeighbour, -1 });
								point->plane->edges[j]->pointsWithDir.erase(point->plane->edges[j]->pointsWithDir.begin() + k + l + 2);
								break;
							}
						}
						break;
					}
				}
				break;
			}
		}
		return createdNeighbour;
	}
	Vec3<double> dir = { 0,0,0 };
	switch (areNeighbours(point, neighbour, pointCloudIndex)) {
	case 1:
	dir = point->plane->pointDirections[0];
	break;
	case 2:
	dir = point->plane->pointDirections[1];
	break;
	case 3:
	dir = point->plane->pointDirections[0] * -1;
	break;
	case 4:
	dir = point->plane->pointDirections[1] * -1;
	break;
	default:
	break;
	}
	Vec3<double> newPointPos = point->position - dir * Vec3<double>::dot_product(point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf || isnan(newPointPos.x) || isnan(newPointPos.y) || isnan(newPointPos.z))
		return nullptr;
	Vec3<double> dirToNew = newPointPos - point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold &&
		(point->position - newPointPos).length() > planeDistanceTreshold)
		return nullptr;
	Vec3<double> neighbourDir = { 0,0,0 };
	switch (areNeighbours(neighbour, point, pointCloudIndex)) {
	case 1:
	neighbourDir = plane->pointDirections[0];
	break;
	case 2:
	neighbourDir = plane->pointDirections[1];
	break;
	case 3:
	neighbourDir = plane->pointDirections[0] * -1;
	break;
	case 4:
	neighbourDir = plane->pointDirections[1] * -1;
	break;
	default:
	break;
	}
	Vec3<double> neighbourNewPointPos = neighbour->position - neighbourDir * Vec3<double>::dot_product(neighbour->position
		- point->plane->planePointPos, point->plane->normal) / Vec3<double>::dot_product(neighbourDir, point->plane->normal);
	if (abs(neighbourNewPointPos.x) > inf || abs(neighbourNewPointPos.y) > inf || abs(neighbourNewPointPos.z) > inf || isnan(neighbourNewPointPos.x) ||
		isnan(neighbourNewPointPos.y) || isnan(neighbourNewPointPos.z))
		return nullptr;
	Vec3<double> dirToNewNeighbour = neighbourNewPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(neighbourDir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold &&
		(neighbour->position - neighbourNewPointPos).length() > planeDistanceTreshold)
		return nullptr;
	if ((newPointPos - neighbourNewPointPos).length() > twoPointDifferenceTreshold)
		return nullptr;
	auto newPos = (newPointPos + neighbourNewPointPos) / 2;
	Point* newPoint;
	Point* newNeighbourPoint;
	newPoint = createNewPoint(newPos, point, { neighbour }, addedCount, pointCloudIndex);
	point->createdNeighbourPoints[neighbourIndex] = newPoint;
	for (size_t i = 0; i < neighbour->neighbourPlaneNeighbours.size(); i++) {
		if (neighbour->neighbourPlaneNeighbours[i] == point) {
			newNeighbourPoint = createNewPoint(newPos, neighbour, { point }, 0, false);
			neighbour->createdNeighbourPoints[i] = newNeighbourPoint;
			break;
		}
	}
	newPoint->neighbourPlaneNeighbours[1] = newNeighbourPoint;
	newNeighbourPoint->neighbourPlaneNeighbours[1] = newPoint;
	return newPoint;
}

void findPlaneConnections(size_t pointCloudIndex)
{
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		for (size_t j = 0; j < planes[pointCloudIndex][i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[pointCloudIndex][i]->edges[j]->pointsWithDir.size(); k++) {
				Point* point = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first;
				if (point->horizontalIndex == 339 && point->verticalIndex == 14) {
					//std::cout << "asd";
				}
				int direction = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].second;
				size_t x = point->horizontalIndex;
				size_t y = point->verticalIndex;
				Point* neighbourPoints[4] = { points[pointCloudIndex][getOffset(x + 1, y)], points[pointCloudIndex][getOffset(x, y + 1)], points[pointCloudIndex][getOffset(x - 1, y)],
					points[pointCloudIndex][getOffset(x, y - 1)] };
				for (size_t i = 0; i < 4; i++) {
					if ((y > 0 || direction != 3) && (y < verticalCount - 1 || direction != 1) && neighbourPoints[direction]) {
						point->neighbourPlaneNeighbours[i] = neighbourPoints[direction];
					}
					direction += direction == 3 ? -3 : 1;
				}
			}
		}
	}
}

void createPlaneCorner(Point* point, std::vector<Point*> point1, std::vector<Point*> point2)
{
	Plane* p1 = point->plane;
	Plane* p2 = point1[2]->plane;
	Plane* p3 = point2[2]->plane;
	auto cross1 = Vec3<double>::crossProduct(p2->normal, p3->normal);
	auto cross2 = Vec3<double>::crossProduct(p3->normal, p1->normal);
	auto cross3 = Vec3<double>::crossProduct(p1->normal, p2->normal);

	auto denom = Vec3<double>::dot_product(p1->normal, cross1);

	auto planeDist1 = Vec3<double>::dot_product(p1->planePointPos, p1->normal);
	auto planeDist2 = Vec3<double>::dot_product(p2->planePointPos, p2->normal);
	auto planeDist3 = Vec3<double>::dot_product(p3->planePointPos, p3->normal);

	cross1 = cross1 * planeDist1;
	cross2 = cross2 * planeDist2;
	cross3 = cross3 * planeDist3;

	auto cornerPoint = (cross1 + cross2 + cross3) / denom;

	createNewPoint(cornerPoint, point1[0], { point1[2], point2[2] }, 0, false, true);
	createNewPoint(cornerPoint, point1[1], { point }, 0, true, true);
	createNewPoint(cornerPoint, point2[1], { point }, 0, false, true);
}

void connectPlanes(size_t pointCloudIndex)
{
	std::vector<Point*> createdPoints;
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		for (size_t j = 0; j < planes[pointCloudIndex][i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[pointCloudIndex][i]->edges[j]->pointsWithDir.size(); k++) {
				auto point = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first;
				if (point->horizontalIndex == 666 && point->verticalIndex == 29) {
					//std::cout << "asd";
				}
				if (point->verticalIndex == verticalCount) {
					continue;
				}
				size_t addedCount = 0;
				if (point->isCorner) {
					for (size_t l = 0; l < point->neighbourPlaneNeighbours.size(); l++) {
						Point* newPoint = nullptr;
						auto neighbourPoint = point->neighbourPlaneNeighbours[l];
						if (neighbourPoint) {
							if (neighbourPoint->plane && neighbourPoint->outlineId > 0 && neighbourPoint->outlineId != point->outlineId &&
								neighbourPoint->plane != point->plane) {
								newPoint = addNewPoint(point, neighbourPoint, neighbourPoint->plane, addedCount, l, pointCloudIndex);
								if (newPoint) {
									if (k == 0 && l == 0) planes[pointCloudIndex][i]->edges[planes[pointCloudIndex][i]->edges.size() - 1]->wasFirstGenerated = true;
									planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->isCorner = false;
									createdPoints.push_back(newPoint);
									addedCount++;
								}
								else {
									planes[pointCloudIndex][i]->edges[j]->pointsWithDir.insert(planes[pointCloudIndex][i]->edges[j]->pointsWithDir.begin() + k + 1 + addedCount, { nullptr, -1 });
									addedCount++;
								}
							}
						}
						if (!neighbourPoint || !neighbourPoint->plane) {
							planes[pointCloudIndex][i]->edges[j]->pointsWithDir.insert(planes[pointCloudIndex][i]->edges[j]->pointsWithDir.begin() + k + 1 + addedCount, { nullptr, -1 });
							addedCount++;
						}
					}
					k += addedCount;
				}
			}
		}
	}
}

void createCorners(size_t pointCloudIndex)
{
	const double normalDiffTreshold = 0.1;
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		for (size_t j = 0; j < planes[pointCloudIndex][i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[pointCloudIndex][i]->edges[j]->pointsWithDir.size(); k++) {
				auto point = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first;
				if (point) {
					for (size_t l = 0; l < planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints.size(); l++) {
						auto createdPoint1 = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints[l];
						auto createdPoint2 = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints[l == 3 ? 0 : (l + 1)];
						if (createdPoint1 && createdPoint2 && createdPoint1->neighbourPlaneNeighbours[1]->plane !=
							createdPoint2->neighbourPlaneNeighbours[1]->plane
							&& (planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->plane->normal -
								createdPoint1->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold &&
							(planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->plane->normal -
								createdPoint2->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold &&
							(createdPoint1->neighbourPlaneNeighbours[1]->plane->normal -
								createdPoint2->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold) {
							createPlaneCorner(planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first, { createdPoint1, createdPoint1->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l] }, { createdPoint2, createdPoint2->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l == 3 ? 0 : (l + 1)] });
							break;
						}
					}
				}
			}
		}
	}
}

void filterEdgePoints(size_t pointCloudIndex)
{
	std::vector<std::vector<std::vector< std::pair<bool, std::pair<size_t, size_t>>>>> cornersToDelete;
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		cornersToDelete.push_back({});
		for (size_t j = 0; j < planes[pointCloudIndex][i]->edges.size(); j++) {
			cornersToDelete[i].push_back({});
			for (size_t k = 0; k < planes[pointCloudIndex][i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first) {
					if (planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->verticalIndex >= verticalCount) {
						for (size_t l = 0; l < planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours.size(); l++) {
							if (planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l] &&
								planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l]->verticalIndex < verticalCount)
								cornersToDelete[i][j].push_back({ planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->verticalIndex > verticalCount,
									{ k,  planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l]->plane->id } });
						}
					}
				}
				else {
					cornersToDelete[i][j].push_back({ false, { 0, 0 } });
					planes[pointCloudIndex][i]->edges[j]->pointsWithDir.erase(planes[pointCloudIndex][i]->edges[j]->pointsWithDir.begin() + k);
					k--;
				}
			}
			for (int k = 0; k < cornersToDelete[i][j].size(); k++) {
				if (k < ((int)cornersToDelete[i][j].size() - 2) && cornersToDelete[i][j][k + 1].first && cornersToDelete[i][j][k + 2].first) {
					cornersToDelete[i][j][k + 1].second = { cornersToDelete[i][j][k + 1].second.first, cornersToDelete[i][j][k].second.second };
					size_t index = k + 2;
					while (index < cornersToDelete[i][j].size() && cornersToDelete[i][j][index].first) {
						cornersToDelete[i][j].erase(cornersToDelete[i][j].begin() + index);
					}
					cornersToDelete[i][j].insert(cornersToDelete[i][j].begin() + index, { true, {cornersToDelete[i][j][k + 1].second.first,
						index < cornersToDelete[i][j].size() ? cornersToDelete[i][j][index].second.second : cornersToDelete[i][j][0].second.second} });
				}
			}
			int newPointStartIndex = planes[pointCloudIndex][i]->edges[j]->wasFirstGenerated ? 1 : 0;
			if ((planes[pointCloudIndex][i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->horizontalIndex < planes[pointCloudIndex][i]->edges[j]->startPoint->horizontalIndex ||
				planes[pointCloudIndex][i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->verticalIndex < planes[pointCloudIndex][i]->edges[j]->startPoint->verticalIndex) &&
				planes[pointCloudIndex][i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->createdNeighbourPoints[0])
				newPointStartIndex++;
			int newPointEndIndex = newPointStartIndex;
			if (currentFrame == 4 && i == 0) {
				std::cout << "asd";
			}
			Point* endPoint = nullptr;
			while (newPointEndIndex < (int)cornersToDelete[i][j].size() - 1) {
				if (cornersToDelete[i][j][newPointStartIndex].second.second > 0) {
					while (newPointEndIndex + 1 < cornersToDelete[i][j].size()
						&& cornersToDelete[i][j][newPointStartIndex].second.second == cornersToDelete[i][j][newPointEndIndex + 1].second.second) {
						newPointEndIndex++;
					}
					for (size_t k = cornersToDelete[i][j][newPointStartIndex].second.first + 1; k <
						cornersToDelete[i][j][newPointEndIndex].second.first; k++) {
						planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->isCorner = false;
					}

					auto startPoint = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[cornersToDelete[i][j][newPointStartIndex].second.first].first;
					for (size_t k = 0; k < 2; k++) {
						if (endPoint && (startPoint->position - endPoint->position).length() < 0.1) {
							if (startPoint->verticalIndex == verticalCount + 1 && endPoint->verticalIndex != verticalCount + 1)
								endPoint->isCorner = false;
							else if (endPoint->verticalIndex == verticalCount + 1 && startPoint->verticalIndex != verticalCount + 1)
								startPoint->isCorner = false;
						}
						endPoint = planes[pointCloudIndex][i]->edges[j]->pointsWithDir[cornersToDelete[i][j][newPointEndIndex].second.first].first;
					}
					if (newPointEndIndex == cornersToDelete[i][j].size() - 1 && newPointStartIndex != newPointEndIndex &&
						planes[pointCloudIndex][i]->edges[j]->wasFirstGenerated && cornersToDelete[i][j][0].second.second ==
						cornersToDelete[i][j][newPointStartIndex].second.second)
						planes[pointCloudIndex][i]->edges[j]->pointsWithDir[cornersToDelete[i][j][cornersToDelete[i][j].size() - 1].second.first].first->isCorner = false;
				}
				newPointStartIndex = newPointEndIndex + 1;
				newPointEndIndex = newPointStartIndex;
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex == verticalCount && planes[i]->edges[j]->pointsWithDir[k].first->isCorner
					&& planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1] &&
					planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1]->verticalIndex == verticalCount) {
					planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1]->isCorner = true;
				}
			}
		}
	}*/
}

void egoCarSegmentation(size_t pointCloudIndex)
{
	for (size_t i = 0; i < points[pointCloudIndex].size(); i++) {
		if (points[pointCloudIndex][i] && points[pointCloudIndex][i]->position.x <= 1 && points[pointCloudIndex][i]->position.x >= -1 &&
			points[pointCloudIndex][i]->position.y <= 1 && points[pointCloudIndex][i]->position.y >= -1 &&
			points[pointCloudIndex][i]->position.z <= 2.5 + egoCarPos[pointCloudIndex].z && points[pointCloudIndex][i]->position.z >= -2.5 + egoCarPos[pointCloudIndex].z)
			points[pointCloudIndex][i] = nullptr;
	}
}

double angleOfVectors(Vec3<double> v1, Vec3<double> v2, bool isBackward)
{
	auto v1Angle = atan2(v1.x, v1.y) / PI;
	auto v2Angle = atan2(v2.x, v2.y) / PI;
	auto angle = abs(v1Angle - v2Angle);
	if ((!isBackward && v1Angle < v2Angle) || (isBackward && v1Angle > v2Angle)) angle = 2 - angle;
	return angle * 180;
}

__host__ __device__
Vec3<double> intersectionOfLines(Vec3<double> p1, Vec3<double> p2, Vec3<double> q1, Vec3<double> q2, size_t& onEdgeType, double& distanceRatioOnEdge)
{
	const double acceptTreshold = 0.0000001;
	auto dir1 = p2 - p1;
	auto dir2 = q2 - q1;
	double R = (p1.y * dir1.x + q1.x * dir1.y - p1.x * dir1.y - q1.y * dir1.x) / (dir2.y * dir1.x - dir1.y * dir2.x);
	Vec3<double> intersectionPos = { q1.x + R * dir2.x, q1.y + R * dir2.y, 0 };
	auto side1Length = (p2 - p1).length() + acceptTreshold;
	auto side2Length = (q2 - q1).length() + acceptTreshold;
	if ((intersectionPos - p1).length() < side1Length && (intersectionPos - p2).length() < side1Length &&
		(intersectionPos - q1).length() < side2Length && (intersectionPos - q2).length() < side2Length) {
		if ((intersectionPos - p1).length() < acceptTreshold)
			onEdgeType = 2;
		else if ((intersectionPos - p2).length() < acceptTreshold)
			onEdgeType = 3;
		else if ((intersectionPos - q1).length() < acceptTreshold)
			onEdgeType = 4;
		else if ((intersectionPos - q2).length() < acceptTreshold)
			onEdgeType = 5;
		else onEdgeType = 1;
		distanceRatioOnEdge = (intersectionPos - p1).length() / (p2 - p1).length();
	}
	return intersectionPos;
}

size_t isPointInsidePolygon(std::vector<Point*>polygon, Vec3<double> point, std::pair<double, double> xBounds, std::pair<double, double> yBounds,
	bool checkOnEdge = false, double onEdgetreshold = 0.0000001)
{
	// 0 - outside
	// 1 - inside
	// 2 - onEdge
	size_t notInUse = 0;
	double notInUseRatio = 0;
	if (point.x > xBounds.first && point.x < xBounds.second &&
		point.y > yBounds.first && point.y < yBounds.second) {
		int rigthCounter = 0;
		for (size_t i = 0; i < polygon.size(); i++) {
			auto p1 = polygon[i]->projected2DPosition;
			auto p2 = polygon[(i + 1) % polygon.size()]->projected2DPosition;
			if ((p1.x < point.x && p2.x < point.x) || (p1.y > point.y && p2.y > point.y) || (p1.y < point.y && p2.y < point.y) || (p1.y == p2.y))
				continue;
			else if (point == p1)
				return false;
			else {
				/*auto smallerAngle = (p1.x < p2.x ? p1 - point : (p2 - point));
				auto biggerAngle = (p1.x < p2.x ? p2 - point : (p1 - point));
				if((p1.x > point.x && p2.x > point.x) || angleOfVectors(smallerAngle, biggerAngle, smallerAngle.y > point.y || biggerAngle.y < point.y)
					< 180)*/
				auto intersection = intersectionOfLines(point, point + Vec3<double>({ 1,0,0 }), p1, p2, notInUse, notInUseRatio);
				if (abs(intersection.x - point.x) < onEdgetreshold && checkOnEdge)
					return 2;
				if (intersection.x <= point.x)
					continue;
				else {
					rigthCounter++;
					if (p2.y == point.y)
						i++;
				}
			}
		}
		return rigthCounter % 2 == 1;
	}
	return false;
}

void changeBaseTo2D(Edge* edge, std::pair<Vec3<double>, Vec3<double>> newBase = { {0,0,0}, {0,0,0} })
{
	edge->xBounds2D = { 100000, -100000 };
	edge->yBounds2D = { 100000, -100000 };
	auto normal = newBase.first == Vec3<double>({ 0,0,0 }) ? edge->pointsWithDir[0].first->plane->normal : newBase.first;
	auto x = newBase.first == Vec3<double>({ 0,0,0 }) ? edge->pointsWithDir[0].first->plane->pointDirections[0] : newBase.second;
	x = Vec3<double>::normalize(x - normal * Vec3<double>::dot_product(x, normal));
	auto y = Vec3<double>::crossProduct(x, normal);
	for (size_t i = 0; i < edge->pointsWithDir.size(); i++) {
		edge->pointsWithDir[i].first->projected2DPosition = { Vec3<double>::dot_product(edge->pointsWithDir[i].first->position, x),
			Vec3<double>::dot_product(edge->pointsWithDir[i].first->position, y), 0 };
		auto pos2D = edge->pointsWithDir[i].first->projected2DPosition;
		if (pos2D.x < edge->xBounds2D.first) edge->xBounds2D.first = pos2D.x;
		if (pos2D.x > edge->xBounds2D.second) edge->xBounds2D.second = pos2D.x;
		if (pos2D.y < edge->yBounds2D.first) edge->yBounds2D.first = pos2D.y;
		if (pos2D.y > edge->yBounds2D.second) edge->yBounds2D.second = pos2D.y;
	}
	for (size_t i = 0; i < edge->closestNeighbourPoints.size(); i++) {
		edge->closestNeighbourPoints[i].second = { Vec3<double>::dot_product(edge->closestNeighbourPoints[i].first, x),
			Vec3<double>::dot_product(edge->closestNeighbourPoints[i].first, y), 0 };
	}
}

void deleteSelfIntersections(std::vector<Plane*> planes)
{
	double notInUseRatio = 0;
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t l = 0; l < planes[k]->edges.size(); l++) {
			for (int i = 0; i < planes[k]->edges[l]->pointsWithDir.size(); i++) {
				for (int j = (i == (planes[k]->edges[l]->pointsWithDir.size() - 1) ? 1 : 0); j < i - 1; j++) {
					size_t onEdgeType = 0;
					auto intersection = intersectionOfLines(planes[k]->edges[l]->pointsWithDir[j].first->projected2DPosition,
						planes[k]->edges[l]->pointsWithDir[j + 1].first->projected2DPosition,
						planes[k]->edges[l]->pointsWithDir[i].first->projected2DPosition, planes[k]->edges[l]->pointsWithDir[(i + 1) %
						planes[k]->edges[l]->pointsWithDir.size()].first->projected2DPosition,
						onEdgeType, notInUseRatio);
					if (onEdgeType > 0) {
						double routeLength1 = 0;
						for (size_t m = j + 1; m < i; m++) {
							routeLength1 += (planes[k]->edges[l]->pointsWithDir[m + 1].first->projected2DPosition -
								planes[k]->edges[l]->pointsWithDir[m].first->projected2DPosition).length();
						}
						double routeLength2 = 0;
						size_t m = (i + 1) % planes[k]->edges[l]->pointsWithDir.size();
						while (m != j) {
							routeLength2 += (planes[k]->edges[l]->pointsWithDir[(m + 1) % planes[k]->edges[l]->pointsWithDir.size()].first->projected2DPosition -
								planes[k]->edges[l]->pointsWithDir[m].first->projected2DPosition).length();
							m = (m + 1) % planes[k]->edges[l]->pointsWithDir.size();
						}
						if (routeLength1 < routeLength2) {
							if (planes[k]->edges[l]->pointsWithDir[j + 1].first->verticalIndex == verticalCount + 1) {
								planes[k]->edges[l]->pointsWithDir[i].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + i);
							}
							else {
								planes[k]->edges[l]->pointsWithDir[j + 1].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + j + 1);
							}
						}
						else {
							if (planes[k]->edges[l]->pointsWithDir[j].first->verticalIndex == verticalCount + 1) {
								planes[k]->edges[l]->pointsWithDir[(i + 1) % planes[k]->edges[l]->pointsWithDir.size()].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + ((i + 1) % planes[k]->edges[l]->pointsWithDir.size()));
							}
							else {
								planes[k]->edges[l]->pointsWithDir[j].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + j);
							}
						}
						i--;
						break;
					}
				}
			}
		}
	}
}

void deleteTigthEdges(size_t pointCloudIndex)
{
	return;
	for (size_t k = 0; k < planes[pointCloudIndex].size(); k++) {
		for (size_t l = 0; l < planes[pointCloudIndex][k]->edges.size(); l++) {
			if (isEdgeTooTigth(planes[pointCloudIndex][k]->edges[l])) {
				planes[pointCloudIndex][k]->edges.erase(planes[pointCloudIndex][k]->edges.begin() + l);
				l--;
			}
		}
		if (planes[pointCloudIndex][k]->edges.size() == 0) 
		{
			planes[pointCloudIndex].erase(planes[pointCloudIndex].begin() + k);
			k--;
		}
	}
}

bool isClockwise(std::vector<Point*>& points)
{
	double angleSum = 0;
	for (size_t i = 0; i < points.size(); i++) {
		auto  a = angleOfVectors(points[(i + points.size() - 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition,
			points[(i + 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition, false);
		angleSum += angleOfVectors(points[(i + points.size() - 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition,
			points[(i + 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition, false);
	}
	return angleSum < (double)points.size() * 360.0 / 2;
}

void extract2DPolygon(size_t pointCloudIndex)
{
	for (size_t i = 0; i < planes[pointCloudIndex].size(); i++) {
		for (size_t j = 0; j < planes[pointCloudIndex][i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[pointCloudIndex][i]->edges[j]->pointsWithDir.size(); k++) {
				if (!planes[pointCloudIndex][i]->edges[j]->pointsWithDir[k].first->isCorner) {
					planes[pointCloudIndex][i]->edges[j]->pointsWithDir.erase(planes[pointCloudIndex][i]->edges[j]->pointsWithDir.begin() + k);
					k--;
				}
			}
			changeBaseTo2D(planes[pointCloudIndex][i]->edges[j]);
		}
	}
}

bool isDesiredEdge(std::vector<Point*> savedEdge, std::vector<Point*> newEdge, std::pair<double, double> newEdgeXBounds2D, std::pair<double, double> newEdgeYBounds2D,
	size_t vertexIndex, bool isInner)
{
	if (savedEdge[vertexIndex]->isMarked2) {
		auto isMiddlePointDesired = isPointInsidePolygon(newEdge, (savedEdge[vertexIndex]->projected2DPosition +
			savedEdge[(vertexIndex + 1)
			% savedEdge.size()]->projected2DPosition) / 2, newEdgeXBounds2D, newEdgeYBounds2D, true, 0.00001) == isInner;
		return isMiddlePointDesired;
	}
	return false;
}

Point* decideIfDesiredEdgeGood(Edge* savedEdge, Edge* newEdge, Point* currentPoint, std::vector<Point*> savedEdgePoints, std::vector<Point*> newEdgePoints,
	bool isOnSavedEdge, bool checkForInsidePoints, std::vector<std::pair<Vec3<double>, Vec3<double>>> allNeighbours, /*out*/ std::vector<Point*>& pointsOnDesiredEdge)
{
	std::pair<double, double> xBounds = { 100000, -1000000 };
	std::pair<double, double> yBounds = { 100000, -1000000 };
	std::vector<Point*> pointsOnAddedPolygonPart;
	Point* endPoint = nullptr;
	size_t currentIndex = 0;
	bool isForward = true;
	do {
		for (size_t i = 0; i < (isOnSavedEdge ? newEdgePoints : savedEdgePoints).size(); i++) {
			if ((isOnSavedEdge ? newEdgePoints : savedEdgePoints)[i] == currentPoint) {
				currentIndex = i;
				currentPoint = (isOnSavedEdge ? newEdgePoints : savedEdgePoints)[currentIndex];
				break;
			}
		}
		isOnSavedEdge = !isOnSavedEdge;
		do {
			if (isForward && pointsOnDesiredEdge.size() == pointsOnAddedPolygonPart.size()) {
				pointsOnDesiredEdge.push_back(currentPoint);
				currentPoint->isMarked = false;
			}
			pointsOnAddedPolygonPart.push_back(currentPoint);
			auto pos2D = currentPoint->projected2DPosition;
			if (pos2D.x < xBounds.first) xBounds.first = pos2D.x;
			if (pos2D.x > xBounds.second) xBounds.second = pos2D.x;
			if (pos2D.y < yBounds.first) yBounds.first = pos2D.y;
			if (pos2D.y > yBounds.second) yBounds.second = pos2D.y;
			currentIndex = (currentIndex + (isForward ? 1 : ((isOnSavedEdge ? savedEdgePoints : newEdgePoints).size() - 1))) % (isOnSavedEdge ?
				savedEdgePoints : newEdgePoints).size();
			currentPoint = (isOnSavedEdge ? savedEdgePoints : newEdgePoints)[currentIndex];
		} while (!currentPoint->isMarked2);
		if (pointsOnDesiredEdge.size() == pointsOnAddedPolygonPart.size()) endPoint = currentPoint;
		isForward = !isForward;
	} while (currentPoint != pointsOnDesiredEdge[0] && checkForInsidePoints);
	if (checkForInsidePoints) {
		for (const auto p : allNeighbours) {
			if (isPointInsidePolygon(pointsOnAddedPolygonPart, p.second, xBounds, yBounds, true, savedEdge->isHole ? 0.00 : 0.0) == 1) {
				/*points.push_back(new Point(p.first, 0, 0, nullptr));
				for (const auto p2 : (isOnSavedEdge ? savedEdge : newEdge)->closestNeighbourPoints) {
					//points.push_back(new Point(p2.first, 0, 0, nullptr));
				}*/
				return nullptr;
			}
		}
	}
	return endPoint;
}

void calculateNewNeighbours(std::vector<Edge*> createdEdges, std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours)
{
	std::vector < std::vector<std::pair<std::vector<std::pair<std::pair<Vec3<double>, Vec3<double>>, double>>, double>>>
		neighboursClosestToEdge(createdEdges.size());
	for (size_t i = 0; i < createdEdges.size(); i++) {
		neighboursClosestToEdge[i] = std::vector< std::pair<std::vector<std::pair<std::pair<Vec3<double>, Vec3<double>>, double>>, double>>(
			createdEdges[i]->pointsWithDir.size(), { {}, 100000 });
	}
	for (size_t i = 0; i < neighbours.size(); i++) {
		double minDist = 100000;
		size_t closestEdgeIndex = 0;
		size_t closestPointIndex = 0;
		for (size_t k = 0; k < createdEdges.size(); k++) {
			for (size_t j = 0; j < createdEdges[k]->pointsWithDir.size(); j++) {
				double newDist = (createdEdges[k]->pointsWithDir[j].first->projected2DPosition - neighbours[i].second).length();
				if (newDist < minDist) {
					minDist = newDist;
					closestEdgeIndex = k;
					closestPointIndex = j;
				}
			}
		}
		neighboursClosestToEdge[closestEdgeIndex][closestPointIndex].first.push_back({ neighbours[i] , minDist });
		if (minDist < neighboursClosestToEdge[closestEdgeIndex][closestPointIndex].second)
			neighboursClosestToEdge[closestEdgeIndex][closestPointIndex].second = minDist;
	}
	for (size_t i = 0; i < neighboursClosestToEdge.size(); i++) {
		createdEdges[i]->closestNeighbourPoints.clear();
		for (size_t j = 0; j < neighboursClosestToEdge[i].size(); j++) {
			for (size_t k = 0; k < neighboursClosestToEdge[i][j].first.size(); k++) {
				if (neighboursClosestToEdge[i][j].first[k].second < neighboursClosestToEdge[i][j].second * 2)
					createdEdges[i]->closestNeighbourPoints.push_back(neighboursClosestToEdge[i][j].first[k].first);
			}
		}
	}
}

bool areEdgesIntersect(Plane* savedPlane, Plane* newPlane, Edge* savedEdge, Edge* newEdge)
{
	bool hasFoundIntersection = false;
	auto savedEdgePoints = savedEdge->getPoints();
	auto newEdgePoints = newEdge->getPoints();
	for (size_t j = 0; j < newEdge->pointsWithDir.size(); j++) {
		if (isPointInsidePolygon(savedEdgePoints, newEdge->pointsWithDir[j].first->projected2DPosition, savedEdge->xBounds2D, savedEdge->yBounds2D, true)) {
			hasFoundIntersection = true;
			break;
		}
		for (size_t i = 0; i < savedEdge->pointsWithDir.size(); i++) {
			if (isPointInsidePolygon(newEdgePoints, savedEdge->pointsWithDir[i].first->projected2DPosition, newEdge->xBounds2D, newEdge->yBounds2D, true)) {
				hasFoundIntersection = true;
				break;
			}
			size_t onEdgeType = 0;
			double distanceRatio = 0;
			auto intersectionPos = intersectionOfLines(savedEdge->pointsWithDir[i].first->projected2DPosition, savedEdge->pointsWithDir[(i + 1) %
				savedEdge->pointsWithDir.size()].first->projected2DPosition, newEdge->pointsWithDir[j].first->projected2DPosition,
				newEdge->pointsWithDir[(j + 1) % newEdge->pointsWithDir.size()].first->projected2DPosition, onEdgeType, distanceRatio);
			if (onEdgeType > 0) {
				hasFoundIntersection = true;
				break;
			}
		}
		if (hasFoundIntersection)
			break;
	}
	return hasFoundIntersection;
}

bool hasNewEdgeNeighbourInsideSavedEdge(Edge* savedEdge, Edge* newEdge)
{
	for (size_t i = 0; i < newEdge->closestNeighbourPoints.size(); i++) {
		if (isPointInsidePolygon(savedEdge->getPoints(), newEdge->closestNeighbourPoints[i].second, savedEdge->xBounds2D, savedEdge->yBounds2D, true) == 1) {
			return true;
		}
	}
	return false;
}

void mergePolygons(Plane* savedPlane, Edge* savedEdge, Edge* newEdge, std::vector<std::pair<Vec3<double>, Vec3<double>>> allNeighbours,
	/*out*/ std::vector<Edge*>& outputEdges, bool& createdNewPolygon)
{
	std::vector<Point*> savedEdgePoints = savedEdge->getPoints();
	std::vector<Point*> newEdgePoints = newEdge->getPoints();
	bool isHole = newEdge->isHole;
	for (size_t j = 0; j < newEdgePoints.size(); j++) {
		for (size_t i = 0; i < savedEdgePoints.size(); i++) {
			size_t onEdgeType = 0;
			double distanceRatio = 0;
			if ((newEdgePoints[j]->isMarked2 && (newEdgePoints[j] == savedEdgePoints[i] || newEdgePoints[j] == savedEdgePoints[(i + 1) % savedEdgePoints.size()])) ||
				(newEdgePoints[(j + 1) % newEdgePoints.size()]->isMarked2 && (newEdgePoints[(j + 1) % newEdgePoints.size()] == savedEdgePoints[i] ||
					newEdgePoints[(j + 1) % newEdgePoints.size()] == savedEdgePoints[(i + 1) % savedEdgePoints.size()])))
				continue;
			auto intersectionPos = intersectionOfLines(savedEdgePoints[i]->projected2DPosition, savedEdgePoints[(i + 1) %
				savedEdgePoints.size()]->projected2DPosition, newEdgePoints[j]->projected2DPosition,
				newEdgePoints[(j + 1) % newEdgePoints.size()]->projected2DPosition, onEdgeType, distanceRatio);
			if (onEdgeType > 0) {
				if (onEdgeType == 1) {
					Vec3<double> pos3D = savedEdgePoints[i]->position + (savedEdgePoints[(i + 1) %
						savedEdgePoints.size()]->position - savedEdgePoints[i]->position) * distanceRatio;
					Point* intersectionPoint = new Point(pos3D, horizontalCount + 2, verticalCount + 2, savedEdge->startPoint->plane);
					intersectionPoint->projected2DPosition = intersectionPos;
					intersectionPoint->isCorner = true;
					intersectionPoint->isMarked2 = true;
					savedEdgePoints.insert(savedEdgePoints.begin() + i + 1, intersectionPoint);
					newEdgePoints.insert(newEdgePoints.begin() + j + 1, intersectionPoint);
					fitAddedPoints.push_back(intersectionPoint);
					i++;
				}
				else {
					bool isOnSavedEdge = onEdgeType < 4;
					size_t selectedIndex = (onEdgeType % 2) == 0 ? (isOnSavedEdge ? i : j) : (isOnSavedEdge ?
						((i + 1) % savedEdgePoints.size()) : ((j + 1) % newEdgePoints.size()));
					auto selectedEdgePoints = isOnSavedEdge ? savedEdgePoints : newEdgePoints;
					auto otherEdgePoints = isOnSavedEdge ? newEdgePoints : savedEdgePoints;
					auto selectedPoint = selectedEdgePoints[selectedIndex];
					bool isAlreadyAdded = false;
					for (size_t k = 0; k < otherEdgePoints.size(); k++) {
						if (otherEdgePoints[k] == selectedPoint) {
							isAlreadyAdded = true;
							break;
						}
					}
					if (!isAlreadyAdded) {
						selectedPoint->isMarked2 = true;
						(isOnSavedEdge ? newEdgePoints : savedEdgePoints).insert((isOnSavedEdge ? newEdgePoints : savedEdgePoints).begin() + (isOnSavedEdge ? j : i) + 1,
							selectedPoint);
					}
				}
			}
		}
	}
	if (newEdgePoints[0]->horizontalIndex == 168 && newEdgePoints[0]->verticalIndex == 18) {
		std::cout << "asd";
	}
	std::vector<Point*> desiredEdges;
	Point* currentPoint = nullptr;
	for (size_t i = 0; i < 2; i++) //filtering
	{
		std::vector<Point*>& currentEdge = (i == 0 ? savedEdgePoints : newEdgePoints);
		for (size_t j = 0; j < currentEdge.size(); j++) {
			if (currentEdge[j]->projected2DPosition == currentEdge[(j + 1) % currentEdge.size()]->projected2DPosition) {
				if (!currentEdge[j]->isMarked2) {
					currentEdge.erase(currentEdge.begin() + j);
					j--;
				}
				else if (!currentEdge[(j + 1) % currentEdge.size()]->isMarked2) {
					currentEdge.erase(currentEdge.begin() + (j + 1) % currentEdge.size());
					j--;
				}
			}
		}
	}
	for (size_t i = 0; i < savedEdgePoints.size(); i++) //filtering
	{
		if (savedEdgePoints[i]->isMarked2) {
			auto pos1 = (savedEdgePoints[i]->projected2DPosition + savedEdgePoints[(i +
				savedEdgePoints.size() - 1) % savedEdgePoints.size()]->projected2DPosition) / 2;
			auto pos2 = (savedEdgePoints[i]->projected2DPosition + savedEdgePoints[(i + 1) %
				savedEdgePoints.size()]->projected2DPosition) / 2;
			if (isPointInsidePolygon(newEdgePoints, pos1, newEdge->xBounds2D, newEdge->yBounds2D, true) ==
				isPointInsidePolygon(newEdgePoints, pos2, newEdge->xBounds2D, newEdge->yBounds2D, true)) {
				savedEdgePoints[i]->isMarked2 = false;
			}
		}
		if (savedEdgePoints[i]->isMarked2 && savedEdgePoints[(i + 1) % savedEdgePoints.size()]->isMarked2)
			//&& savedEdgePoints[i]->verticalIndex != verticalCount + 2 && savedEdgePoints[(i + 1) % 
			//savedEdgePoints.size()]->verticalIndex != verticalCount + 2)
		{
			for (size_t j = 0; j < newEdgePoints.size(); j++) {
				if ((newEdgePoints[j] == savedEdgePoints[i] && newEdgePoints[(j + 1) %
					newEdgePoints.size()] == savedEdgePoints[(i + 1) % savedEdgePoints.size()]) || (newEdgePoints[j] == savedEdgePoints[(i + 1) % savedEdgePoints.size()]
						&& newEdgePoints[(j + 1) % newEdgePoints.size()] == savedEdgePoints[i])) {
					auto pos1 = (newEdgePoints[(j + newEdgePoints.size() - 1) % newEdgePoints.size()]->projected2DPosition + newEdgePoints[j]->projected2DPosition) / 2;
					auto pos2 = (newEdgePoints[(j + 2) % newEdgePoints.size()]->projected2DPosition + newEdgePoints[(j + 1) % newEdgePoints.size()]->projected2DPosition) / 2;
					if (isPointInsidePolygon(savedEdgePoints, pos1, savedEdge->xBounds2D, savedEdge->yBounds2D, true, 0.01) ==
						isPointInsidePolygon(savedEdgePoints, pos2, savedEdge->xBounds2D, savedEdge->yBounds2D, true, 0.01)) {
						newEdgePoints[j]->isMarked2 = false;
					}
					newEdgePoints[(j + 1) % newEdgePoints.size()]->isMarked2 = false;
					break;
				}
			}
		}
	}
	int dbgCounter = 0;
	for (size_t i = 0; i < savedEdgePoints.size(); i++) {
		if (savedEdgePoints[i]->isMarked2)
			dbgCounter++;
		if (isDesiredEdge(savedEdgePoints, newEdgePoints, newEdge->xBounds2D, newEdge->yBounds2D, i, isHole)) {
			desiredEdges.push_back(savedEdgePoints[i]);
			savedEdgePoints[i]->isMarked = true;
			if (savedEdgePoints[i]->isMarked2 && !currentPoint) {
				currentPoint = savedEdgePoints[i];
			}
		}
	}
	if (dbgCounter % 2 != 0) {
		/*/for (size_t y = 0; y < savedEdge->pointsWithDir.size(); y++) {
			savedEdge->pointsWithDir[y].first->cornerId = currentCornerId;
			savedEdge->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
			savedPoints.push_back(savedEdge->pointsWithDir[y].first);
			savedEdge->pointsWithDir[y].first->position = savedEdge->pointsWithDir[y].first->projected2DPosition;
			currentCornerIndex++;
		}
		currentCornerId++;
		currentCornerIndex = 0;
		for (size_t y = 0; y < newEdge->pointsWithDir.size(); y++) {
			newEdge->pointsWithDir[y].first->cornerId = currentCornerId;
			newEdge->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
			savedPoints.push_back(newEdge->pointsWithDir[y].first);
			newEdge->pointsWithDir[y].first->position = newEdge->pointsWithDir[y].first->projected2DPosition;
			currentCornerIndex++;
		}
		currentCornerId++;
		writeData(4);*/
		std::cout << "INTERSECTION COUNT IS NOT EVEN!" << std::endl;
	}

	while (desiredEdges.size() > 0) {
		bool isOnSavedEdge = false;
		Edge* unionPolygon = new Edge();
		fitEdges.push_back(unionPolygon);
		unionPolygon->startPoint = currentPoint;
		unionPolygon->isHole = isHole;
		do {
			std::vector<Point*> pointsOnNextEdge;
			auto helperPoint = currentPoint;
			bool helperIsOnEdge = isOnSavedEdge;
			currentPoint = decideIfDesiredEdgeGood(savedEdge, newEdge, currentPoint, savedEdgePoints, newEdgePoints, isOnSavedEdge, true, allNeighbours, pointsOnNextEdge);
			if (!currentPoint) {
				pointsOnNextEdge.clear();
				isOnSavedEdge = !isOnSavedEdge;
				currentPoint = decideIfDesiredEdgeGood(savedEdge, newEdge, helperPoint, savedEdgePoints, newEdgePoints, isOnSavedEdge, false, allNeighbours,
					pointsOnNextEdge);
			}
			for (size_t i = 0; i < pointsOnNextEdge.size(); i++) {
				pointsOnNextEdge[i]->isMarked = false;
				unionPolygon->pointsWithDir.push_back({ pointsOnNextEdge[i], -1 });
				auto pos2D = pointsOnNextEdge[i]->projected2DPosition;
				if (pos2D.x < unionPolygon->xBounds2D.first) unionPolygon->xBounds2D.first = pos2D.x;
				if (pos2D.x > unionPolygon->xBounds2D.second) unionPolygon->xBounds2D.second = pos2D.x;
				if (pos2D.y < unionPolygon->yBounds2D.first) unionPolygon->yBounds2D.first = pos2D.y;
				if (pos2D.y > unionPolygon->yBounds2D.second) unionPolygon->yBounds2D.second = pos2D.y;
			}
			isOnSavedEdge = !helperIsOnEdge;
		} while (currentPoint != unionPolygon->pointsWithDir[0].first);
		outputEdges.push_back(unionPolygon);
		currentPoint = nullptr;
		for (size_t i = 0; i < desiredEdges.size(); i++) {
			if (!desiredEdges[i]->isMarked) {
				desiredEdges.erase(desiredEdges.begin() + i);
				i--;
			}
			else if (currentPoint == nullptr && desiredEdges[i]->isMarked2)
				currentPoint = desiredEdges[i];
		}
	}
	if (outputEdges.size() > 0) {
		if (!isHole) {
			while (outputEdges.size() > 1) {
				if (isPointInsidePolygon(outputEdges[1]->getPoints(), outputEdges[0]->pointsWithDir[0].first->projected2DPosition, outputEdges[1]->xBounds2D,
					outputEdges[1]->yBounds2D))
					outputEdges.erase(outputEdges.begin());
				else
					outputEdges.erase(outputEdges.begin() + 1);
			}
		}
		else {
			double maxLength = 0;
			size_t maxIndex = 0;
			for (size_t i = 0; i < outputEdges.size(); i++) {
				double length = 0;
				for (size_t j = 0; j < outputEdges[i]->pointsWithDir.size(); j++) {
					length += (outputEdges[i]->pointsWithDir[j].first->projected2DPosition - outputEdges[i]->pointsWithDir[(j + 1) %
						outputEdges[i]->pointsWithDir.size()].first->projected2DPosition).length();
				}
				if (length > maxLength) {
					maxIndex = i;
					maxLength = length;
				}
			}
			auto helper = outputEdges[maxIndex];
			outputEdges.clear();
			outputEdges.push_back(helper);
		}
	}
	else {
		bool hasFoundPointInside = false;
		for (size_t j = 0; j < 2; j++) {
			for (size_t i = 0; i < (j == 0 ? newEdgePoints : savedEdgePoints).size(); i++) {
				if (isPointInsidePolygon((j == 0 ? savedEdgePoints : newEdgePoints), (j == 0 ? newEdgePoints :
					savedEdgePoints)[i]->projected2DPosition, (j == 0 ? savedEdge : newEdge)->xBounds2D, (j == 0 ? savedEdge : newEdge)->yBounds2D,
					true) == 1) {
					bool hasFoundNeighbourInside = false;
					for (size_t k = 0; k < (((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? newEdge : savedEdge)->closestNeighbourPoints.size(); k++) {
						if (isPointInsidePolygon((((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? savedEdge : newEdge)->getPoints(),
							(((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? newEdge : savedEdge)->closestNeighbourPoints[k].second,
							(((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? savedEdge : newEdge)->xBounds2D,
							(((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? savedEdge : newEdge)->yBounds2D, true) == 1) {
							hasFoundNeighbourInside = true;
							break;
						}
					}
					hasFoundPointInside = true;
					auto acceptedEdge = (((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) && !hasFoundNeighbourInside) ? savedEdge : newEdge;
					Edge* unionPolygon = new Edge();
					fitEdges.push_back(unionPolygon);
					unionPolygon->startPoint = acceptedEdge->startPoint;
					unionPolygon->isHole = acceptedEdge->isHole;
					unionPolygon->pointsWithDir = acceptedEdge->pointsWithDir;
					unionPolygon->xBounds2D = acceptedEdge->xBounds2D;
					unionPolygon->yBounds2D = acceptedEdge->yBounds2D;
					outputEdges.push_back(unionPolygon);
					break;
				}
			}
			if (hasFoundPointInside) break;
		}
		if (!hasFoundPointInside) {
			createdNewPolygon = false;
			outputEdges.push_back(savedEdge);
			outputEdges.push_back(newEdge);
		}
	}
	if (createdNewPolygon) {
		std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours;
		neighbours.reserve(savedEdge->closestNeighbourPoints.size() + newEdge->closestNeighbourPoints.size());
		neighbours.insert(neighbours.end(), savedEdge->closestNeighbourPoints.begin(), savedEdge->closestNeighbourPoints.end());
		neighbours.insert(neighbours.end(), newEdge->closestNeighbourPoints.begin(), newEdge->closestNeighbourPoints.end());
		calculateNewNeighbours(outputEdges, neighbours);
		for (size_t i = 0; i < outputEdges.size(); i++) {
			for (size_t j = 0; j < outputEdges[i]->pointsWithDir.size(); j++) {
				outputEdges[i]->pointsWithDir[j].first->plane = savedPlane;
			}
		}
	}
	setPointsMarked(savedEdgePoints, false, false);
	setPointsMarked(newEdgePoints, false, false);
}

bool isEdgeTooTigth(Edge* edge)
{
	double acceptTreshold = 0.1;
	double maxDistance = 0;
	Vec3<double> normal = { 0,0,0 };
	Vec3<double> linePoint = { 0,0,0 };
	for (size_t i = 0; i < edge->pointsWithDir.size() - 1; i++) {
		for (size_t j = i + 1; j < edge->pointsWithDir.size(); j++) {
			double currentDistance = (edge->pointsWithDir[i].first->projected2DPosition - edge->pointsWithDir[j].first->projected2DPosition).length();
			if (currentDistance > maxDistance) {
				maxDistance = currentDistance;
				normal = edge->pointsWithDir[i].first->projected2DPosition - edge->pointsWithDir[j].first->projected2DPosition;
				linePoint = edge->pointsWithDir[i].first->projected2DPosition;
			}
		}
	}
	normal = { -normal.y, normal.x, 0 };
	for (size_t i = 0; i < edge->pointsWithDir.size() - 1; i++) {
		if (abs(Vec3<double>::dot_product(normal, edge->pointsWithDir[i].first->projected2DPosition - linePoint)) > acceptTreshold)
			return false;
	}
	return true;
}

void mergeArrayOfEdges(Plane* savedPlane, std::vector<Edge*>& createdEdges, std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours, size_t pointCloudIndex, bool dbg = false)
{
	std::vector<Edge*> tempCreatedEdges;
	bool createdNewPolygon = true;
	bool foundNewIntersection = true;
	int counter = 0;
	while (foundNewIntersection) {
		foundNewIntersection = false;
		while (createdEdges.size() > 0) {

			for (size_t i = 1; i < createdEdges.size(); i++) {
				if (createdEdges[0]->isHole == createdEdges[i]->isHole) {
					std::vector<Edge*> innerCreatedEdges;
					/*if (false && dbg && i == 24 && counter == 13) {// currentFrame == 4 && savedEdge->isHole && counter == 3) {
						savedPoints.clear();
						addedPoints.clear();
						points.clear();
						changeBaseTo2D(createdEdges[0], { savedPlane->normal, savedPlane->pointDirections.first });
						changeBaseTo2D(createdEdges[i], { savedPlane->normal, savedPlane->pointDirections.first });
						for (size_t y = 0; y < createdEdges[0]->pointsWithDir.size(); y++) {
							createdEdges[0]->pointsWithDir[y].first->cornerId = currentCornerId;
							createdEdges[0]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
							savedPoints.push_back(createdEdges[0]->pointsWithDir[y].first);
							createdEdges[0]->pointsWithDir[y].first->position = createdEdges[0]->pointsWithDir[y].first->projected2DPosition;
							currentCornerIndex++;
						}
						currentCornerId++;
						currentCornerIndex = 0;
						for (size_t y = 0; y < createdEdges[i]->pointsWithDir.size(); y++) {
							createdEdges[i]->pointsWithDir[y].first->cornerId = currentCornerId;
							createdEdges[i]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
							savedPoints.push_back(createdEdges[i]->pointsWithDir[y].first);
							createdEdges[i]->pointsWithDir[y].first->position = createdEdges[i]->pointsWithDir[y].first->projected2DPosition;
							currentCornerIndex++;
						}

						for (size_t x = 0; x < createdEdges.size(); x++) {
							for (size_t y = 0; y < createdEdges[x]->closestNeighbourPoints.size(); y++) {
								savedPoints.push_back(new Point(createdEdges[x]->closestNeighbourPoints[y].first, 0, 0, nullptr));
							}
							currentCornerIndex = 0;
							for (size_t y = 0; y < createdEdges[x]->pointsWithDir.size(); y++) {
								createdEdges[x]->pointsWithDir[y].first->cornerId = 0;
								createdEdges[x]->pointsWithDir[y].first->cornerIndex = 0;
							}
							for (size_t y = 0; y < createdEdges[x]->pointsWithDir.size(); y++) {
								if (createdEdges[x]->pointsWithDir[y].first->isCorner && createdEdges[x]->pointsWithDir[y].first->cornerId == 0) {
									createdEdges[x]->pointsWithDir[y].first->cornerId = currentCornerId;
									createdEdges[x]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
									savedPoints.push_back(createdEdges[x]->pointsWithDir[y].first);
									currentCornerIndex++;
								}
								else if (createdEdges[x]->pointsWithDir[y].first->cornerId != 0) {
									std::cout << "asd";
								}
							}
							currentCornerId++;
						}
						writeData(4);
						std::cout << "asd";
					}*/
					createdNewPolygon = true;
					mergePolygons(savedPlane, createdEdges[0], createdEdges[i], neighbours, innerCreatedEdges, createdNewPolygon);

					if (createdNewPolygon) {
						foundNewIntersection = true;
						for (size_t m = 0; m < innerCreatedEdges.size(); m++) {
							if (!isEdgeTooTigth(innerCreatedEdges[m]))
								tempCreatedEdges.push_back(innerCreatedEdges[m]);
						}
						createdEdges.erase(createdEdges.begin() + i);
						createdEdges.erase(createdEdges.begin());
						break;
					}
				}
				else
					createdNewPolygon = false;
			}
			counter++;
			if (!createdNewPolygon || createdEdges.size() == 1) {
				tempCreatedEdges.push_back(createdEdges[0]);
				createdEdges.erase(createdEdges.begin());
			}
		}
		createdEdges = tempCreatedEdges;
		tempCreatedEdges.clear();
	}
}

void filterFittedPlanes()
{
	int dbgCounter = 0;
	double normalDistTreshold = 0.1;
	double distTreshold = 0.01;
	double newDirectionTreshold = 0.1;
	for (size_t i = 0; i < allPlanes.size(); i++) {
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			bool deletedPoint = true;
			dbgCounter = 0;
			while (deletedPoint && allPlanes[i]->edges[j]->pointsWithDir.size() > 2) {
				dbgCounter++;
				deletedPoint = false;
				auto normal = Vec3<double>::normalize(allPlanes[i]->edges[j]->pointsWithDir[allPlanes[i]->edges[j]->pointsWithDir.size() - 1].first->projected2DPosition -
					allPlanes[i]->edges[j]->pointsWithDir[allPlanes[i]->edges[j]->pointsWithDir.size() - 2].first->projected2DPosition);
				normal = { -normal.y, normal.x, 0 };
				auto linePoint = allPlanes[i]->edges[j]->pointsWithDir[allPlanes[i]->edges[j]->pointsWithDir.size() - 2].first->projected2DPosition;
				for (size_t k = 0; k < allPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
					auto verticalIndex = allPlanes[i]->edges[j]->pointsWithDir[k].first->verticalIndex;
					auto previousIndex = (k + allPlanes[i]->edges[j]->pointsWithDir.size() - 1) % allPlanes[i]->edges[j]->pointsWithDir.size();
					auto previousPoint = allPlanes[i]->edges[j]->pointsWithDir[previousIndex].first;
					auto previousVerticalIndex = previousPoint->verticalIndex;
					double dist = (allPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition - previousPoint->projected2DPosition).length();
					if (dist < distTreshold) {
						if (previousVerticalIndex < verticalCount || verticalIndex >= previousVerticalIndex) {
							allPlanes[i]->edges[j]->pointsWithDir.erase(allPlanes[i]->edges[j]->pointsWithDir.begin() + previousIndex);
							if (previousIndex < k) k--;
						}
						else {
							allPlanes[i]->edges[j]->pointsWithDir.erase(allPlanes[i]->edges[j]->pointsWithDir.begin() + k);
							k--;
						}
						deletedPoint = true;
						continue;
					}
					double normalDist = abs(Vec3<double>::dot_product(normal, allPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition - linePoint));
					if (normalDist < normalDistTreshold) {
						if (previousVerticalIndex < verticalCount || verticalIndex >= previousVerticalIndex) {
							allPlanes[i]->edges[j]->pointsWithDir.erase(allPlanes[i]->edges[j]->pointsWithDir.begin() + previousIndex);
							if (previousIndex < k) k--;
						}
						else if ((Vec3<double>::normalize(allPlanes[i]->edges[j]->pointsWithDir[(k + 1) % allPlanes[i]->edges[j]->pointsWithDir.size()].first->projected2DPosition -
							previousPoint->projected2DPosition) - Vec3<double>::normalize(allPlanes[i]->edges[j]->pointsWithDir[(k
								+ 1) % allPlanes[i]->edges[j]->pointsWithDir.size()].first->projected2DPosition -
								allPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition)).length() < newDirectionTreshold) {
							allPlanes[i]->edges[j]->pointsWithDir.erase(allPlanes[i]->edges[j]->pointsWithDir.begin() + k);
							k--;
						}
						else
							continue;
						deletedPoint = true;
					}
					else {
						normal = Vec3<double>::normalize(allPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition - previousPoint->projected2DPosition);
						normal = { -normal.y, normal.x, 0 };
						linePoint = previousPoint->projected2DPosition;
					}
				}
			}
			if (allPlanes[i]->edges[j]->pointsWithDir.size() < 3) {
				allPlanes[i]->edges.erase(allPlanes[i]->edges.begin() + j);
				j--;
			}
		}
	}
}

void checkIfHolesAreContained()
{
	for (size_t i = 0; i < allPlanes.size(); i++) {
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			if (allPlanes[i]->edges[j]->isHole) {
				bool isCointained = false;
				for (size_t k = 0; k < allPlanes[i]->edges.size(); k++) {
					if (!allPlanes[i]->edges[k]->isHole) {
						for (size_t l = 0; l < allPlanes[i]->edges[j]->pointsWithDir.size(); l++) {
							if (isPointInsidePolygon(allPlanes[i]->edges[k]->getPoints(),
								allPlanes[i]->edges[j]->pointsWithDir[l].first->projected2DPosition, allPlanes[i]->edges[k]->xBounds2D,
								allPlanes[i]->edges[k]->yBounds2D)) {
								isCointained = true;
							}
							else {
								isCointained = false;
								break;
							}
						}
						if (isCointained) {
							double notInUse = 0;
							for (size_t l = 0; l < allPlanes[i]->edges[j]->pointsWithDir.size(); l++) {
								for (size_t m = 0; m < allPlanes[i]->edges[k]->pointsWithDir.size(); m++) {
									size_t onEdgeType = 0;
									intersectionOfLines(allPlanes[i]->edges[j]->pointsWithDir[l].first->projected2DPosition,
										allPlanes[i]->edges[j]->pointsWithDir[(l + 1) % allPlanes[i]->edges[j]->pointsWithDir.size()].first->projected2DPosition,
										allPlanes[i]->edges[k]->pointsWithDir[m].first->projected2DPosition,
										allPlanes[i]->edges[k]->pointsWithDir[(m + 1) % allPlanes[i]->edges[k]->pointsWithDir.size()].first->projected2DPosition, onEdgeType, notInUse);
									if (onEdgeType > 0) {
										isCointained = false;
										break;
									}
								}
								if (!isCointained)
									break;
							}
							break;
						}
					}
				}
				allPlanes[i]->edges[j]->isInUse = isCointained;
			}
		}
	}
}

bool arePlanesIntersecting(Plane* plane1, Plane* plane2)
{
	double normalTreshold = 0.1;
	double distanceTreshold = 0.1;
	if ((plane1->normal - plane2->normal).length() < normalTreshold &&
		(abs(Vec3<double>::dot_product(plane1->normal, plane2->furthestNormalPoints[0] - plane1->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane1->normal, plane2->furthestNormalPoints[1] - plane1->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane2->normal, plane1->furthestNormalPoints[0] - plane2->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane2->normal, plane1->furthestNormalPoints[1] - plane2->planePointPos)) < distanceTreshold)) {
		for (int k = 0; k < plane1->edges.size(); k++) {
			changeBaseTo2D(plane1->edges[k], { plane1->normal, plane1->pointDirections[0] });
		}
		for (int k = 0; k < plane2->edges.size(); k++) {
			changeBaseTo2D(plane2->edges[k], { plane1->normal, plane1->pointDirections[0] });
		}
		for (int k = 0; k < plane1->edges.size(); k++) {
			for (int l = 0; l < plane2->edges.size(); l++) {
				if (plane1->edges[k]->isHole == plane2->edges[l]->isHole && plane1->edges[k]->canIntersectWithEdge(plane2->edges[l])) {
					if (areEdgesIntersect(plane2, plane1, plane2->edges[l], plane1->edges[k])) {
						return true;
					}
				}
			}
		}
		for (int k = 0; k < plane2->edges.size(); k++) {
			changeBaseTo2D(plane2->edges[k], { plane2->normal, plane2->pointDirections[0] });
		}
	}
	return false;
}

__global__
void relocateNeighbours(double* positions, bool* isInsideHole, double* polygon, size_t neighbourCount, size_t polygonSize, double xBoundsMin, double xBoundsMax,
	double yBoundsMin, double yBoundsMax, bool checkOnEdge = false, double onEdgetreshold = 0.0000001)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < neighbourCount) 
	{
		// 0 - outside
		// 1 - inside
		// 2 - onEdge
		size_t notInUse = 0;
		double notInUseRatio = 0;
		size_t type = 0;
		Vec3<double> point = { positions[index], positions[index + 1], positions[index + 2] };
		if (point.x > xBoundsMin && point.x < xBoundsMax &&
			point.y > yBoundsMin && point.y < yBoundsMax) {
			int rigthCounter = 0;
			for (size_t i = 0; i < polygonSize; i++) {
				Vec3<double> p1 = { polygon[i], polygon[i + 1], polygon[i + 2] };
				Vec3<double> p2 = { polygon[(i + 3) % (polygonSize * 3)], polygon[(i + 4) % (polygonSize * 3)], polygon[(i + 5) % (polygonSize * 3)] };
				if ((p1.x < point.x && p2.x < point.x) || (p1.y > point.y && p2.y > point.y) || (p1.y < point.y && p2.y < point.y) || (p1.y == p2.y))
					continue;
				else if (point == p1)
					break;
				else {
					auto intersection = intersectionOfLines(point, point + Vec3<double>({ 1,0,0 }), p1, p2, notInUse, notInUseRatio);
					if (abs(intersection.x - point.x) < onEdgetreshold && checkOnEdge)
					{
						type = 2;
						break;
					}
					if (intersection.x <= point.x)
						continue;
					else {
						rigthCounter++;
						if (p2.y == point.y)
							i++;
					}
				}
			}
			if(type == 0)
				type = rigthCounter % 2 == 1;
		}
		if (type > 0) 
		{
			isInsideHole[index] = true;
		}
	}
}

void relocateHoleNeighbours(Plane* plane)
{
	for (size_t i = 0; i < plane->edges.size(); i++) {
		if (plane->edges[i]->isHole) {
			for (size_t k = 0; k < plane->closestNeighbourPointsNonHole.size(); k++) {
				if (isPointInsidePolygon(plane->edges[i]->getPoints(), plane->closestNeighbourPointsNonHole[k].second,
					plane->edges[i]->xBounds2D, plane->edges[i]->yBounds2D, true, 0.01) > 0) {
					plane->closestNeighbourPointsHole.push_back(plane->closestNeighbourPointsNonHole[k]);
					plane->closestNeighbourPointsNonHole.erase(plane->closestNeighbourPointsNonHole.begin() + k);
					k--;
				}
			}
		}
	}
}

void relocateHoleNeighbours2(Plane* plane)
{
	double* nonHoleNeighboursCoords;
	hipMallocManaged(&nonHoleNeighboursCoords, plane->closestNeighbourPointsNonHole.size() * sizeof(double) * 3);
	bool* isInsideHole;
	hipMallocManaged(&isInsideHole, plane->closestNeighbourPointsNonHole.size() * sizeof(bool));
	
	for (size_t i = 0; i < plane->closestNeighbourPointsNonHole.size(); i++)
	{
		isInsideHole[i] = false;
		nonHoleNeighboursCoords[i * 3] = plane->closestNeighbourPointsNonHole[i].second.x;
		nonHoleNeighboursCoords[i * 3 + 1] = plane->closestNeighbourPointsNonHole[i].second.y;
		nonHoleNeighboursCoords[i * 3 + 2] = plane->closestNeighbourPointsNonHole[i].second.z;
	}
	
	for (size_t i = 0; i < plane->edges.size(); i++) {
		if (plane->edges[i]->isHole) {

			int blockSize = 256;
			int numBlocks = (plane->closestNeighbourPointsNonHole.size() + blockSize - 1) / blockSize;
			
			double* polygon;
			hipMallocManaged(&polygon, plane->edges[i]->pointsWithDir.size() * sizeof(double) * 3);

			for (size_t i = 0; i < plane->edges[i]->pointsWithDir.size(); i++) 
			{
				polygon[i * 3] = plane->edges[i]->pointsWithDir[i].first->projected2DPosition.x;
				polygon[i * 3 + 1] = plane->edges[i]->pointsWithDir[i].first->projected2DPosition.y;
				polygon[i * 3 + 2] = plane->edges[i]->pointsWithDir[i].first->projected2DPosition.z;
			}
			
			relocateNeighbours <<<numBlocks, blockSize>>> (nonHoleNeighboursCoords, isInsideHole, polygon, plane->closestNeighbourPointsNonHole.size(),
				plane->edges[i]->pointsWithDir.size(), plane->edges[i]->xBounds2D.first, plane->edges[i]->xBounds2D.second, plane->edges[i]->yBounds2D.first, 
				plane->edges[i]->yBounds2D.second, true, 0.01);

			hipDeviceSynchronize();
			hipFree(polygon);
			/*for (size_t k = 0; k < plane->closestNeighbourPointsNonHole.size(); k++) {
				if (isPointInsidePolygon(plane->edges[i]->getPoints(), plane->closestNeighbourPointsNonHole[k].second,
					plane->edges[i]->xBounds2D, plane->edges[i]->yBounds2D, true, 0.01) > 0) {
					plane->closestNeighbourPointsHole.push_back(plane->closestNeighbourPointsNonHole[k]);
					plane->closestNeighbourPointsNonHole.erase(plane->closestNeighbourPointsNonHole.begin() + k);
					k--;
				}
			}*/
		}
	}
	for (size_t i = 0; i < plane->closestNeighbourPointsNonHole.size(); i++)
	{
		if (isInsideHole[i]) 
		{
			plane->closestNeighbourPointsHole.push_back(plane->closestNeighbourPointsNonHole[i]);
			plane->closestNeighbourPointsNonHole.erase(plane->closestNeighbourPointsNonHole.begin() + i);
			i--;
		}
	}
	hipFree(nonHoleNeighboursCoords);
	hipFree(isInsideHole);
}

__device__
void changeBaseTo2D_CUDA(Edge_CUDA* edge, Vec3<double> normal, Vec3<double> x)
{
	edge->xBounds2DMin = 100000;
	edge->xBounds2DMax = -100000;
	edge->yBounds2DMin = 100000;
	edge->yBounds2DMax = -100000;
	x = Vec3<double>::normalize(x - normal * Vec3<double>::dot_product(x, normal));
	auto y = Vec3<double>::crossProduct(x, normal);
	for (size_t i = 0; i < edge->pointsSize; i++) {
		edge->pointsWithDir[i]->projected2DPosition = { Vec3<double>::dot_product(edge->pointsWithDir[i]->position, x),
			Vec3<double>::dot_product(edge->pointsWithDir[i]->position, y), 0 };
		auto pos2D = edge->pointsWithDir[i]->projected2DPosition;
		if (pos2D.x < edge->xBounds2DMin) edge->xBounds2DMin = pos2D.x;
		if (pos2D.x > edge->xBounds2DMax) edge->xBounds2DMax = pos2D.x;
		if (pos2D.y < edge->yBounds2DMin) edge->yBounds2DMin = pos2D.y;
		if (pos2D.y > edge->yBounds2DMax) edge->yBounds2DMax = pos2D.y;
	}
}

__device__
size_t isPointInsidePolygon_CUDA(Point_CUDA** polygon, size_t polygonSize, Vec3<double> point, double xBoundsMin, double xBoundsMax, double yBoundsMin, double yBoundsMax,
	bool checkOnEdge = false, double onEdgetreshold = 0.0000001)
{
	// 0 - outside
	// 1 - inside
	// 2 - onEdge
	size_t notInUse = 0;
	double notInUseRatio = 0;
	if (point.x > xBoundsMin && point.x < xBoundsMax &&
		point.y > yBoundsMin && point.y < yBoundsMax) {
		int rigthCounter = 0;
		for (size_t i = 0; i < polygonSize; i++) {
			auto p1 = polygon[i]->projected2DPosition;
			auto p2 = polygon[(i + 1) % polygonSize]->projected2DPosition;
			if ((p1.x < point.x && p2.x < point.x) || (p1.y > point.y && p2.y > point.y) || (p1.y < point.y && p2.y < point.y) || (p1.y == p2.y))
				continue;
			else if (point == p1)
				return false;
			else {
				auto intersection = intersectionOfLines(point, point + Vec3<double>({ 1,0,0 }), p1, p2, notInUse, notInUseRatio);
				if (abs(intersection.x - point.x) < onEdgetreshold && checkOnEdge)
					return 2;
				if (intersection.x <= point.x)
					continue;
				else {
					rigthCounter++;
					if (p2.y == point.y)
						i++;
				}
			}
		}
		return rigthCounter % 2 == 1;
	}
	return false;
}

__device__
bool areEdgesIntersect_CUDA(Plane_CUDA* savedPlane, Plane_CUDA* newPlane, Edge_CUDA* savedEdge, Edge_CUDA* newEdge)
{
	bool hasFoundIntersection = false;
	auto savedEdgePoints = savedEdge->getPoints();
	auto newEdgePoints = newEdge->getPoints();
	for (size_t j = 0; j < newEdge->pointsSize; j++) {
		if (isPointInsidePolygon_CUDA(savedEdgePoints, savedEdge->pointsSize, newEdge->pointsWithDir[j]->projected2DPosition, savedEdge->xBounds2DMin, savedEdge->xBounds2DMax,
			savedEdge->yBounds2DMin, savedEdge->yBounds2DMin, true)) {
			hasFoundIntersection = true;
			break;
		}
		for (size_t i = 0; i < savedEdge->pointsSize; i++) {
			if (isPointInsidePolygon_CUDA(newEdgePoints, newEdge->pointsSize, savedEdge->pointsWithDir[i]->projected2DPosition, newEdge->xBounds2DMin, newEdge->xBounds2DMax,
				newEdge->yBounds2DMin, newEdge->yBounds2DMin, true)) {
				hasFoundIntersection = true;
				break;
			}
			size_t onEdgeType = 0;
			double distanceRatio = 0;
			auto intersectionPos = intersectionOfLines(savedEdge->pointsWithDir[i]->projected2DPosition, savedEdge->pointsWithDir[(i + 1) %
				savedEdge->pointsSize]->projected2DPosition, newEdge->pointsWithDir[j]->projected2DPosition,
				newEdge->pointsWithDir[(j + 1) % newEdge->pointsSize]->projected2DPosition, onEdgeType, distanceRatio);
			if (onEdgeType > 0) {
				hasFoundIntersection = true;
				break;
			}
		}
		if (hasFoundIntersection)
			break;
	}
	return hasFoundIntersection;
}

__device__
bool arePlanesIntersecting_CUDA(Plane_CUDA* plane1, Plane_CUDA* plane2)
{
	double normalTreshold = 0.1;
	double distanceTreshold = 0.1;
	if ((plane1->normal - plane2->normal).length() < normalTreshold &&
		(abs(Vec3<double>::dot_product(plane1->normal, plane2->furthestNormalPoints[0] - plane1->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane1->normal, plane2->furthestNormalPoints[1] - plane1->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane2->normal, plane1->furthestNormalPoints[0] - plane2->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane2->normal, plane1->furthestNormalPoints[1] - plane2->planePointPos)) < distanceTreshold)) 
	{
		for (int k = 0; k < plane1->edgesSize; k++) {
			changeBaseTo2D_CUDA(plane1->edges[k], plane1->normal, plane1->pointDirections[0]);
		}
		for (int k = 0; k < plane2->edgesSize; k++) {
			changeBaseTo2D_CUDA(plane2->edges[k], plane1->normal, plane1->pointDirections[0]);
		}
		for (int k = 0; k < plane1->edgesSize; k++) {
			for (int l = 0; l < plane2->edgesSize; l++) {
				if (plane1->edges[k]->isHole == plane2->edges[l]->isHole && plane1->edges[k]->canIntersectWithEdge(plane2->edges[l])) {
					if (areEdgesIntersect_CUDA(plane2, plane1, plane2->edges[l], plane1->edges[k])) {
						return true;
					}
				}
			}
		}
		for (int k = 0; k < plane2->edgesSize; k++) {
			changeBaseTo2D_CUDA(plane2->edges[k], plane2->normal, plane2->pointDirections[0]);
		}
	}
	return false;
}

__global__ 
void findPlaneGroups(size_t* groupPlaneIndices, Plane_CUDA** allPlanes, size_t planeCount)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < (planeCount * planeCount))
	{
		size_t index1 = index / planeCount;
		size_t index2 = index % planeCount;
		if (index1 < index2 && groupPlaneIndices[index2] == planeCount && index1 != index2)
		{
			if (arePlanesIntersecting_CUDA(allPlanes[index1], allPlanes[index2]))
				groupPlaneIndices[index2] = index1;
		}
	}
}

void groupPlanes()
{
	size_t allSize = 0;
	for (size_t i = 0; i < planes.size(); i++) {
		allSize += planes[i].size();
	}
	allPlanes.reserve(allSize);
	for (size_t i = 0; i < planes.size(); i++) {
		allPlanes.insert(allPlanes.end(), planes[i].begin(), planes[i].end());
	}
	for (size_t i = 0; i < allPlanes.size(); i++) {
		if (allPlanes[i]->edges.size() == 0) {
			delete allPlanes[i];
			allPlanes.erase(allPlanes.begin() + i);
			i--;
		}
	}
	auto start = std::chrono::steady_clock::now();

	std::vector<Plane*> tempPlanes;
	bool foundNewConnectionAtAll = true;
	while (foundNewConnectionAtAll) {
		foundNewConnectionAtAll = false;
		while (allPlanes.size() > 0) {
			bool foundNewConnection = true;
			while (foundNewConnection) {
				foundNewConnection = false;
				for (size_t i = 1; i < allPlanes.size(); i++) {
					bool foundNewConnection = arePlanesIntersecting(allPlanes[0], allPlanes[i]);
					if (foundNewConnection) {
						for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
							allPlanes[0]->edges.push_back(allPlanes[i]->edges[j]);
						}
						allPlanes[0]->presentInFrames++;
						foundNewConnectionAtAll = true;
						delete allPlanes[i];
						allPlanes.erase(allPlanes.begin() + i);
						break;
					}
				}
			}
			tempPlanes.push_back(allPlanes[0]);
			allPlanes.erase(allPlanes.begin());
		}
		allPlanes = tempPlanes;
		tempPlanes.clear();
	}

	/*size_t* groupPlaneIndices;
	hipMallocManaged(&groupPlaneIndices, allPlanes.size() * sizeof(size_t));
	for (size_t i = 0; i < allPlanes.size(); i++) {
		groupPlaneIndices[i] = allPlanes.size();
	}

	Plane_CUDA** allPlanes_CUDA;
	hipMallocManaged(&allPlanes_CUDA, allPlanes.size() * sizeof(Plane_CUDA*));
	for (size_t i = 0; i < allPlanes.size(); i++) {
		Plane_CUDA* newPlane_CUDA;
		hipMallocManaged(&newPlane_CUDA, sizeof(Plane_CUDA*));
		allPlanes_CUDA[i] = newPlane_CUDA;
		newPlane_CUDA->copyFromPlane(allPlanes[i]);
	}

	int blockSize = 256;
	int numBlocks = (pow(allPlanes.size(), 2) + blockSize - 1) / blockSize;

	findPlaneGroups << <numBlocks, blockSize >> > (groupPlaneIndices, allPlanes_CUDA, allPlanes.size());
	hipDeviceSynchronize();


	for (size_t i = 0; i < allPlanes.size(); i++)
	{
		break;
		std::cout << groupPlaneIndices[i] << std::endl;
	}

	hipFree(groupPlaneIndices);
	hipFree(allPlanes_CUDA);*/

	auto end = std::chrono::steady_clock::now();
	std::cout << "Planes grouped --- Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;
	start = std::chrono::steady_clock::now();
	for (size_t i = 0; i < allPlanes.size(); i++) {
		if (allPlanes[i]->presentInFrames < 2) {
			delete allPlanes[i];
			allPlanes.erase(allPlanes.begin() + i);
			i--;
			continue;
		}
		size_t neighboursSizeNonHole = 0;
		size_t neighboursSizeHole = 0;
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			if (allPlanes[i]->edges[j]->isHole)
				neighboursSizeHole += allPlanes[i]->edges[j]->closestNeighbourPoints.size();
			else
				neighboursSizeNonHole += allPlanes[i]->edges[j]->closestNeighbourPoints.size();
		}
		allPlanes[i]->closestNeighbourPointsNonHole.reserve(neighboursSizeNonHole);
		allPlanes[i]->closestNeighbourPointsHole.reserve(neighboursSizeHole);
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			if (allPlanes[i]->edges[j]->isHole)
				allPlanes[i]->closestNeighbourPointsHole.insert(allPlanes[i]->closestNeighbourPointsHole.end(), allPlanes[i]->edges[j]->closestNeighbourPoints.begin(),
					allPlanes[i]->edges[j]->closestNeighbourPoints.end());
			else
				allPlanes[i]->closestNeighbourPointsNonHole.insert(allPlanes[i]->closestNeighbourPointsNonHole.end(), allPlanes[i]->edges[j]->closestNeighbourPoints.begin(),
					allPlanes[i]->edges[j]->closestNeighbourPoints.end());
		}
		relocateHoleNeighbours(allPlanes[i]);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "Relocated neighbours: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;
}

void filterNeighbours()
{
	auto start = std::chrono::steady_clock::now();
	for (size_t l = 0; l < 2; l++) {
		for (size_t i = 0; i < allPlanes.size(); i++) {
			for (size_t j = 0; j < (l == 0 ? allPlanes[i]->closestNeighbourPointsNonHole : allPlanes[i]->closestNeighbourPointsHole).size(); j++) {
				bool hasFoundEdge = false;
				for (size_t k = 0; k < allPlanes[i]->edges.size(); k++) {
					if (allPlanes[i]->edges[k]->isHole == (l != 0) && (isPointInsidePolygon(allPlanes[i]->edges[k]->getPoints(), (l == 0 ?
						allPlanes[i]->closestNeighbourPointsNonHole : allPlanes[i]->closestNeighbourPointsHole)[j].second,
						allPlanes[i]->edges[k]->xBounds2D, allPlanes[i]->edges[k]->yBounds2D, true, 0.0) == 0) == (l != 0)) 
					{
						hasFoundEdge = true;
						break;
					}
				}
				if (!hasFoundEdge) {
					(l == 0 ? allPlanes[i]->closestNeighbourPointsNonHole : allPlanes[i]->closestNeighbourPointsHole).erase((l == 0 ?
						allPlanes[i]->closestNeighbourPointsNonHole : allPlanes[i]->closestNeighbourPointsHole).begin() + j);
					j--;
				}
			}
		}
	}
	auto end = std::chrono::steady_clock::now();
	std::cout << "Neighbours filtered: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;
}

void fitPlanes()
{
	auto start = std::chrono::steady_clock::now();
	for (size_t j = 0; j < allPlanes.size(); j++) {
		std::vector<Edge*> tempCreatedEdges;
		auto holeNeighbours = allPlanes[j]->closestNeighbourPointsHole;
		auto nonHoleNeighbours = allPlanes[j]->closestNeighbourPointsNonHole;
		bool createdNewPolygon = true;
		bool foundNewIntersection = true;
		int counter = 0;
		while (foundNewIntersection) {
			foundNewIntersection = false;
			while (allPlanes[j]->edges.size() > 0) {
				for (size_t i = 1; i < allPlanes[j]->edges.size(); i++) {
					if (allPlanes[j]->edges[0]->isHole == allPlanes[j]->edges[i]->isHole) {
						std::vector<Edge*> innerCreatedEdges;
						/*if (false && dbg && i == 24 && counter == 13) {// currentFrame == 4 && savedEdge->isHole && counter == 3) {
							savedPoints.clear();
							addedPoints.clear();
							points.clear();
							changeBaseTo2D(createdEdges[0], { savedPlane->normal, savedPlane->pointDirections.first });
							changeBaseTo2D(createdEdges[i], { savedPlane->normal, savedPlane->pointDirections.first });
							for (size_t y = 0; y < createdEdges[0]->pointsWithDir.size(); y++) {
								createdEdges[0]->pointsWithDir[y].first->cornerId = currentCornerId;
								createdEdges[0]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
								savedPoints.push_back(createdEdges[0]->pointsWithDir[y].first);
								createdEdges[0]->pointsWithDir[y].first->position = createdEdges[0]->pointsWithDir[y].first->projected2DPosition;
								currentCornerIndex++;
							}
							currentCornerId++;
							currentCornerIndex = 0;
							for (size_t y = 0; y < createdEdges[i]->pointsWithDir.size(); y++) {
								createdEdges[i]->pointsWithDir[y].first->cornerId = currentCornerId;
								createdEdges[i]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
								savedPoints.push_back(createdEdges[i]->pointsWithDir[y].first);
								createdEdges[i]->pointsWithDir[y].first->position = createdEdges[i]->pointsWithDir[y].first->projected2DPosition;
								currentCornerIndex++;
							}

							for (size_t x = 0; x < createdEdges.size(); x++) {
								for (size_t y = 0; y < createdEdges[x]->closestNeighbourPoints.size(); y++) {
									savedPoints.push_back(new Point(createdEdges[x]->closestNeighbourPoints[y].first, 0, 0, nullptr));
								}
								currentCornerIndex = 0;
								for (size_t y = 0; y < createdEdges[x]->pointsWithDir.size(); y++) {
									createdEdges[x]->pointsWithDir[y].first->cornerId = 0;
									createdEdges[x]->pointsWithDir[y].first->cornerIndex = 0;
								}
								for (size_t y = 0; y < createdEdges[x]->pointsWithDir.size(); y++) {
									if (createdEdges[x]->pointsWithDir[y].first->isCorner && createdEdges[x]->pointsWithDir[y].first->cornerId == 0) {
										createdEdges[x]->pointsWithDir[y].first->cornerId = currentCornerId;
										createdEdges[x]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
										savedPoints.push_back(createdEdges[x]->pointsWithDir[y].first);
										currentCornerIndex++;
									}
									else if (createdEdges[x]->pointsWithDir[y].first->cornerId != 0) {
										std::cout << "asd";
									}
								}
								currentCornerId++;
							}
							writeData(4);
							std::cout << "asd";
						}*/
						createdNewPolygon = true;
						mergePolygons(allPlanes[j], allPlanes[j]->edges[0], allPlanes[j]->edges[i], allPlanes[j]->edges[0]->isHole ? holeNeighbours : nonHoleNeighbours, 
							innerCreatedEdges, createdNewPolygon);

						if (createdNewPolygon) {
							foundNewIntersection = true;
							for (size_t m = 0; m < innerCreatedEdges.size(); m++) {
								if (!isEdgeTooTigth(innerCreatedEdges[m]))
									tempCreatedEdges.push_back(innerCreatedEdges[m]);
							}
							allPlanes[j]->edges.erase(allPlanes[j]->edges.begin() + i);
							allPlanes[j]->edges.erase(allPlanes[j]->edges.begin());
							break;
						}
					}
					else
						createdNewPolygon = false;
				}
				counter++;
				if (!createdNewPolygon || allPlanes[j]->edges.size() == 1) {
					tempCreatedEdges.push_back(allPlanes[j]->edges[0]);
					allPlanes[j]->edges.erase(allPlanes[j]->edges.begin());
				}
			}
			allPlanes[j]->edges = tempCreatedEdges;
			tempCreatedEdges.clear();
		}
		std::cout << "Planes processed: " << allPlanes.size() << "/" << j + 1 << std::endl;
	}
	auto end = std::chrono::steady_clock::now();
	std::cout << "Planes fit: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;
}

void convexSegmentation()
{
	double notInUseRatio = 0;
	const std::pair<double, double> acceptAngle = { 181, 359 };
	size_t currentConvexId = 1;
	for (size_t i = 0; i < allPlanes.size(); i++) {
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			for (size_t k = 0; k < allPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				allPlanes[i]->edges[j]->pointsWithDir[k].first->convexId.clear();
				allPlanes[i]->edges[j]->pointsWithDir[k].first->convexIndex.clear();
			}
		}
		allPlanes[i]->convexFaces.clear();
	}
	for (size_t i = 0; i < allPlanes.size(); i++) {
		std::vector<Edge*> holeEdges;
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			if (allPlanes[i]->edges[j]->isHole && allPlanes[i]->edges[j]->isInUse) {
				holeEdges.push_back(allPlanes[i]->edges[j]);
			}
		}
		for (size_t x = 0; x < allPlanes[i]->edges.size(); x++) {
			if (!allPlanes[i]->edges[x]->isHole) {
				std::vector<Point*> remainingPoints;
				for (size_t k = 0; k < allPlanes[i]->edges[x]->pointsWithDir.size(); k++) {
					remainingPoints.push_back(allPlanes[i]->edges[x]->pointsWithDir[k].first);
				}
				if (!isClockwise(remainingPoints)) {
					for (size_t k = 0; k < remainingPoints.size(); k++) {
						remainingPoints.push_back(remainingPoints[remainingPoints.size() - 1 - k]);
						remainingPoints.erase(remainingPoints.begin() + remainingPoints.size() - 2 - k);
					}
				}
				while (remainingPoints.size() > 3) {
					std::vector<Point*> remainingPointsHelper(remainingPoints);
					std::vector<Point*> L = { remainingPointsHelper[0], remainingPointsHelper[1] };
					std::vector<Point*> remainingPointsHelperSave;
					std::vector<Point*> LSave;

					remainingPointsHelper.erase(remainingPointsHelper.begin(), remainingPointsHelper.begin() + 2);
					std::pair<double, double> xBounds = { std::min(L[0]->projected2DPosition.x, L[1]->projected2DPosition.x),
						std::max(L[0]->projected2DPosition.x, L[1]->projected2DPosition.x) };
					std::pair<double, double> yBounds = { std::min(L[0]->projected2DPosition.y, L[1]->projected2DPosition.y),
						std::max(L[0]->projected2DPosition.y, L[1]->projected2DPosition.y) };

					for (int j = 0; j < 2; j++) {
						bool isForward = j == 0;
						while (remainingPointsHelper.size() > 0) {
							auto newPoint = isForward ? remainingPointsHelper[0] : remainingPointsHelper[remainingPointsHelper.size() - 1];
							auto v1 = isForward ? L[L.size() - 1]->projected2DPosition - L[L.size() - 2]->projected2DPosition :
								(L[0]->projected2DPosition - L[1]->projected2DPosition);
							auto v2 = isForward ? newPoint->projected2DPosition - L[L.size() - 1]->projected2DPosition :
								(newPoint->projected2DPosition - L[0]->projected2DPosition);
							auto vecToBegin = isForward ? L[0]->projected2DPosition - newPoint->projected2DPosition :
								(L[L.size() - 1]->projected2DPosition - newPoint->projected2DPosition);
							auto vecAtBegin = isForward ? L[1]->projected2DPosition - L[0]->projected2DPosition :
								(L[L.size() - 2]->projected2DPosition - L[L.size() - 1]->projected2DPosition);
							auto temp = angleOfVectors(v1 * -1, v2, !isForward);
							auto temp2 = angleOfVectors(v2 * -1, vecToBegin, !isForward);
							auto temp3 = angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward);
							if ((angleOfVectors(v1 * -1, v2, !isForward) <= acceptAngle.first || angleOfVectors(v1 * -1, v2, !isForward) >=
								acceptAngle.second) &&
								(angleOfVectors(v2 * -1, vecToBegin, !isForward) <= acceptAngle.first || angleOfVectors(v2 * -1, vecToBegin, !isForward) >=
									acceptAngle.second) &&
								(angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward) <= acceptAngle.first || angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward) >=
									acceptAngle.second)) {
								L.insert(isForward ? L.end() : L.begin(), newPoint);
								remainingPointsHelper.erase(isForward ? remainingPointsHelper.begin() : remainingPointsHelper.end() - 1);
								if (newPoint->projected2DPosition.x < xBounds.first)
									xBounds.first = newPoint->projected2DPosition.x;
								if (newPoint->projected2DPosition.x > xBounds.second)
									xBounds.second = newPoint->projected2DPosition.x;
								if (newPoint->projected2DPosition.y < yBounds.first)
									yBounds.first = newPoint->projected2DPosition.y;
								if (newPoint->projected2DPosition.y > yBounds.second)
									yBounds.second = newPoint->projected2DPosition.y;
							}
							else {
								if (L.size() > 2) {
									bool containsCorner = true;
									while (containsCorner && L.size() > 2) {
										containsCorner = false;
										for (size_t k = 0; k < remainingPointsHelper.size(); k++) {
											if (isPointInsidePolygon(L, remainingPointsHelper[k]->projected2DPosition, xBounds, yBounds)) {
												containsCorner = true;
												break;
											}
										}
										if (containsCorner) {
											remainingPointsHelper.insert(isForward ? remainingPointsHelper.begin() : remainingPointsHelper.end(),
												isForward ? L[L.size() - 1] : L[0]);
											L.erase(isForward ? L.end() - 1 : L.begin());
										}
									}
									int absoluteClosestEdgeIndex = -1;
									size_t absoluteClosestPointIndex = 0;
									bool isNewPointFound = true;
									auto lastPointPos = L[L.size() - 1]->projected2DPosition;
									if (L.size() > 2) {
										while (isNewPointFound) {
											double minIntersectionDistance = 1000;
											int closestEdgeIndex = -1;
											size_t closestPointIndex = 0;
											auto LTemp = L;
											if (absoluteClosestEdgeIndex != -1)
												LTemp.insert(LTemp.begin(), holeEdges[absoluteClosestEdgeIndex]->pointsWithDir[absoluteClosestPointIndex].first);
											isNewPointFound = false;
											for (size_t l = 0; l < holeEdges.size(); l++) {
												std::vector<bool> isPointsInside;
												auto edgePoints = holeEdges[l];
												size_t holeType = 0;
												for (size_t m = 0; m < edgePoints->pointsWithDir.size(); m++) {
													bool isHolePointInside = isPointInsidePolygon(LTemp,
														edgePoints->pointsWithDir[m].first->projected2DPosition, xBounds, yBounds);
													isPointsInside.push_back(isHolePointInside);
													if (isHolePointInside && holeType == 0)
														holeType = 1;
													if (m > 0 && isPointsInside[m - 1] != isPointsInside[m])
														holeType = 2;
												}
												if (holeType > 0) {
													for (size_t m = 0; m < edgePoints->pointsWithDir.size(); m++) {
														if (holeType == 2 && isPointsInside[m] != isPointsInside[(m + 1) % isPointsInside.size()] &&
															LTemp[0] != edgePoints->pointsWithDir[m].first && LTemp[0] !=
															edgePoints->pointsWithDir[(m + 1) % isPointsInside.size()].first) {
															size_t onEdgeType = 0;
															auto intersection = intersectionOfLines(LTemp[0]->projected2DPosition, lastPointPos,
																edgePoints->pointsWithDir[m].first->projected2DPosition,
																edgePoints->pointsWithDir[(m + 1) % isPointsInside.size()].first->projected2DPosition,
																onEdgeType, notInUseRatio);
															if (onEdgeType > 0 && (intersection - lastPointPos).length() < minIntersectionDistance) {
																minIntersectionDistance = (intersection - lastPointPos).length();
																closestEdgeIndex = l;
																if (((edgePoints->pointsWithDir[m].first->projected2DPosition - lastPointPos).length() <
																	(edgePoints->pointsWithDir[(m + 1) % isPointsInside.size()].first->projected2DPosition
																		- lastPointPos).length() &&
																	isPointInsidePolygon(LTemp, edgePoints->pointsWithDir[m].first->projected2DPosition,
																		xBounds, yBounds)) || ((edgePoints->pointsWithDir[m].first->projected2DPosition
																			- lastPointPos).length() >= (edgePoints->pointsWithDir[(m + 1) %
																				isPointsInside.size()].first->projected2DPosition - lastPointPos).length() &&
																			!isPointInsidePolygon(LTemp, edgePoints->pointsWithDir[(m + 1) %
																				isPointsInside.size()].first->projected2DPosition,
																				xBounds, yBounds)))
																	closestPointIndex = m;
																else
																	closestPointIndex = ((m + 1) % isPointsInside.size());
															}
														}
														else if (holeType == 1 && absoluteClosestEdgeIndex == -1) {
															if ((edgePoints->pointsWithDir[m].first->projected2DPosition - lastPointPos).length() <
																minIntersectionDistance) {
																minIntersectionDistance = (edgePoints->pointsWithDir[m].first->projected2DPosition -
																	lastPointPos).length();
																closestEdgeIndex = l;
																closestPointIndex = m;
															}
														}
													}
												}
											}
											if (closestEdgeIndex != -1) {
												absoluteClosestEdgeIndex = closestEdgeIndex;
												absoluteClosestPointIndex = closestPointIndex;
												isNewPointFound = true;
											}
										}
										if (absoluteClosestEdgeIndex >= 0) {
											if (!isForward) {
												remainingPointsHelper = remainingPointsHelperSave;
												L = LSave;
											}
											remainingPointsHelper.insert(remainingPointsHelper.begin(), L[L.size() - 1]);
											for (size_t l = 0; l < holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size() + 1; l++) {
												remainingPointsHelper.insert(remainingPointsHelper.begin() + l,
													holeEdges[absoluteClosestEdgeIndex]->pointsWithDir[(absoluteClosestPointIndex + l) %
													holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size()].first);
											}
											for (size_t m = 0; m < remainingPoints.size(); m++) {
												if (remainingPoints[m] == L[L.size() - 1]) {
													remainingPoints.insert(remainingPoints.begin() + m + 1, L[L.size() - 1]);
													for (size_t l = 0; l < holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size() + 1; l++) {
														remainingPoints.insert(remainingPoints.begin() + m + 1 + l,
															holeEdges[absoluteClosestEdgeIndex]->pointsWithDir[(absoluteClosestPointIndex + l) %
															holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size()].first);
													}
													break;
												}
											}
											holeEdges.erase(holeEdges.begin() + absoluteClosestEdgeIndex);
											j = -1;
											break;
										}
									}
								}
								if (isForward) {
									LSave = L;
									remainingPointsHelperSave = remainingPointsHelper;
								}
								else {
									if (L.size() > 2) {
										remainingPointsHelper.insert(remainingPointsHelper.begin(), L[L.size() - 1]);
										remainingPointsHelper.insert(remainingPointsHelper.end(), L[0]);
									}
									else {
										L.clear();
										remainingPoints.push_back(remainingPoints[0]);
										remainingPoints.erase(remainingPoints.begin());
										remainingPointsHelper = remainingPoints;
									}
								}
								break;
							}
						}
					}
					if (remainingPointsHelper.size() == 0) //last convex polygon hole test
					{
						double minDistance = 1000;
						int closestEdgeIndex = -1;
						size_t closestPointIndex = 0;
						for (size_t l = 0; l < holeEdges.size(); l++) {
							auto edgePoints = holeEdges[l];
							for (size_t m = 0; m < edgePoints->pointsWithDir.size(); m++) {
								if (isPointInsidePolygon(L, edgePoints->pointsWithDir[m].first->projected2DPosition, xBounds, yBounds) && minDistance >
									(L[0]->projected2DPosition - edgePoints->pointsWithDir[m].first->projected2DPosition).length()) {
									minDistance = (L[0]->projected2DPosition - edgePoints->pointsWithDir[m].first->projected2DPosition).length();
									closestEdgeIndex = l;
									closestPointIndex = m;
								}
							}
						}
						if (closestEdgeIndex >= 0) {
							remainingPointsHelper.insert(remainingPointsHelper.begin(), L[0]);
							for (size_t l = 0; l < holeEdges[closestEdgeIndex]->pointsWithDir.size() + 1; l++) {
								remainingPointsHelper.insert(remainingPointsHelper.begin() + l + 1,
									holeEdges[closestEdgeIndex]->pointsWithDir[(closestPointIndex + l) % holeEdges[closestEdgeIndex]->pointsWithDir.size()].first);
							}
							for (size_t l = 0; l < L.size(); l++) {
								remainingPointsHelper.insert(remainingPointsHelper.end(), L[l]);
							}
							L.clear();
							holeEdges.erase(holeEdges.begin() + closestEdgeIndex);
							remainingPoints = remainingPointsHelper;
						}
					}
					if (L.size() > 0) {
						std::vector<Point*> convexFace;
						for (size_t j = 0; j < L.size(); j++) {
							L[j]->convexId.push_back(currentConvexId);
							L[j]->convexIndex.push_back(j);
							convexFace.push_back(L[j]);
						}
						allPlanes[i]->convexFaces.push_back(convexFace);
						/*if (currentFrame == 36 && i == 7 && allPlanes[i]->convexFaces.size() == 81)
						{
							if (false) {
								savedPoints.clear();
								addedPoints.clear();
								points.clear();
								//changeBaseTo2D(createdEdges[0], { savedPlane->normal, savedPlane->pointDirections.first });
								//changeBaseTo2D(createdEdges[i], { savedPlane->normal, savedPlane->pointDirections.first });
								for (size_t y = 0; y < allPlanes[i]->edges.size(); y++) {
									currentCornerIndex = 0;
									for (size_t j = 0; j < allPlanes[i]->edges[y]->pointsWithDir.size(); j++) {
										allPlanes[i]->edges[y]->pointsWithDir[j].first->cornerId = currentCornerId;
										allPlanes[i]->edges[y]->pointsWithDir[j].first->cornerIndex = currentCornerIndex;
										savedPoints.push_back(allPlanes[i]->edges[y]->pointsWithDir[j].first);
										allPlanes[i]->edges[y]->pointsWithDir[j].first->position = allPlanes[i]->edges[y]->pointsWithDir[j].first->projected2DPosition;
										currentCornerIndex++;
									}
									currentCornerId++;
								}
								writeData(4);
								std::cout << "asd";
							}
							//return;
						}*/
						remainingPoints = remainingPointsHelper;
						currentConvexId++;
					}
				}
				if (remainingPoints.size() == 3) {
					std::vector<Point*> convexFace;
					for (size_t j = 0; j < remainingPoints.size(); j++) {
						remainingPoints[j]->convexId.push_back(currentConvexId);
						remainingPoints[j]->convexIndex.push_back(j);
						convexFace.push_back(remainingPoints[j]);
					}
					allPlanes[i]->convexFaces.push_back(convexFace);
					currentConvexId++;
				}
			}
		}
	}
}

void fitPointsToPlane()
{
	for (size_t i = 0; i < allPlanes.size(); i++) {
		auto normal = allPlanes[i]->normal;
		auto planePointPos = allPlanes[i]->planePointPos;
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			for (size_t k = 0; k < allPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				double dist = Vec3<double>::dot_product(normal, allPlanes[i]->edges[j]->pointsWithDir[k].first->position - planePointPos);
				allPlanes[i]->edges[j]->pointsWithDir[k].first->position = allPlanes[i]->edges[j]->pointsWithDir[k].first->position - normal * dist;
			}
		}
	}
}

void exportObjects(size_t pointCloudIndex)
{
	size_t objCounter = 0;
	size_t currentCornerId = 1;
	for (size_t i = 0; i < allPlanes.size(); i++) {
		std::vector<Point*> corners;
		size_t currentCornerIndex = 0;
		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			for (size_t k = 0; k < allPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (allPlanes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
					allPlanes[i]->edges[j]->pointsWithDir[k].first->cornerId = currentCornerId;
					allPlanes[i]->edges[j]->pointsWithDir[k].first->cornerIndex = currentCornerIndex;
					corners.push_back(allPlanes[i]->edges[j]->pointsWithDir[k].first);
					currentCornerIndex++;
				}
			}
			currentCornerId++;
		}
		if (corners.size() < 3 || allPlanes[i]->convexFaces.size() == 0)
			continue;
		std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models_test/processed_obj_"
			+ std::to_string(objCounter) + ".obj");
		MyFile << "o Mesh" << std::endl;
		for (size_t k = 0; k < corners.size(); k++) {
			MyFile << "v " << std::to_string(-corners[k]->position.x) << " " << std::to_string(corners[k]->position.y)
				<< " " << std::to_string(corners[k]->position.z) << std::endl;
		}

		for (size_t j = 0; j < allPlanes[i]->convexFaces.size(); j++) {
			MyFile << "f ";
			for (size_t k = 0; k < allPlanes[i]->convexFaces[j].size(); k++) {
				MyFile << allPlanes[i]->convexFaces[j][k]->cornerIndex + 1 << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (int k = allPlanes[i]->convexFaces[j].size() - 1; k >= 0; k--) {
				MyFile << allPlanes[i]->convexFaces[j][k]->cornerIndex + 1 << " ";
			}
			MyFile << std::endl;
		}

		for (size_t j = 0; j < allPlanes[i]->edges.size(); j++) {
			int indexShift = -1;
			for (size_t k = 0; k < allPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (allPlanes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
					if (indexShift == -1)
						indexShift = allPlanes[i]->edges[j]->pointsWithDir[k].first->cornerIndex;
					allPlanes[i]->edges[j]->pointsWithDir[k].first->cornerIndex -= indexShift;
				}
			}
		}
		MyFile << std::endl;
		MyFile.close();
		objCounter++;
	}
}

void processData(size_t pointCloudIndex)
{
	auto start = std::chrono::steady_clock::now();
	size_t endIndex = (pointCloudTestIndex == -1 ? std::max<size_t>(1, pointCloudCount) : 1);
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) 
	{
		groundSegmentation(currentFrame);
	}
	auto end = std::chrono::steady_clock::now();
	std::cout << "groundSegmentation: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		egoCarSegmentation(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "egoCarSegmentation: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		findPlanes(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "findPlanes: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;


	std::cout << "dbgCounter: Elapsed time in seconds : "
		<< dbgTimeCounter
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		findEdgePoints(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "findEdgePoints: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		findCorners(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "findCorners: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		findPlaneConnections(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "findPlaneConnections: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		connectPlanes(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "connectPlanes: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		createCorners(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "createCorners: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		filterEdgePoints(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "filterEdgePoints: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		extract2DPolygon(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "extract2DPolygon: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		deleteSelfIntersections(planes[currentFrame]);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "deleteSelfIntersections: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;

	start = std::chrono::steady_clock::now();
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		deleteTigthEdges(currentFrame);
	}
	end = std::chrono::steady_clock::now();
	std::cout << "deleteTigthEdges: Elapsed time in seconds : "
		<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
		<< " sec" << std::endl;
	


	/*groundSegmentation(pointCloudIndex);
	egoCarSegmentation(pointCloudIndex);
	findPlanes(pointCloudIndex);
	findEdgePoints(pointCloudIndex);
	findCorners(pointCloudIndex);
	findPlaneConnections(pointCloudIndex);
	connectPlanes(pointCloudIndex);
	createCorners(pointCloudIndex);
	filterEdgePoints(pointCloudIndex);
	extract2DPolygon(pointCloudIndex);
	deleteSelfIntersections(planes[pointCloudIndex]);
	deleteTigthEdges(pointCloudIndex);*/
}

void connectData()
{
	groupPlanes();
	//filterNeighbours();
	fitPlanes();
	filterFittedPlanes();
	checkIfHolesAreContained();
	deleteSelfIntersections(allPlanes);
	convexSegmentation();
	fitPointsToPlane();
	exportObjects(currentFrame);
}

int main()
{
	size_t endIndex = (pointCloudTestIndex == -1 ? std::max<size_t>(1, pointCloudCount) : 1);
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		auto start = std::chrono::steady_clock::now();
		readData(currentFrame);
		auto end = std::chrono::steady_clock::now();
		std::cout << "Read " + std::to_string(currentFrame) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
	}
	processData(currentFrame);
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		break;
		auto start = std::chrono::steady_clock::now();
		processData(currentFrame);
		auto end = std::chrono::steady_clock::now();
		std::cout << "It. " + std::to_string(currentFrame) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
	}
	connectData();
	return 0;
}