#include "hip/hip_runtime.h"
#include "kernel.h"

void processData()
{
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	findEdgePoints();
	findCorners();
	findPlaneConnections();
	connectPlanes();
	createCorners();
	filterEdgePoints();
	extract2DPolygon();
	deleteSelfIntersections(planes);
	deleteTigthEdges();
	fitPlanes();
	deleteSelfIntersections(savedPlanes);
	saveSavedPoints();
}

void nonSequential()
{
	double avarageComputeTime = 0;
	size_t endIndex = (pointCloudTestIndex == -1 ? std::max<size_t>(1, pointCloudCount) : 1);
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
		currentFrame < endIndex; currentFrame++) {
		auto start = std::chrono::steady_clock::now();
		readData(currentFrame);
		auto end = std::chrono::steady_clock::now();
		std::cout << "Read " + std::to_string(currentFrame) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
		start = std::chrono::steady_clock::now();
		processData();
		end = std::chrono::steady_clock::now();
		std::cout << "It. " + std::to_string(currentFrame) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
		avarageComputeTime += (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000;
		if (currentFrame == endIndex - 1) {
			start = std::chrono::steady_clock::now();
			convexSegmentation();
			fitPointsToPlane();
			exportObjects(currentFrame);
			writeData(currentFrame);
			end = std::chrono::steady_clock::now();
			std::cout << "Write " + std::to_string(currentFrame) + " Elapsed time in seconds : "
				<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
				<< " sec" << std::endl;
		}
		//clearMemory();
	}
	avarageComputeTime /= endIndex - (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0);
	std::cout << "Avg computation time: " << avarageComputeTime << std::endl;
}