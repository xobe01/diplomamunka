#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>
#include "kernel.h"

struct Point
{
	size_t horizontalIndex;
	size_t verticalIndex;
	Vec3<double> position;
	Plane* plane{ 0 };
	size_t cornerId{ 0 };
	size_t outlineId{ 0 };
	size_t cornerIndex{ 0 };
	Vec3<double> normal = { 0, 0, 0 };
	bool isMarked{ false };
	bool isMarked2{ false };
	std::vector<Point*> neighbourPlaneNeighbours;
	bool isCorner;
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, Plane* _plane) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), plane(_plane)
	{
		isCorner = false;
	};
};

struct Plane {
	std::vector<Point*> points;
	std::pair<double, double> horizontalBounds;
	std::pair<double, double> verticalBounds;
	std::vector<std::vector<Point*>> edges;
	Vec3<double> planePointPos;
	Vec3<double> normal;
	std::pair<Vec3<double>, Vec3<double>> pointDirections;
	size_t id;
};

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Plane*> planes;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPlaneId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData()
{
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}		
	}
    MyReadFile.close();
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i])
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
			';' << (points[i]->plane ? points[i]->plane->id : 0) << ';' << points[i]->outlineId << ';' << points[i]->cornerId
			<< ';' << points[i]->cornerIndex << std::endl;
	}
}

void writeData()
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	MyFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

bool isSpike(Point* p)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;

	//fel-le
	bool isUpNotNeightbour = y == 0 || !points[getOffset(x, y - 1)] || points[getOffset(x, y - 1)]->plane != p->plane;
	bool isDownNotNeightbour = y == verticalCount - 1 || !points[getOffset(x, y + 1)] || points[getOffset(x, y + 1)]->plane != p->plane;

	//jobbra-balra
	bool isLeftNotNeightbour = !points[getOffset(x - 1, y)] || points[getOffset(x - 1, y)]->plane != p->plane;
	bool isRighttNotNeightbour = !points[getOffset(x + 1, y)] || points[getOffset(x + 1, y)]->plane != p->plane;
	if (((isLeftNotNeightbour && isRighttNotNeightbour) || (isUpNotNeightbour && isDownNotNeightbour))) {
		return true;
	}
	return false;
}

bool checkIfBridge(Point* p)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && isSpike(neighbourPoints[j]))
			neighbourCount++;
	}
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || points[getOffset(x - 1, y - 1)]->plane != p->plane)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || points[getOffset(x + 1, y + 1)]->plane != p->plane))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || points[getOffset(x + 1, y - 1)]->plane != p->plane)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || points[getOffset(x - 1, y + 1)]->plane != p->plane))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

void checkForGaps(/*out*/ std::vector<Point*>& chosenPoints)
{  
	for (auto p : chosenPoints) p->isMarked = true;
	while (isThereBridge(chosenPoints)) {}
	std::vector<Point*> nonProcessedPoints(chosenPoints);
	std::vector<Point*> nextStepPoints;
	std::vector<Point*> bestPartitionPoints;
	while (bestPartitionPoints.size() < nonProcessedPoints.size()) {
		std::vector<Point*> currentPartitionPoints;
		currentPartitionPoints.push_back(nonProcessedPoints[0]);
		nextStepPoints.push_back(nonProcessedPoints[0]);
		nonProcessedPoints[0]->isMarked = false;
		while (nextStepPoints.size() > 0) {
			std::vector<Point*> tempNextStepPoints;
			for (size_t i = 0; i < nextStepPoints.size(); i++) {
				size_t x = nextStepPoints[i]->horizontalIndex;
				size_t y = nextStepPoints[i]->verticalIndex;
				Point* neighbourPoint = points[getOffset(x, y - 1)];
				if (y > 0 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x, y + 1)];
				if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x - 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x + 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
			}
			nextStepPoints = tempNextStepPoints;
		}
		if (currentPartitionPoints.size() > bestPartitionPoints.size()) bestPartitionPoints = currentPartitionPoints;
		std::vector<Point*> tempNonProcessedPoints;
		for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
			if (nonProcessedPoints[i]->isMarked) tempNonProcessedPoints.push_back(nonProcessedPoints[i]);
		}
		nonProcessedPoints = tempNonProcessedPoints;
	}
	for (size_t i = 0; i < chosenPoints.size(); i++) {
		chosenPoints[i]->isMarked = false;
	}
	chosenPoints = bestPartitionPoints;
}

#include <chrono>

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(center->position - p1->position, center->position - p2->position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x - 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x + 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

void choosePoints(const Vec3<Point*> planePoints, double acceptTreshold, Vec3<double> normal, /*out*/ Plane* plane)
{
	if (normal == Vec3<double>{0, 0, 0}) {
		normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.x->position -
			planePoints.y->position, planePoints.z->position - planePoints.y->position));
	}
	Vec3<double> horizontalDirection = { 0,0,0 };
	Vec3<double> verticalDirection = { 0,0,0 };
	Point* neighbours[2] = { planePoints.y, planePoints.z };
	for each (auto neighbour in neighbours) {
		switch (areNeighbours(planePoints.x, neighbour)) {
		case 1:
		horizontalDirection = neighbour->position - planePoints.x->position;
		break;
		case 2:
		horizontalDirection = planePoints.x->position - neighbour->position;
		break;
		case 3:
		verticalDirection = neighbour->position - planePoints.x->position;
		break;
		case 4:
		verticalDirection = planePoints.x->position - neighbour->position;
		break;
		default:
		break;
		}
	}
	plane->pointDirections = {horizontalDirection, verticalDirection };
	plane->normal = normal;
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(normal, neighbourPoints[j]->position - planePoints.x ->position));
					if (dist <= acceptTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void calculateBounds(Plane& plane)
{
	plane.horizontalBounds = std::make_pair(plane.points[0]->horizontalIndex, plane.points[0]->horizontalIndex);
	plane.verticalBounds = std::make_pair(plane.points[0]->verticalIndex, plane.points[0]->verticalIndex);
	for (size_t i = 1; i < plane.points.size(); i++) 
	{
		if (plane.points[i]->horizontalIndex > plane.horizontalBounds.second)
			plane.horizontalBounds.second = plane.points[i]->horizontalIndex;
		else if (plane.points[i]->horizontalIndex < plane.horizontalBounds.first)
			plane.horizontalBounds.first = plane.points[i]->horizontalIndex;
		if (plane.points[i]->verticalIndex > plane.verticalBounds.second)
			plane.verticalBounds.second = plane.points[i]->verticalIndex;
		else if (plane.points[i]->verticalIndex < plane.verticalBounds.first)
			plane.verticalBounds.first = plane.points[i]->verticalIndex;
	}
}

void findPlanes()
{
	auto start = std::chrono::steady_clock::now();
	size_t minPointCount = 10;
	size_t counter = 1;
	double planeDistanceTreshold = 0.1;
	double normalTreshold = 0.1;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = true;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = true;
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			nextStepPoints.push_back(points[j]);
			calculateNormal(points[j]);
			while (nextStepPoints.size() > 0) {
				Plane* plane = new Plane();
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x - 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x + 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k < verticalCount - 1 || y < 3) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								if (neighbourPoints[k]->normal.length() == 0) {
									calculateNormal(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if ((normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {

								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, planeDistanceTreshold, {0,0,0}, 
									plane);
								break;
							}
						}
					}
				}
				if (plane->points.size() > 0)
				{
					planes.push_back(plane);
					currentPlaneId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		int originalSize = planes[i]->points.size();
		while (isThereBridge(planes[i]->points)) {}		
		if (originalSize != planes[i]->points.size() && planes[i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[i]->points[0], nullptr, nullptr }, planeDistanceTreshold, planes[i]->normal,
					plane);
				if (plane->points.size() < planes[i]->points.size()) {
					for (size_t j = 0; j < planes[i]->points.size(); j++) {
						if (planes[i]->points[j]->plane != planes[i]) {
							planes[i]->points.erase(planes[i]->points.begin() + j);
							j--;
						}
					}
					planes.push_back(plane);
					currentPlaneId++;
				}
				else 
				{
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[i];
					}
					delete plane;
					break;
				}
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		calculateBounds(planes[i]);	
	}/**/
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec" << std::endl;
}

void findNextPoint(Point* startPoint, bool isOuterEdge, bool isPreviousSpike, bool wasThereNonSpike, /*out*/ std::vector<Point*>& currentEdge)
{
	size_t direction = isOuterEdge ? 0 : 1;
	Point* currentPoint = nullptr;
	while (currentPoint != startPoint) 
	{
		if (!currentPoint)
			currentPoint = startPoint;
		Point* neighbourPoint = nullptr;
		size_t x = currentPoint->horizontalIndex;
		size_t y = currentPoint->verticalIndex;
		if (currentPoint->isMarked) {
			currentEdge.push_back(currentPoint);
			for (size_t i = 0; i < 4; i++) {
				switch (direction) {
				case 0: //to right
				neighbourPoint = points[getOffset(x + 1, y)];
				break;
				case 1: //to down
				neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
				break;
				case 2: //to left
				neighbourPoint = points[getOffset(x - 1, y)];
				break;
				case 3: //to up
				neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
				break;
				default:
				break;
				}
				if (neighbourPoint && neighbourPoint->plane != startPoint->plane && neighbourPoint->plane != nullptr) {
					currentPoint->neighbourPlaneNeighbours.push_back(neighbourPoint);
				}
				direction += direction == 3 ? -3 : 1;
			}
		}
		size_t initialDirection = direction;
		for (size_t i = 0; i < 4; i++) {
			switch (direction) {
			case 0: //to right
			neighbourPoint = points[getOffset(x + 1, y)];
			break;
			case 1: //to down
			neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
			break;
			case 2: //to left
			neighbourPoint = points[getOffset(x - 1, y)];
			break;
			case 3: //to up
			neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
			break;
			default:
			break;
			}
			if (neighbourPoint == startPoint) {
				currentPoint->isMarked = false;
				currentPoint = startPoint;
				break;
			}
			if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr) {
				if (wasThereNonSpike && currentEdge.size() > 1 && isPreviousSpike && !isSpike(currentPoint) && currentPoint->isMarked) {
					currentEdge.pop_back();
					currentPoint = currentEdge[currentEdge.size() - 1];
					direction = (initialDirection + 2) % 4;
					break;
				}
				else {
					if (!wasThereNonSpike && !isSpike(currentPoint)) {
						startPoint = currentPoint;
						wasThereNonSpike = true;
					}
					currentPoint->isMarked = false;
					currentPoint = neighbourPoint;
					direction = (direction + 3) % 4;
					break;
				}
			}
			direction += direction == 3 ? -3 : 1;
		}
	}
}

bool isPoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

bool isStraightPoint(size_t pointIndex, std::vector<Point*>& edge, bool& isPreviousConcave)
{
	bool previousConcaveStore = isPreviousConcave;
	size_t neighbourCount = 0;
	Point* point = edge[pointIndex];
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Plane* plane = point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i < 3) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane && (neighbourPoints[i]->outlineId == 0 || neighbourPoints[i]->outlineId == point->outlineId)) {
			neighbourCount++;
			isNeighbour[i] = true;
		}
	}
	isPreviousConcave = neighbourCount == 4;
	if (neighbourCount == 3 || (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3])))) {
		return true;
	}
	if (neighbourCount == 4 && (pointIndex == 0 ? edge[edge.size() - 1] : edge[pointIndex - 1])->isCorner)
		return true;
	if (previousConcaveStore) {
		edge[pointIndex - 1] -> isCorner = false;
	}
	return false;
}

void findPoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> PointsInPlane;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) if(isPoint(planes[i]->points[j])) PointsInPlane.push_back(planes[i]->points[j]);
		bool isFirstEdge = true;
		while (PointsInPlane.size() > 0) 
		{
			std::vector<Point*> tempPointsInPlane;
			Point* startPoint = PointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < PointsInPlane.size(); j++) {
				if ((PointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - PointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| PointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = PointsInPlane[j]->horizontalIndex;
					minVerticalCoord = PointsInPlane[j]->verticalIndex;
					startPoint = PointsInPlane[j];
				}
				else if (PointsInPlane[j]->horizontalIndex == minHorizontalCoord && PointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = PointsInPlane[j]->verticalIndex;
					startPoint = PointsInPlane[j];
				}
			}
			std::vector<Point*> currentEdge;
			findNextPoint(startPoint, isFirstEdge, false, false, currentEdge);
			if (!isFirstEdge) {
				currentEdge.insert(currentEdge.begin(), currentEdge[currentEdge.size() - 1]);
				currentEdge.pop_back();
			}
			for (size_t k = 0; k < currentEdge.size(); k++) {
				currentEdge[k]->outlineId = currentOutlineId;
			}
			for (size_t j = 0; j < PointsInPlane.size(); j++) {
				if (PointsInPlane[j]->isMarked) tempPointsInPlane.push_back(PointsInPlane[j]);
			}
			PointsInPlane = tempPointsInPlane;
			currentOutlineId++;
			planes[i]->edges.push_back(currentEdge);
			isFirstEdge = false;
		}

		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = false;
	}
}

bool arePlanesNeighbours(Plane p1, Plane p2, std::pair<int, int>& horizontalCommonBounds, std::pair<int, int>& verticalCommonBounds)
{
	if (p1.horizontalBounds.first > p1.horizontalBounds.second)
		p1.horizontalBounds.second += horizontalCount;
	double p1HorizontalSize = p1.horizontalBounds.second - p1.horizontalBounds.first;
	double p1VerticalSize = p1.verticalBounds.second - p1.verticalBounds.first;
	std::pair<int, int> p1Center = {(int)(p1.horizontalBounds.first + p1HorizontalSize / 2) % horizontalCount, 
		p1.verticalBounds.first + p1VerticalSize / 2 };

	if (p2.horizontalBounds.first > p2.horizontalBounds.second)
		p2.horizontalBounds.second += horizontalCount;
	double p2HorizontalSize = p2.horizontalBounds.second - p2.horizontalBounds.first;
	double p2VerticalSize = p2.verticalBounds.second - p2.verticalBounds.first;
	std::pair<int, int> p2Center = { (int)(p2.horizontalBounds.first + p2HorizontalSize / 2) % horizontalCount,
		p2.verticalBounds.first + p2VerticalSize / 2 };

	if (abs(p1Center.first - p2Center.first) > horizontalCount / 2) 
	{
		if (p1Center.first > p2Center.first)
			p2Center.first += horizontalCount;
		if (p1Center.first < p2Center.first)
			p1Center.first += horizontalCount;
	}
	if (abs(p1Center.first - p2Center.first) <= (p1HorizontalSize + p2HorizontalSize) / 2 + 1 &&
		abs(p1Center.second - p2Center.second) <= (p1VerticalSize + p2VerticalSize) / 2 + 1) 
	{
		horizontalCommonBounds = {abs(p1.horizontalBounds.first - p2.horizontalBounds.first) > horizontalCount / 2
			? std::min(p1.horizontalBounds.first, p2.horizontalBounds.first) : std::max(p1.horizontalBounds.first, p2.horizontalBounds.first),
			abs(p1.horizontalBounds.second - p2.horizontalBounds.second) > horizontalCount / 2
			? std::max(p1.horizontalBounds.second, p2.horizontalBounds.second) : std::min(p1.horizontalBounds.second, p2.horizontalBounds.second)};
		if (horizontalCommonBounds.second < horizontalCommonBounds.first)
			horizontalCommonBounds = { horizontalCommonBounds.second, horizontalCommonBounds.first };

		verticalCommonBounds = { std::max(p1.verticalBounds.first, p2.verticalBounds.first),
			std::min(p1.verticalBounds.second, p2.verticalBounds.second) };
		if (verticalCommonBounds.second < verticalCommonBounds.first)
			verticalCommonBounds = { verticalCommonBounds.second, verticalCommonBounds.first };
		return true;
	}
	return false;
}

const double newPointAcceptTreshold = 0.95;
const double inf = 1000000;

Point* createNewPoint(Vec3<double> newPointPos, Point* point, size_t addedCount)
{
	Point* newPoint = new Point(newPointPos, point->horizontalIndex, verticalCount, point->plane);
	addedPoints.push_back(newPoint);
	newPoint->isCorner = true;
	newPoint->outlineId = point->outlineId;
	for (size_t j = 0; j < point->plane->edges.size(); j++) {
		if (point->plane->edges[j][0]->outlineId == point->outlineId) {
			for (size_t k = 0; k < point->plane->edges[j].size(); k++) {
				if (point->plane->edges[j][k] == point) {
					point->plane->edges[j].insert(point->plane->edges[j].begin() + k + 1 + addedCount, newPoint);
					break;
				}
			}
			break;
		}
	}
	return newPoint;
}

bool addNewPoint(Point* point, Point*& neighbour, Plane* plane, size_t addedCount)
{
	if (neighbour->verticalIndex == verticalCount) { //created by other plane
		for (size_t j = 0; j < point->plane->edges.size(); j++) {
			if (point->plane->edges[j][0]->outlineId == point->outlineId) {
				for (size_t k = 0; k < point->plane->edges[j].size(); k++) {
					if (point->plane->edges[j][k] == point) {
						for (size_t l = 0; l < 4; l++) 
						{
							if (point->plane->edges[j][k + l + 1] == neighbour) {
								point->plane->edges[j].insert(point->plane->edges[j].begin() + k + 1 + addedCount, neighbour);
								point->plane->edges[j].erase(point->plane->edges[j].begin() + k + l + 2);
								break;
							}
						}
						break;
					}
				}
				break;
			}
		}
		return true;
	}
	Vec3<double> dir = { 0,0,0 };
	switch (areNeighbours(point, neighbour)) {
	case 1:
	dir = point->plane->pointDirections.first;
	break;
	case 2:
	dir = point->plane->pointDirections.first * -1;
	break;
	case 3:
	dir = point->plane->pointDirections.second;
	break;
	case 4:
	dir = point->plane->pointDirections.second * -1;
	break;
	default:
	break;
	}
	Vec3<double> newPointPos = point->position - dir * Vec3<double>::dot_product(point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf || isnan(newPointPos.x) || isnan(newPointPos.y) || isnan(newPointPos.z))
		return false;
	Vec3<double> dirToNew = newPointPos - point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold)
		return false;
	Vec3<double> neighbourDir = { 0,0,0 };
	switch (areNeighbours(neighbour, point)) {
	case 1:
	neighbourDir = plane->pointDirections.first;
	break;
	case 2:
	neighbourDir = plane->pointDirections.first * -1;
	break;
	case 3:
	neighbourDir = plane->pointDirections.second;
	break;
	case 4:
	neighbourDir = plane->pointDirections.second * -1;
	break;
	default:
	break;
	}	
	Vec3<double> neighbourNewPointPos = neighbour->position - neighbourDir * Vec3<double>::dot_product(neighbour->position
		- point->plane->planePointPos, point->plane->normal) / Vec3<double>::dot_product(neighbourDir, point->plane->normal);
	if (abs(neighbourNewPointPos.x) > inf || abs(neighbourNewPointPos.y) > inf || abs(neighbourNewPointPos.z) > inf || isnan(neighbourNewPointPos.x) ||
		isnan(neighbourNewPointPos.y) || isnan(neighbourNewPointPos.z))
		return false;
	Vec3<double> dirToNewNeighbour = neighbourNewPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(neighbourDir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold) 
		return false;
	auto newPos = (newPointPos + neighbourNewPointPos) / 2;
	for (size_t i = 0; i < neighbour->neighbourPlaneNeighbours.size(); i++) 
	{
		if (neighbour->neighbourPlaneNeighbours[i] == point) 		
		{
			auto newPoint = createNewPoint(newPointPos, neighbour, 0);
			neighbour->neighbourPlaneNeighbours[i] = newPoint;
			break;
		}
	}
	std::cout << point->plane->edges[0].size() << std::endl;
	createNewPoint(newPointPos, point, addedCount);
	std::cout << point->plane->edges[0].size() << std::endl;
	std::cout << newPointPos.x << std::endl;
	return true;
}

void connectPlanes()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		for (size_t j = 0; j < planes[i]->edges.size(); j++) 
		{
			bool wasFirstPointGenerated = false;
			bool wasPreviousSelected = true;
			Point* previousPoint = nullptr;
			Point* previousNeighbourPoint = nullptr;
			size_t previousIndex = 0;
			size_t previousAddedCount = 0;
			size_t previousId = 0;
			std::vector<std::pair<size_t, size_t>> cornersToDelete;
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {
				if (planes[i]->edges[j][k]->verticalIndex == verticalCount)
					continue;
				size_t addedCount = 0;
				for (auto neighbour : planes[i]->edges[j][k]->neighbourPlaneNeighbours) 
				{
					if (!previousNeighbourPoint || neighbour->plane->id != previousId) 
					{
						if (!wasPreviousSelected)
						{
							auto newPoint = addNewPoint(previousPoint, previousNeighbourPoint, previousNeighbourPoint->plane, previousAddedCount);
							if (newPoint) {
								cornersToDelete.push_back({ k,  previousNeighbourPoint->plane->id});
								k++;
							}
						}
						auto newPoint = addNewPoint(planes[i]->edges[j][k], neighbour, neighbour->plane, addedCount);
						if (newPoint) 
						{
							if (k == 0 && ((planes[i]->edges[j][k]->horizontalIndex - neighbour->horizontalIndex + horizontalCount) % horizontalCount
								== 1)) wasFirstPointGenerated = true;
							cornersToDelete.push_back({ k, neighbour->plane->id });
							k++;
							addedCount++;
							previousId = neighbour->plane->id;
						}
						wasPreviousSelected = true;
					}
					else
						wasPreviousSelected = false;
					previousPoint = planes[i]->edges[j][k];
					previousIndex = k;
					previousAddedCount = addedCount;
					if(k > 0 || !wasFirstPointGenerated)
						previousNeighbourPoint = neighbour;
				}
			}
			if (!wasPreviousSelected && (cornersToDelete.size() == 0 || previousNeighbourPoint->plane->id != cornersToDelete[0].second || !wasFirstPointGenerated)) {
				auto newPoint = addNewPoint(previousPoint, previousNeighbourPoint, previousNeighbourPoint->plane, previousAddedCount);
				if (newPoint) {
					cornersToDelete.push_back({ previousIndex, previousNeighbourPoint->plane->id});
				}
			}
			for (size_t k = 0; k < cornersToDelete.size(); k++)
			{
				if (cornersToDelete[k].second == cornersToDelete[k == cornersToDelete.size() - 1 ? 0 : k + 1].second && (wasFirstPointGenerated || k < cornersToDelete.size() - 1) &&
					(!wasFirstPointGenerated || k > 0))
				{
					for (size_t l = cornersToDelete[k].first; l <= (k == cornersToDelete.size() - 1 ? planes[i]->edges[j].size() - 1 : cornersToDelete[k + 1].first); l++) {
						if(planes[i]->edges[j][l]->verticalIndex < verticalCount)
							planes[i]->edges[j][l]->isCorner = false;
					}
				}
				else planes[i]->edges[j][cornersToDelete[k].first]->isCorner = false;
			}
		}
	}
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k]->edges.size(); i++) {
			bool isPreviousConcave = false;
			for (size_t j = 0; j < planes[k]->edges[i].size(); j++) {
				if (!isStraightPoint(j, planes[k]->edges[i], isPreviousConcave)) {
					planes[k]->edges[i][j]->isCorner = true;
				}
			}
		}
	}
}

double distancePointFromLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line)
{
	Vec3<double> directionVector = line.first - line.second;
	return Vec3<double>::crossProduct(p - line.first, directionVector).length() / directionVector.length();
}

bool isOnLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line, double treshold)
{
	return distancePointFromLine(p, line) < treshold;
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i] = nullptr;
	}
}

void exportObjects()
{
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models/processed_obj_" 
				+ std::to_string(currentCornerId - 1) + ".obj");
			MyFile << "o Mesh" << std::endl;
			std::vector<Point*> corners;
			currentCornerIndex = 0;
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {
				if (planes[i]->edges[j][k]->isCorner) {
					planes[i]->edges[j][k]->cornerId = currentCornerId;
					planes[i]->edges[j][k]->cornerIndex = currentCornerIndex;
					corners.push_back(planes[i]->edges[j][k]);
					currentCornerIndex++;
				}
			}
			for (size_t k = 0; k < corners.size(); k++) {
				MyFile << "v " << -corners[k]->position.x << " " << corners[k]->position.y << " " << corners[k]->position.z << std::endl;
			}
			MyFile << "f ";
			for (size_t j = 1; j < corners.size() + 1; j++) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (size_t j = corners.size(); j > 0; j--) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile.close();
			currentCornerId++;
		}
	}
}

void processData() {
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	findPoints();
	findCorners();
	connectPlanes();
	exportObjects();
	writeData();
}

int main()
{
    readData();
	processData();
    return 0;
}