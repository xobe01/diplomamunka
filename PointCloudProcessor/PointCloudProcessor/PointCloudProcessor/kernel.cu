#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>
#include "kernel.h"
#include <chrono>

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Point*> savedPoints;
std::vector<Plane*> planes;
std::vector<Plane*> savedPlanes;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPlaneId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;
const double planeDistanceTreshold = 0.05;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData(size_t pointCloudIndex)
{
	points.clear();
	addedPoints.clear();
	savedPoints.clear();
	planes.clear();
	verticalCounts.clear();
	currentCornerId = 1;
	currentSeparatedObjectId = 1;
	currentPlaneId = 1;
	currentOutlineId = 1;
	currentCornerIndex = 0;
	verticalCounts.push_back(0);
    std::string myText;		  
	
	
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw_" + 
		(pointCloudTestIndex == -1 ? (pointCloudCount == 0 ? "test" : std::to_string(pointCloudIndex)) : std::to_string(pointCloudTestIndex)) + ".txt");
	getline(MyReadFile, myText);
	std::replace(myText.begin(), myText.end(), ',', '.');
	std::stringstream ss(myText);
	std::string _x, _y, _z;
	std::getline(ss, _x, ';');
	std::getline(ss, _y, ';');
	std::getline(ss, _z, ';');
	egoCarPos = { std::stof(_x),  std::stof(_y),  std::stof(_z) };
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}		
	}
    MyReadFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}

	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel + 0.1) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

void setPointsMarked(std::vector<Point*> points, bool isMarked, bool isMarked2)
{
	for (size_t i = 0; i < points.size(); i++) {
		points[i]->isMarked = isMarked;
		points[i]->isMarked2 = isMarked2;
	}
}

int spikeType(Point* p, int arriveDirection, bool onlyMarkedNeighbours)
{
	//arriveDirection
	// 0 - from left
	// 1 - from up
	// 2 - from rigth
	// 3 - from down
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	size_t neighbourCount = 0;
	bool isNeighbour[4] = { false, false, false, false };
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };

	Point* diagNeighbourPoints[4] = { points[getOffset(x - 1, y - 1)], points[getOffset(x + 1, y - 1)], points[getOffset(x + 1, y + 1)], points[getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->plane == p->plane && 
			(!onlyMarkedNeighbours || neighbourPoints[j]->isMarked)) {
			neighbourCount++;
			isNeighbour[j] = true;
		}
	}
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarkedNeighbours || diagNeighbourPoints[j]->isMarked)) {
			diagIsNeighbour[j] = true;
		}
	}
	if (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[2] && !diagIsNeighbour[0]) || (isNeighbour[0] && isNeighbour[3] && !diagIsNeighbour[1]) 
		|| (isNeighbour[1] && isNeighbour[2] && !diagIsNeighbour[3]) || (isNeighbour[1] && isNeighbour[3] && !diagIsNeighbour[2])))
		return -1;
	if ((neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3]))) || (neighbourCount > 1 && 
		((arriveDirection == 0 && !isNeighbour[1]) || (arriveDirection == 1 && !isNeighbour[2]) || (arriveDirection == 2 && !isNeighbour[0])
			|| (arriveDirection == 3 && !isNeighbour[3]))))
		return 0;
	if (neighbourCount == 1)
		return 1;
	if (neighbourCount > 1)
		return 2;
	return 3;
}

bool checkIfBridge(Point* p, bool onlyMarked)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && neighbourPoints[j]->plane == p->plane && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) &&
			spikeType(neighbourPoints[j], -1, onlyMarked) > 1 && (!onlyMarked || neighbourPoints[j]->isMarked))
			neighbourCount++;
	}
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* diagNeighbourPoints[4] = { points[getOffset(x - 1, y - 1)], points[getOffset(x + 1, y - 1)], points[getOffset(x + 1, y + 1)],
		points[getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarked || diagNeighbourPoints[j]->isMarked)) {
			diagIsNeighbour[j] = true;
		}
	}


	if (((!diagIsNeighbour[0] && !diagIsNeighbour[2]) || (!diagIsNeighbour[1] && !diagIsNeighbour[3])) && neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p, false)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(p1->position - center->position, p2->position - center-> position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x + 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x - 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

void choosePoints(const Vec3<Point*> planePoints, Plane* basePlane, /*out*/ Plane* plane)
{
	if (basePlane) {
		plane->pointDirections = basePlane->pointDirections;
		plane->normal = basePlane->normal;
	}
	else 
	{
		auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.y->position -
			planePoints.x->position, planePoints.z->position - planePoints.x->position));
		Vec3<double> horizontalDirection = { 0,0,0 };
		Vec3<double> verticalDirection = { 0,0,0 };
		Point* neighbours[2] = { planePoints.y, planePoints.z };
		for each (auto neighbour in neighbours) {
			switch (areNeighbours(planePoints.x, neighbour)) {
			case 1:
			horizontalDirection = neighbour->position - planePoints.x->position;
			break;
			case 2:
			verticalDirection = neighbour->position - planePoints.x->position;
			break;
			case 3:
			horizontalDirection = planePoints.x->position - neighbour->position;
			break;
			case 4:
			verticalDirection = planePoints.x->position - neighbour->position;
			break;
			default:
			break;
			}
		}
		plane->pointDirections = { horizontalDirection, verticalDirection };
		plane->normal = normal;
	}
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(plane->normal, neighbourPoints[j]->position - plane->planePointPos));
					if (dist <= planeDistanceTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
						if ((neighbourPoints[j]->normal - plane->normal).length() < 0.05) {
							plane->normal = Vec3<double>::normalize(plane->normal * (plane->points.size() - 1) + neighbourPoints[j]->normal);
						}
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void findPlanes()
{
	size_t minPointCount = 10;
	size_t counter = 1;
	double normalTreshold = 0.01;
	for (size_t i = 0; i < points.size(); i++)
		if (points[i]) {
			points[i]->isMarked = true;
			points[i]->isMarked2 = true;
			calculateNormal(points[i]);
		}
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			nextStepPoints.push_back(points[j]);
			while (nextStepPoints.size() > 0) {
				Plane* plane = new Plane();
				bool planeMerged;
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x + 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x - 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k != 2 || y < verticalCount - 1) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if (normals[k].length() > 0 && normals[(k + 1) % 4].length() > 0 &&
								(normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {
								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, nullptr,
									plane);
								break;
							}
						}
						if (plane->points.size() > 0)
							break;
					}
				}
				if (plane->points.size() > 0)
				{
					planes.push_back(plane);
					currentPlaneId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		int originalSize = planes[i]->points.size();
		while (isThereBridge(planes[i]->points)) {}		
		if (originalSize != planes[i]->points.size() && planes[i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[i]->points[0], nullptr, nullptr }, planes[i], plane);
				if (plane->points.size() < planes[i]->points.size()) {
					for (size_t j = 0; j < planes[i]->points.size(); j++) {
						if (planes[i]->points[j]->plane != planes[i]) {
							planes[i]->points.erase(planes[i]->points.begin() + j);
							j--;
						}
					}
					planes.push_back(plane);
					currentPlaneId++;
				}
				else 
				{
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[i];
					}
					delete plane;
					break;
				}
			}
		}
	}
	for (size_t i = 0; i < planes.size(); i++)
		planes[i]->calculateAvaragePointPos();
}

bool hasNonSpykeNeighbour(size_t x, size_t y)
{
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 4; i++) {
		switch (i) {
		case 0: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && spikeType(neighbourPoint, -1, false) == 2)
			return true;
	}
	return false;
}

void findNextPoint(Point*& startPoint, size_t direction, /*out*/ 
	std::vector<std::pair<Point*, int>>& currentEdge, std::vector<Point*>& spikePoints, size_t dbgPlaneIndex, std::vector<Plane*> dbgPlanes)
{
	//isMarked -- turned off after the edge is complete
	//isMarked2 -- turned off when point added to edge
	
	Point* currentPoint = nullptr;
	std::pair<Point*, size_t> previousSavedPoint = {nullptr, 0};
	bool isFirstPoint = true;
	bool comeFromDeadEnd = false;
	bool isPreviousSpike = false;
	bool wasThereNonSpike = false;
	bool isHole = direction == 1;
	while (currentPoint != startPoint || comeFromDeadEnd)
	{
		
		if (!currentPoint)
			currentPoint = startPoint;
		if (currentPoint->horizontalIndex == 366 && currentPoint->verticalIndex == 1) {
			std::cout << "asd";
		}
		Point* neighbourPoint = nullptr;
		size_t x = currentPoint->horizontalIndex;
		size_t y = currentPoint->verticalIndex;
		isPreviousSpike = currentPoint != previousSavedPoint.first && spikeType(currentPoint, (!wasThereNonSpike || currentPoint == startPoint) ? -1 : ((direction + 1) % 4), !isHole) == 0;
		if (!isPreviousSpike || currentEdge.size() == 0) {
			if (currentPoint->isMarked2) {
				currentEdge.push_back({ currentPoint, direction });
			}
		}
		else spikePoints.push_back(currentPoint);
		if (isHole && isPreviousSpike)
		{
			currentPoint = previousSavedPoint.first;
			direction = (previousSavedPoint.second + 1) % 4;
			continue;
		}
		if(true || !isHole) currentPoint->isMarked2 = false;		
		for (size_t i = 0; i < 4; i++) {
			/*if (checkIfBridge(currentPoint)) {
				i += 2;
				direction = (direction + 2) % 4;
			}*/
			switch (direction) {
			case 0: //to right
			neighbourPoint = points[getOffset(x + 1, y)];
			break;
			case 1: //to down
			neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
			break;
			case 2: //to left
			neighbourPoint = points[getOffset(x - 1, y)];
			break;
			case 3: //to up
			neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
			break;
			default:
			break;
			}
			if (neighbourPoint == startPoint) {
				currentPoint = startPoint;
				currentEdge[0].second = (direction + 3) % 4;
				comeFromDeadEnd = false;
				break;
			}
			if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr && neighbourPoint->isMarked2
				&& (isPreviousSpike || spikeType(neighbourPoint, -1, !isHole) <= 1 || spikeType(currentPoint, direction, !isHole) > 0))
			{
				if (isFirstPoint) {
					currentEdge[0].second = (direction + 1) % 4 ;
					isFirstPoint = false;
				}
				if (!wasThereNonSpike && spikeType(currentPoint, -1, !isHole) > 1) {
					if (currentEdge.size() > 1) 
					{
						auto helper = currentEdge[0];
						currentEdge[0] = currentEdge[1];
						currentEdge[1] = helper;
						startPoint = currentPoint;
						if (!hasNonSpykeNeighbour(startPoint->horizontalIndex, startPoint->verticalIndex)) {
							for (size_t j = 0; j < currentEdge.size(); j++) {
								spikePoints.push_back(currentEdge[j].first);
								currentEdge[j].first->plane = nullptr;
							}
							currentEdge.clear();
							return;
						}
					}
					isPreviousSpike = false;
					wasThereNonSpike = true;
				}
				if (!isPreviousSpike)
					previousSavedPoint = { currentPoint, (direction + (4 - i)) % 4 };
				auto neighbourSpikeType = spikeType(neighbourPoint, -1, !isHole);
				if ((!isHole && checkIfBridge(neighbourPoint, true)) || neighbourSpikeType == -1 || neighbourSpikeType == 1 || (wasThereNonSpike && 
					currentEdge.size() > 1 && isPreviousSpike && neighbourSpikeType == 2 && neighbourPoint->isMarked)) {
					auto savedPoint = neighbourSpikeType == 1 ? neighbourPoint : currentPoint;
					if(savedPoint->isMarked2 || isPreviousSpike)
						currentEdge.push_back({ savedPoint, (direction + 3) % 4 });
					if (neighbourSpikeType == -1 || (!isHole && checkIfBridge(neighbourPoint, true)))
					{
						neighbourPoint->isMarked = false;
						neighbourPoint->isMarked2 = false;
						neighbourPoint->plane = nullptr;
					}
					savedPoint->isMarked2 = false;
					currentPoint = previousSavedPoint.first;
					direction = previousSavedPoint.second;
					isPreviousSpike = false;
					comeFromDeadEnd = true;
					break;
				}
				else {
					comeFromDeadEnd = false;
					currentPoint = neighbourPoint;
				}
				direction = (direction + 3) % 4;
				break;
			}
			direction += direction == 3 ? -3 : 1;
			if (i == 3) {
				if (!wasThereNonSpike) {
					for (size_t j = 0; j < currentEdge.size(); j++) {
						currentEdge[j].first->plane = nullptr;
						spikePoints.push_back(currentEdge[j].first);
					}
					currentEdge.clear();
					return;
				}
				if (currentPoint == startPoint)
					return;
				currentPoint = previousSavedPoint.first;
				direction = previousSavedPoint.second;
				std::cout << "INVALID EDGE SEARCH" << std::endl;;
			}
		}
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

Vec3<double> getRay(size_t hoizontalIndex, size_t verticalIndex)
{
	hoizontalIndex = (hoizontalIndex + horizontalCount) % horizontalCount;
	hoizontalIndex = (hoizontalIndex + horizontalCount) % horizontalCount;
	return  Vec3<double>::normalize({ sin(2 * PI * ((double)hoizontalIndex / horizontalCount)),
				rayAngles.second / 45 + (rayAngles.first - rayAngles.second) / 45 / verticalCount * (double)verticalIndex,
				cos(2 * PI * ((double)hoizontalIndex / horizontalCount)) });
}

void saveEdgeNeighbours(Edge* edge)
{
	const double backgroundPlaneDistTreshold = 0.5;
	auto normal = Vec3<double>::crossProduct(edge->pointsWithDir[0].first->plane->pointDirections.first,
		edge->pointsWithDir[0].first->plane->pointDirections.second);//edge->pointsWithDir[0].first->plane->normal;
	auto planePointPos = edge->pointsWithDir[0].first->plane->planePointPos;
	for (size_t i = 0; i < edge->pointsWithDir.size(); i++) 
	{
		auto point = edge->pointsWithDir[i].first;
		size_t x = point->horizontalIndex;
		size_t y = point->verticalIndex;
		Point* neighbourPoints[4] = { points[getOffset(x + 1, y)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
			points[getOffset(x, y - 1)] };
		Vec3<double> neighbourRays[4] = { getRay(x + 1, y), getRay(x, y + 1), getRay(x - 1, y), getRay(x, y - 1) };
		for (size_t j = 0; j < 4; j++) {
			if ((y > 0 || j != 3) && (y < verticalCount - 1 || j != 1) && (!neighbourPoints[j] || neighbourPoints[j]->plane !=
				edge->pointsWithDir[i].first->plane))
			{
				double d = Vec3<double>::dot_product(normal, planePointPos);
				if (Vec3<double>::dot_product(normal, neighbourRays[j]) == 0) { // No intersection, the line is parallel to the plane
					continue;
				}
				float x = (Vec3<double>::dot_product(normal, planePointPos) - Vec3<double>::dot_product(normal, egoCarPos)) /
					Vec3<double>::dot_product(normal, neighbourRays[j]);
				auto intersection = egoCarPos + neighbourRays[j] * x;
				if (!neighbourPoints[j] || ((intersection - egoCarPos).length() + backgroundPlaneDistTreshold) <
					(neighbourPoints[j]->position - egoCarPos).length())
				{
 					edge->closestNeighbourPoints.push_back({ intersection, {0,0,0} });
				}
			}
		}
	}
}	

void findEdgePoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> outerConnectedEdgePoints;
		std::vector<Point*> holeConnectedEdgePoints;
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) if(isEdgePoint(planes[i]->points[j])) edgePointsInPlane.push_back(planes[i]->points[j]);
		while (edgePointsInPlane.size() > 0) 
		{
			Edge* currentEdge = new Edge();
			currentEdge->intersectedEdges.push_back(currentEdge);
			std::vector<Point*> tempEdgePointsInPlane;
			Point* startPoint = edgePointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
				if ((edgePointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - edgePointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| edgePointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = edgePointsInPlane[j]->horizontalIndex;
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
				else if (edgePointsInPlane[j]->horizontalIndex == minHorizontalCoord && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
			}
			size_t direction = 2;
			if (spikeType(startPoint, -1, true) == -1) {
				size_t x = startPoint->horizontalIndex;
				size_t y = startPoint->verticalIndex;
				startPoint->plane = nullptr;
				startPoint->isMarked = false;
				startPoint->isMarked2 = false;
				for (size_t j = 0; j < planes[i]->points.size(); j++) {
					if (planes[i]->points[j] == startPoint) {
						planes[i]->points[j]->plane = nullptr;
						planes[i]->points.erase(planes[i]->points.begin() + j);
						break;
					}
				}
				minVerticalCoord = points[getOffset(x + 1, y)]->verticalIndex;
				for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
					if (edgePointsInPlane[j]->horizontalIndex == x + 1 && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) 			
					{
						minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
						startPoint = edgePointsInPlane[j];
					}
				}
			}
			else if (startPoint->verticalIndex > 0 && points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)] &&
				points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)]->plane == startPoint->plane &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex - 1)] &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex - 1)]->plane == startPoint->plane &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex)] &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex)]->plane == startPoint->plane)
			{
				direction = 1;
				currentEdge->isHole = true;
			}			
			if (i == 8 && planes[i]->edges.size() == 1){
				std::cout << "asd";
			}
			std::vector<Point*> spikePoints;
			setPointsMarked(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints, false, false);
			setPointsMarked(currentEdge->isHole ? outerConnectedEdgePoints : holeConnectedEdgePoints, true, false);
			findNextPoint(startPoint, direction, currentEdge->pointsWithDir, spikePoints, i, planes);
			setPointsMarked(outerConnectedEdgePoints, false, false);
			setPointsMarked(holeConnectedEdgePoints, false, false);
			for (size_t j = 0; j < currentEdge->pointsWithDir.size(); j++) {
				currentEdge->pointsWithDir[j].first->isMarked = false;
				(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints).push_back(currentEdge->pointsWithDir[j].first);
			}
			for (size_t j = 0; j < spikePoints.size(); j++) {
				spikePoints[j]->isMarked = false;
				(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints).push_back(spikePoints[j]);
			}
			for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
				if (edgePointsInPlane[j]->isMarked) tempEdgePointsInPlane.push_back(edgePointsInPlane[j]);
			}
			edgePointsInPlane = tempEdgePointsInPlane;
			if (currentEdge->pointsWithDir.size() > 3) {
				if (direction == 1) {
					currentEdge->pointsWithDir.insert(currentEdge->pointsWithDir.begin(),
						currentEdge->pointsWithDir[currentEdge->pointsWithDir.size() - 1]);
					currentEdge->pointsWithDir.pop_back();
				}
				for (size_t k = 0; k < currentEdge->pointsWithDir.size(); k++) {
					currentEdge->pointsWithDir[k].first->outlineId = currentOutlineId;
				}
				currentOutlineId++;
				currentEdge->startPoint = startPoint;
				planes[i]->edges.push_back(currentEdge);
				saveEdgeNeighbours(currentEdge);
			}
		}
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = false;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = false;
	}
}

const double newPointAcceptTreshold = 0.95;
const double inf = 1000000;

bool isStraightPoint(size_t pointIndex, Edge* edge, size_t& previousNeighbourCount, 
	Vec3<double>& straigthDir)
{
	const double newDirTreshold = 0.1;
	Point* point = edge->pointsWithDir[pointIndex].first;
	if (point->horizontalIndex == 666 && point->verticalIndex == 29) {
		//std::cout << "asd";
	}
	size_t previousNeighbourCountStore = previousNeighbourCount;
	size_t neighbourCount = 0;
	size_t neighbourEdgeCount = 0;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Plane* plane = point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i != 1) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane) {
			neighbourCount++;
			isNeighbour[i] = true;
			if (neighbourPoints[i]->outlineId > 0)
				neighbourEdgeCount++;
		}
	}
	previousNeighbourCount = neighbourCount;
	if (pointIndex > 0 && areNeighbours(pointIndex < (edge->pointsWithDir.size() - 1) ? edge->pointsWithDir[pointIndex + 1].first : edge->startPoint,
		point) == 0 && neighbourCount < 3) { //deadend
		auto previousPoint = edge->pointsWithDir[pointIndex - 1];
		auto previousPreviousPoint = pointIndex > 1 ? edge->pointsWithDir[pointIndex - 2] : edge->pointsWithDir[edge->pointsWithDir.size() - 2 
			+ pointIndex];
		straigthDir = { 0,0,0 };
		if (areNeighbours(previousPoint.first, previousPreviousPoint.first) > 0 && ((previousPoint.first->verticalIndex ==
			previousPreviousPoint.first->verticalIndex && previousPoint.first->horizontalIndex == point->horizontalIndex &&
			previousPoint.first->verticalIndex != point->verticalIndex) || (previousPoint.first->horizontalIndex ==
				previousPreviousPoint.first->horizontalIndex && previousPoint.first->verticalIndex == point->verticalIndex &&
				previousPoint.first->horizontalIndex != point->horizontalIndex))) //if curve swap with previous
		{
			point->isCorner = true;
			edge->pointsWithDir[pointIndex - 1] = edge->pointsWithDir[pointIndex];
			edge->pointsWithDir[pointIndex] = previousPoint;
			if (pointIndex > 1) edge->pointsWithDir[pointIndex - 2].first->isCorner = true;
			return true;
		}
		return false;
	}
	if (neighbourCount == 3 && (neighbourEdgeCount == 2 && (pointIndex == edge->pointsWithDir.size() - 1 || 
		areNeighbours(point, edge->pointsWithDir[pointIndex + 1].first)
		> 0)))
	{
		if (straigthDir.length() == 0) {
			if (pointIndex > 0)
				straigthDir = Vec3<double>::normalize(point->position - edge->pointsWithDir[pointIndex - 1].first->position);
		}
		else
		{
			auto newDir = Vec3<double>::normalize(point->position - edge->pointsWithDir[pointIndex - 1].first->position);
			if ((straigthDir - newDir).length() > newDirTreshold) {
				straigthDir = newDir;
				return false;
			}
		}
		return true;
	}
	straigthDir = { 0,0,0 };
	if (neighbourCount == 4 && (pointIndex == 0 ? edge->pointsWithDir[edge->pointsWithDir.size() - 1].first : 
		edge->pointsWithDir[pointIndex - 1].first)->isCorner &&
		previousNeighbourCountStore == 2)
		return true;
	if (previousNeighbourCountStore == 4) {
		edge->pointsWithDir[pointIndex - 1].first->isCorner = false;
	}
	return false;
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k]->edges.size(); i++) {
			size_t previousNeighbourCount = 0;
			Vec3<double> straightDir = { 0,0,0 };
			for (size_t j = 0; j < planes[k]->edges[i]->pointsWithDir.size(); j++) {
				if (!isStraightPoint(j, planes[k]->edges[i], previousNeighbourCount, straightDir))
				{
					planes[k]->edges[i]->pointsWithDir[j].first->isCorner = true;
				}
			}
		}
	}
}

Point* createNewPoint(Vec3<double> newPointPos, Point* point, std::vector<Point*> neighbours, size_t addedCount, bool createBeforePoint = false, 
	bool isCornerPoint = false)
{
	double deleteDurroundingCornersTreshold = 0.1;
	Point* newPoint = new Point(newPointPos, neighbours[0]->horizontalIndex, isCornerPoint ? verticalCount + 1 : verticalCount, point->plane);
	addedPoints.push_back(newPoint);
	newPoint->isCorner = true;
	newPoint->outlineId = point->outlineId;
	for (size_t j = 0; j < point->plane->edges.size(); j++) {
		if (point->plane->edges[j]->pointsWithDir[0].first->outlineId == point->outlineId) {
			for (size_t k = 0; k < point->plane->edges[j]->pointsWithDir.size(); k++) {
				if (point->plane->edges[j]->pointsWithDir[k].first == point) {
					auto index = k + (createBeforePoint ? 0 : 1) + addedCount;
					point->plane->edges[j]->pointsWithDir.insert(point->plane->edges[j]->pointsWithDir.begin() + index, { newPoint, -1 });
					break;
				}
			}
			break;
		}
	}
	for (size_t i = 0; i < neighbours.size(); i++) {
		newPoint->neighbourPlaneNeighbours[i] = neighbours[i];
	}
	return newPoint;
}

const double twoPointDifferenceTreshold = 0.5;

Point* addNewPoint(Point* point, Point*& neighbour, Plane* plane, size_t addedCount, size_t neighbourIndex)
{
	if (point->createdNeighbourPoints[neighbourIndex] != nullptr) { //created by other plane
		auto createdNeighbour = point->createdNeighbourPoints[neighbourIndex];
		createdNeighbour->isCorner = true;
		for (size_t j = 0; j < point->plane->edges.size(); j++) {
			if (point->plane->edges[j]->pointsWithDir[0].first->outlineId == point->outlineId) {
				for (size_t k = 0; k < point->plane->edges[j]->pointsWithDir.size(); k++) {
					if (point->plane->edges[j]->pointsWithDir[k].first == point) {
						for (size_t l = 0; l < 4; l++) 
						{
							if (point->plane->edges[j]->pointsWithDir[k + l + 1].first == createdNeighbour) {
								point->plane->edges[j]->pointsWithDir.insert(point->plane->edges[j]->pointsWithDir.begin() + k + 1 + addedCount,
									{ createdNeighbour, -1 });
								point->plane->edges[j]->pointsWithDir.erase(point->plane->edges[j]->pointsWithDir.begin() + k + l + 2);
								break;
							}
						}
						break;
					}
				}
				break;
			}
		}
		return createdNeighbour;
	}
	Vec3<double> dir = { 0,0,0 };
	switch (areNeighbours(point, neighbour)) {
	case 1:
	dir = point->plane->pointDirections.first;
	break;
	case 2:
	dir = point->plane->pointDirections.second;
	break;
	case 3:
	dir = point->plane->pointDirections.first * -1;
	break;
	case 4:
	dir = point->plane->pointDirections.second * -1;
	break;
	default:
	break;
	}
	Vec3<double> newPointPos = point->position - dir * Vec3<double>::dot_product(point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf || isnan(newPointPos.x) || isnan(newPointPos.y) || isnan(newPointPos.z))
		return nullptr;
	Vec3<double> dirToNew = newPointPos - point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold && 
		(point->position - newPointPos).length() > planeDistanceTreshold)
		return nullptr;
	Vec3<double> neighbourDir = { 0,0,0 };
	switch (areNeighbours(neighbour, point)) {
	case 1:
	neighbourDir = plane->pointDirections.first;
	break;
	case 2:
	neighbourDir = plane->pointDirections.second;
	break;
	case 3:
	neighbourDir = plane->pointDirections.first * -1;
	break;
	case 4:
	neighbourDir = plane->pointDirections.second * -1;
	break;
	default:
	break;
	}	
	Vec3<double> neighbourNewPointPos = neighbour->position - neighbourDir * Vec3<double>::dot_product(neighbour->position
		- point->plane->planePointPos, point->plane->normal) / Vec3<double>::dot_product(neighbourDir, point->plane->normal);
	if (abs(neighbourNewPointPos.x) > inf || abs(neighbourNewPointPos.y) > inf || abs(neighbourNewPointPos.z) > inf || isnan(neighbourNewPointPos.x) ||
		isnan(neighbourNewPointPos.y) || isnan(neighbourNewPointPos.z))
		return nullptr;
	Vec3<double> dirToNewNeighbour = neighbourNewPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(neighbourDir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold &&
		(neighbour->position - neighbourNewPointPos).length() > planeDistanceTreshold)
		return nullptr;
	if ((newPointPos - neighbourNewPointPos).length() > twoPointDifferenceTreshold)
		return nullptr;
	auto newPos = (newPointPos + neighbourNewPointPos) / 2;
	Point* newPoint;
	Point* newNeighbourPoint;
	newPoint = createNewPoint(newPos, point, { neighbour }, addedCount);
	point->createdNeighbourPoints[neighbourIndex] = newPoint;
	for (size_t i = 0; i < neighbour->neighbourPlaneNeighbours.size(); i++) 
	{
		if (neighbour->neighbourPlaneNeighbours[i] == point) 		
		{
			newNeighbourPoint = createNewPoint(newPos, neighbour, { point }, 0, false);
			neighbour->createdNeighbourPoints[i] = newNeighbourPoint;
			break;
		}
	}
	newPoint->neighbourPlaneNeighbours[1] = newNeighbourPoint;
	newNeighbourPoint->neighbourPlaneNeighbours[1] = newPoint;
	return newPoint;
}

void findPlaneConnections()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		for (size_t j = 0; j < planes[i]->edges.size(); j++) 
		{
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++)
			{
				Point* point = planes[i]->edges[j]->pointsWithDir[k].first;
				if (point->horizontalIndex == 339 && point->verticalIndex == 14) 	
				{
					//std::cout << "asd";
				}
				int direction = planes[i]->edges[j]->pointsWithDir[k].second;
				size_t x = point->horizontalIndex;
				size_t y = point->verticalIndex;
				Point* neighbourPoints[4] = { points[getOffset(x + 1, y)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
					points[getOffset(x, y - 1)] };
				for (size_t i = 0; i < 4; i++) {
					if ((y > 0 || direction != 3) && (y < verticalCount - 1 || direction != 1) && neighbourPoints[direction]) {
						point->neighbourPlaneNeighbours[i] = neighbourPoints[direction];
					}
					direction += direction == 3 ? -3 : 1;
				}
			}
		}
	}
}

void createPlaneCorner(Point* point, std::vector<Point*> point1, std::vector<Point*> point2)
{
	Plane* p1 = point->plane;
	Plane* p2 = point1[2]->plane;
	Plane* p3 = point2[2]->plane;
	auto cross1 = Vec3<double>::crossProduct(p2->normal, p3->normal);
	auto cross2 = Vec3<double>::crossProduct(p3->normal, p1->normal);
	auto cross3 = Vec3<double>::crossProduct(p1->normal, p2->normal);

	auto denom = Vec3<double>::dot_product(p1->normal, cross1);

	auto planeDist1 = Vec3<double>::dot_product(p1->planePointPos, p1->normal);
	auto planeDist2 = Vec3<double>::dot_product(p2->planePointPos, p2->normal);
	auto planeDist3 = Vec3<double>::dot_product(p3->planePointPos, p3->normal);

	cross1 = cross1 * planeDist1;
	cross2 = cross2 * planeDist2;
	cross3 = cross3 * planeDist3;

	auto cornerPoint = (cross1 + cross2 + cross3) / denom;

	createNewPoint(cornerPoint, point1[0], {point1[2], point2[2]}, 0, false, true);
	createNewPoint(cornerPoint, point1[1], { point }, 0, true, true);
	createNewPoint(cornerPoint, point2[1], { point }, 0, false, true);
}

void connectPlanes()
{
	std::vector<Point*> createdPoints;
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				auto point = planes[i]->edges[j]->pointsWithDir[k].first;
				if (point->horizontalIndex == 666 && point->verticalIndex == 29) {
					//std::cout << "asd";
				}
				if (point->verticalIndex == verticalCount)
				{
					continue;
				}
				size_t addedCount = 0;
				if(point->isCorner)
				{					
					for (size_t l = 0; l < point->neighbourPlaneNeighbours.size(); l++) {
						Point* newPoint = nullptr;
						auto neighbourPoint = point->neighbourPlaneNeighbours[l];
						if (neighbourPoint)
						{
							if (neighbourPoint->plane && neighbourPoint->outlineId > 0 && neighbourPoint->outlineId != point->outlineId &&
									neighbourPoint->plane != point->plane) 
							{
								newPoint = addNewPoint(point, neighbourPoint, neighbourPoint->plane, addedCount, l);
								if (newPoint) {
									if (k == 0 && l == 0) planes[i]->edges[planes[i]->edges.size() - 1]->wasFirstGenerated = true;
									planes[i]->edges[j]->pointsWithDir[k].first->isCorner = false;
									createdPoints.push_back(newPoint);
									addedCount++;
								}
								else 
								{
									planes[i]->edges[j]->pointsWithDir.insert(planes[i]->edges[j]->pointsWithDir.begin() + k + 1 + addedCount, { nullptr, -1 });
									addedCount++;
								}
							}
						}	
						if (!neighbourPoint || !neighbourPoint->plane) {
							planes[i]->edges[j]->pointsWithDir.insert(planes[i]->edges[j]->pointsWithDir.begin() + k + 1 + addedCount, { nullptr, -1 });
							addedCount++;
						}
					}
					k += addedCount;
				}
			}			
		}
	}
}

void createCorners()
{
	const double normalDiffTreshold = 0.1;
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {	
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				auto point = planes[i]->edges[j]->pointsWithDir[k].first;
				if (point) {
					for (size_t l = 0; l < planes[i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints.size(); l++) {
						auto createdPoint1 = planes[i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints[l];
						auto createdPoint2 = planes[i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints[l == 3 ? 0 : (l + 1)];
						if (createdPoint1 && createdPoint2 && createdPoint1->neighbourPlaneNeighbours[1]->plane !=
							createdPoint2->neighbourPlaneNeighbours[1]->plane
							&& (planes[i]->edges[j]->pointsWithDir[k].first->plane->normal -
							createdPoint1->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold && 
							(planes[i]->edges[j]->pointsWithDir[k].first->plane->normal -
								createdPoint2->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold && 
							(createdPoint1->neighbourPlaneNeighbours[1]->plane->normal -
									createdPoint2->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold) {
							createPlaneCorner(planes[i]->edges[j]->pointsWithDir[k].first, { createdPoint1, createdPoint1->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l] }, { createdPoint2, createdPoint2->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l == 3 ? 0 : (l + 1)] });
							break;
						}
					}
				}
			}
		}
	}
}

void filterEdgePoints()
{
	std::vector<std::vector<std::vector< std::pair<bool, std::pair<size_t, size_t>>>>> cornersToDelete;
	for (size_t i = 0; i < planes.size(); i++) {
		cornersToDelete.push_back({});
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			cornersToDelete[i].push_back({});
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first) {
					if (planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex >= verticalCount) {
						for (size_t l = 0; l < planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours.size(); l++) {
							if (planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l] &&
								planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l]->verticalIndex < verticalCount)
								cornersToDelete[i][j].push_back({ planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex > verticalCount,
									{ k,  planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l]->plane->id } });
						}
					}
				}
				else {
					cornersToDelete[i][j].push_back({ false, { 0, 0 } });
					planes[i]->edges[j]->pointsWithDir.erase(planes[i]->edges[j]->pointsWithDir.begin() + k);
					k--;
				}
			}
			for (int k = 0; k < cornersToDelete[i][j].size(); k++) 
			{
				if (k < ((int)cornersToDelete[i][j].size() - 2) && cornersToDelete[i][j][k + 1].first && cornersToDelete[i][j][k + 2].first) 
				{
					cornersToDelete[i][j][k + 1].second = { cornersToDelete[i][j][k + 1].second.first, cornersToDelete[i][j][k].second.second };
					size_t index = k + 2;
					while (index < cornersToDelete[i][j].size() && cornersToDelete[i][j][index].first)
					{
						cornersToDelete[i][j].erase(cornersToDelete[i][j].begin() + index);
					}
					cornersToDelete[i][j].insert(cornersToDelete[i][j].begin() + index, { true, {cornersToDelete[i][j][k + 1].second.first,
						index < cornersToDelete[i][j].size() ? cornersToDelete[i][j][index].second.second : cornersToDelete[i][j][0].second.second} });
				}
			}
			int newPointStartIndex = planes[i]->edges[j]->wasFirstGenerated ? 1 : 0;
			if ((planes[i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->horizontalIndex < planes[i]->edges[j]->startPoint->horizontalIndex ||
				planes[i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->verticalIndex < planes[i]->edges[j]->startPoint->verticalIndex) &&
				planes[i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->createdNeighbourPoints[0])
				newPointStartIndex++;
			int newPointEndIndex = newPointStartIndex;
			if (currentFrame == 4 && i == 0) {
				std::cout << "asd";
			}
			Point* endPoint = nullptr;
			while (newPointEndIndex < (int)cornersToDelete[i][j].size() - 1) {
				if (cornersToDelete[i][j][newPointStartIndex].second.second > 0) {
					while (newPointEndIndex + 1 < cornersToDelete[i][j].size()
						&& cornersToDelete[i][j][newPointStartIndex].second.second == cornersToDelete[i][j][newPointEndIndex + 1].second.second) {
						newPointEndIndex++;
					}
					for (size_t k = cornersToDelete[i][j][newPointStartIndex].second.first + 1; k <
						cornersToDelete[i][j][newPointEndIndex].second.first; k++) {
						planes[i]->edges[j]->pointsWithDir[k].first->isCorner = false;
					}

					auto startPoint = planes[i]->edges[j]->pointsWithDir[cornersToDelete[i][j][newPointStartIndex].second.first].first;
					for (size_t k = 0; k < 2; k++) {
						if (endPoint && (startPoint->position - endPoint->position).length() < 0.1) {
							if (startPoint->verticalIndex == verticalCount + 1 && endPoint->verticalIndex != verticalCount + 1)
								endPoint->isCorner = false;
							else if (endPoint->verticalIndex == verticalCount + 1 && startPoint->verticalIndex != verticalCount + 1)
								startPoint->isCorner = false;
						}
						endPoint = planes[i]->edges[j]->pointsWithDir[cornersToDelete[i][j][newPointEndIndex].second.first].first;
					}				
					if (newPointEndIndex == cornersToDelete[i][j].size() - 1 && newPointStartIndex != newPointEndIndex && 
						planes[i]->edges[j]->wasFirstGenerated && cornersToDelete[i][j][0].second.second ==
						cornersToDelete[i][j][newPointStartIndex].second.second)
						planes[i]->edges[j]->pointsWithDir[cornersToDelete[i][j][cornersToDelete[i][j].size() - 1].second.first].first->isCorner = false;
				}
				newPointStartIndex = newPointEndIndex + 1;
				newPointEndIndex = newPointStartIndex;
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex == verticalCount && planes[i]->edges[j]->pointsWithDir[k].first->isCorner
					&& planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1] &&
					planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1]->verticalIndex == verticalCount) {
					planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1]->isCorner = true;
				}
			}
		}
	}*/
}

void egoCarSegmentation()
{	
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 + egoCarPos.z && points[i]->position.z >= -2.5 + egoCarPos.z)
			points[i] = nullptr;
	}
}

double angleOfVectors(Vec3<double> v1, Vec3<double> v2, bool isBackward)
{
	auto v1Angle = atan2(v1.x, v1.y) / PI;
	auto v2Angle = atan2(v2.x, v2.y) / PI;
	auto angle = abs(v1Angle - v2Angle);
	if ((!isBackward && v1Angle < v2Angle) || (isBackward && v1Angle > v2Angle)) angle = 2 - angle;
	return angle * 180;
}

Vec3<double> intersectionOfLines(Vec3<double> p1, Vec3<double> p2, Vec3<double> q1, Vec3<double> q2, size_t& onEdgeType, double& distanceRatioOnEdge)
{
	const double acceptTreshold = 0.0000001;
	auto dir1 = p2 - p1;
	auto dir2 = q2 - q1;
	double R = (p1.y * dir1.x + q1.x * dir1.y - p1.x * dir1.y - q1.y * dir1.x) / (dir2.y * dir1.x - dir1.y * dir2.x);
	Vec3<double> intersectionPos = { q1.x + R * dir2.x, q1.y + R * dir2.y, 0 };
	auto side1Length = (p2 - p1).length() + acceptTreshold;
	auto side2Length = (q2 - q1).length() + acceptTreshold;
	if ((intersectionPos - p1).length() < side1Length && (intersectionPos - p2).length() < side1Length &&
		(intersectionPos - q1).length() < side2Length && (intersectionPos - q2).length() < side2Length)
	{
		if ((intersectionPos - p1).length() < acceptTreshold)
			onEdgeType = 2;
		else if ((intersectionPos - p2).length() < acceptTreshold)
			onEdgeType = 3;
		else if ((intersectionPos - q1).length() < acceptTreshold)
			onEdgeType = 4;
		else if ((intersectionPos - q2).length() < acceptTreshold)
			onEdgeType = 5;
		else onEdgeType = 1;
		distanceRatioOnEdge = (intersectionPos - p1).length() / (p2 - p1).length();
	}
	return intersectionPos;
}

size_t isPointInsidePolygon(std::vector<Point*>polygon, Vec3<double> point, std::pair<double, double> xBounds, std::pair<double, double> yBounds,
	bool checkOnEdge = false, double onEdgetreshold = 0.0000001)
{
	// 0 - outside
	// 1 - inside
	// 2 - onEdge
	size_t notInUse = 0;
	double notInUseRatio = 0;
	if (point.x > xBounds.first && point.x < xBounds.second &&
		point.y > yBounds.first && point.y < yBounds.second) 
	{
		int rigthCounter = 0;
		for (size_t i = 0; i < polygon.size(); i++) 
		{
			auto p1 = polygon[i]->projected2DPosition;
			auto p2 = polygon[(i + 1) % polygon.size()]->projected2DPosition;
			if ((p1.x < point.x && p2.x < point.x) || (p1.y > point.y && p2.y > point.y) || (p1.y < point.y && p2.y < point.y) || (p1.y == p2.y ))
				continue;
			else if (point == p1)
				return false;
			else
			{
				/*auto smallerAngle = (p1.x < p2.x ? p1 - point : (p2 - point));
				auto biggerAngle = (p1.x < p2.x ? p2 - point : (p1 - point));
				if((p1.x > point.x && p2.x > point.x) || angleOfVectors(smallerAngle, biggerAngle, smallerAngle.y > point.y || biggerAngle.y < point.y)
					< 180)*/
				auto intersection = intersectionOfLines(point, point + Vec3<double>({ 1,0,0 }), p1, p2, notInUse, notInUseRatio);
				if (abs(intersection.x - point.x) < onEdgetreshold && checkOnEdge)
					return 2;
				if (intersection.x <= point.x)
					continue;
				else
				{
					rigthCounter++;
					if (p2.y == point.y)
						i++;
				}
			}
		}
		return rigthCounter % 2 == 1;
	}
	return false;
}

void changeBaseTo2D(Edge* edge, std::pair<Vec3<double>, Vec3<double>> newBase = { {0,0,0}, {0,0,0} })
{
	edge->xBounds2D = { 100000, -100000 };
	edge->yBounds2D = { 100000, -100000 };
	auto normal = newBase.first == Vec3<double>({0,0,0}) ? edge->pointsWithDir[0].first->plane->normal : newBase.first;
	auto x = newBase.first == Vec3<double>({ 0,0,0 }) ? edge->pointsWithDir[0].first->plane->pointDirections.first : newBase.second;
	x = Vec3<double>::normalize(x - normal * Vec3<double>::dot_product(x, normal));
	auto y = Vec3<double>::crossProduct(x, normal);
	for (size_t i = 0; i < edge->pointsWithDir.size(); i++) {
		edge->pointsWithDir[i].first->projected2DPosition = { Vec3<double>::dot_product(edge->pointsWithDir[i].first->position, x), 
			Vec3<double>::dot_product(edge->pointsWithDir[i].first->position, y), 0 };
		auto pos2D = edge->pointsWithDir[i].first->projected2DPosition;
		if (pos2D.x < edge->xBounds2D.first) edge->xBounds2D.first = pos2D.x;
		if (pos2D.x > edge->xBounds2D.second) edge->xBounds2D.second = pos2D.x;
		if (pos2D.y < edge->yBounds2D.first) edge->yBounds2D.first = pos2D.y;
		if (pos2D.y > edge->yBounds2D.second) edge->yBounds2D.second = pos2D.y;
	}	
	for (size_t i = 0; i < edge->closestNeighbourPoints.size(); i++) {
		edge->closestNeighbourPoints[i].second = { Vec3<double>::dot_product(edge->closestNeighbourPoints[i].first, x),
			Vec3<double>::dot_product(edge->closestNeighbourPoints[i].first, y), 0 };
	}
}

void deleteSelfIntersections(std::vector<Plane*> planes)
{
	double notInUseRatio = 0;
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t l = 0; l < planes[k]->edges.size(); l++) {
			for (int i = 0; i < planes[k]->edges[l]->pointsWithDir.size(); i++) {
				for (int j = (i == (planes[k]->edges[l]->pointsWithDir.size() - 1) ? 1 : 0); j < i - 1; j++) {
					size_t onEdgeType = 0;
					auto intersection = intersectionOfLines(planes[k]->edges[l]->pointsWithDir[j].first->projected2DPosition,
						planes[k]->edges[l]->pointsWithDir[j + 1].first->projected2DPosition,
						planes[k]->edges[l]->pointsWithDir[i].first->projected2DPosition, planes[k]->edges[l]->pointsWithDir[(i + 1) % 
						planes[k]->edges[l]->pointsWithDir.size()].first->projected2DPosition,
						onEdgeType, notInUseRatio);
					if (onEdgeType > 0) {
						double routeLength1 = 0;
						for (size_t m = j + 1; m < i; m++) 
						{
							routeLength1 += (planes[k]->edges[l]->pointsWithDir[m + 1].first->projected2DPosition -
								planes[k]->edges[l]->pointsWithDir[m].first->projected2DPosition).length();
						}
						double routeLength2 = 0;
						size_t m = (i + 1) % planes[k]->edges[l]->pointsWithDir.size();
						while (m != j) 
						{
							routeLength2 += (planes[k]->edges[l]->pointsWithDir[(m + 1) % planes[k]->edges[l]->pointsWithDir.size()].first->projected2DPosition -
								planes[k]->edges[l]->pointsWithDir[m].first->projected2DPosition).length();
							m = (m + 1) % planes[k]->edges[l]->pointsWithDir.size();
						}
						if (routeLength1 < routeLength2) 
						{
							if (planes[k]->edges[l]->pointsWithDir[j + 1].first->verticalIndex == verticalCount + 1) 
							{
								planes[k]->edges[l]->pointsWithDir[i].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + i);
							}
							else 
							{
								planes[k]->edges[l]->pointsWithDir[j + 1].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + j + 1);
							}
						}
						else {
							if (planes[k]->edges[l]->pointsWithDir[j].first->verticalIndex == verticalCount + 1) 
							{
								planes[k]->edges[l]->pointsWithDir[(i + 1) % planes[k]->edges[l]->pointsWithDir.size()].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + ((i + 1) % planes[k]->edges[l]->pointsWithDir.size()));
							}
							else 
							{
								planes[k]->edges[l]->pointsWithDir[j].first->isCorner = false;
								planes[k]->edges[l]->pointsWithDir.erase(planes[k]->edges[l]->pointsWithDir.begin() + j);
							}
						}	
						i--;
						break;
					}
				}
			}
		}
	}
}

bool isClockwise(std::vector<Point*>& points)
{
	double angleSum = 0;
	for (size_t i = 0; i < points.size(); i++) {
		auto  a = angleOfVectors(points[(i + points.size() - 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition,
			points[(i + 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition, false);
		angleSum += angleOfVectors(points[(i + points.size() - 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition,
			points[(i + 1) % points.size()]->projected2DPosition - points[i]->projected2DPosition, false);
	}
	return angleSum < (double)points.size() * 360.0 / 2;
}

void extract2DPolygon()
{
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (!planes[i]->edges[j]->pointsWithDir[k].first->isCorner)
				{
					planes[i]->edges[j]->pointsWithDir.erase(planes[i]->edges[j]->pointsWithDir.begin() + k);
					k--;
				}
			}
			changeBaseTo2D(planes[i]->edges[j]);			
		}
	}
}

bool isDesiredEdge(std::vector<Point*> savedEdge, std::vector<Point*> newEdge, std::pair<double, double> newEdgeXBounds2D, std::pair<double, double> newEdgeYBounds2D,
	size_t vertexIndex, bool isInner)
{
	if (savedEdge[vertexIndex]->isMarked2) {
		auto isMiddlePointDesired = isPointInsidePolygon(newEdge, (savedEdge[vertexIndex]->projected2DPosition +
			savedEdge[(vertexIndex + 1)
			% savedEdge.size()]->projected2DPosition) / 2, newEdgeXBounds2D, newEdgeYBounds2D, true, 0.00001) == isInner;
		return isMiddlePointDesired;
	}
	return false;
}

Point* decideIfDesiredEdgeGood(Edge* savedEdge, Edge* newEdge, Point* currentPoint, std::vector<Point*> savedEdgePoints, std::vector<Point*> newEdgePoints,
	bool isOnSavedEdge, bool checkForInsidePoints, std::vector<std::pair<Vec3<double>, Vec3<double>>> allNeighbours, /*out*/ std::vector<Point*>& pointsOnDesiredEdge)
{
	std::pair<double, double> xBounds = { 100000, -1000000 };
	std::pair<double, double> yBounds = { 100000, -1000000 };
	std::vector<Point*> pointsOnAddedPolygonPart;
	Point* endPoint = nullptr;
	size_t currentIndex = 0;
	bool isForward = true;
	do {
		for (size_t i = 0; i < (isOnSavedEdge ? newEdgePoints : savedEdgePoints).size(); i++) {
			if ((isOnSavedEdge ? newEdgePoints : savedEdgePoints)[i] == currentPoint) {
				currentIndex = i;
				currentPoint = (isOnSavedEdge ? newEdgePoints : savedEdgePoints)[currentIndex];
				break;
			}
		}
		isOnSavedEdge = !isOnSavedEdge;
		do {
			if (isForward && pointsOnDesiredEdge.size() == pointsOnAddedPolygonPart.size()) {
				pointsOnDesiredEdge.push_back(currentPoint);
				currentPoint->isMarked = false;
			}
			pointsOnAddedPolygonPart.push_back(currentPoint);
			auto pos2D = currentPoint->projected2DPosition;
			if (pos2D.x < xBounds.first) xBounds.first = pos2D.x;
			if (pos2D.x > xBounds.second) xBounds.second = pos2D.x;
			if (pos2D.y < yBounds.first) yBounds.first = pos2D.y;
			if (pos2D.y > yBounds.second) yBounds.second = pos2D.y;
			currentIndex = (currentIndex + (isForward ? 1 : ((isOnSavedEdge ? savedEdgePoints : newEdgePoints).size() - 1))) % (isOnSavedEdge ?
				savedEdgePoints : newEdgePoints).size();
			currentPoint = (isOnSavedEdge ? savedEdgePoints : newEdgePoints)[currentIndex];
		} while (!currentPoint->isMarked2);	
		if (pointsOnDesiredEdge.size() == pointsOnAddedPolygonPart.size()) endPoint = currentPoint;
		isForward = !isForward;
	} while (currentPoint != pointsOnDesiredEdge[0] && checkForInsidePoints);
	if (checkForInsidePoints) {
		for (const auto p : allNeighbours.size() > 0 ? allNeighbours : (isOnSavedEdge ? savedEdge : newEdge)->closestNeighbourPoints) {
			if (isPointInsidePolygon(pointsOnAddedPolygonPart, p.second, xBounds, yBounds, true, savedEdge->isHole ? 0.00 : 0.0) == 1) {
				points.push_back(new Point(p.first, 0, 0, nullptr));
				for (const auto p2 : (isOnSavedEdge ? savedEdge : newEdge)->closestNeighbourPoints) {
					//points.push_back(new Point(p2.first, 0, 0, nullptr));
				}
				return nullptr;
			}
		}
	}
	return endPoint;
}

void calculateNewNeighbours(std::vector<Edge*> createdEdges, std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours)
{
	std::vector < std::vector<std::pair<std::vector<std::pair<std::pair<Vec3<double>, Vec3<double>>, double>>, double>>>
		neighboursClosestToEdge(createdEdges.size());
	for (size_t i = 0; i < createdEdges.size(); i++) {
		neighboursClosestToEdge[i] = std::vector< std::pair<std::vector<std::pair<std::pair<Vec3<double>, Vec3<double>>, double>>, double>>(
			createdEdges[i]->pointsWithDir.size(), { {}, 100000 });
	}
	for (size_t i = 0; i < neighbours.size(); i++) {
		double minDist = 100000;
		size_t closestEdgeIndex = 0;
		size_t closestPointIndex = 0;
		for (size_t k = 0; k < createdEdges.size(); k++) {
			for (size_t j = 0; j < createdEdges[k]->pointsWithDir.size(); j++) {
				double newDist = (createdEdges[k]->pointsWithDir[j].first->projected2DPosition - neighbours[i].second).length();
				if (newDist < minDist) {
					minDist = newDist;
					closestEdgeIndex = k;
					closestPointIndex = j;
				}
			}
		}
		neighboursClosestToEdge[closestEdgeIndex][closestPointIndex].first.push_back({ neighbours[i] , minDist });
		if (minDist < neighboursClosestToEdge[closestEdgeIndex][closestPointIndex].second)
			neighboursClosestToEdge[closestEdgeIndex][closestPointIndex].second = minDist;
	}
	for (size_t i = 0; i < neighboursClosestToEdge.size(); i++) {
		createdEdges[i]->closestNeighbourPoints.clear();
		for (size_t j = 0; j < neighboursClosestToEdge[i].size(); j++) {
			for (size_t k = 0; k < neighboursClosestToEdge[i][j].first.size(); k++) {
				if(neighboursClosestToEdge[i][j].first[k].second < neighboursClosestToEdge[i][j].second * 2)
					createdEdges[i]->closestNeighbourPoints.push_back(neighboursClosestToEdge[i][j].first[k].first);
			}
		}
	}
}

void relocateHoleNeighbours(Plane* plane1, Plane* plane2)
{
	for (size_t i = 0; i < plane1->edges.size(); i++) 
	{
		if (!plane1->edges[i]->isHole) 			
		{
			for (size_t j = 0; j < plane2->edges.size(); j++) 	
			{
				if (plane2->edges[j]->isHole)
				{
					for (size_t k = 0; k < plane1->edges[i]->closestNeighbourPoints.size(); k++) 
					{
						if (isPointInsidePolygon(plane2->edges[j]->getPoints(), plane1->edges[i]->closestNeighbourPoints[k].second,
							plane2->edges[j]->xBounds2D, plane2->edges[j]->yBounds2D, true, 0.01) > 0)
						{
							plane2->edges[j]->closestNeighbourPoints.push_back(plane1->edges[i]->closestNeighbourPoints[k]);
							plane1->edges[i]->closestNeighbourPoints.erase(plane1->edges[i]->closestNeighbourPoints.begin() + k);
							k--;
						}
					}
				}
			}
		}
	}
}

bool areEdgesIntersect(Plane* savedPlane, Plane* newPlane, Edge* savedEdge, Edge* newEdge, bool& hasRelocatedHolePoints)
{
	bool hasFoundIntersection = false;
	auto savedEdgePoints = savedEdge->getPoints();
	auto newEdgePoints = newEdge->getPoints();
	for (size_t j = 0; j < newEdge->pointsWithDir.size(); j++) {
		if (isPointInsidePolygon(savedEdgePoints, newEdge->pointsWithDir[j].first->projected2DPosition, savedEdge->xBounds2D, savedEdge->yBounds2D, true)) {
			hasFoundIntersection = true;
			break;
		}
		for (size_t i = 0; i < savedEdge->pointsWithDir.size(); i++) {
			if (isPointInsidePolygon(newEdgePoints, savedEdge->pointsWithDir[i].first->projected2DPosition, newEdge->xBounds2D, newEdge->yBounds2D, true)) {
				hasFoundIntersection = true;
				break;
			}
			size_t onEdgeType = 0;
			double distanceRatio = 0;
			auto intersectionPos = intersectionOfLines(savedEdge->pointsWithDir[i].first->projected2DPosition, savedEdge->pointsWithDir[(i + 1) %
				savedEdge->pointsWithDir.size()].first->projected2DPosition, newEdge->pointsWithDir[j].first->projected2DPosition,
				newEdge->pointsWithDir[(j + 1) % newEdge->pointsWithDir.size()].first->projected2DPosition, onEdgeType, distanceRatio);
			if (onEdgeType > 0) {
				hasFoundIntersection = true;
				break;
			}
		}
		if (hasFoundIntersection)
			break;
	}
	if (hasFoundIntersection && !hasRelocatedHolePoints)
	{
		relocateHoleNeighbours(savedPlane, newPlane);
		relocateHoleNeighbours(newPlane, savedPlane);
		hasRelocatedHolePoints = true;
	}
	return hasFoundIntersection;
}

bool hasNewEdgeNeighbourInsideSavedEdge(Edge* savedEdge, Edge* newEdge)
{
	for (size_t i = 0; i < newEdge->closestNeighbourPoints.size(); i++) {
		if (isPointInsidePolygon(savedEdge->getPoints(), newEdge->closestNeighbourPoints[i].second, savedEdge->xBounds2D, savedEdge->yBounds2D, true) == 1) {
			return true;
		}
	}
	return false;
}

void mergePolygons(Plane* savedPlane, Edge* savedEdge, Edge* newEdge, std::vector<std::pair<Vec3<double>, Vec3<double>>> allNeighbours,
	/*out*/ std::vector<Edge*>& outputEdges, bool& createdNewPolygon)
{
	std::vector<Point*> savedEdgePoints = savedEdge->getPoints();
	std::vector<Point*> newEdgePoints = newEdge->getPoints();
	bool isHole = newEdge->isHole;	
	for (size_t j = 0; j < newEdgePoints.size(); j++) {
		for (size_t i = 0; i < savedEdgePoints.size(); i++) {
			size_t onEdgeType = 0;
			double distanceRatio = 0;
			if((newEdgePoints[j]->isMarked2 && (newEdgePoints[j] == savedEdgePoints[i] || newEdgePoints[j] == savedEdgePoints[(i + 1) % savedEdgePoints.size()])) ||
				(newEdgePoints[(j + 1) % newEdgePoints.size()]->isMarked2 && (newEdgePoints[(j + 1) % newEdgePoints.size()] == savedEdgePoints[i] ||
					newEdgePoints[(j + 1) % newEdgePoints.size()] == savedEdgePoints[(i + 1) % savedEdgePoints.size()])))
				continue;
			auto intersectionPos = intersectionOfLines(savedEdgePoints[i]->projected2DPosition, savedEdgePoints[(i + 1) %
				savedEdgePoints.size()]->projected2DPosition, newEdgePoints[j]->projected2DPosition,
				newEdgePoints[(j + 1) % newEdgePoints.size()]->projected2DPosition, onEdgeType, distanceRatio);
			if (onEdgeType > 0) {
				if (onEdgeType == 1) {
					Vec3<double> pos3D = savedEdgePoints[i]->position + (savedEdgePoints[(i + 1) %	
						savedEdgePoints.size()]->position - savedEdgePoints[i]->position) * distanceRatio;
					Point* intersectionPoint = new Point(pos3D, horizontalCount + 2, verticalCount + 2, savedEdge->startPoint->plane);
					intersectionPoint->projected2DPosition = intersectionPos;
					intersectionPoint->isCorner = true;
					intersectionPoint->isMarked2 = true;
					savedEdgePoints.insert(savedEdgePoints.begin() + i + 1, intersectionPoint);
					newEdgePoints.insert(newEdgePoints.begin() + j + 1, intersectionPoint);
					i++;
				}
				else 
				{
					bool isOnSavedEdge = onEdgeType < 4;
					size_t selectedIndex = (onEdgeType % 2) == 0 ? (isOnSavedEdge ? i : j) : (isOnSavedEdge ?
						((i + 1) % savedEdgePoints.size()) : ((j + 1) % newEdgePoints.size()));
					auto selectedEdgePoints = isOnSavedEdge ? savedEdgePoints : newEdgePoints;
					auto otherEdgePoints = isOnSavedEdge ? newEdgePoints : savedEdgePoints;
					auto selectedPoint = selectedEdgePoints[selectedIndex];
					bool isAlreadyAdded = false;
					for (size_t k = 0; k < otherEdgePoints.size(); k++) 
					{
						if (otherEdgePoints[k] == selectedPoint) {
							isAlreadyAdded = true;
							break;
						}
					}
					if (!isAlreadyAdded)
					{
						selectedPoint->isMarked2 = true;
						(isOnSavedEdge ? newEdgePoints : savedEdgePoints).insert((isOnSavedEdge ? newEdgePoints : savedEdgePoints).begin() + (isOnSavedEdge ? j : i) + 1,
							selectedPoint);
					}
				}
			}
		}
	}	
	std::vector<Point*> desiredEdges;
	Point* currentPoint = nullptr;
	for (size_t i = 0; i < savedEdgePoints.size(); i++) {
		if (savedEdgePoints[i]->isMarked2) 
		{
			auto pos1 = (savedEdgePoints[i]->projected2DPosition + savedEdgePoints[(i +
				savedEdgePoints.size() - 1) % savedEdgePoints.size()]->projected2DPosition) / 2;
			auto pos2 = (savedEdgePoints[i]->projected2DPosition + savedEdgePoints[(i + 1) %
				savedEdgePoints.size()]->projected2DPosition) / 2;
			if (isPointInsidePolygon(newEdgePoints, pos1, newEdge->xBounds2D, newEdge->yBounds2D, true) ==
				isPointInsidePolygon(newEdgePoints, pos2, newEdge->xBounds2D, newEdge->yBounds2D, true)) 	
			{
				savedEdgePoints[i]->isMarked2 = false;
			}
		}
		if (savedEdgePoints[i]->isMarked2 && savedEdgePoints[(i + 1) % savedEdgePoints.size()]->isMarked2 &&
			savedEdgePoints[i]->verticalIndex != verticalCount + 2 && savedEdgePoints[(i + 1) % 
			savedEdgePoints.size()]->verticalIndex != verticalCount + 2)
		{
			for (size_t j = 0; j < newEdgePoints.size(); j++) {
				if ((newEdgePoints[j] == savedEdgePoints[i] && newEdgePoints[(j + 1) %
					newEdgePoints.size()] == savedEdgePoints[(i + 1) % savedEdgePoints.size()]) || (newEdgePoints[j] == savedEdgePoints[(i + 1) % savedEdgePoints.size()]
						&& newEdgePoints[(j + 1) % newEdgePoints.size()] == savedEdgePoints[i]))
				{
					auto pos1 = (newEdgePoints[(j + newEdgePoints.size() - 1) % newEdgePoints.size()]->projected2DPosition + newEdgePoints[j]->projected2DPosition) / 2;
					auto pos2 = (newEdgePoints[(j + 2) % newEdgePoints.size()]->projected2DPosition + newEdgePoints[j]->projected2DPosition) / 2;
					if (isPointInsidePolygon(savedEdgePoints, pos1, savedEdge->xBounds2D, savedEdge->yBounds2D) ==
						isPointInsidePolygon(savedEdgePoints, pos2, savedEdge->xBounds2D, savedEdge->yBounds2D))
					{
						newEdgePoints[j]->isMarked2 = false;
					}
					newEdgePoints[(j + 1) % newEdgePoints.size()]->isMarked2 = false;
					break;
				}
			}
		}
	}	
	for (size_t i = 0; i < savedEdgePoints.size(); i++) {
		if (isDesiredEdge(savedEdgePoints, newEdgePoints, newEdge->xBounds2D, newEdge->yBounds2D, i, isHole)) {
			desiredEdges.push_back(savedEdgePoints[i]);
			savedEdgePoints[i]->isMarked = true;
			if (!currentPoint && savedEdgePoints[i]->isMarked2)
				currentPoint = savedEdgePoints[i];
		}
	}
	while (desiredEdges.size() > 0) {	
		bool isOnSavedEdge = false;
		Edge* unionPolygon = new Edge();
		unionPolygon->startPoint = currentPoint;
		unionPolygon->isHole = isHole;
		do {
			std::vector<Point*> pointsOnNextEdge;
			auto helperPoint = currentPoint;
			bool helperIsOnEdge = isOnSavedEdge;
			currentPoint = decideIfDesiredEdgeGood(savedEdge, newEdge, currentPoint, savedEdgePoints, newEdgePoints, isOnSavedEdge, true, allNeighbours, pointsOnNextEdge);
			if (!currentPoint)
			{
				pointsOnNextEdge.clear();
				isOnSavedEdge = !isOnSavedEdge;
				currentPoint = decideIfDesiredEdgeGood(savedEdge, newEdge, helperPoint, savedEdgePoints, newEdgePoints, isOnSavedEdge, false, allNeighbours,
					pointsOnNextEdge);
			}
			for (size_t i = 0; i < pointsOnNextEdge.size(); i++) {
				pointsOnNextEdge[i]->isMarked = false;
				unionPolygon->pointsWithDir.push_back({ pointsOnNextEdge[i], -1 });
				auto pos2D = pointsOnNextEdge[i]->projected2DPosition;
				if (pos2D.x < unionPolygon->xBounds2D.first) unionPolygon->xBounds2D.first = pos2D.x;
				if (pos2D.x > unionPolygon->xBounds2D.second) unionPolygon->xBounds2D.second = pos2D.x;
				if (pos2D.y < unionPolygon->yBounds2D.first) unionPolygon->yBounds2D.first = pos2D.y;
				if (pos2D.y > unionPolygon->yBounds2D.second) unionPolygon->yBounds2D.second = pos2D.y;
			}
			isOnSavedEdge = !helperIsOnEdge;
		} while (currentPoint != unionPolygon->pointsWithDir[0].first);
		outputEdges.push_back(unionPolygon);
		currentPoint = nullptr;
		for (size_t i = 0; i < desiredEdges.size(); i++) {
			if (!desiredEdges[i]->isMarked) {
				desiredEdges.erase(desiredEdges.begin() + i);
				i--;
			}
			else if (currentPoint == nullptr && desiredEdges[i]->isMarked2)
				currentPoint = desiredEdges[i];
		}
	}
	if (outputEdges.size() > 0) 
	{
		if (!isHole) {
			while (outputEdges.size() > 1) {
				if (isPointInsidePolygon(outputEdges[1]->getPoints(), outputEdges[0]->pointsWithDir[0].first->projected2DPosition, outputEdges[1]->xBounds2D,
					outputEdges[1]->yBounds2D))
					outputEdges.erase(outputEdges.begin());
				else
					outputEdges.erase(outputEdges.begin() + 1);
			}
		}
		else {
			double maxLength = 0;
			size_t maxIndex = 0;
			for (size_t i = 0; i < outputEdges.size(); i++) {
				double length = 0;
				for (size_t j = 0; j < outputEdges[i]->pointsWithDir.size(); j++) {
					length += (outputEdges[i]->pointsWithDir[j].first->projected2DPosition - outputEdges[i]->pointsWithDir[(j + 1) %
						outputEdges[i]->pointsWithDir.size()].first->projected2DPosition).length();
				}
				if (length > maxLength) {
					maxIndex = i;
					maxLength = length;
				}
			}
			auto helper = outputEdges[maxIndex];
			outputEdges.clear();
			outputEdges.push_back(helper);
		}
	}
	else
	{
		bool hasFoundPointInside = false;
		for (size_t j = 0; j < 2; j++) {
			for (size_t i = 0; i < (j == 0 ? newEdgePoints : savedEdgePoints).size(); i++) {
				if (isPointInsidePolygon((j == 0 ? savedEdgePoints : newEdgePoints), (j == 0 ? newEdgePoints : 
					savedEdgePoints)[i]->projected2DPosition, (j == 0 ? savedEdge : newEdge)->xBounds2D, (j == 0 ? savedEdge : newEdge)->yBounds2D,
					true) == 1) 
				{
					bool hasFoundNeighbourInside = false;
					for (size_t k = 0; k < (((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? newEdge : savedEdge)->closestNeighbourPoints.size(); k++)
					{
						if (isPointInsidePolygon((((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? savedEdge : newEdge)->getPoints(), 
							(((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? newEdge : savedEdge)->closestNeighbourPoints[k].second,
							(((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? savedEdge : newEdge)->xBounds2D,
							(((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) ? savedEdge : newEdge)->yBounds2D, true) == 1)
						{
							hasFoundNeighbourInside = true;
							break;
						}
					}
					hasFoundPointInside = true;
					auto acceptedEdge = (((j == 0 && !newEdge->isHole) || (j == 1 && newEdge->isHole)) && !hasFoundNeighbourInside) ? savedEdge : newEdge;
					Edge* unionPolygon = new Edge();
					unionPolygon->startPoint = acceptedEdge->startPoint;
					unionPolygon->isHole = acceptedEdge->isHole;
					unionPolygon->pointsWithDir = acceptedEdge->pointsWithDir;
					unionPolygon->xBounds2D = acceptedEdge->xBounds2D;
					unionPolygon->yBounds2D = acceptedEdge->yBounds2D;
					outputEdges.push_back(unionPolygon);
					break;
				}
			}
			if (hasFoundPointInside) break;
		}
		if(!hasFoundPointInside)
		{
			createdNewPolygon = false;
			outputEdges.push_back(savedEdge);
			outputEdges.push_back(newEdge);
		}
	}
	if (createdNewPolygon) 
	{
		std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours;
		neighbours.reserve(savedEdge->closestNeighbourPoints.size() + newEdge->closestNeighbourPoints.size());
		neighbours.insert(neighbours.end(), savedEdge->closestNeighbourPoints.begin(), savedEdge->closestNeighbourPoints.end());
		neighbours.insert(neighbours.end(), newEdge->closestNeighbourPoints.begin(), newEdge->closestNeighbourPoints.end());
		calculateNewNeighbours(outputEdges, neighbours);
		for (size_t i = 0; i < outputEdges.size(); i++) {
			outputEdges[i]->intersectedEdges.reserve(savedEdge->intersectedEdges.size() + newEdge->intersectedEdges.size());
			outputEdges[i]->intersectedEdges.insert(outputEdges[i]->intersectedEdges.end(), savedEdge->intersectedEdges.begin(), savedEdge->intersectedEdges.end());
			outputEdges[i]->intersectedEdges.insert(outputEdges[i]->intersectedEdges.end(), newEdge->intersectedEdges.begin(), newEdge->intersectedEdges.end());
			for (size_t j = 0; j < outputEdges[i]->pointsWithDir.size(); j++) {
				outputEdges[i]->pointsWithDir[j].first->plane = savedPlane;
			}
		}
	}
	setPointsMarked(savedEdgePoints, false, false);
	setPointsMarked(newEdgePoints, false, false);
}

void mergeArrayOfEdges(Plane* savedPlane, std::vector<Edge*>& createdEdges, std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours, bool dbg = false)
{
	std::vector<Edge*> tempCreatedEdges;
	bool createdNewPolygon = true;
	bool foundNewIntersection = true;
	int counter = 0;
	for (size_t i = 0; i < createdEdges.size(); i++) 
	{
		for (size_t j = 0; j < createdEdges[i]->intersectedEdges.size(); j++) {
			changeBaseTo2D(createdEdges[i]->intersectedEdges[j], { savedPlane->normal, savedPlane->pointDirections.first });
		}
	}
	while (foundNewIntersection) {
		foundNewIntersection = false;
		while (createdEdges.size() > 0) {
			
			for (size_t i = 1; i < createdEdges.size(); i++) {
				if (createdEdges[0]->isHole == createdEdges[i]->isHole) {
					std::vector<Edge*> innerCreatedEdges;
					/*if (false && dbg && i == 24 && counter == 13) {// currentFrame == 4 && savedEdge->isHole && counter == 3) {
						savedPoints.clear();
						addedPoints.clear();
						points.clear();
						changeBaseTo2D(createdEdges[0], { savedPlane->normal, savedPlane->pointDirections.first });
						changeBaseTo2D(createdEdges[i], { savedPlane->normal, savedPlane->pointDirections.first });
						for (size_t y = 0; y < createdEdges[0]->pointsWithDir.size(); y++) {
							createdEdges[0]->pointsWithDir[y].first->cornerId = currentCornerId;
							createdEdges[0]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
							savedPoints.push_back(createdEdges[0]->pointsWithDir[y].first);
							createdEdges[0]->pointsWithDir[y].first->position = createdEdges[0]->pointsWithDir[y].first->projected2DPosition;
							currentCornerIndex++;
						}
						currentCornerId++;
						currentCornerIndex = 0;
						for (size_t y = 0; y < createdEdges[i]->pointsWithDir.size(); y++) {
							createdEdges[i]->pointsWithDir[y].first->cornerId = currentCornerId;
							createdEdges[i]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
							savedPoints.push_back(createdEdges[i]->pointsWithDir[y].first);
							createdEdges[i]->pointsWithDir[y].first->position = createdEdges[i]->pointsWithDir[y].first->projected2DPosition;
							currentCornerIndex++;
						}

						for (size_t x = 0; x < createdEdges.size(); x++) {
							for (size_t y = 0; y < createdEdges[x]->closestNeighbourPoints.size(); y++) {
								savedPoints.push_back(new Point(createdEdges[x]->closestNeighbourPoints[y].first, 0, 0, nullptr));
							}
							currentCornerIndex = 0;
							for (size_t y = 0; y < createdEdges[x]->pointsWithDir.size(); y++) {
								createdEdges[x]->pointsWithDir[y].first->cornerId = 0;
								createdEdges[x]->pointsWithDir[y].first->cornerIndex = 0;
							}
							for (size_t y = 0; y < createdEdges[x]->pointsWithDir.size(); y++) {
								if (createdEdges[x]->pointsWithDir[y].first->isCorner && createdEdges[x]->pointsWithDir[y].first->cornerId == 0) {
									createdEdges[x]->pointsWithDir[y].first->cornerId = currentCornerId;
									createdEdges[x]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
									savedPoints.push_back(createdEdges[x]->pointsWithDir[y].first);
									currentCornerIndex++;
								}
								else if (createdEdges[x]->pointsWithDir[y].first->cornerId != 0) {
									std::cout << "asd";
								}
							}
							currentCornerId++;
						}
						writeData(4);
 						std::cout << "asd";
					}*/
					createdNewPolygon = true;
					mergePolygons(savedPlane, createdEdges[0], createdEdges[i], neighbours, innerCreatedEdges, createdNewPolygon);

					if (createdNewPolygon) {
						foundNewIntersection = true;
						for (size_t m = 0; m < innerCreatedEdges.size(); m++) {
							tempCreatedEdges.push_back(innerCreatedEdges[m]);
						}
						createdEdges.erase(createdEdges.begin() + i);
						createdEdges.erase(createdEdges.begin());
						break;
					}
				}
				else
					createdNewPolygon = false;
			}
			counter++;
			if (!createdNewPolygon || createdEdges.size() == 1) {
				tempCreatedEdges.push_back(createdEdges[0]);
				createdEdges.erase(createdEdges.begin());
			}
		}
		createdEdges = tempCreatedEdges;
		tempCreatedEdges.clear();
	}
}

bool mergeTwoPlaneEdges(Plane* newPlane, Plane* savedPlane, std::vector<Edge*>& newEdges, std::vector<Edge*>& savedEdges, bool isOuter)
{
	bool hasFoundIntersection = false;
	bool hasRelocatedNeighbours = false;
	
	for (int k = 0; k < newEdges.size(); k++) {
		for (int l = 0; l < savedEdges.size(); l++) {
			if (newEdges[k]->isHole == savedEdges[l]->isHole &&
				newEdges[k]->canIntersectWithEdge(savedEdges[l])) {
				std::vector<Edge*> createdEdges;
				if (areEdgesIntersect(savedPlane, newPlane, savedEdges[l], newEdges[k], hasRelocatedNeighbours)) 
				{
					bool createdNewPolygon = true;					
					if (hasNewEdgeNeighbourInsideSavedEdge(savedEdges[l], newEdges[k])) 
					{
						std::vector<std::pair<Vec3<double>, Vec3<double>>> neighbours;
						neighbours.reserve(savedEdges[l]->closestNeighbourPoints.size() + newEdges[k]->closestNeighbourPoints.size());
						neighbours.insert(neighbours.end(), savedEdges[l]->closestNeighbourPoints.begin(), savedEdges[l]->closestNeighbourPoints.end());
						neighbours.insert(neighbours.end(), newEdges[k]->closestNeighbourPoints.begin(), newEdges[k]->closestNeighbourPoints.end());
						createdEdges = savedEdges[l]->intersectedEdges;
						createdEdges.push_back(newEdges[k]);
						mergeArrayOfEdges(savedPlane, createdEdges, neighbours, currentFrame == 34 && k == 0 && l == 9);
						createdNewPolygon = true;
						calculateNewNeighbours(createdEdges, neighbours);
					}
					else 
					{
						mergePolygons(savedPlane, savedEdges[l], newEdges[k], {}, createdEdges, createdNewPolygon);
					}
					if (createdNewPolygon) {
						savedEdges.erase(savedEdges.begin() + l);
						newEdges.erase(newEdges.begin() + k);
						l--;
						k--;
						//if (isOuter)
							//mergeTwoPlanes(savedPlane, newPlane, createdEdges, newEdges, false);
						hasFoundIntersection = true;
						for (size_t m = 0; m < createdEdges.size(); m++) {
							newEdges.push_back(createdEdges[m]);
						}
						break;
					}
				}
			}
		}
	}
	/*if (false && currentFrame == 36 && savedEdges.size() > 0)
	{
		for (size_t y = 0; y < savedEdges[0]->pointsWithDir.size(); y++) {
			savedEdges[0]->pointsWithDir[y].first->cornerId = currentCornerId;
			savedEdges[0]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
			savedPoints.push_back(savedEdges[0]->pointsWithDir[y].first);
			//savedEdges[0]->pointsWithDir[y].first->position = savedEdges[0]->pointsWithDir[y].first->projected2DPosition;
			currentCornerIndex++;
		}
		currentCornerId++;
		currentCornerIndex = 0;
		for (size_t i = 0; i < newEdges.size(); i++) {
			for (size_t y = 0; y < newEdges[i]->pointsWithDir.size(); y++) {
				newEdges[i]->pointsWithDir[y].first->cornerId = currentCornerId;
				newEdges[i]->pointsWithDir[y].first->cornerIndex = currentCornerIndex;
				savedPoints.push_back(newEdges[i]->pointsWithDir[y].first);
				//newEdges[i]->pointsWithDir[y].first->position = newEdges[i]->pointsWithDir[y].first->projected2DPosition;
				currentCornerIndex++;
			}
			currentCornerId++;
		}
		writeData(4);
		auto temp = 0;
	}*/
	if (hasFoundIntersection) 
	{
		mergeArrayOfEdges(savedPlane, newEdges, {});
	}
	return hasFoundIntersection;
}

void filterFittedPlanes()
{
	int dbgCounter = 0;
	double normalDistTreshold = 0.1;
	double distTreshold = 0.01;
	double newDirectionTreshold = 0.1;
	for (size_t i = 0; i < savedPlanes.size(); i++) {
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
			bool deletedPoint = true;
			dbgCounter = 0;
			while (deletedPoint && savedPlanes[i]->edges[j]->pointsWithDir.size() > 2)
			{
				dbgCounter++;
				deletedPoint = false;
				auto normal = Vec3<double>::normalize(savedPlanes[i]->edges[j]->pointsWithDir[savedPlanes[i]->edges[j]->pointsWithDir.size() - 1].first->projected2DPosition -
					savedPlanes[i]->edges[j]->pointsWithDir[savedPlanes[i]->edges[j]->pointsWithDir.size() - 2].first->projected2DPosition);
				normal = { -normal.y, normal.x, 0 };
				auto linePoint = savedPlanes[i]->edges[j]->pointsWithDir[savedPlanes[i]->edges[j]->pointsWithDir.size() - 2].first->projected2DPosition;
				for (size_t k = 0; k < savedPlanes[i]->edges[j]->pointsWithDir.size(); k++)
				{
					auto verticalIndex = savedPlanes[i]->edges[j]->pointsWithDir[k].first->verticalIndex;
					auto previousIndex = (k + savedPlanes[i]->edges[j]->pointsWithDir.size() - 1) % savedPlanes[i]->edges[j]->pointsWithDir.size();
					auto previousPoint = savedPlanes[i]->edges[j]->pointsWithDir[previousIndex].first;
					auto previousVerticalIndex = previousPoint->verticalIndex;
					double dist = (savedPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition - previousPoint->projected2DPosition).length();
					if (dist < distTreshold)
					{
						if (previousVerticalIndex < verticalCount || verticalIndex >= previousVerticalIndex)
						{
							savedPlanes[i]->edges[j]->pointsWithDir.erase(savedPlanes[i]->edges[j]->pointsWithDir.begin() + previousIndex);
							if (previousIndex < k) k--;
						}
						else
						{
							savedPlanes[i]->edges[j]->pointsWithDir.erase(savedPlanes[i]->edges[j]->pointsWithDir.begin() + k);
							k--;
						}
						deletedPoint = true;
						continue;
					}
					double normalDist = abs(Vec3<double>::dot_product(normal, savedPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition - linePoint));
					if (normalDist < normalDistTreshold)
					{
						if (previousVerticalIndex < verticalCount || verticalIndex >= previousVerticalIndex) 
						{
							savedPlanes[i]->edges[j]->pointsWithDir.erase(savedPlanes[i]->edges[j]->pointsWithDir.begin() + previousIndex);
							if (previousIndex < k) k--;
						}
						else if ((Vec3<double>::normalize(savedPlanes[i]->edges[j]->pointsWithDir[(k + 1) % savedPlanes[i]->edges[j]->pointsWithDir.size()].first->projected2DPosition -
							previousPoint->projected2DPosition) - Vec3<double>::normalize(savedPlanes[i]->edges[j]->pointsWithDir[(k
								+ 1) % savedPlanes[i]->edges[j]->pointsWithDir.size()].first->projected2DPosition -
								savedPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition)).length() < newDirectionTreshold) 
						{
							savedPlanes[i]->edges[j]->pointsWithDir.erase(savedPlanes[i]->edges[j]->pointsWithDir.begin() + k);
							k--;
						}
						else
							continue;
						deletedPoint = true;
					}
					else {
						normal = Vec3<double>::normalize(savedPlanes[i]->edges[j]->pointsWithDir[k].first->projected2DPosition - previousPoint->projected2DPosition);
						normal = { -normal.y, normal.x, 0 };
						linePoint = previousPoint->projected2DPosition;
					}
				}
			} 
			if (savedPlanes[i]->edges[j]->pointsWithDir.size() < 3) 
			{
				savedPlanes[i]->edges.erase(savedPlanes[i]->edges.begin() + j);
				j--;
			}
		}
	}
}

void checkIfHolesAreContained()
{
	for (size_t i = 0; i < savedPlanes.size(); i++) 
	{
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
			if (savedPlanes[i]->edges[j]->isHole) {
				bool isCointained = false;
				for (size_t k = 0; k < savedPlanes[i]->edges.size(); k++) {
					if (!savedPlanes[i]->edges[k]->isHole) 
					{
						for (size_t l = 0; l < savedPlanes[i]->edges[j]->pointsWithDir.size(); l++) 
						{
							if (isPointInsidePolygon(savedPlanes[i]->edges[k]->getPoints(),
								savedPlanes[i]->edges[j]->pointsWithDir[l].first->projected2DPosition, savedPlanes[i]->edges[k]->xBounds2D,
								savedPlanes[i]->edges[k]->yBounds2D)) {
								isCointained = true;
							}
							else
							{
								isCointained = false;
								break;
							}
						}
						if (isCointained)
						{
							double notInUse = 0;
							for (size_t l = 0; l < savedPlanes[i]->edges[j]->pointsWithDir.size(); l++)
							{
								for (size_t m = 0; m < savedPlanes[i]->edges[k]->pointsWithDir.size(); m++) 
								{
									size_t onEdgeType = 0;
									intersectionOfLines(savedPlanes[i]->edges[j]->pointsWithDir[l].first->projected2DPosition, 
										savedPlanes[i]->edges[j]->pointsWithDir[(l + 1) % savedPlanes[i]->edges[j]->pointsWithDir.size()].first->projected2DPosition,
										savedPlanes[i]->edges[k]->pointsWithDir[m].first->projected2DPosition,
										savedPlanes[i]->edges[k]->pointsWithDir[(m + 1) % savedPlanes[i]->edges[k]->pointsWithDir.size()].first->projected2DPosition, onEdgeType, notInUse);
									if (onEdgeType > 0)
									{
										isCointained = false;
										break;
									}
								}
								if (!isCointained)
									break;
							}
							break;
						}							
					}
				}
				savedPlanes[i]->edges[j]->isInUse = isCointained;
			}
		}
	}
}

bool mergeTwoPlanes(Plane* plane, Plane* savedPlane)
{
	bool hasFoundIntersectionAtAll = false;
	double normalTreshold = 0.1;
	double distanceTreshold = 0.1;
	if ((plane->normal - savedPlane->normal).length() < normalTreshold &&
		(abs(Vec3<double>::dot_product(plane->normal, savedPlane->furthestNormalPoints.first - plane->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(plane->normal, savedPlane->furthestNormalPoints.second - plane->planePointPos)) < distanceTreshold || 
			abs(Vec3<double>::dot_product(savedPlane->normal, plane->furthestNormalPoints.first - savedPlane->planePointPos)) < distanceTreshold ||
			abs(Vec3<double>::dot_product(savedPlane->normal, plane->furthestNormalPoints.second - savedPlane->planePointPos)) < distanceTreshold)) 
	{
		for (int k = 0; k < plane->edges.size(); k++) {
			changeBaseTo2D(plane->edges[k], { savedPlane->normal, savedPlane->pointDirections.first });
		}
		for (int k = 0; k < savedPlane->edges.size(); k++) {
			changeBaseTo2D(savedPlane->edges[k], { savedPlane->normal, savedPlane->pointDirections.first });
		}
		bool hasFoundIntersection = mergeTwoPlaneEdges(plane, savedPlane, plane->edges, savedPlane->edges, true);
		if (hasFoundIntersection) {
			hasFoundIntersectionAtAll = true;
			for (size_t k = 0; k < plane->edges.size(); k++) {
				for (size_t l = 0; l < plane->edges[k]->pointsWithDir.size(); l++) {
					plane->edges[k]->pointsWithDir[l].first->plane = savedPlane;
				}
				savedPlane->edges.push_back(plane->edges[k]);
			}
			plane->edges.clear();
			savedPlane->isNewlyCreated = false;
		}
		else {
			for (int k = 0; k < plane->edges.size(); k++) {
				changeBaseTo2D(plane->edges[k], { plane->normal, plane->pointDirections.first });
			}
		}
	}
	return hasFoundIntersectionAtAll;
}

void fitPlanes()
{
	if (currentFrame == 36) {
		/*or (size_t i = 0; i < planes.size(); i++) {
			for (size_t j = 0; j < planes[i]->edges.size(); j++) {
				for (auto p : planes[i]->edges[j]->closestNeighbourPoints)
				{
					points.push_back(new Point(p.first, 0, 0, nullptr));
				}
			}
		}
		for (size_t i = 0; i < savedPlanes.size(); i++) {
			for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
				for (auto p : savedPlanes[i]->edges[j]->closestNeighbourPoints) {
					points.push_back(new Point(p.first, 0, 0, nullptr));
				}
			}
		}*/
		std::cout << "asd";
		//savedPlanes = planes;
		//return;
	}
	for (int i = 0; i < savedPlanes.size(); i++) {
		savedPlanes[i]->calculateAvaragePointPos();
	}
	for (int i = 0; i < planes.size(); i++) {
		for (int j = 0; j < savedPlanes.size(); j++) {
			mergeTwoPlanes(planes[i], savedPlanes[j]);
		}
	}
	for (int i = 0; i < (int)savedPlanes.size() - 1; i++) {
		for (int j = i + 1; j < (int)savedPlanes.size(); j++) {
			bool foundConnection = mergeTwoPlanes(savedPlanes[i], savedPlanes[j]);
			if (foundConnection) {
				savedPlanes.erase(savedPlanes.begin() + i);
				i--;
				break;
			}
		}
	}
	for (int i = 0; i < savedPlanes.size(); i++) {
		if (savedPlanes[i]->isNewlyCreated) {
			savedPlanes.erase(savedPlanes.begin() + i);
			i--;
		}
	}
	for (int i = 0; i < planes.size(); i++) {
		if (planes[i]->edges.size() > 0) {
			savedPlanes.push_back(planes[i]);
		}
	}
	filterFittedPlanes();
	checkIfHolesAreContained();
}

void convexSegmentation()
{	
	double notInUseRatio = 0;
	const std::pair<double, double> acceptAngle = { 181, 359 };
	size_t currentConvexId = 1;
	for (size_t i = 0; i < savedPlanes.size(); i++) {
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
			for (size_t k = 0; k < savedPlanes[i]->edges[j]->pointsWithDir.size(); k++) 
			{
				savedPlanes[i]->edges[j]->pointsWithDir[k].first->convexId.clear();
				savedPlanes[i]->edges[j]->pointsWithDir[k].first->convexIndex.clear();
			}
		}
		savedPlanes[i]->convexFaces.clear();
	}
	for (size_t i = 0; i < savedPlanes.size(); i++) {
		std::vector<Edge*> holeEdges;
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {			
			if (savedPlanes[i]->edges[j]->isHole && savedPlanes[i]->edges[j]->isInUse) {
				holeEdges.push_back(savedPlanes[i]->edges[j]);
			}
 		}		
		for (size_t x = 0; x < savedPlanes[i]->edges.size(); x++) {
			if (!savedPlanes[i]->edges[x]->isHole) {
				std::vector<Point*> remainingPoints;
				for (size_t k = 0; k < savedPlanes[i]->edges[x]->pointsWithDir.size(); k++) {
					remainingPoints.push_back(savedPlanes[i]->edges[x]->pointsWithDir[k].first);
				}
				if (!isClockwise(remainingPoints)) 
				{
					for (size_t k = 0; k < remainingPoints.size(); k++) 
					{
						remainingPoints.push_back(remainingPoints[remainingPoints.size() - 1 - k]);
						remainingPoints.erase(remainingPoints.begin() + remainingPoints.size() - 2 - k);
					}
				}
				while (remainingPoints.size() > 3) {
					std::vector<Point*> remainingPointsHelper(remainingPoints);
					std::vector<Point*> L = { remainingPointsHelper[0], remainingPointsHelper[1] };
					std::vector<Point*> remainingPointsHelperSave;
					std::vector<Point*> LSave;

					remainingPointsHelper.erase(remainingPointsHelper.begin(), remainingPointsHelper.begin() + 2);
					std::pair<double, double> xBounds = { std::min(L[0]->projected2DPosition.x, L[1]->projected2DPosition.x),
						std::max(L[0]->projected2DPosition.x, L[1]->projected2DPosition.x) };
					std::pair<double, double> yBounds = { std::min(L[0]->projected2DPosition.y, L[1]->projected2DPosition.y),
						std::max(L[0]->projected2DPosition.y, L[1]->projected2DPosition.y) };

					for (int j = 0; j < 2; j++) {
						bool isForward = j == 0;
						while (remainingPointsHelper.size() > 0) {
							auto newPoint = isForward ? remainingPointsHelper[0] : remainingPointsHelper[remainingPointsHelper.size() - 1];
							auto v1 = isForward ? L[L.size() - 1]->projected2DPosition - L[L.size() - 2]->projected2DPosition :
								(L[0]->projected2DPosition - L[1]->projected2DPosition);
							auto v2 = isForward ? newPoint->projected2DPosition - L[L.size() - 1]->projected2DPosition :
								(newPoint->projected2DPosition - L[0]->projected2DPosition);
							auto vecToBegin = isForward ? L[0]->projected2DPosition - newPoint->projected2DPosition :
								(L[L.size() - 1]->projected2DPosition - newPoint->projected2DPosition);
							auto vecAtBegin = isForward ? L[1]->projected2DPosition - L[0]->projected2DPosition :
								(L[L.size() - 2]->projected2DPosition - L[L.size() - 1]->projected2DPosition);
							auto temp = angleOfVectors(v1 * -1, v2, !isForward);
							auto temp2 = angleOfVectors(v2 * -1, vecToBegin, !isForward);
							auto temp3 = angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward);
							if ((angleOfVectors(v1 * -1, v2, !isForward) <= acceptAngle.first || angleOfVectors(v1 * -1, v2, !isForward) >= 
								acceptAngle.second) &&
								(angleOfVectors(v2 * -1, vecToBegin, !isForward) <= acceptAngle.first || angleOfVectors(v2 * -1, vecToBegin, !isForward) >=
								acceptAngle.second) && 
								(angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward) <= acceptAngle.first || angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward) >=
								acceptAngle.second)) {
								L.insert(isForward ? L.end() : L.begin(), newPoint);
								remainingPointsHelper.erase(isForward ? remainingPointsHelper.begin() : remainingPointsHelper.end() - 1);
								if (newPoint->projected2DPosition.x < xBounds.first)
									xBounds.first = newPoint->projected2DPosition.x;
								if (newPoint->projected2DPosition.x > xBounds.second)
									xBounds.second = newPoint->projected2DPosition.x;
								if (newPoint->projected2DPosition.y < yBounds.first)
									yBounds.first = newPoint->projected2DPosition.y;
								if (newPoint->projected2DPosition.y > yBounds.second)
									yBounds.second = newPoint->projected2DPosition.y;
							}
							else {
								if (L.size() > 2) {
									bool containsCorner = true;
									while (containsCorner && L.size() > 2) {
										containsCorner = false;
										for (size_t k = 0; k < remainingPointsHelper.size(); k++) {
											if (isPointInsidePolygon(L, remainingPointsHelper[k]->projected2DPosition, xBounds, yBounds)) {
												containsCorner = true;
												break;
											}
										}
										if (containsCorner) {
											remainingPointsHelper.insert(isForward ? remainingPointsHelper.begin() : remainingPointsHelper.end(),
												isForward ? L[L.size() - 1] : L[0]);
											L.erase(isForward ? L.end() - 1 : L.begin());
										}
									}
									int absoluteClosestEdgeIndex = -1;
									size_t absoluteClosestPointIndex = 0;
									bool isNewPointFound = true;
									auto lastPointPos = L[L.size() - 1]->projected2DPosition;
									if (L.size() > 2) {
										while (isNewPointFound) {
											double minIntersectionDistance = 1000;
											int closestEdgeIndex = -1;
											size_t closestPointIndex = 0;
											auto LTemp = L;
											if (absoluteClosestEdgeIndex != -1)
												LTemp.insert(LTemp.begin(), holeEdges[absoluteClosestEdgeIndex]->pointsWithDir[absoluteClosestPointIndex].first);
											isNewPointFound = false;
											for (size_t l = 0; l < holeEdges.size(); l++) {
												std::vector<bool> isPointsInside;
												auto edgePoints = holeEdges[l];
												size_t holeType = 0;
												for (size_t m = 0; m < edgePoints->pointsWithDir.size(); m++) {
													bool isHolePointInside = isPointInsidePolygon(LTemp,
														edgePoints->pointsWithDir[m].first->projected2DPosition, xBounds, yBounds);
													isPointsInside.push_back(isHolePointInside);
													if (isHolePointInside && holeType == 0)
														holeType = 1;
													if (m > 0 && isPointsInside[m - 1] != isPointsInside[m])
														holeType = 2;
												}
												if (holeType > 0) {
													for (size_t m = 0; m < edgePoints->pointsWithDir.size(); m++) {
														if (holeType == 2 && isPointsInside[m] != isPointsInside[(m + 1) % isPointsInside.size()] &&
															LTemp[0] != edgePoints->pointsWithDir[m].first && LTemp[0] != 
															edgePoints->pointsWithDir[(m + 1) % isPointsInside.size()].first) {
															size_t onEdgeType = 0;
															auto intersection = intersectionOfLines(LTemp[0]->projected2DPosition, lastPointPos, 
																edgePoints->pointsWithDir[m].first->projected2DPosition,
																edgePoints->pointsWithDir[(m + 1) % isPointsInside.size()].first->projected2DPosition,
																onEdgeType, notInUseRatio);
															if (onEdgeType > 0 && (intersection - lastPointPos).length() < minIntersectionDistance) {
																minIntersectionDistance = (intersection - lastPointPos).length();
																closestEdgeIndex = l;
																if (((edgePoints->pointsWithDir[m].first->projected2DPosition - lastPointPos).length() <
																	(edgePoints->pointsWithDir[(m + 1) % isPointsInside.size()].first->projected2DPosition
																		- lastPointPos).length() &&
																	isPointInsidePolygon(LTemp, edgePoints->pointsWithDir[m].first->projected2DPosition,
																		xBounds, yBounds)) || ((edgePoints->pointsWithDir[m].first->projected2DPosition
																			- lastPointPos).length() >= (edgePoints->pointsWithDir[(m + 1) % 
																				isPointsInside.size()].first->projected2DPosition - lastPointPos).length() &&
																		!isPointInsidePolygon(LTemp, edgePoints->pointsWithDir[(m + 1) %
																			isPointsInside.size()].first->projected2DPosition,
																			xBounds, yBounds)))
																	closestPointIndex = m;
																else
																	closestPointIndex = ((m + 1) % isPointsInside.size());
															}
														}
														else if (holeType == 1 && absoluteClosestEdgeIndex == -1) {
															if ((edgePoints->pointsWithDir[m].first->projected2DPosition - lastPointPos).length() <
																minIntersectionDistance) {
																minIntersectionDistance = (edgePoints->pointsWithDir[m].first->projected2DPosition -
																	lastPointPos).length();
																closestEdgeIndex = l;
																closestPointIndex = m;
															}
														}
													}
												}
											}
											if (closestEdgeIndex != -1) {
												absoluteClosestEdgeIndex = closestEdgeIndex;
												absoluteClosestPointIndex = closestPointIndex;
												isNewPointFound = true;
											}
										}
										if (absoluteClosestEdgeIndex >= 0) {
											if (!isForward) {
												remainingPointsHelper = remainingPointsHelperSave;
												L = LSave;
											}
											remainingPointsHelper.insert(remainingPointsHelper.begin(), L[L.size() - 1]);
											for (size_t l = 0; l < holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size() + 1; l++) {
												remainingPointsHelper.insert(remainingPointsHelper.begin() + l,
													holeEdges[absoluteClosestEdgeIndex]->pointsWithDir[(absoluteClosestPointIndex + l) % 
													holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size()].first);
											}
											for (size_t m = 0; m < remainingPoints.size(); m++) {
												if (remainingPoints[m] == L[L.size() - 1]) {
													remainingPoints.insert(remainingPoints.begin() + m + 1, L[L.size() - 1]);
													for (size_t l = 0; l < holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size() + 1; l++) {
														remainingPoints.insert(remainingPoints.begin() + m + 1 + l,
															holeEdges[absoluteClosestEdgeIndex]->pointsWithDir[(absoluteClosestPointIndex + l) %
															holeEdges[absoluteClosestEdgeIndex]->pointsWithDir.size()].first);
													}
													break;
												}
											}
											holeEdges.erase(holeEdges.begin() + absoluteClosestEdgeIndex);
											j = -1;
											break;
										}
									}
								}
								if (isForward) {
									LSave = L;
									remainingPointsHelperSave = remainingPointsHelper;
								}
								else {
									if (L.size() > 2) {
										remainingPointsHelper.insert(remainingPointsHelper.begin(), L[L.size() - 1]);
										remainingPointsHelper.insert(remainingPointsHelper.end(), L[0]);
									}
									else {
										L.clear();
										remainingPoints.push_back(remainingPoints[0]);
										remainingPoints.erase(remainingPoints.begin());
										remainingPointsHelper = remainingPoints;
									}
								}
								break;
							}
						}
					}
					if (remainingPointsHelper.size() == 0) //last convex polygon hole test
					{
						double minDistance = 1000;
						int closestEdgeIndex = -1;
						size_t closestPointIndex = 0;
						for (size_t l = 0; l < holeEdges.size(); l++) {
							auto edgePoints = holeEdges[l];
							for (size_t m = 0; m < edgePoints->pointsWithDir.size(); m++) {
								if (isPointInsidePolygon(L, edgePoints->pointsWithDir[m].first->projected2DPosition, xBounds, yBounds) && minDistance >
									(L[0]->projected2DPosition - edgePoints->pointsWithDir[m].first->projected2DPosition).length()) {
									minDistance = (L[0]->projected2DPosition - edgePoints->pointsWithDir[m].first->projected2DPosition).length();
									closestEdgeIndex = l;
									closestPointIndex = m;
								}
							}
						}
						if (closestEdgeIndex >= 0) {
							remainingPointsHelper.insert(remainingPointsHelper.begin(), L[0]);
							for (size_t l = 0; l < holeEdges[closestEdgeIndex]->pointsWithDir.size() + 1; l++) {
								remainingPointsHelper.insert(remainingPointsHelper.begin() + l + 1,
									holeEdges[closestEdgeIndex]->pointsWithDir[(closestPointIndex + l) % holeEdges[closestEdgeIndex]->pointsWithDir.size()].first);
							}
							for (size_t l = 0; l < L.size(); l++) {
								remainingPointsHelper.insert(remainingPointsHelper.end(), L[l]);
							}
							L.clear();
							holeEdges.erase(holeEdges.begin() + closestEdgeIndex);
							remainingPoints = remainingPointsHelper;
						}
					}
					if (L.size() > 0) {
						std::vector<Point*> convexFace;
						for (size_t j = 0; j < L.size(); j++) {
							L[j]->convexId.push_back(currentConvexId);
							L[j]->convexIndex.push_back(j);
							convexFace.push_back(L[j]);
						}
						savedPlanes[i]->convexFaces.push_back(convexFace);
						if (currentFrame == 36 && i == 7 && savedPlanes[i]->convexFaces.size() == 81)
						{
							if (false) {
								savedPoints.clear();
								addedPoints.clear();
								points.clear();
								//changeBaseTo2D(createdEdges[0], { savedPlane->normal, savedPlane->pointDirections.first });
								//changeBaseTo2D(createdEdges[i], { savedPlane->normal, savedPlane->pointDirections.first });
								for (size_t y = 0; y < savedPlanes[i]->edges.size(); y++) {
									currentCornerIndex = 0;
									for (size_t j = 0; j < savedPlanes[i]->edges[y]->pointsWithDir.size(); j++) {
										savedPlanes[i]->edges[y]->pointsWithDir[j].first->cornerId = currentCornerId;
										savedPlanes[i]->edges[y]->pointsWithDir[j].first->cornerIndex = currentCornerIndex;
										savedPoints.push_back(savedPlanes[i]->edges[y]->pointsWithDir[j].first);
										savedPlanes[i]->edges[y]->pointsWithDir[j].first->position = savedPlanes[i]->edges[y]->pointsWithDir[j].first->projected2DPosition;
										currentCornerIndex++;
									}
									currentCornerId++;
								}
								writeData(4);
								std::cout << "asd";
							}
							//return;
						}
						remainingPoints = remainingPointsHelper;
						currentConvexId++;
					}
				}
				if (remainingPoints.size() == 3) 
				{
					std::vector<Point*> convexFace;
					for (size_t j = 0; j < remainingPoints.size(); j++) {
						remainingPoints[j]->convexId.push_back(currentConvexId);
						remainingPoints[j]->convexIndex.push_back(j);
						convexFace.push_back(remainingPoints[j]);
					}
					savedPlanes[i]->convexFaces.push_back(convexFace);
					currentConvexId++;
				}
			}
		}
	}
}

void saveSavedPoints()
{
	for (size_t i = 0; i < savedPlanes.size(); i++) {
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
			//deleteSelfIntersections(savedPlanes[i]->edges[j]);
			if (savedPlanes[i]->edges[j]->pointsWithDir.size() < 3) 				
			{
				savedPlanes[i]->edges.erase(savedPlanes[i]->edges.begin() + j);
				j--;
				continue;
			}
			for (size_t k = 0; k < savedPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				savedPoints.push_back(savedPlanes[i]->edges[j]->pointsWithDir[k].first);
			}
		}
	}
}

void exportObjects(size_t pointCloudIndex)
{	
	if (currentFrame == 21) {
		std::cout << "asd";
	}
	std::string name = "C:\\Users\\ungbo\\Desktop\\BME\\_Diplomamunka\\Diplomamunka\\Diplomamunka\\Assets\\Resources\\Generated_Models_" + 
		((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + "\\processed_obj_0.obj";
	std::ifstream f(name);
	size_t counter = 0;
	while (f.good())
	{
		f.close();
		remove(name.c_str());
		counter++;
		name = "C:\\Users\\ungbo\\Desktop\\BME\\_Diplomamunka\\Diplomamunka\\Diplomamunka\\Assets\\Resources\\Generated_Models_" +
			((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + "\\processed_obj_"
			+ std::to_string(counter) + ".obj";
		f = std::ifstream(name.c_str());
	}
	size_t objCounter = 0;
	for (size_t i = 0; i < savedPlanes.size(); i++) {
		std::vector<Point*> corners;
		currentCornerIndex = 0;		
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
			for (size_t k = 0; k < savedPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (savedPlanes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
					savedPlanes[i]->edges[j]->pointsWithDir[k].first->cornerId = currentCornerId;
					savedPlanes[i]->edges[j]->pointsWithDir[k].first->cornerIndex = currentCornerIndex;
					corners.push_back(savedPlanes[i]->edges[j]->pointsWithDir[k].first);
					currentCornerIndex++;
				}
			}
			currentCornerId++;
		}
		if (corners.size() < 3)
			continue;
		//std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models_test/processed_obj_"
			//+ std::to_string(objCounter) + ".obj");
		std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models_" +
			((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + "/processed_obj_"
			+ std::to_string(objCounter) + ".obj");
		MyFile << "o Mesh" << std::endl;
		for (size_t k = 0; k < corners.size(); k++) {
			MyFile << "v " << std::to_string(-corners[k]->position.x) << " " << std::to_string(corners[k]->position.y)
				<< " " << std::to_string(corners[k]->position.z) << std::endl;
		}

		for (size_t j = 0; j < savedPlanes[i]->convexFaces.size(); j++) {
			MyFile << "f ";
			for (size_t k = 0; k < savedPlanes[i]->convexFaces[j].size(); k++) {
				MyFile << savedPlanes[i]->convexFaces[j][k]->cornerIndex + 1 << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (int k = savedPlanes[i]->convexFaces[j].size() - 1; k >= 0; k--) {
				MyFile << savedPlanes[i]->convexFaces[j][k]->cornerIndex + 1 << " ";
			}
			MyFile << std::endl;
		}
		
		for (size_t j = 0; j < savedPlanes[i]->edges.size(); j++) {
			int indexShift = -1;
			for (size_t k = 0; k < savedPlanes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (savedPlanes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
					if (indexShift == -1)
						indexShift = savedPlanes[i]->edges[j]->pointsWithDir[k].first->cornerIndex;
					savedPlanes[i]->edges[j]->pointsWithDir[k].first->cornerIndex -= indexShift;
				}
			}
		}
		MyFile << std::endl;
		MyFile.close();
		objCounter++;
	}
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i]) {
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
				';' << (points[i]->plane ? points[i]->plane->id : 0) << ';' << points[i]->outlineId << ';' << points[i]->cornerId
				<< ';' << points[i]->cornerIndex;
			for (size_t j = 0; j < points[i]->convexId.size(); j++) {
				MyFile << ";" << points[i]->convexId[j] << ";" << points[i]->convexIndex[j];
			}
			MyFile << std::endl;
		}
	}
}

void writePlanes(std::ofstream& MyFile)
{
	MyFile << planes.size() << std::endl;
	for (size_t i = 0; i < planes.size(); i++) {
		MyFile << planes[i]->planePointPos.x << ';' << planes[i]->planePointPos.y << ';' << planes[i]->planePointPos.z << ';'
			<< planes[i]->normal.x << ';' << planes[i]->normal.y << ';' << planes[i]->normal.z << std::endl;
	}
}

void writeData(size_t pointCloudIndex)
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed_" +
		((pointCloudCount == 0 || pointCloudTestIndex != -1 || true) ? "test" : std::to_string(pointCloudIndex)) + ".txt");
	writePlanes(MyFile);
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	writePoints(savedPoints, MyFile);
	MyFile.close();
}

void processData() {
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	findEdgePoints();
	findCorners();
	findPlaneConnections();
	connectPlanes();
	createCorners();
	filterEdgePoints();
	extract2DPolygon();
	deleteSelfIntersections(planes);
	fitPlanes();
	deleteSelfIntersections(savedPlanes);
	saveSavedPoints();
}

int main()
{
	size_t endIndex = (pointCloudTestIndex == -1 ? std::max<size_t>(1, pointCloudCount) : 1);
	for (currentFrame = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0); 
		currentFrame < endIndex; currentFrame++)
	{
		auto start = std::chrono::steady_clock::now();
		readData(currentFrame);
		auto end = std::chrono::steady_clock::now();
		std::cout << "Read " + std::to_string(currentFrame) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
		start = std::chrono::steady_clock::now();
		processData();
		end = std::chrono::steady_clock::now();
		std::cout << "It. " + std::to_string(currentFrame) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
		if (true || currentFrame == endIndex - 1) 
		{
			start = std::chrono::steady_clock::now();
			convexSegmentation();
			exportObjects(currentFrame);
			writeData(currentFrame);
			end = std::chrono::steady_clock::now();
			std::cout << "Write " + std::to_string(currentFrame) + " Elapsed time in seconds : "
				<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
				<< " sec" << std::endl;
		}
	}
	return 0;
}