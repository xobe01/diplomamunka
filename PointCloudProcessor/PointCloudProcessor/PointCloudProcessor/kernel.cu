#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>

struct Point
{
	Vec3<double> position;
	size_t horizontalIndex;
	size_t verticalIndex;
	size_t id{ 0 };
	size_t separatedObjectId{ 0 };
	size_t lineId{ 0 };
	size_t outlineId{ 0 };
	size_t planeId { 0 };
	size_t lineIndex{ 0 };
	Vec3<double> normal = {0, 0, 0};
	bool isMarked{ false };
	bool isMarked2{ false };
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, size_t _id) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), id(_id) {};
};

std::vector<Point*> points;
std::vector<std::vector<Point*>> separatedPoints;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentLineId = 1;
int currentSeparatedObjectId = 1;
int currentPointId = 1;
int currentOutlineId = 1;
int currentLineIndex = 0;
const double objectPointDistance = 5;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void ReadData()
{
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, id);
		}		
	}
    MyReadFile.close();
}

void WriteData()
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i])
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
			';' << points[i]->id << ';' << points[i]->separatedObjectId << ';' << points[i]->outlineId << ';' << points[i]->lineId
			<< ';' << points[i]->lineIndex << std::endl;
	}
	MyFile.close();
}

void GroundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel) {
			points[i] = nullptr;
		}
	}
}

void choosePoints(const Vec3<Point*> planePoints, const std::vector<Point*>& nonProcessedPoints, double acceptTreshold, 
	/*out*/ std::vector<Point*>& chosenPoints, /*out*/ double std)
{
	std = 0;
	std::vector<double> distances;
	double avg = 0;
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.x->position - 
		planePoints.y->position, planePoints.z->position -planePoints.y->position));
	for (size_t i = 0; i < nonProcessedPoints.size(); i++)
	{
		double dist = abs(Vec3<double>::dot_product(normal, nonProcessedPoints[i]->position - planePoints.y->position));
		if (dist <= acceptTreshold) 
		{
			avg += dist;
			distances.push_back(dist);
			chosenPoints.push_back(nonProcessedPoints[i]);
		}
	}
	avg /= chosenPoints.size();
	for (size_t i = 0; i < chosenPoints.size(); i++)
	{
		std += pow(distances[i] - avg, 2);
	}
	std /= chosenPoints.size();
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

bool checkIfBridge(Point* p)
{
	if (!p->isMarked) return false;
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;
	//fel-le
	bool isUpNotNeightbour = y == 0 || !points[getOffset(x, y - 1)] || !points[getOffset(x, y - 1)]->isMarked;
	bool isDownNotNeightbour = y == verticalCount - 1 || !points[getOffset(x, y + 1)] || !points[getOffset(x, y + 1)]->isMarked;
	if (!isUpNotNeightbour)
		neighbourCount++;
	if (!isDownNotNeightbour)
		neighbourCount++;
	//jobbra-balra
	bool isLeftNotNeightbour = !points[getOffset(x - 1, y)] || !points[getOffset(x - 1, y)]->isMarked;
	bool isRighttNotNeightbour = !points[getOffset(x + 1, y)] || !points[getOffset(x + 1, y)]->isMarked;
	if (!isLeftNotNeightbour)
		neighbourCount++;
	if (!isRighttNotNeightbour)
		neighbourCount++;
	if (((isLeftNotNeightbour && isRighttNotNeightbour) || (isUpNotNeightbour && isDownNotNeightbour))) return true;

	//átlósan
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || !points[getOffset(x - 1, y - 1)]->isMarked)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || !points[getOffset(x + 1, y + 1)]->isMarked))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || !points[getOffset(x + 1, y - 1)]->isMarked)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || !points[getOffset(x - 1, y + 1)]->isMarked))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(const std::vector<Point*>& checkPoints)
{
	bool theresBridge = false;
	for (auto p : checkPoints) 
		if (checkIfBridge(p)) {
			theresBridge = true;
			p->isMarked = false;
		}
	return theresBridge;
}

void checkForGaps(/*out*/ std::vector<Point*>& chosenPoints)
{  
	for (auto p : chosenPoints) p->isMarked = true;
	while (isThereBridge(chosenPoints)) {}
	std::vector<Point*> nonProcessedPoints(chosenPoints);
	std::vector<Point*> nextStepPoints;
	std::vector<Point*> bestPartitionPoints;
	while (bestPartitionPoints.size() < nonProcessedPoints.size()) {
		std::vector<Point*> currentPartitionPoints;
		currentPartitionPoints.push_back(nonProcessedPoints[0]);
		nextStepPoints.push_back(nonProcessedPoints[0]);
		nonProcessedPoints[0]->isMarked = false;
		while (nextStepPoints.size() > 0) {
			std::vector<Point*> tempNextStepPoints;
			for (size_t i = 0; i < nextStepPoints.size(); i++) {
				size_t x = nextStepPoints[i]->horizontalIndex;
				size_t y = nextStepPoints[i]->verticalIndex;
				Point* neighbourPoint = points[getOffset(x, y - 1)];
				if (y > 0 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x, y + 1)];
				if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x - 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x + 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
			}
			nextStepPoints = tempNextStepPoints;
		}
		if (currentPartitionPoints.size() > bestPartitionPoints.size()) bestPartitionPoints = currentPartitionPoints;
		std::vector<Point*> tempNonProcessedPoints;
		for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
			if (nonProcessedPoints[i]->isMarked) tempNonProcessedPoints.push_back(nonProcessedPoints[i]);
		}
		nonProcessedPoints = tempNonProcessedPoints;
	}
	for (size_t i = 0; i < chosenPoints.size(); i++) {
		chosenPoints[i]->isMarked = false;
	}
	chosenPoints = bestPartitionPoints;
}

void RANSAC(std::vector<Point*>& nonProcessedPoints)
{
	/*size_t maxPointInPLane = nonProcessedPoints.size()/10;
	double findChance = 0.9;
	double eps = 1 - (double)maxPointInPLane / nonProcessedPoints.size();
	size_t N = log(1 - findChance) / log(1 - pow(1 - eps, 3));*/
	size_t N = 1000;
	double bestStd = 100000;
	double acceptTreshold = 0.1;
	std::vector<Point*> bestPoints;
	size_t counter = 0;
	for (size_t i = 0; i < N; i++) 
	{
		std::vector<Point*> chosenPoints;
		std::vector<Point*> chosenPoints2;
		double std = 0;
		auto pickedPoints = pick3Points(nonProcessedPoints);
		choosePoints(pickedPoints, nonProcessedPoints, acceptTreshold, chosenPoints, std);
		checkForGaps(chosenPoints); //TODO std
		if (chosenPoints.size() >= 3) {
			choosePoints(pick3Points(chosenPoints), nonProcessedPoints, acceptTreshold, chosenPoints2, std);
			checkForGaps(chosenPoints2); //TODO std
			chosenPoints = chosenPoints2;
		}		
		counter++;
		if (chosenPoints.size() > bestPoints.size() || (chosenPoints.size() == bestPoints.size() && std < bestStd))
		{
			bestPoints = chosenPoints;
			bestStd = std;
			counter = 0;
		}
		//if (counter > 100) break;
	}
	for (size_t i = 0; i < bestPoints.size(); i++) {
		bestPoints[i]->isMarked = true;
	}
}

void ObjectSegmentation()
{
	double tresholdMax = 10;
	double tresholdMin = 1;
	double maxDistance = 200.0;
	for (size_t i = 0; i < points.size(); i++) if(points[i]) points[i]->isMarked = true;
	std::vector<Point*>nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) 
	{
		if (points[j] && points[j]->isMarked) {
			std::vector<Point*> objectPoints;
			objectPoints.push_back(points[j]);
			points[j]->separatedObjectId = currentSeparatedObjectId;
			nextStepPoints.push_back(points[j]);
			points[j]->isMarked = false;
			while (nextStepPoints.size() > 0) {
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					double currentTreshold = tresholdMin + nextStepPoints[i]->position.length() / maxDistance * (tresholdMax - tresholdMin);
					size_t x = nextStepPoints[i]->horizontalIndex;
					size_t y = nextStepPoints[i]->verticalIndex;
					Point* neighbourPoint = points[getOffset(x, y - 1)];
					if (neighbourPoint && y > 0 &&  neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position,
						neighbourPoint->position) < currentTreshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
						neighbourPoint->separatedObjectId = currentSeparatedObjectId;
					}
					neighbourPoint = points[getOffset(x, y + 1)];
					if (neighbourPoint && y < verticalCount - 1 && neighbourPoint->isMarked &&
						Vec3<double>::distance(nextStepPoints[i]->position, neighbourPoint->position) < currentTreshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
						neighbourPoint->separatedObjectId = currentSeparatedObjectId;
					}
					neighbourPoint = points[getOffset(x - 1, y)];
					if (neighbourPoint && neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position,
						neighbourPoint->position) < currentTreshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
						neighbourPoint->separatedObjectId = currentSeparatedObjectId;
					}
					neighbourPoint = points[getOffset(x + 1, y)];
					if (neighbourPoint && neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position,
						neighbourPoint->position) < currentTreshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
						neighbourPoint->separatedObjectId = currentSeparatedObjectId;
					}
				}
				nextStepPoints = tempNextStepPoints;
			}
			if(objectPoints.size() > 2)
				separatedPoints.push_back(objectPoints);
			currentSeparatedObjectId++;
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
}

#include <chrono>

void findPlanes(std::vector<std::vector<Point*>>& planes)
{
	auto start = std::chrono::steady_clock::now();
	std::vector<Point*> nonProcessedPoints;
	size_t minPointCount = 10;
	size_t counter = 1;
	for (size_t j = 0; j < separatedPoints.size(); j++) {
		nonProcessedPoints = separatedPoints[j];
		while (true) {
			std::vector<Point*> plane;
			if (nonProcessedPoints.size() >= minPointCount)
				RANSAC(nonProcessedPoints);
			else break;
			std::vector<Point*> tempPoints;
			for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
				if (nonProcessedPoints[i]->isMarked) {
					nonProcessedPoints[i]->isMarked = false;
					plane.push_back(nonProcessedPoints[i]);
					nonProcessedPoints[i]->id = currentPointId;
				}
				else {
					tempPoints.push_back(nonProcessedPoints[i]);
				}
			}
		currentPointId++;
		if (plane.size() < minPointCount) break;
		planes.push_back(plane);
		nonProcessedPoints = tempPoints;
		std::cout << counter << ": " << plane.size() << std::endl;
		counter++;
		}
	}
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec"<< std::endl;
}

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(center->position - p1->position, center->position - p2->position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x - 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x + 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

void choosePoints2(const Vec3<Point*> planePoints, double acceptTreshold, /*out*/ std::vector<Point*>& chosenPoints)
{
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.x->position -
		planePoints.y->position, planePoints.z->position - planePoints.y->position));
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->id = currentPointId;
	chosenPoints.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(normal, neighbourPoints[j]->position - planePoints.y->position));
					if (dist <= acceptTreshold) {
						chosenPoints.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->id = currentPointId;
						tempNextStepPoints.push_back(neighbourPoints[j]);
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void findPlanes2(std::vector<std::vector<Point*>>& planes)
{
	auto start = std::chrono::steady_clock::now();
	size_t minPointCount = 10;
	size_t counter = 1;
	double planeDistanceTreshold = 0.1;
	double normalTreshold = 0.1;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = true;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = true;
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			std::vector<Point*> planePoints;			
			nextStepPoints.push_back(points[j]);
			calculateNormal(points[j]);
			while (nextStepPoints.size() > 0) {
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x - 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x + 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k < verticalCount - 1 || y < 3) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								if (neighbourPoints[k]->normal.length() == 0) {
									calculateNormal(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if ((normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {
								choosePoints2({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, planeDistanceTreshold,
									planePoints);
								break;
							}
						}
					}
				}
				if (planePoints.size() > 0) 
				{
					planes.push_back(planePoints);
					planePoints.clear();
					currentPointId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec" << std::endl;
}

void findNextEdgePoint(Point* startPoint, Point* currentPoint, std::vector<Point*>& edgePoints, size_t direction)
{
	edgePoints.push_back(currentPoint);
	size_t x = currentPoint->horizontalIndex;
	size_t y = currentPoint->verticalIndex;
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 4; i++) {
		switch (direction) {
		case 0: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && neighbourPoint->id == startPoint->id) {
			if (neighbourPoint == startPoint) {
			}
			else 
			{
				findNextEdgePoint(startPoint, neighbourPoint, edgePoints, (direction + 3) % 4);
			}
			break;
		}
		direction += direction == 3 ? -3 : 1;
	}
	if (neighbourPoint == nullptr) {
		std::cout << "PLANE WITHOUT RELEVANT OUTLINE!" << std::endl;
	}
}

bool isEdgePoint(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

bool isConcaveCornerPoint(Point* point)
{
	size_t neighbourCount = 0;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	size_t planeId = point->planeId;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i < 3) && neighbourPoints[i] && neighbourPoints[i]->planeId == planeId) {
			neighbourCount++;
			isNeighbour[i] = true;
		}
	}
	if (neighbourCount > 2 || (isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3])) {
		return true;
	}
	return false;
}

void findEdgePoints(std::vector<std::vector<Point*>>& planes, /*out*/ std::vector<std::vector<Point*>>& filteredCorners)
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i].size(); j++) {
			planes[i][j]->isMarked = true;
			planes[i][j]->planeId = i + 1;
		}
		for (size_t j = 0; j < planes[i].size(); j++) if(isEdgePoint(planes[i][j])) edgePointsInPlane.push_back(planes[i][j]);
		
		bool isFirstEdge = true;
		for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
			if (edgePointsInPlane[j]->isMarked) {
				std::vector<Point*> edgePoints;
				findNextEdgePoint(edgePointsInPlane[j], edgePointsInPlane[j], edgePoints, isFirstEdge ? 0 : 1);
				if (!isFirstEdge) {
					edgePoints.insert(edgePoints.begin(), edgePoints[edgePoints.size() - 1]);
					edgePoints.pop_back();
				}
				bool edgeFound = false;
				std::vector<Point*> realEdge;
				for (size_t k = 0; k < edgePoints.size(); k++) 
				{
					edgePoints[k]->isMarked = false;
					edgePoints[k]->outlineId = currentOutlineId;
					if (!isConcaveCornerPoint(edgePoints[k]))
					{
						realEdge.push_back(edgePoints[k]);
					}
				}
				currentOutlineId++;
				if (realEdge.size() >= 3) {
					filteredCorners.push_back(realEdge);
					for (size_t k = 0; k < realEdge.size(); k++) {
						realEdge[k]->lineId = currentLineId;
						realEdge[k]->lineIndex = currentLineIndex;
						currentLineIndex++;
					}
					currentLineIndex = 0;
					currentLineId++;
				}
				isFirstEdge = false;
			}
		}
		currentPointId++;
	}
}

double distancePointFromLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line)
{
	Vec3<double> directionVector = line.first - line.second;
	return Vec3<double>::crossProduct(p - line.first, directionVector).length() / directionVector.length();
}

bool isOnLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line, double treshold)
{
	return distancePointFromLine(p, line) < treshold;
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i] = nullptr;
	}
}

void exportObjects(std::vector<std::vector<Point*>> filteredCorners)
{
	for (size_t i = 0; i < filteredCorners.size(); i++) {
		std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models/processed_obj_" + std::to_string(i) + ".obj");
		MyFile << "o Mesh" << std::endl;
		for (size_t j = 0; j < filteredCorners[i].size(); j++) {
			MyFile << "v " << -filteredCorners[i][j]->position.x << " " << filteredCorners[i][j]->position.y << " " <<
				filteredCorners[i][j]->position.z << std::endl;
		}
		MyFile << "f ";
		for (size_t j = 1; j < filteredCorners[i].size() + 1; j++) {
			MyFile << j << " ";
		}
		MyFile << std::endl;
		MyFile << "f ";
		for (size_t j = filteredCorners[i].size(); j > 0; j--) {
			MyFile << j << " ";
		}
		MyFile << std::endl;
		MyFile.close();
	}
}

void ProcessData() {
	GroundSegmentation();
	egoCarSegmentation();
	//ObjectSegmentation();
	for (auto t : separatedPoints) {
		std::cout << t.size() << std::endl;
	}
	std::vector<std::vector<Point*>> planes;
	findPlanes2(planes);
	std::vector<std::vector<std::vector<Point*>>> edges;
	std::vector<std::vector<Point*>> filteredCorners;
	findEdgePoints(planes, filteredCorners);
	exportObjects(filteredCorners);
	WriteData();
}

int main()
{
    ReadData();
	ProcessData();
    return 0;
}