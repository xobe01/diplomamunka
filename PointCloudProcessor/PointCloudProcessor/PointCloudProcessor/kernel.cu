#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>
#include "kernel.h"

struct Point
{
	size_t horizontalIndex;
	size_t verticalIndex;
	Vec3<double> position;
	Plane* plane{ 0 };
	size_t cornerId{ 0 };
	size_t outlineId{ 0 };
	size_t cornerIndex{ 0 };
	Vec3<double> normal = { 0, 0, 0 };
	bool isMarked{ false };
	bool isMarked2{ false };
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, Plane* _plane) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), plane(_plane)
	{};
};

struct EdgePoint {
	Point* point;
	std::vector<Point*> neighbourPlaneNeighbours;
	bool isCorner;
	EdgePoint(Point* _point, std::vector<Point*> _neighbourPlaneNeighbours, bool _isCorner) : point(_point), neighbourPlaneNeighbours(_neighbourPlaneNeighbours), 
		isCorner(_isCorner) {};
};

struct Plane {
	std::vector<Point*> points;
	std::pair<double, double> horizontalBounds;
	std::pair<double, double> verticalBounds;
	std::vector<std::vector<EdgePoint*>> edges;
	Vec3<double> planePointPos;
	Vec3<double> normal;
	std::pair<Vec3<double>, Vec3<double>> pointDirections;
	size_t id;
};

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Plane*> planes;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPlaneId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData()
{
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}		
	}
    MyReadFile.close();
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i])
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
			';' << (points[i]->plane ? points[i]->plane->id : 0) << ';' << points[i]->outlineId << ';' << points[i]->cornerId
			<< ';' << points[i]->cornerIndex << std::endl;
	}
}

void writeData()
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	MyFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

bool isSpike(Point* p)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;

	//fel-le
	bool isUpNotNeightbour = y == 0 || !points[getOffset(x, y - 1)] || points[getOffset(x, y - 1)]->plane != p->plane;
	bool isDownNotNeightbour = y == verticalCount - 1 || !points[getOffset(x, y + 1)] || points[getOffset(x, y + 1)]->plane != p->plane;

	//jobbra-balra
	bool isLeftNotNeightbour = !points[getOffset(x - 1, y)] || points[getOffset(x - 1, y)]->plane != p->plane;
	bool isRighttNotNeightbour = !points[getOffset(x + 1, y)] || points[getOffset(x + 1, y)]->plane != p->plane;
	if (((isLeftNotNeightbour && isRighttNotNeightbour) || (isUpNotNeightbour && isDownNotNeightbour))) {
		return true;
	}
	return false;
}

bool checkIfBridge(Point* p)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && isSpike(neighbourPoints[j]))
			neighbourCount++;
	}
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || points[getOffset(x - 1, y - 1)]->plane != p->plane)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || points[getOffset(x + 1, y + 1)]->plane != p->plane))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || points[getOffset(x + 1, y - 1)]->plane != p->plane)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || points[getOffset(x - 1, y + 1)]->plane != p->plane))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

void checkForGaps(/*out*/ std::vector<Point*>& chosenPoints)
{  
	for (auto p : chosenPoints) p->isMarked = true;
	while (isThereBridge(chosenPoints)) {}
	std::vector<Point*> nonProcessedPoints(chosenPoints);
	std::vector<Point*> nextStepPoints;
	std::vector<Point*> bestPartitionPoints;
	while (bestPartitionPoints.size() < nonProcessedPoints.size()) {
		std::vector<Point*> currentPartitionPoints;
		currentPartitionPoints.push_back(nonProcessedPoints[0]);
		nextStepPoints.push_back(nonProcessedPoints[0]);
		nonProcessedPoints[0]->isMarked = false;
		while (nextStepPoints.size() > 0) {
			std::vector<Point*> tempNextStepPoints;
			for (size_t i = 0; i < nextStepPoints.size(); i++) {
				size_t x = nextStepPoints[i]->horizontalIndex;
				size_t y = nextStepPoints[i]->verticalIndex;
				Point* neighbourPoint = points[getOffset(x, y - 1)];
				if (y > 0 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x, y + 1)];
				if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x - 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x + 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
			}
			nextStepPoints = tempNextStepPoints;
		}
		if (currentPartitionPoints.size() > bestPartitionPoints.size()) bestPartitionPoints = currentPartitionPoints;
		std::vector<Point*> tempNonProcessedPoints;
		for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
			if (nonProcessedPoints[i]->isMarked) tempNonProcessedPoints.push_back(nonProcessedPoints[i]);
		}
		nonProcessedPoints = tempNonProcessedPoints;
	}
	for (size_t i = 0; i < chosenPoints.size(); i++) {
		chosenPoints[i]->isMarked = false;
	}
	chosenPoints = bestPartitionPoints;
}

#include <chrono>

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(center->position - p1->position, center->position - p2->position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x - 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x + 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

void choosePoints(const Vec3<Point*> planePoints, double acceptTreshold, Vec3<double> normal, /*out*/ Plane* plane)
{
	if (normal == Vec3<double>{0, 0, 0}) {
		normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.x->position -
			planePoints.y->position, planePoints.z->position - planePoints.y->position));
	}
	plane->pointDirections = { Vec3<double>::normalize(planePoints.x->position - planePoints.y->position),
	 Vec3<double>::normalize(planePoints.x->position - planePoints.z->position) };
	plane->normal = normal;
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(normal, neighbourPoints[j]->position - planePoints.x ->position));
					if (dist <= acceptTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void calculateBounds(Plane& plane)
{
	plane.horizontalBounds = std::make_pair(plane.points[0]->horizontalIndex, plane.points[0]->horizontalIndex);
	plane.verticalBounds = std::make_pair(plane.points[0]->verticalIndex, plane.points[0]->verticalIndex);
	for (size_t i = 1; i < plane.points.size(); i++) 
	{
		if (plane.points[i]->horizontalIndex > plane.horizontalBounds.second)
			plane.horizontalBounds.second = plane.points[i]->horizontalIndex;
		else if (plane.points[i]->horizontalIndex < plane.horizontalBounds.first)
			plane.horizontalBounds.first = plane.points[i]->horizontalIndex;
		if (plane.points[i]->verticalIndex > plane.verticalBounds.second)
			plane.verticalBounds.second = plane.points[i]->verticalIndex;
		else if (plane.points[i]->verticalIndex < plane.verticalBounds.first)
			plane.verticalBounds.first = plane.points[i]->verticalIndex;
	}
}

void findPlanes()
{
	auto start = std::chrono::steady_clock::now();
	size_t minPointCount = 10;
	size_t counter = 1;
	double planeDistanceTreshold = 0.1;
	double normalTreshold = 0.1;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = true;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = true;
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			nextStepPoints.push_back(points[j]);
			calculateNormal(points[j]);
			while (nextStepPoints.size() > 0) {
				Plane* plane = new Plane();
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x - 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x + 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k < verticalCount - 1 || y < 3) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								if (neighbourPoints[k]->normal.length() == 0) {
									calculateNormal(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if ((normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {

								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, planeDistanceTreshold, {0,0,0}, 
									plane);
								break;
							}
						}
					}
				}
				if (plane->points.size() > 0)
				{
					planes.push_back(plane);
					currentPlaneId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		int originalSize = planes[i]->points.size();
		while (isThereBridge(planes[i]->points)) {}		
		if (originalSize != planes[i]->points.size() && planes[i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[i]->points[0], nullptr, nullptr }, planeDistanceTreshold, planes[i]->normal,
					plane);
				if (plane->points.size() < planes[i]->points.size()) {
					for (size_t j = 0; j < planes[i]->points.size(); j++) {
						if (planes[i]->points[j]->plane != planes[i]) {
							planes[i]->points.erase(planes[i]->points.begin() + j);
							j--;
						}
					}
					planes.push_back(plane);
					currentPlaneId++;
				}
				else 
				{
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[i];
					}
					delete plane;
					break;
				}
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		calculateBounds(planes[i]);	
	}/**/
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec" << std::endl;
}

void findNextEdgePoint(Point* startPoint, EdgePoint* currentPoint, std::vector<EdgePoint*>& edgePoints, size_t direction, bool isPreviousSpike,
	bool wasThereNonSpike)
{
	
	if(currentPoint->point->isMarked)
		edgePoints.push_back(currentPoint);
	size_t x = currentPoint->point->horizontalIndex;
	size_t y = currentPoint->point->verticalIndex;
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 4; i++) { //Check for neighbours on other planes
		switch (direction) {
		case 0: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 1: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		case 2: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 3: //to down		
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && neighbourPoint->plane != startPoint->plane && neighbourPoint->plane != nullptr) {
			currentPoint->neighbourPlaneNeighbours.push_back(neighbourPoint);
		}
		direction += direction == 3 ? -3 : 1;
	}	
	for (size_t i = 0; i < 4; i++) {
		switch (direction) {
		case 0: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint == startPoint) {
			currentPoint->point->isMarked = false;
			break;
		}
		if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr && (neighbourPoint->isMarked 
			|| i == 3)) {
			if (wasThereNonSpike && edgePoints.size() > 1 && isPreviousSpike && !isSpike(currentPoint->point) && currentPoint->point->isMarked) {
				edgePoints[edgePoints.size() - 1] = nullptr;
				return;
			}
			else {
				if (!edgePoints[edgePoints.size() - 1])
					edgePoints.pop_back();
				if (!wasThereNonSpike && !isSpike(currentPoint->point)) 					 
				{
					startPoint = currentPoint->point;
					wasThereNonSpike = true;
				}
				currentPoint->point->isMarked = false;
				findNextEdgePoint(startPoint, new EdgePoint(neighbourPoint, {}, false), edgePoints, (direction + 3) % 4, isSpike(currentPoint->point),
					wasThereNonSpike);
			}
			if (edgePoints[edgePoints.size() - 1])
				break;
		}
		direction += direction == 3 ? -3 : 1;
	}
	if (neighbourPoint == nullptr) {
		std::cout << "PLANE WITHOUT RELEVANT OUTLINE!" << std::endl;
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

bool isStraightEdgePoint(size_t pointIndex, std::vector<EdgePoint*>& edge, bool& isPreviousConcave)
{
	bool previousConcaveStore = isPreviousConcave;
	size_t neighbourCount = 0;
	EdgePoint* point = edge[pointIndex];
	size_t x = point->point->horizontalIndex;
	size_t y = point->point->verticalIndex;
	Plane* plane = point->point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i < 3) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane && (neighbourPoints[i]->outlineId == 0 || neighbourPoints[i]->outlineId == point->point->outlineId)) {
			neighbourCount++;
			isNeighbour[i] = true;
		}
	}
	isPreviousConcave = neighbourCount == 4;
	if (neighbourCount == 3 || (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3])))) {
		return true;
	}
	if (neighbourCount == 4 && (pointIndex == 0 ? edge[edge.size() - 1] : edge[pointIndex - 1])->isCorner)
		return true;
	if (previousConcaveStore) {
		edge[pointIndex - 1] -> isCorner = false;
	}
	return false;
}

void findEdgePoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) if(isEdgePoint(planes[i]->points[j])) edgePointsInPlane.push_back(planes[i]->points[j]);
		bool isFirstEdge = true;
		while (edgePointsInPlane.size() > 0) 
		{
			std::vector<Point*> tempEdgePointsInPlane;
			Point* startPoint = edgePointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
				if ((edgePointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - edgePointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| edgePointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = edgePointsInPlane[j]->horizontalIndex;
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
				else if (edgePointsInPlane[j]->horizontalIndex == minHorizontalCoord && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
			}
			std::vector<EdgePoint*> edgePoints;
			findNextEdgePoint(startPoint, new EdgePoint(startPoint, {}, false), edgePoints, isFirstEdge ? 0 : 1, false, false);
			if (!isFirstEdge) {
				edgePoints.insert(edgePoints.begin(), edgePoints[edgePoints.size() - 1]);
				edgePoints.pop_back();
			}
			for (size_t k = 0; k < edgePoints.size(); k++) {
				edgePoints[k]->point->outlineId = currentOutlineId;
			}
			for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
				if (edgePointsInPlane[j]->isMarked) tempEdgePointsInPlane.push_back(edgePointsInPlane[j]);
			}
			edgePointsInPlane = tempEdgePointsInPlane;
			currentOutlineId++;
			planes[i]->edges.push_back(edgePoints);
			isFirstEdge = false;
		}

		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = false;
	}
}

bool arePlanesNeighbours(Plane p1, Plane p2, std::pair<int, int>& horizontalCommonBounds, std::pair<int, int>& verticalCommonBounds)
{
	if (p1.horizontalBounds.first > p1.horizontalBounds.second)
		p1.horizontalBounds.second += horizontalCount;
	double p1HorizontalSize = p1.horizontalBounds.second - p1.horizontalBounds.first;
	double p1VerticalSize = p1.verticalBounds.second - p1.verticalBounds.first;
	std::pair<int, int> p1Center = {(int)(p1.horizontalBounds.first + p1HorizontalSize / 2) % horizontalCount, 
		p1.verticalBounds.first + p1VerticalSize / 2 };

	if (p2.horizontalBounds.first > p2.horizontalBounds.second)
		p2.horizontalBounds.second += horizontalCount;
	double p2HorizontalSize = p2.horizontalBounds.second - p2.horizontalBounds.first;
	double p2VerticalSize = p2.verticalBounds.second - p2.verticalBounds.first;
	std::pair<int, int> p2Center = { (int)(p2.horizontalBounds.first + p2HorizontalSize / 2) % horizontalCount,
		p2.verticalBounds.first + p2VerticalSize / 2 };

	if (abs(p1Center.first - p2Center.first) > horizontalCount / 2) 
	{
		if (p1Center.first > p2Center.first)
			p2Center.first += horizontalCount;
		if (p1Center.first < p2Center.first)
			p1Center.first += horizontalCount;
	}
	if (abs(p1Center.first - p2Center.first) <= (p1HorizontalSize + p2HorizontalSize) / 2 + 1 &&
		abs(p1Center.second - p2Center.second) <= (p1VerticalSize + p2VerticalSize) / 2 + 1) 
	{
		horizontalCommonBounds = {abs(p1.horizontalBounds.first - p2.horizontalBounds.first) > horizontalCount / 2
			? std::min(p1.horizontalBounds.first, p2.horizontalBounds.first) : std::max(p1.horizontalBounds.first, p2.horizontalBounds.first),
			abs(p1.horizontalBounds.second - p2.horizontalBounds.second) > horizontalCount / 2
			? std::max(p1.horizontalBounds.second, p2.horizontalBounds.second) : std::min(p1.horizontalBounds.second, p2.horizontalBounds.second)};
		if (horizontalCommonBounds.second < horizontalCommonBounds.first)
			horizontalCommonBounds = { horizontalCommonBounds.second, horizontalCommonBounds.first };

		verticalCommonBounds = { std::max(p1.verticalBounds.first, p2.verticalBounds.first),
			std::min(p1.verticalBounds.second, p2.verticalBounds.second) };
		if (verticalCommonBounds.second < verticalCommonBounds.first)
			verticalCommonBounds = { verticalCommonBounds.second, verticalCommonBounds.first };
		return true;
	}
	return false;
}

const double newPointAcceptTreshold = 0.9;
const double inf = 1000000;

EdgePoint* addNewPoint(EdgePoint* point, Point* neighbour, Plane* plane, std::vector<EdgePoint*>& edge, size_t pointIndex)
{
	int offset = getOffset(point->point->horizontalIndex - (neighbour->horizontalIndex -
		point->point->horizontalIndex), point->point->verticalIndex - (neighbour->verticalIndex - point->point->verticalIndex));
	Vec3<double> dir = areNeighbours(point->point, neighbour) < 2 ? plane->pointDirections.second : plane->pointDirections.second;
	Vec3<double> newPointPos = point->point->position - dir * Vec3<double>::dot_product(point->point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf)
		return nullptr;
	Vec3<double> dirToNew = newPointPos - point->point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold)
		return nullptr;
	int offsetNeighbour = getOffset(neighbour->horizontalIndex - (point->point->horizontalIndex -
		neighbour->horizontalIndex), neighbour->verticalIndex - (point->point->verticalIndex - neighbour->verticalIndex));
	Vec3<double> dirToNewNeighbour = newPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold)
		return nullptr;
	Point* newPoint = new Point(newPointPos, 0, 0, point->point->plane);
	addedPoints.push_back(newPoint);
	return new EdgePoint(newPoint, {}, true);
}

void connectPlanes()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		for (size_t j = 0; j < planes[i]->edges.size(); j++) 
		{
			bool wasFirstPointGenerated = false;
			bool wasPreviousSelected = true;
			size_t neighbourPlaneId = 0;
			EdgePoint* previousPoint = nullptr;
			Point* previousNeighbourPoint = nullptr;
			size_t previousIndex = 0;
			std::vector<std::pair<std::pair<EdgePoint*, size_t>, size_t>> newPoints;
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {

				for (auto neighbour : planes[i]->edges[j][k]->neighbourPlaneNeighbours) 
				{
					if (!previousNeighbourPoint || neighbour->plane != previousNeighbourPoint->plane) 
					{
						if (!wasPreviousSelected)
						{
							auto newPoint = addNewPoint(previousPoint, previousNeighbourPoint, previousNeighbourPoint->plane, planes[i]->edges[j], k);
							if (newPoint) {								
								newPoints.push_back({ { newPoint , k },  previousNeighbourPoint->plane->id});
							}
						}
						auto newPoint = addNewPoint(planes[i]->edges[j][k], neighbour, neighbour->plane, planes[i]->edges[j], k);
						if (newPoint) 
						{
							if (k == 0 && ((planes[i]->edges[j][k]->point->horizontalIndex - neighbour->horizontalIndex + horizontalCount) % horizontalCount
								== 1)) wasFirstPointGenerated = true;
							newPoints.push_back({ { newPoint, k }, neighbour->plane->id });
						}
						neighbourPlaneId = neighbour->plane->id;
						wasPreviousSelected = true;
					}
					else
						wasPreviousSelected = false;
					previousPoint = planes[i]->edges[j][k];
					previousIndex = k;
					if(k > 0 || !wasFirstPointGenerated)
						previousNeighbourPoint = neighbour;
				}
			}
			if (!wasPreviousSelected && (newPoints.size() == 0 || previousNeighbourPoint->plane->id != newPoints[0].second || !wasFirstPointGenerated)) {
				auto newPoint = addNewPoint(previousPoint, previousNeighbourPoint, previousNeighbourPoint->plane, planes[i]->edges[j], previousIndex);
				if (newPoint) {
					newPoints.push_back({{ newPoint, previousIndex }, previousNeighbourPoint->plane->id});
				}
			}
			for (size_t k = 0; k < newPoints.size(); k++) 
			{
				size_t realIndex = k + newPoints[k].first.second;
				if (newPoints[k].second == newPoints[k == newPoints.size() - 1 ? 0 : k + 1].second && (wasFirstPointGenerated || k < newPoints.size() - 1) &&
					(!wasFirstPointGenerated || k > 0))
				{
					for (size_t l = realIndex; l <= (k == newPoints.size() - 1 ? planes[i]->edges[j].size() - 1 : k + newPoints[k + 1].first.second); l++) {
						planes[i]->edges[j][l]->isCorner = false;
					}
				}
				else planes[i]->edges[j][realIndex]->isCorner = false;
				planes[i]->edges[j].insert(planes[i]->edges[j].begin() + realIndex, newPoints[k].first.first);
			}
		}
	}
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k]->edges.size(); i++) {
			bool isPreviousConcave = false;
			for (size_t j = 0; j < planes[k]->edges[i].size(); j++) {
				if (!isStraightEdgePoint(j, planes[k]->edges[i], isPreviousConcave)) {
					planes[k]->edges[i][j]->isCorner = true;
				}
			}
		}
	}
}

double distancePointFromLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line)
{
	Vec3<double> directionVector = line.first - line.second;
	return Vec3<double>::crossProduct(p - line.first, directionVector).length() / directionVector.length();
}

bool isOnLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line, double treshold)
{
	return distancePointFromLine(p, line) < treshold;
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i] = nullptr;
	}
}

void exportObjects()
{
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models/processed_obj_" 
				+ std::to_string(currentCornerId) + ".obj");
			MyFile << "o Mesh" << std::endl;
			std::vector<EdgePoint*> corners;
			currentCornerIndex = 0;
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {
				if (planes[i]->edges[j][k]->isCorner) {
					planes[i]->edges[j][k]->point->cornerId = currentCornerId;
					planes[i]->edges[j][k]->point->cornerIndex = currentCornerIndex;
					corners.push_back(planes[i]->edges[j][k]);
					currentCornerIndex++;
				}
			}
			for (size_t k = 0; k < corners.size(); k++) {
				MyFile << "v " << -corners[k]->point->position.x << " " << corners[k]->point->position.y << " " << corners[k]->point->position.z << std::endl;
			}
			MyFile << "f ";
			for (size_t j = 1; j < corners.size() + 1; j++) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (size_t j = corners.size(); j > 0; j--) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile.close();
			currentCornerId++;
		}
	}
}

void processData() {
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	findEdgePoints();
	findCorners();
	connectPlanes();
	exportObjects();
	writeData();
}

int main()
{
    readData();
	processData();
    return 0;
}