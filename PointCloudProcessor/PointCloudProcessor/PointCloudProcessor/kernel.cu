#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>
#include "kernel.h"
#include <chrono>

struct Point
{
	size_t horizontalIndex;
	size_t verticalIndex;
	Vec3<double> position;
	Plane* plane{ 0 };
	size_t cornerId{ 0 };
	size_t outlineId{ 0 };
	size_t cornerIndex{ 0 };
	Vec3<double> normal = { 0, 0, 0 };
	bool isMarked{ false };
	bool isMarked2{ false };
	std::vector<Point*> neighbourPlaneNeighbours;
	std::vector<Point*> createdNeighbourPoints;
	std::vector<size_t> convexId;
	std::vector<size_t> convexIndex;
	bool isCorner;
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, Plane* _plane) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), plane(_plane)
	{
		isCorner = false;
		neighbourPlaneNeighbours = { nullptr, nullptr, nullptr, nullptr };
		createdNeighbourPoints = { nullptr, nullptr, nullptr, nullptr };
		convexId = {};
		convexIndex = {};
	};
	std::string printCoordinates()
	{
		return std::to_string(horizontalIndex) + " " + std::to_string(verticalIndex) + '\n';
	}
};

struct Edge
{
	bool isHole;
	bool wasFirstGenerated;
	Point* startPoint;
	std::vector<std::pair<Point*, int>> pointsWithDir;
	Edge() : isHole(false), wasFirstGenerated(false) {}
};

struct Plane {
	std::vector<Point*> points;
	std::vector<Edge*> edges; //wasFirstGenerated, startpoint, points, direction
	Vec3<double> planePointPos;
	Vec3<double> normal;
	std::pair<Vec3<double>, Vec3<double>> pointDirections;
	size_t id;
	std::vector<std::vector<Point*>> convexFaces;
};

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Plane*> planes;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPlaneId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;
const double planeDistanceTreshold = 0.05;

const size_t pointCloudCount = 100;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData(size_t pointCloudIndex)
{
	points.clear();
	addedPoints.clear();
	planes.clear();
	verticalCounts.clear();
	currentCornerId = 1;
	currentSeparatedObjectId = 1;
	currentPlaneId = 1;
	currentOutlineId = 1;
	currentCornerIndex = 0;
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw_" + 
		(pointCloudTestIndex == -1 ? (pointCloudCount == 0 ? "test" : std::to_string(pointCloudIndex)) : std::to_string(pointCloudTestIndex)) + ".txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}		
	}
    MyReadFile.close();
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i]) {
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
				';' << (points[i]->plane ? points[i]->plane->id : 0) << ';' << points[i]->outlineId << ';' << points[i]->cornerId
				<< ';' << points[i]->cornerIndex;
			for (size_t j = 0; j < points[i]->convexId.size(); j++) {
				MyFile << ";" << points[i]->convexId[j] << ";" << points[i]->convexIndex[j];
			}
			MyFile << std::endl;
		}
	}
}

void writePlanes(std::ofstream& MyFile)
{
	MyFile << planes.size() << std::endl;
	for (size_t i = 0; i < planes.size(); i++) {
		MyFile << planes[i]->planePointPos.x << ';' << planes[i]->planePointPos.y << ';' << planes[i]->planePointPos.z << ';' 
			<< planes[i]->normal.x << ';' << planes[i]->normal.y << ';' << planes[i]->normal.z << std::endl;
	}
}

void writeData(size_t pointCloudIndex)
{
	std::cout << pointCloudIndex << std::endl;
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed_" +
		((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + ".txt");
	writePlanes(MyFile);
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	MyFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}

	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel + 0.1) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

void setPointsMarked(std::vector<Point*> points, bool isMarked, bool isMarked2)
{
	for (size_t i = 0; i < points.size(); i++) {
		points[i]->isMarked = isMarked;
		points[i]->isMarked2 = isMarked2;
	}
}

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

int spikeType(Point* p, int arriveDirection, bool onlyMarkedNeighbours)
{
	//arriveDirection
	// 0 - from left
	// 1 - from up
	// 2 - from rigth
	// 3 - from down
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	size_t neighbourCount = 0;
	bool isNeighbour[4] = { false, false, false, false };
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };

	Point* diagNeighbourPoints[4] = { points[getOffset(x - 1, y - 1)], points[getOffset(x + 1, y - 1)], points[getOffset(x + 1, y + 1)], points[getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->plane == p->plane && 
			(!onlyMarkedNeighbours || neighbourPoints[j]->isMarked)) {
			neighbourCount++;
			isNeighbour[j] = true;
		}
	}
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarkedNeighbours || diagNeighbourPoints[j]->isMarked)) {
			diagIsNeighbour[j] = true;
		}
	}
	if (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[2] && !diagIsNeighbour[0]) || (isNeighbour[0] && isNeighbour[3] && !diagIsNeighbour[1]) 
		|| (isNeighbour[1] && isNeighbour[2] && !diagIsNeighbour[3]) || (isNeighbour[1] && isNeighbour[3] && !diagIsNeighbour[2])))
		return -1;
	if ((neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3]))) || (neighbourCount > 1 && 
		((arriveDirection == 0 && !isNeighbour[1]) || (arriveDirection == 1 && !isNeighbour[2]) || (arriveDirection == 2 && !isNeighbour[0])
			|| (arriveDirection == 3 && !isNeighbour[3]))))
		return 0;
	if (neighbourCount == 1)
		return 1;
	if (neighbourCount > 1)
		return 2;
	return 3;
}

bool checkIfBridge(Point* p, bool onlyMarked)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && neighbourPoints[j]->plane == p->plane && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) &&
			spikeType(neighbourPoints[j], -1, false) > 1 && (!onlyMarked || neighbourPoints[j]->isMarked))
			neighbourCount++;
	}
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* diagNeighbourPoints[4] = { points[getOffset(x - 1, y - 1)], points[getOffset(x + 1, y - 1)], points[getOffset(x + 1, y + 1)],
		points[getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarked || diagNeighbourPoints[j]->isMarked)) {
			diagIsNeighbour[j] = true;
		}
	}


	if (((!diagIsNeighbour[0] && !diagIsNeighbour[2]) || (!diagIsNeighbour[1] && !diagIsNeighbour[3])) && neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p, false)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(p1->position - center->position, p2->position - center-> position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x + 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x - 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

void choosePoints(const Vec3<Point*> planePoints, Plane* basePlane, /*out*/ Plane* plane)
{
	if (basePlane) {
		plane->pointDirections = basePlane->pointDirections;
		plane->normal = basePlane->normal;
	}
	else 
	{
		auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.y->position -
			planePoints.x->position, planePoints.z->position - planePoints.x->position));
		Vec3<double> horizontalDirection = { 0,0,0 };
		Vec3<double> verticalDirection = { 0,0,0 };
		Point* neighbours[2] = { planePoints.y, planePoints.z };
		for each (auto neighbour in neighbours) {
			switch (areNeighbours(planePoints.x, neighbour)) {
			case 1:
			horizontalDirection = neighbour->position - planePoints.x->position;
			break;
			case 2:
			verticalDirection = neighbour->position - planePoints.x->position;
			break;
			case 3:
			horizontalDirection = planePoints.x->position - neighbour->position;
			break;
			case 4:
			verticalDirection = planePoints.x->position - neighbour->position;
			break;
			default:
			break;
			}
		}
		plane->pointDirections = { horizontalDirection, verticalDirection };
		plane->normal = normal;
	}
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(plane->normal, neighbourPoints[j]->position - plane->planePointPos));
					if (dist <= planeDistanceTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
						if ((neighbourPoints[j]->normal - plane->normal).length() < 0.05) {
							plane->normal = Vec3<double>::normalize(plane->normal * (plane->points.size() - 1) + neighbourPoints[j]->normal);
						}
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void findPlanes()
{
	size_t minPointCount = 10;
	size_t counter = 1;
	double normalTreshold = 0.01;
	for (size_t i = 0; i < points.size(); i++)
		if (points[i]) {
			points[i]->isMarked = true;
			points[i]->isMarked2 = true;
			calculateNormal(points[i]);
		}
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			nextStepPoints.push_back(points[j]);
			while (nextStepPoints.size() > 0) {
				Plane* plane = new Plane();
				bool planeMerged;
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x + 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x - 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k != 2 || y < verticalCount - 1) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if (normals[k].length() > 0 && normals[(k + 1) % 4].length() > 0 &&
								(normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {
								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, nullptr,
									plane);
								break;
							}
						}
						if (plane->points.size() > 0)
							break;
					}
				}
				if (plane->points.size() > 0)
				{
					planes.push_back(plane);
					currentPlaneId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		int originalSize = planes[i]->points.size();
		while (isThereBridge(planes[i]->points)) {}		
		if (originalSize != planes[i]->points.size() && planes[i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[i]->points[0], nullptr, nullptr }, planes[i], plane);
				if (plane->points.size() < planes[i]->points.size()) {
					for (size_t j = 0; j < planes[i]->points.size(); j++) {
						if (planes[i]->points[j]->plane != planes[i]) {
							planes[i]->points.erase(planes[i]->points.begin() + j);
							j--;
						}
					}
					planes.push_back(plane);
					currentPlaneId++;
				}
				else 
				{
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[i];
					}
					delete plane;
					break;
				}
			}
		}
	}
}

bool hasNonSpykeNeighbour(size_t x, size_t y)
{
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 4; i++) {
		switch (i) {
		case 0: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && spikeType(neighbourPoint, -1, false) == 2)
			return true;
	}
	return false;
}

void findNextPoint(Point*& startPoint, size_t direction, /*out*/ 
	std::vector<std::pair<Point*, int>>& currentEdge, std::vector<Point*>& spikePoints, size_t dbgPlaneIndex, std::vector<Plane*> dbgPlanes)
{
	//isMarked -- turned off after the edge is complete
	//isMarked -- turned off when point added to edge
	
	Point* currentPoint = nullptr;
	std::pair<Point*, size_t> previousSavedPoint = {nullptr, 0};
	bool isFirstPoint = true;
	bool comeFromDeadEnd = false;
	bool isPreviousSpike = false;
	bool wasThereNonSpike = false;
	bool isHole = direction == 1;
	while (currentPoint != startPoint || comeFromDeadEnd)
	{
		
		if (!currentPoint)
			currentPoint = startPoint;
		if (currentPoint->horizontalIndex == 77 && currentPoint->verticalIndex == 95) {
			std::cout << "asd";
		}
		Point* neighbourPoint = nullptr;
		size_t x = currentPoint->horizontalIndex;
		size_t y = currentPoint->verticalIndex;
		isPreviousSpike = spikeType(currentPoint, (!wasThereNonSpike || currentPoint == startPoint) ? -1 : ((direction + 1) % 4), false) == 0;
		if (!isPreviousSpike || currentEdge.size() == 0) {
			if (currentPoint->isMarked2) {
				currentEdge.push_back({ currentPoint, direction });
			}
		}
		else spikePoints.push_back(currentPoint);
		if(true || !isHole) currentPoint->isMarked2 = false;		
		for (size_t i = 0; i < 4; i++) {
			/*if (checkIfBridge(currentPoint)) {
				i += 2;
				direction = (direction + 2) % 4;
			}*/
			switch (direction) {
			case 0: //to right
			neighbourPoint = points[getOffset(x + 1, y)];
			break;
			case 1: //to down
			neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
			break;
			case 2: //to left
			neighbourPoint = points[getOffset(x - 1, y)];
			break;
			case 3: //to up
			neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
			break;
			default:
			break;
			}
			if (neighbourPoint == startPoint) {
				currentPoint = startPoint;
				currentEdge[0].second = (direction + 3) % 4;
				comeFromDeadEnd = false;
				break;
			}
			if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr && neighbourPoint->isMarked2
				&& (isPreviousSpike || spikeType(neighbourPoint, -1, false) <= 1 || spikeType(currentPoint, direction, false) > 0))
			{
				if (isFirstPoint) {
					currentEdge[0].second = (direction + 1) % 4 ;
					isFirstPoint = false;
				}
				if (!wasThereNonSpike && spikeType(currentPoint, -1, false) > 1) {
					if (currentEdge.size() > 1) 
					{
						auto helper = currentEdge[0];
						currentEdge[0] = currentEdge[1];
						currentEdge[1] = helper;
						startPoint = currentPoint;
						if (!hasNonSpykeNeighbour(startPoint->horizontalIndex, startPoint->verticalIndex)) {
							for (size_t j = 0; j < currentEdge.size(); j++) {
								spikePoints.push_back(currentEdge[j].first);
								currentEdge[j].first->plane = nullptr;
							}
							currentEdge.clear();
							return;
						}
					}
					isPreviousSpike = false;
					wasThereNonSpike = true;
				}
				if (!isPreviousSpike)
					previousSavedPoint = { currentPoint, (direction + (4 - i)) % 4 };
				auto neighbourSpikeType = spikeType(neighbourPoint, -1, false);
				if ((!isHole && checkIfBridge(neighbourPoint, true)) || neighbourSpikeType == -1 || neighbourSpikeType == 1 || (wasThereNonSpike && 
					currentEdge.size() > 1 && isPreviousSpike &&  neighbourSpikeType == 2 && neighbourPoint->isMarked)) {
					auto savedPoint = neighbourSpikeType == 1 ? neighbourPoint : currentPoint;
					if(savedPoint->isMarked2)
						currentEdge.push_back({ savedPoint, (direction + 3) % 4 });
					if (neighbourSpikeType == -1 || (!isHole && checkIfBridge(neighbourPoint, true)))
					{
						neighbourPoint->isMarked = false;
						neighbourPoint->isMarked2 = false;
						neighbourPoint->plane = nullptr;
					}
					savedPoint->isMarked2 = false;
					currentPoint = previousSavedPoint.first;
					direction = previousSavedPoint.second;
					isPreviousSpike = false;
					comeFromDeadEnd = true;
					break;
				}
				else {
					comeFromDeadEnd = false;
					currentPoint = neighbourPoint;
				}
				direction = (direction + 3) % 4;
				break;
			}
			direction += direction == 3 ? -3 : 1;
			if (i == 3) {
				if (!wasThereNonSpike) {
					for (size_t j = 0; j < currentEdge.size(); j++) {
						currentEdge[j].first->plane = nullptr;
						spikePoints.push_back(currentEdge[j].first);
					}
					currentEdge.clear();
					return;
				}
				if (currentPoint == startPoint)
					return;
				currentPoint = currentEdge[currentEdge.size() - 2].first;
				direction = currentEdge[currentEdge.size() - 2].second;
				std::cout << "INVALID EDGE SEARCH" << std::endl;;
			}
		}
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

void findEdgePoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> outerConnectedEdgePoints;
		std::vector<Point*> holeConnectedEdgePoints;
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) if(isEdgePoint(planes[i]->points[j])) edgePointsInPlane.push_back(planes[i]->points[j]);
		while (edgePointsInPlane.size() > 0) 
		{
			Edge* currentEdge = new Edge();
			std::vector<Point*> tempEdgePointsInPlane;
			Point* startPoint = edgePointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
				if ((edgePointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - edgePointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| edgePointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = edgePointsInPlane[j]->horizontalIndex;
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
				else if (edgePointsInPlane[j]->horizontalIndex == minHorizontalCoord && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
			}
			size_t direction = 2;
			if (spikeType(startPoint, -1, true) == -1) {
				size_t x = startPoint->horizontalIndex;
				size_t y = startPoint->verticalIndex;
				startPoint->plane = nullptr;
				startPoint->isMarked = false;
				startPoint->isMarked2 = false;
				for (size_t j = 0; j < planes[i]->points.size(); j++) {
					if (planes[i]->points[j] == startPoint) {
						planes[i]->points[j]->plane = nullptr;
						planes[i]->points.erase(planes[i]->points.begin() + j);
						break;
					}
				}
				minVerticalCoord = points[getOffset(x + 1, y)]->verticalIndex;
				for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
					if (edgePointsInPlane[j]->horizontalIndex == x + 1 && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) 			
					{
						minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
						startPoint = edgePointsInPlane[j];
					}
				}
			}
			else if (startPoint->verticalIndex > 0 && points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)] &&
				points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)]->plane == startPoint->plane &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex - 1)] &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex - 1)]->plane == startPoint->plane &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex)] &&
				points[getOffset(startPoint->horizontalIndex - 1, startPoint->verticalIndex)]->plane == startPoint->plane)
			{
				direction = 1;
				currentEdge->isHole = true;
			}			
			if (i == 28) {
				std::cout << "asd";
			}
			std::vector<Point*> spikePoints;
			setPointsMarked(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints, false, false);
			setPointsMarked(currentEdge->isHole ? outerConnectedEdgePoints : holeConnectedEdgePoints, true, false);
			findNextPoint(startPoint, direction, currentEdge->pointsWithDir, spikePoints, i, planes);
			setPointsMarked(outerConnectedEdgePoints, false, false);
			setPointsMarked(holeConnectedEdgePoints, false, false);
			for (size_t j = 0; j < currentEdge->pointsWithDir.size(); j++) {
				currentEdge->pointsWithDir[j].first->isMarked = false;
				(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints).push_back(currentEdge->pointsWithDir[j].first);
			}
			for (size_t j = 0; j < spikePoints.size(); j++) {
				spikePoints[j]->isMarked = false;
				(currentEdge->isHole ? holeConnectedEdgePoints : outerConnectedEdgePoints).push_back(spikePoints[j]);
			}
			for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
				if (edgePointsInPlane[j]->isMarked) tempEdgePointsInPlane.push_back(edgePointsInPlane[j]);
			}
			edgePointsInPlane = tempEdgePointsInPlane;
			if (currentEdge->pointsWithDir.size() > 3) {
				if (direction == 1) {
					currentEdge->pointsWithDir.insert(currentEdge->pointsWithDir.begin(),
						currentEdge->pointsWithDir[currentEdge->pointsWithDir.size() - 1]);
					currentEdge->pointsWithDir.pop_back();
				}
				for (size_t k = 0; k < currentEdge->pointsWithDir.size(); k++) {
					currentEdge->pointsWithDir[k].first->outlineId = currentOutlineId;
				}
				currentOutlineId++;
				currentEdge->startPoint = startPoint;
				planes[i]->edges.push_back(currentEdge);
			}
		}
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = false;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = false;
	}
}

const double newPointAcceptTreshold = 0.95;
const double inf = 1000000;

bool isStraightPoint(size_t pointIndex, Edge*& edge, size_t& previousNeighbourCount, 
	Vec3<double>& straigthDir)
{
	const double newDirTreshold = 0.1;
	Point* point = edge->pointsWithDir[pointIndex].first;
	if (point->horizontalIndex == 666 && point->verticalIndex == 29) {
		//std::cout << "asd";
	}
	size_t previousNeighbourCountStore = previousNeighbourCount;
	size_t neighbourCount = 0;
	size_t neighbourEdgeCount = 0;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Plane* plane = point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i != 1) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane) {
			neighbourCount++;
			isNeighbour[i] = true;
			if (neighbourPoints[i]->outlineId > 0)
				neighbourEdgeCount++;
		}
	}
	previousNeighbourCount = neighbourCount;
	if (pointIndex > 0 && areNeighbours(pointIndex < (edge->pointsWithDir.size() - 1) ? edge->pointsWithDir[pointIndex + 1].first : edge->startPoint,
		point) == 0 && neighbourCount < 3) { //deadend
		auto previousPoint = edge->pointsWithDir[pointIndex - 1];
		auto previousPreviousPoint = pointIndex > 1 ? edge->pointsWithDir[pointIndex - 2] : edge->pointsWithDir[edge->pointsWithDir.size() - 2 
			+ pointIndex];
		straigthDir = { 0,0,0 };
		if (areNeighbours(previousPoint.first, previousPreviousPoint.first) > 0 && ((previousPoint.first->verticalIndex ==
			previousPreviousPoint.first->verticalIndex && previousPoint.first->horizontalIndex == point->horizontalIndex &&
			previousPoint.first->verticalIndex != point->verticalIndex) || (previousPoint.first->horizontalIndex ==
				previousPreviousPoint.first->horizontalIndex && previousPoint.first->verticalIndex == point->verticalIndex &&
				previousPoint.first->horizontalIndex != point->horizontalIndex))) //if curve swap with previous
		{
			point->isCorner = true;
			edge->pointsWithDir[pointIndex - 1] = edge->pointsWithDir[pointIndex];
			edge->pointsWithDir[pointIndex] = previousPoint;
			return true;
		}
		return false;
	}
	if (neighbourCount == 3 && (neighbourEdgeCount == 2 && (pointIndex == edge->pointsWithDir.size() - 1 || 
		areNeighbours(point, edge->pointsWithDir[pointIndex + 1].first)
		> 0)))
	{
		if (straigthDir.length() == 0) {
			if (pointIndex > 0)
				straigthDir = Vec3<double>::normalize(point->position - edge->pointsWithDir[pointIndex - 1].first->position);
		}
		else
		{
			auto newDir = Vec3<double>::normalize(point->position - edge->pointsWithDir[pointIndex - 1].first->position);
			if ((straigthDir - newDir).length() > newDirTreshold) {
				straigthDir = newDir;
				return false;
			}
		}
		return true;
	}
	straigthDir = { 0,0,0 };
	if (neighbourCount == 4 && (pointIndex == 0 ? edge->pointsWithDir[edge->pointsWithDir.size() - 1].first : 
		edge->pointsWithDir[pointIndex - 1].first)->isCorner &&
		previousNeighbourCountStore == 2)
		return true;
	if (previousNeighbourCountStore == 4) {
		edge->pointsWithDir[pointIndex - 1].first->isCorner = false;
	}
	return false;
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k]->edges.size(); i++) {
			size_t previousNeighbourCount = 0;
			Vec3<double> straightDir = { 0,0,0 };
			for (size_t j = 0; j < planes[k]->edges[i]->pointsWithDir.size(); j++) {
				if (k == 86 && i == 0) {
				//	std::cout << "asd";
				}
				if (!isStraightPoint(j, planes[k]->edges[i], previousNeighbourCount, straightDir)) {
					planes[k]->edges[i]->pointsWithDir[j].first->isCorner = true;
				}
			}
		}
	}
}

Point* createNewPoint(Vec3<double> newPointPos, Point* point, std::vector<Point*> neighbours, size_t addedCount, bool createBeforePoint = false, 
	bool isCornerPoint = false)
{
	double deleteDurroundingCornersTreshold = 0.1;
	Point* newPoint = new Point(newPointPos, neighbours[0]->horizontalIndex, isCornerPoint ? verticalCount + 1 : verticalCount, point->plane);
	addedPoints.push_back(newPoint);
	newPoint->isCorner = true;
	newPoint->outlineId = point->outlineId;
	for (size_t j = 0; j < point->plane->edges.size(); j++) {
		if (point->plane->edges[j]->pointsWithDir[0].first->outlineId == point->outlineId) {
			for (size_t k = 0; k < point->plane->edges[j]->pointsWithDir.size(); k++) {
				if (point->plane->edges[j]->pointsWithDir[k].first == point) {
					auto index = k + (createBeforePoint ? 0 : 1) + addedCount;
					point->plane->edges[j]->pointsWithDir.insert(point->plane->edges[j]->pointsWithDir.begin() + index, { newPoint, -1 });
					break;
				}
			}
			break;
		}
	}
	for (size_t i = 0; i < neighbours.size(); i++) {
		newPoint->neighbourPlaneNeighbours[i] = neighbours[i];
	}
	return newPoint;
}

const double twoPointDifferenceTreshold = 0.5;

Point* addNewPoint(Point* point, Point*& neighbour, Plane* plane, size_t addedCount, size_t neighbourIndex)
{
	if (point->createdNeighbourPoints[neighbourIndex] != nullptr) { //created by other plane
		auto createdNeighbour = point->createdNeighbourPoints[neighbourIndex];
		createdNeighbour->isCorner = true;
		for (size_t j = 0; j < point->plane->edges.size(); j++) {
			if (point->plane->edges[j]->pointsWithDir[0].first->outlineId == point->outlineId) {
				for (size_t k = 0; k < point->plane->edges[j]->pointsWithDir.size(); k++) {
					if (point->plane->edges[j]->pointsWithDir[k].first == point) {
						for (size_t l = 0; l < 4; l++) 
						{
							if (point->plane->edges[j]->pointsWithDir[k + l + 1].first == createdNeighbour) {
								point->plane->edges[j]->pointsWithDir.insert(point->plane->edges[j]->pointsWithDir.begin() + k + 1 + addedCount,
									{ createdNeighbour, -1 });
								point->plane->edges[j]->pointsWithDir.erase(point->plane->edges[j]->pointsWithDir.begin() + k + l + 2);
								break;
							}
						}
						break;
					}
				}
				break;
			}
		}
		return createdNeighbour;
	}
	Vec3<double> dir = { 0,0,0 };
	switch (areNeighbours(point, neighbour)) {
	case 1:
	dir = point->plane->pointDirections.first;
	break;
	case 2:
	dir = point->plane->pointDirections.second;
	break;
	case 3:
	dir = point->plane->pointDirections.first * -1;
	break;
	case 4:
	dir = point->plane->pointDirections.second * -1;
	break;
	default:
	break;
	}
	Vec3<double> newPointPos = point->position - dir * Vec3<double>::dot_product(point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf || isnan(newPointPos.x) || isnan(newPointPos.y) || isnan(newPointPos.z))
		return nullptr;
	Vec3<double> dirToNew = newPointPos - point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold && 
		(point->position - newPointPos).length() > planeDistanceTreshold)
		return nullptr;
	Vec3<double> neighbourDir = { 0,0,0 };
	switch (areNeighbours(neighbour, point)) {
	case 1:
	neighbourDir = plane->pointDirections.first;
	break;
	case 2:
	neighbourDir = plane->pointDirections.second;
	break;
	case 3:
	neighbourDir = plane->pointDirections.first * -1;
	break;
	case 4:
	neighbourDir = plane->pointDirections.second * -1;
	break;
	default:
	break;
	}	
	Vec3<double> neighbourNewPointPos = neighbour->position - neighbourDir * Vec3<double>::dot_product(neighbour->position
		- point->plane->planePointPos, point->plane->normal) / Vec3<double>::dot_product(neighbourDir, point->plane->normal);
	if (abs(neighbourNewPointPos.x) > inf || abs(neighbourNewPointPos.y) > inf || abs(neighbourNewPointPos.z) > inf || isnan(neighbourNewPointPos.x) ||
		isnan(neighbourNewPointPos.y) || isnan(neighbourNewPointPos.z))
		return nullptr;
	Vec3<double> dirToNewNeighbour = neighbourNewPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(neighbourDir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold &&
		(neighbour->position - neighbourNewPointPos).length() > planeDistanceTreshold)
		return nullptr;
	if ((newPointPos - neighbourNewPointPos).length() > twoPointDifferenceTreshold)
		return nullptr;
	auto newPos = (newPointPos + neighbourNewPointPos) / 2;
	Point* newPoint;
	Point* newNeighbourPoint;
	newPoint = createNewPoint(newPos, point, { neighbour }, addedCount);
	point->createdNeighbourPoints[neighbourIndex] = newPoint;
	for (size_t i = 0; i < neighbour->neighbourPlaneNeighbours.size(); i++) 
	{
		if (neighbour->neighbourPlaneNeighbours[i] == point) 		
		{
			newNeighbourPoint = createNewPoint(newPos, neighbour, { point }, 0, false);
			neighbour->createdNeighbourPoints[i] = newNeighbourPoint;
			break;
		}
	}
	newPoint->neighbourPlaneNeighbours[1] = newNeighbourPoint;
	newNeighbourPoint->neighbourPlaneNeighbours[1] = newPoint;
	return newPoint;
}

void findPlaneConnections()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		for (size_t j = 0; j < planes[i]->edges.size(); j++) 
		{
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++)
			{
				Point* point = planes[i]->edges[j]->pointsWithDir[k].first;
				if (point->horizontalIndex == 339 && point->verticalIndex == 14) 	
				{
					//std::cout << "asd";
				}
				int direction = planes[i]->edges[j]->pointsWithDir[k].second;
				size_t x = point->horizontalIndex;
				size_t y = point->verticalIndex;
				Point* neighbourPoints[4] = { points[getOffset(x + 1, y)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
					points[getOffset(x, y - 1)] };
				for (size_t i = 0; i < 4; i++) {
					if ((y > 0 || direction != 3) && (y < verticalCount - 1 || direction != 1) && neighbourPoints[direction] && 
						neighbourPoints[direction]->outlineId > 0 && neighbourPoints[direction]->outlineId != point->outlineId && 
						neighbourPoints[direction]->plane != point->plane) {
						point->neighbourPlaneNeighbours[i] = neighbourPoints[direction];
					}
					else if((y == 0 && direction == 3) || (y == verticalCount - 1 && direction == 1) || !neighbourPoints[direction] ||
						neighbourPoints[direction]->plane == nullptr)
						point->neighbourPlaneNeighbours[i] = nullptr;
					direction += direction == 3 ? -3 : 1;
				}
			}
		}
	}
}

void createPlaneCorner(Point* point, std::vector<Point*> point1, std::vector<Point*> point2)
{
	Plane* p1 = point->plane;
	Plane* p2 = point1[2]->plane;
	Plane* p3 = point2[2]->plane;
	auto cross1 = Vec3<double>::crossProduct(p2->normal, p3->normal);
	auto cross2 = Vec3<double>::crossProduct(p3->normal, p1->normal);
	auto cross3 = Vec3<double>::crossProduct(p1->normal, p2->normal);

	auto denom = Vec3<double>::dot_product(p1->normal, cross1);

	auto planeDist1 = Vec3<double>::dot_product(p1->planePointPos, p1->normal);
	auto planeDist2 = Vec3<double>::dot_product(p2->planePointPos, p2->normal);
	auto planeDist3 = Vec3<double>::dot_product(p3->planePointPos, p3->normal);

	cross1 = cross1 * planeDist1;
	cross2 = cross2 * planeDist2;
	cross3 = cross3 * planeDist3;

	auto cornerPoint = (cross1 + cross2 + cross3) / denom;

	createNewPoint(cornerPoint, point1[0], {point1[2], point2[2]}, 0, false, true);
	createNewPoint(cornerPoint, point1[1], { point }, 0, true, true);
	createNewPoint(cornerPoint, point2[1], { point }, 0, false, true);
}

void connectPlanes()
{
	std::vector<Point*> createdPoints;
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				auto point = planes[i]->edges[j]->pointsWithDir[k].first;
				if (point->horizontalIndex == 666 && point->verticalIndex == 29) {
					//std::cout << "asd";
				}
				if (point->verticalIndex == verticalCount)
				{
					continue;
				}
				size_t addedCount = 0;
				if(point->isCorner)
				{					
					for (size_t l = 0; l < point->neighbourPlaneNeighbours.size(); l++) {
						Point* newPoint = nullptr;
						if (point->neighbourPlaneNeighbours[l])
						{
							newPoint = addNewPoint(point, point->neighbourPlaneNeighbours[l], point->neighbourPlaneNeighbours[l]->plane, addedCount, l);
							if (newPoint) {
								if (k == 0 && l == 0) planes[i]->edges[planes[i]->edges.size() - 1]->wasFirstGenerated = true;
								planes[i]->edges[j]->pointsWithDir[k].first->isCorner = false;
								createdPoints.push_back(newPoint);
								addedCount++;
							}
							else {
								planes[i]->edges[j]->pointsWithDir.insert(planes[i]->edges[j]->pointsWithDir.begin() + k + 1 + addedCount, { nullptr, -1 });
								addedCount++;
							}
						}						
					}
					k += addedCount;
				}
			}			
		}
	}
}

void createCorners()
{
	const double normalDiffTreshold = 0.1;
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {	
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				auto point = planes[i]->edges[j]->pointsWithDir[k].first;
				if (point) {
					for (size_t l = 0; l < planes[i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints.size(); l++) {
						auto createdPoint1 = planes[i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints[l];
						auto createdPoint2 = planes[i]->edges[j]->pointsWithDir[k].first->createdNeighbourPoints[l == 3 ? 0 : (l + 1)];
						if (createdPoint1 && createdPoint2 && createdPoint1->neighbourPlaneNeighbours[1]->plane !=
							createdPoint2->neighbourPlaneNeighbours[1]->plane
							&& (planes[i]->edges[j]->pointsWithDir[k].first->plane->normal -
							createdPoint1->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold && 
							(planes[i]->edges[j]->pointsWithDir[k].first->plane->normal -
								createdPoint2->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold && 
							(createdPoint1->neighbourPlaneNeighbours[1]->plane->normal -
									createdPoint2->neighbourPlaneNeighbours[1]->plane->normal).length() > normalDiffTreshold) {
							createPlaneCorner(planes[i]->edges[j]->pointsWithDir[k].first, { createdPoint1, createdPoint1->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l] }, { createdPoint2, createdPoint2->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l == 3 ? 0 : (l + 1)] });
							break;
						}
					}
				}
			}
		}
	}
}

void filterEdgePoints()
{

	std::vector<std::vector<std::vector< std::pair<bool, std::pair<size_t, size_t>>>>> cornersToDelete;
	for (size_t i = 0; i < planes.size(); i++) {
		cornersToDelete.push_back({});
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			cornersToDelete[i].push_back({});
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first) {
					if (planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex >= verticalCount) {
						for (size_t l = 0; l < planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours.size(); l++) {
							if (planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l] &&
								planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l]->verticalIndex < verticalCount)
								cornersToDelete[i][j].push_back({ planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex > verticalCount,
									{ k,  planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[l]->plane->id } });
						}
					}
				}
				else {
					cornersToDelete[i][j].push_back({ false, { 0, 0 } });
					planes[i]->edges[j]->pointsWithDir.erase(planes[i]->edges[j]->pointsWithDir.begin() + k);
					k--;
				}
			}
			for (int k = 0; k < cornersToDelete[i][j].size(); k++) 
			{
				if (k < ((int)cornersToDelete[i][j].size() - 2) && cornersToDelete[i][j][k + 1].first && cornersToDelete[i][j][k + 2].first) 
				{
					cornersToDelete[i][j][k + 1].second = { cornersToDelete[i][j][k + 1].second.first, cornersToDelete[i][j][k].second.second };
					size_t index = k + 2;
					while (index < cornersToDelete[i][j].size() && cornersToDelete[i][j][index].first)
					{
						cornersToDelete[i][j].erase(cornersToDelete[i][j].begin() + index);
					}
					cornersToDelete[i][j].insert(cornersToDelete[i][j].begin() + index, { true, {cornersToDelete[i][j][k + 1].second.first,
						index < cornersToDelete[i][j].size() ? cornersToDelete[i][j][index].second.second : cornersToDelete[i][j][0].second.second} });
				}
			}
			int newPointStartIndex = planes[i]->edges[j]->wasFirstGenerated ? 1 : 0;
			if (planes[i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->horizontalIndex < planes[i]->edges[j]->startPoint->horizontalIndex &&
				planes[i]->edges[j]->pointsWithDir[newPointStartIndex + 1].first->createdNeighbourPoints[0])
				newPointStartIndex++;
			int newPointEndIndex = newPointStartIndex;
			if (i == 165) {
				std::cout << "asd";
			}
			while (newPointEndIndex < (int)cornersToDelete[i][j].size() - 1) {
				if (cornersToDelete[i][j][newPointStartIndex].second.second > 0) {
					while (newPointEndIndex + 1 < cornersToDelete[i][j].size()
						&& cornersToDelete[i][j][newPointStartIndex].second.second == cornersToDelete[i][j][newPointEndIndex + 1].second.second) {
						newPointEndIndex++;
					}
					for (size_t k = cornersToDelete[i][j][newPointStartIndex].second.first + 1; k <
						cornersToDelete[i][j][newPointEndIndex].second.first; k++) {
						planes[i]->edges[j]->pointsWithDir[k].first->isCorner = false;
					}
					auto startPoint = planes[i]->edges[j]->pointsWithDir[cornersToDelete[i][j][newPointStartIndex].second.first].first;
					auto endPoint = planes[i]->edges[j]->pointsWithDir[cornersToDelete[i][j][newPointEndIndex].second.first].first;
					if ((startPoint->position - endPoint->position).length() < 0.1) {
						if (startPoint->verticalIndex == verticalCount + 1 && endPoint->verticalIndex != verticalCount + 1)
							endPoint->isCorner = false;
						else if (endPoint->verticalIndex == verticalCount + 1 && startPoint->verticalIndex != verticalCount + 1)
							startPoint->isCorner = false;
					}
					if (newPointEndIndex == cornersToDelete[i][j].size() - 1 && newPointStartIndex != newPointEndIndex && 
						planes[i]->edges[j]->wasFirstGenerated && cornersToDelete[i][j][0].second.second ==
						cornersToDelete[i][j][newPointStartIndex].second.second)
						planes[i]->edges[j]->pointsWithDir[cornersToDelete[i][j][cornersToDelete[i][j].size() - 1].second.first].first->isCorner = false;
				}
				newPointStartIndex = newPointEndIndex + 1;
				newPointEndIndex = newPointStartIndex;
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first->verticalIndex == verticalCount && planes[i]->edges[j]->pointsWithDir[k].first->isCorner
					&& planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1] &&
					planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1]->verticalIndex == verticalCount) {
					planes[i]->edges[j]->pointsWithDir[k].first->neighbourPlaneNeighbours[1]->isCorner = true;
				}
			}
		}
	}*/
}

void egoCarSegmentation(size_t frameIndex)
{
	float egoCarPosZ = -(int)frameIndex * (50.0 / 36.0);
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 + egoCarPosZ && points[i]->position.z >= -2.5 + egoCarPosZ)
			points[i] = nullptr;
	}
}

const double PI = 3.14159265359;

double angleOfVectors(Vec3<double> v1, Vec3<double> v2, bool isBackward)
{
	auto v1Angle = atan2(v1.x, v1.y) / PI;
	auto v2Angle = atan2(v2.x, v2.y) / PI;
	auto angle = abs(v1Angle - v2Angle);
	if ((!isBackward && v1Angle < v2Angle) || (isBackward && v1Angle > v2Angle)) angle = 2 - angle;
	return angle * 180;
}

Vec3<double> intersectionOfLines(Vec3<double> p1, Vec3<double> p2, Vec3<double> q1, Vec3<double> q2, bool& isOnEdge)
{
	auto dir1 = p2 - p1;
	auto dir2 = q2 - q1;
	double R = (p1.y * dir1.x + q1.x * dir1.y - p1.x * dir1.y - q1.y * dir1.x) / (dir2.y * dir1.x - dir1.y * dir2.x);
	Vec3<double> intersectionPos = { q1.x + R * dir2.x, q1.y + R * dir2.y, 0 };
	auto side1Length = (p2 - p1).length();
	auto side2Length = (q2 - q1).length();
	if ((intersectionPos - p1).length() < side1Length && (intersectionPos - p2).length() < side1Length &&
		(intersectionPos - q1).length() < side2Length && (intersectionPos - q2).length() < side2Length)
		isOnEdge = true;
	return intersectionPos;
}

bool isPointInsidePolygon(std::vector<std::pair<Vec3<double>, Point*>>polygon, Vec3<double> point, std::pair<double, double> xBounds,
	std::pair<double, double> yBounds)
{
	bool notInUse = false;
	if (point.x > xBounds.first && point.x < xBounds.second &&
		point.y > yBounds.first && point.y < yBounds.second) 
	{
		int rigthCounter = 0;
		for (size_t i = 0; i < polygon.size(); i++) 
		{
			auto p1 = polygon[i].first;
			auto p2 = polygon[(i + 1) % polygon.size()].first;
			if ((p1.x < point.x && p2.x < point.x) || (p1.y > point.y && p2.y > point.y) || (p1.y < point.y && p2.y < point.y) || (p1.y == p2.y ))
				continue;
			else if (point == p1)
				return false;
			else
			{
				/*auto smallerAngle = (p1.x < p2.x ? p1 - point : (p2 - point));
				auto biggerAngle = (p1.x < p2.x ? p2 - point : (p1 - point));
				if((p1.x > point.x && p2.x > point.x) || angleOfVectors(smallerAngle, biggerAngle, smallerAngle.y > point.y || biggerAngle.y < point.y)
					< 180)*/
				auto intersection = intersectionOfLines(point, point + Vec3<double>({ 1,0,0 }), p1, p2, notInUse);
				if (intersection.x <= point.x)
					continue;
				else
				{
					rigthCounter++;
					if (p2.y == point.y)
						i++;
				}
			}
		}
		return rigthCounter % 2 == 1;
	}
	return false;
}

void changeBaseTo2D(std::vector<std::pair<Vec3<double>, Point*>>& points)
{
	auto normal = points[0].second->plane->normal;
	auto x = points[0].second->plane->pointDirections.first;
	x = Vec3<double>::normalize(x - normal * Vec3<double>::dot_product(x, normal));
	auto y = Vec3<double>::crossProduct(x, normal);
	for (size_t i = 0; i < points.size(); i++) {
		points[i].first = { Vec3<double>::dot_product(points[i].second->position, x), Vec3<double>::dot_product(points[i].second->position, y), 0 };
	}
	for (int i = 0; i < points.size(); i++) {
		for (int j = (i == (points.size() - 1) ? 1 : 0); j < i - 1; j++) {
			bool isOnEdge = false;
			auto intersection = intersectionOfLines(points[j].first, points[(j + 1) % points.size()].first, points[i].first,
				points[(i + 1) % points.size()].first, isOnEdge);
			if(isOnEdge)
			{
				points[i].second->isCorner = false;
				points.erase(points.begin() + i);
				i-= 2;
				break;
			}
		}
	}
}

bool isClockwise(std::vector<std::pair<Vec3<double>, Point*>>& points)
{
	double angleSum = 0;
	for (size_t i = 0; i < points.size(); i++) {
		auto  a = angleOfVectors(points[(i + points.size() - 1) % points.size()].first - points[i].first,
			points[(i + 1) % points.size()].first - points[i].first, false);
		angleSum += angleOfVectors(points[(i + points.size() - 1) % points.size()].first - points[i].first,
			points[(i + 1) % points.size()].first - points[i].first, false);
	}
	return angleSum < (double)points.size() * 360.0 / 2;
}

void convexSegmentation()
{
	const std::pair<double, double> acceptAngle = { 181, 359 };
	size_t currentConvexId = 1;
	for (size_t i = 0; i < planes.size(); i++) {
		std::vector<std::vector<std::pair<Vec3<double>, Point*>>> holeEdges;
		size_t outerCounter = 0;
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			if (planes[i]->edges[j]->isHole) {
				outerCounter++;
				std::vector<std::pair<Vec3<double>, Point*>> holeEdge;
				size_t counter = 0;
				for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
					if (planes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
						holeEdge.push_back({ {0,0,0}, planes[i]->edges[j]->pointsWithDir[k].first });
						counter++;
					}
				}
				if (holeEdge.size() < 4) {
					for (size_t k = 0; k < holeEdge.size(); k++) {
						holeEdge[k].second->isCorner = false;
					}
				}
				else {
					changeBaseTo2D(holeEdge);
					holeEdges.push_back(holeEdge);
				}
			}
		}
		if (i == 51) 			
		{
			std::cout << "asd";
			//return;
		}
		for (size_t x = 0; x < planes[i]->edges.size(); x++) {
			if (!planes[i]->edges[x]->isHole) {
				//std::vector<Vec3<double>> remainingPoints;
				std::vector<std::pair<Vec3<double>, Point*>> remainingPoints;
				size_t counter = 0;
				Vec3<double> pivotPoint = { 10000, 10000, 10000 };
				for (size_t j = 0; j < planes[i]->edges[x]->pointsWithDir.size(); j++) {
					if (planes[i]->edges[x]->pointsWithDir[j].first->isCorner) {
						if (pivotPoint == Vec3<double>({ 10000, 10000, 10000 }))
						{
							pivotPoint = planes[i]->edges[x]->pointsWithDir[j].first->position;
						}
						remainingPoints.push_back({ {},	planes[i]->edges[x]->pointsWithDir[j].first });
						counter++;
					}
				}
				if (remainingPoints.size() < 4) {
					for (size_t k = 0; k < remainingPoints.size(); k++) {
						remainingPoints[k].second->isCorner = false;
					}
					continue;
				}				
				changeBaseTo2D(remainingPoints);
				if (!isClockwise(remainingPoints)) 
				{
					for (size_t k = 0; k < remainingPoints.size(); k++) 
					{
						remainingPoints.push_back(remainingPoints[remainingPoints.size() - 1 - k]);
						remainingPoints.erase(remainingPoints.begin() + remainingPoints.size() - 2 - k);
					}
				}
				while (remainingPoints.size() > 3) {
					if (i == 16) {
						int a = 1;
					}
					//std::vector<Vec3<double>> remainingPointsHelper(remainingPoints);
					//std::vector<Vec3<double>> L = { remainingPointsHelper[0], remainingPointsHelper[1] };

					std::vector<std::pair<Vec3<double>, Point*>> remainingPointsHelper(remainingPoints);
					std::vector<std::pair<Vec3<double>, Point*>> L = { remainingPointsHelper[0], remainingPointsHelper[1] };
						std::vector<std::pair<Vec3<double>, Point*>> remainingPointsHelperSave;
					std::vector<std::pair<Vec3<double>, Point*>> LSave;

					remainingPointsHelper.erase(remainingPointsHelper.begin(), remainingPointsHelper.begin() + 2);
					std::pair<double, double> xBounds = { std::min(L[0].first.x, L[1].first.x), std::max(L[0].first.x, L[1].first.x) };
					std::pair<double, double> yBounds = { std::min(L[0].first.y, L[1].first.y), std::max(L[0].first.y, L[1].first.y) };

					for (int j = 0; j < 2; j++) {
						bool isForward = j == 0;
						while (remainingPointsHelper.size() > 0) {
							auto newPoint = isForward ? remainingPointsHelper[0] : remainingPointsHelper[remainingPointsHelper.size() - 1];
							auto v1 = isForward ? L[L.size() - 1].first - L[L.size() - 2].first : (L[0].first - L[1].first);
							auto v2 = isForward ? newPoint.first - L[L.size() - 1].first : (newPoint.first - L[0].first);
							auto vecToBegin = isForward ? L[0].first - newPoint.first : (L[L.size() - 1].first - newPoint.first);
							auto vecAtBegin = isForward ? L[1].first - L[0].first : (L[L.size() - 2].first - L[L.size() - 1].first);
							auto temp = angleOfVectors(v1 * -1, v2, !isForward);
							auto temp2 = angleOfVectors(v2 * -1, vecToBegin, !isForward);
							auto temp3 = angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward);
							if ((angleOfVectors(v1 * -1, v2, !isForward) <= acceptAngle.first || angleOfVectors(v1 * -1, v2, !isForward) >= 
								acceptAngle.second) &&
								(angleOfVectors(v2 * -1, vecToBegin, !isForward) <= acceptAngle.first || angleOfVectors(v2 * -1, vecToBegin, !isForward) >=
								acceptAngle.second) && 
								(angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward) <= acceptAngle.first || angleOfVectors(vecToBegin * -1, vecAtBegin, !isForward) >=
								acceptAngle.second)) {
								L.insert(isForward ? L.end() : L.begin(), newPoint);
								remainingPointsHelper.erase(isForward ? remainingPointsHelper.begin() : remainingPointsHelper.end() - 1);
								if (newPoint.first.x < xBounds.first)
									xBounds.first = newPoint.first.x;
								if (newPoint.first.x > xBounds.second)
									xBounds.second = newPoint.first.x;
								if (newPoint.first.y < yBounds.first)
									yBounds.first = newPoint.first.y;
								if (newPoint.first.y > yBounds.second)
									yBounds.second = newPoint.first.y;
							}
							else {
								if (L.size() > 2) {
									bool containsCorner = true;
									while (containsCorner && L.size() > 2) {
										containsCorner = false;
										for (size_t k = 0; k < remainingPointsHelper.size(); k++) {
											if (isPointInsidePolygon(L, remainingPointsHelper[k].first, xBounds, yBounds)) {
												containsCorner = true;
												break;
											}
										}
										if (containsCorner) {
											remainingPointsHelper.insert(isForward ? remainingPointsHelper.begin() : remainingPointsHelper.end(),
												isForward ? L[L.size() - 1] : L[0]);
											L.erase(isForward ? L.end() - 1 : L.begin());
										}
									}
									int absoluteClosestEdgeIndex = -1;
									size_t absoluteClosestPointIndex = 0;
									bool isNewPointFound = true;
									auto lastPointPos = L[L.size() - 1].first;
									if (L.size() > 2) {
										while (isNewPointFound) {
											double minIntersectionDistance = 1000;
											int closestEdgeIndex = -1;
											size_t closestPointIndex = 0;
											auto LTemp = L;
											if (absoluteClosestEdgeIndex != -1)
												LTemp.insert(LTemp.begin(), holeEdges[absoluteClosestEdgeIndex][absoluteClosestPointIndex]);
											isNewPointFound = false;
											for (size_t l = 0; l < holeEdges.size(); l++) {
												std::vector<bool> isPointsInside;
												auto edgePoints = holeEdges[l];
												size_t holeType = 0;
												for (size_t m = 0; m < edgePoints.size(); m++) {
													bool isHolePointInside = isPointInsidePolygon(LTemp, edgePoints[m].first, xBounds, yBounds);
													isPointsInside.push_back(isHolePointInside);
													if (isHolePointInside && holeType == 0)
														holeType = 1;
													if (m > 0 && isPointsInside[m - 1] != isPointsInside[m])
														holeType = 2;
												}
												if (holeType > 0) {
													for (size_t m = 0; m < edgePoints.size(); m++) {
														if (holeType == 2 && isPointsInside[m] != isPointsInside[(m + 1) % isPointsInside.size()]) {
															bool isOnEdge = false;
															auto intersection = intersectionOfLines(LTemp[0].first, lastPointPos, edgePoints[m].first,
																edgePoints[(m + 1) % isPointsInside.size()].first, isOnEdge);
															if (isOnEdge && (intersection - lastPointPos).length() < minIntersectionDistance) {
																minIntersectionDistance = (intersection - lastPointPos).length();
																closestEdgeIndex = l;
																if (((edgePoints[m].first - lastPointPos).length() <
																	(edgePoints[(m + 1) % isPointsInside.size()].first - lastPointPos).length() &&
																	isPointInsidePolygon(LTemp, edgePoints[m].first, xBounds, yBounds)) ||
																	((edgePoints[m].first - lastPointPos).length() >=
																		(edgePoints[(m + 1) % isPointsInside.size()].first - lastPointPos).length() &&
																		!isPointInsidePolygon(LTemp, edgePoints[(m + 1) % isPointsInside.size()].first,
																			xBounds, yBounds)))
																	closestPointIndex = m;
																else
																	closestPointIndex = ((m + 1) % isPointsInside.size());
															}
														}
														else if (holeType == 1 && absoluteClosestEdgeIndex == -1) {
															if ((edgePoints[m].first - lastPointPos).length() < minIntersectionDistance) {
																minIntersectionDistance = (edgePoints[m].first - lastPointPos).length();
																closestEdgeIndex = l;
																closestPointIndex = m;
															}
														}
													}
												}
											}
											if (closestEdgeIndex != -1) {
												absoluteClosestEdgeIndex = closestEdgeIndex;
												absoluteClosestPointIndex = closestPointIndex;
												isNewPointFound = true;
											}
										}
										if (absoluteClosestEdgeIndex >= 0) {
											if (!isForward) {
												remainingPointsHelper = remainingPointsHelperSave;
												L = LSave;
											}
											remainingPointsHelper.insert(remainingPointsHelper.begin(), L[L.size() - 1]);
											for (size_t l = 0; l < holeEdges[absoluteClosestEdgeIndex].size() + 1; l++) {
												remainingPointsHelper.insert(remainingPointsHelper.begin() + l,
													holeEdges[absoluteClosestEdgeIndex][(absoluteClosestPointIndex + l) % holeEdges[absoluteClosestEdgeIndex].size()]);
											}
											holeEdges.erase(holeEdges.begin() + absoluteClosestEdgeIndex);
											j = -1;
											break;
										}
									}
								}
								if (isForward) {
									LSave = L;
									remainingPointsHelperSave = remainingPointsHelper;
								}
								else {
									if (L.size() > 2) {
										remainingPointsHelper.insert(remainingPointsHelper.begin(), L[L.size() - 1]);
										remainingPointsHelper.insert(remainingPointsHelper.end(), L[0]);
									}
									else {
										L.clear();
										remainingPoints.push_back(remainingPoints[0]);
										remainingPoints.erase(remainingPoints.begin());
										remainingPointsHelper = remainingPoints;
									}
								}
								break;
							}
						}
					}
					if (remainingPointsHelper.size() == 0) //last convex polygon hole test
					{
						double minDistance = 1000;
						int closestEdgeIndex = -1;
						size_t closestPointIndex = 0;
						for (size_t l = 0; l < holeEdges.size(); l++) {
							auto edgePoints = holeEdges[l];
							for (size_t m = 0; m < edgePoints.size(); m++) {
								if (isPointInsidePolygon(L, edgePoints[m].first, xBounds, yBounds) && minDistance >
									(L[0].first - edgePoints[m].first).length()) {
									minDistance = (L[0].first - edgePoints[m].first).length();
									closestEdgeIndex = l;
									closestPointIndex = m;
								}
							}
						}
						if (closestEdgeIndex >= 0) {
							remainingPointsHelper.insert(remainingPointsHelper.begin(), L[0]);
							for (size_t l = 0; l < holeEdges[closestEdgeIndex].size() + 1; l++) {
								remainingPointsHelper.insert(remainingPointsHelper.begin() + l + 1,
									holeEdges[closestEdgeIndex][(closestPointIndex + l) % holeEdges[closestEdgeIndex].size()]);
							}
							for (size_t l = 0; l < L.size(); l++) {
								remainingPointsHelper.insert(remainingPointsHelper.end(), L[l]);
							}
							L.clear();
							remainingPoints = remainingPointsHelper;
						}
					}
					if (L.size() > 0) {
						std::vector<Point*> convexFace;
						for (size_t j = 0; j < L.size(); j++) {
							L[j].second->convexId.push_back(currentConvexId);
							L[j].second->convexIndex.push_back(j);
							convexFace.push_back(L[j].second);
						}
						planes[i]->convexFaces.push_back(convexFace);
						if (i == 51 && planes[i]->convexFaces.size() == 9) {
							int a = 1;
							//return;
						}
						remainingPoints = remainingPointsHelper;
						currentConvexId++;
					}
				}
				if (remainingPoints.size() == 3) 
				{
					std::vector<Point*> convexFace;
					for (size_t j = 0; j < remainingPoints.size(); j++) {
						remainingPoints[j].second->convexId.push_back(currentConvexId);
						remainingPoints[j].second->convexIndex.push_back(j);
						convexFace.push_back(remainingPoints[j].second);
					}
					planes[i]->convexFaces.push_back(convexFace);
					currentConvexId++;
				}
			}
		}
	}
}

void exportObjects(size_t pointCloudIndex)
{
	std::string name = "C:\\Users\\ungbo\\Desktop\\BME\\_Diplomamunka\\Diplomamunka\\Diplomamunka\\Assets\\Resources\\Generated_Models_" + 
		((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + "\\processed_obj_0.obj";
	std::ifstream f(name);
	size_t counter = 0;
	while (f.good())
	{
		f.close();
		remove(name.c_str());
		counter++;
		name = "C:\\Users\\ungbo\\Desktop\\BME\\_Diplomamunka\\Diplomamunka\\Diplomamunka\\Assets\\Resources\\Generated_Models_" +
			((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + "\\processed_obj_"
			+ std::to_string(counter) + ".obj";
		f = std::ifstream(name.c_str());
	}
	size_t objCounter = 0;
	for (size_t i = 0; i < planes.size(); i++) {
		if (objCounter == 37) 			{
			std::cout << "asd";
		}
		std::vector<Point*> corners;
		currentCornerIndex = 0;
		
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
					planes[i]->edges[j]->pointsWithDir[k].first->cornerId = currentCornerId;
					planes[i]->edges[j]->pointsWithDir[k].first->cornerIndex = currentCornerIndex;
					corners.push_back(planes[i]->edges[j]->pointsWithDir[k].first);
					currentCornerIndex++;
				}
			}
			currentCornerId++;
		}
		if (corners.size() == 0)
			continue;
		std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models_" +
			((pointCloudCount == 0 || pointCloudTestIndex != -1) ? "test" : std::to_string(pointCloudIndex)) + "/processed_obj_"
			+ std::to_string(objCounter) + ".obj");
		MyFile << "o Mesh" << std::endl;
		for (size_t k = 0; k < corners.size(); k++) {
			MyFile << "v " << -corners[k]->position.x << " " << corners[k]->position.y << " " << corners[k]->position.z << std::endl;
		}

		for (size_t j = 0; j < planes[i]->convexFaces.size(); j++) {
			MyFile << "f ";
			for (size_t k = 0; k < planes[i]->convexFaces[j].size(); k++) {
				MyFile << planes[i]->convexFaces[j][k]->cornerIndex + 1 << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (int k = planes[i]->convexFaces[j].size() - 1; k >= 0; k--) {
				MyFile << planes[i]->convexFaces[j][k]->cornerIndex + 1 << " ";
			}
			MyFile << std::endl;
		}
		
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			int indexShift = -1;
			for (size_t k = 0; k < planes[i]->edges[j]->pointsWithDir.size(); k++) {
				if (planes[i]->edges[j]->pointsWithDir[k].first->isCorner) {
					if (indexShift == -1)
						indexShift = planes[i]->edges[j]->pointsWithDir[k].first->cornerIndex;
					planes[i]->edges[j]->pointsWithDir[k].first->cornerIndex -= indexShift;
				}
			}
		}
		MyFile << std::endl;
		MyFile.close();
		objCounter++;
	}
}

void processData(size_t frameIndex) {
	groundSegmentation();
	egoCarSegmentation(frameIndex);
	findPlanes();
	findEdgePoints();
	findCorners();
	findPlaneConnections();
	connectPlanes();
	createCorners();
	filterEdgePoints();
	convexSegmentation();
}

int main()
{
	for (size_t i = (pointCloudTestIndex == -1 ? pointCloudBeginIndex : 0); i < (pointCloudTestIndex == -1 ? std::max<size_t>(1, pointCloudCount) : 1); i++) {
		readData(i);
		auto start = std::chrono::steady_clock::now();
		processData(std::max<int>(pointCloudTestIndex, i));
		auto end = std::chrono::steady_clock::now();
		std::cout << "It. " + std::to_string(i) + " Elapsed time in seconds : "
			<< (double)std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000
			<< " sec" << std::endl;
		exportObjects(i);
		writeData(i);
	}
	return 0;
}