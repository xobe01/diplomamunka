#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>
#include "kernel.h"

struct Point
{
	size_t horizontalIndex;
	size_t verticalIndex;
	Vec3<double> position;
	Plane* plane{ 0 };
	size_t cornerId{ 0 };
	size_t outlineId{ 0 };
	size_t cornerIndex{ 0 };
	Vec3<double> normal = { 0, 0, 0 };
	bool isMarked{ false };
	bool isMarked2{ false };
	std::vector<Point*> neighbourPlaneNeighbours;
	bool isCorner;
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, Plane* _plane) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), plane(_plane)
	{
		isCorner = false;
	};
	std::string printCoordinates()
	{
		return std::to_string(horizontalIndex) + " " + std::to_string(verticalIndex) + '\n';
	}
};

struct Plane {
	std::vector<Point*> points;
	std::pair<double, double> horizontalBounds;
	std::pair<double, double> verticalBounds;
	std::vector<std::vector<Point*>> edges;
	Vec3<double> planePointPos;
	Vec3<double> normal;
	std::pair<Vec3<double>, Vec3<double>> pointDirections;
	size_t id;
};

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Plane*> planes;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPlaneId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData()
{
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}		
	}
    MyReadFile.close();
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i])
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
			';' << (points[i]->plane ? points[i]->plane->id : 0) << ';' << points[i]->outlineId << ';' << points[i]->cornerId
			<< ';' << points[i]->cornerIndex << std::endl;
	}
}

void writeData()
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	MyFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

size_t spikeType(Point* p, int arriveDirection)
{
	//arriveDirection
	// 0 - from left
	// 1 - from up
	// 2 - from rigth
	// 3 - from down
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	size_t neighbourCount = 0;
	bool isNeighbour[4] = { false, false, false, false };
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };

	Point* diagNeighbourPoints[4] = { points[getOffset(x - 1, y - 1)], points[getOffset(x + 1, y - 1)], points[getOffset(x + 1, y + 1)], points[getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->plane == p->plane) {
			neighbourCount++;
			isNeighbour[j] = true;
		}
	}
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane) {
			diagIsNeighbour[j] = true;
		}
	}
	if ((neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3]) ||
		(isNeighbour[0] && isNeighbour[2] && !diagIsNeighbour[0]) || (isNeighbour[0] && isNeighbour[3] && !diagIsNeighbour[1]) ||
		(isNeighbour[1] && isNeighbour[2] && !diagIsNeighbour[3]) || (isNeighbour[1] && isNeighbour[3] && !diagIsNeighbour[2]))) || 
		(neighbourCount > 1 && ((arriveDirection == 0 && !isNeighbour[1]) || (arriveDirection == 1 && !isNeighbour[2]) || (arriveDirection == 2 && !isNeighbour[0])
			|| (arriveDirection == 3 && !isNeighbour[3]))))
		return 0;
	if (neighbourCount == 1)
		return 1;
	if (neighbourCount > 1)
		return 2;
	return 3;
}

bool checkIfBridge(Point* p)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && neighbourPoints[j]->plane == p->plane && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) &&
			spikeType(neighbourPoints[j], -1) > 1)
			neighbourCount++;
	}
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || points[getOffset(x - 1, y - 1)]->plane != p->plane)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || points[getOffset(x + 1, y + 1)]->plane != p->plane))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || points[getOffset(x + 1, y - 1)]->plane != p->plane)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || points[getOffset(x - 1, y + 1)]->plane != p->plane))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

void checkForGaps(/*out*/ std::vector<Point*>& chosenPoints)
{  
	for (auto p : chosenPoints) p->isMarked = true;
	while (isThereBridge(chosenPoints)) {}
	std::vector<Point*> nonProcessedPoints(chosenPoints);
	std::vector<Point*> nextStepPoints;
	std::vector<Point*> bestPartitionPoints;
	while (bestPartitionPoints.size() < nonProcessedPoints.size()) {
		std::vector<Point*> currentPartitionPoints;
		currentPartitionPoints.push_back(nonProcessedPoints[0]);
		nextStepPoints.push_back(nonProcessedPoints[0]);
		nonProcessedPoints[0]->isMarked = false;
		while (nextStepPoints.size() > 0) {
			std::vector<Point*> tempNextStepPoints;
			for (size_t i = 0; i < nextStepPoints.size(); i++) {
				size_t x = nextStepPoints[i]->horizontalIndex;
				size_t y = nextStepPoints[i]->verticalIndex;
				Point* neighbourPoint = points[getOffset(x, y - 1)];
				if (y > 0 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x, y + 1)];
				if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x - 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x + 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
			}
			nextStepPoints = tempNextStepPoints;
		}
		if (currentPartitionPoints.size() > bestPartitionPoints.size()) bestPartitionPoints = currentPartitionPoints;
		std::vector<Point*> tempNonProcessedPoints;
		for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
			if (nonProcessedPoints[i]->isMarked) tempNonProcessedPoints.push_back(nonProcessedPoints[i]);
		}
		nonProcessedPoints = tempNonProcessedPoints;
	}
	for (size_t i = 0; i < chosenPoints.size(); i++) {
		chosenPoints[i]->isMarked = false;
	}
	chosenPoints = bestPartitionPoints;
}

#include <chrono>

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(p1->position - center->position, p2->position - center-> position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x + 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x - 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

void choosePoints(const Vec3<Point*> planePoints, double acceptTreshold, Plane* basePlane, /*out*/ Plane* plane)
{
	if (basePlane) {
		plane->pointDirections = basePlane->pointDirections;
		plane->normal = basePlane->normal;
	}
	else 
	{
		auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.y->position -
			planePoints.x->position, planePoints.z->position - planePoints.x->position));
		Vec3<double> horizontalDirection = { 0,0,0 };
		Vec3<double> verticalDirection = { 0,0,0 };
		Point* neighbours[2] = { planePoints.y, planePoints.z };
		for each (auto neighbour in neighbours) {
			switch (areNeighbours(planePoints.x, neighbour)) {
			case 1:
			horizontalDirection = neighbour->position - planePoints.x->position;
			break;
			case 2:
			horizontalDirection = planePoints.x->position - neighbour->position;
			break;
			case 3:
			verticalDirection = neighbour->position - planePoints.x->position;
			break;
			case 4:
			verticalDirection = planePoints.x->position - neighbour->position;
			break;
			default:
			break;
			}
		}
		plane->pointDirections = { horizontalDirection, verticalDirection };
		plane->normal = normal;
	}
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(plane->normal, neighbourPoints[j]->position - plane->planePointPos));
					if (dist <= acceptTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void calculateBounds(Plane& plane)
{
	plane.horizontalBounds = std::make_pair(plane.points[0]->horizontalIndex, plane.points[0]->horizontalIndex);
	plane.verticalBounds = std::make_pair(plane.points[0]->verticalIndex, plane.points[0]->verticalIndex);
	for (size_t i = 1; i < plane.points.size(); i++) 
	{
		if (plane.points[i]->horizontalIndex > plane.horizontalBounds.second)
			plane.horizontalBounds.second = plane.points[i]->horizontalIndex;
		else if (plane.points[i]->horizontalIndex < plane.horizontalBounds.first)
			plane.horizontalBounds.first = plane.points[i]->horizontalIndex;
		if (plane.points[i]->verticalIndex > plane.verticalBounds.second)
			plane.verticalBounds.second = plane.points[i]->verticalIndex;
		else if (plane.points[i]->verticalIndex < plane.verticalBounds.first)
			plane.verticalBounds.first = plane.points[i]->verticalIndex;
	}
}

void findPlanes()
{
	auto start = std::chrono::steady_clock::now();
	size_t minPointCount = 10;
	size_t counter = 1;
	double planeDistanceTreshold = 0.01;
	double normalTreshold = 0.01;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = true;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = true;
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			nextStepPoints.push_back(points[j]);
			calculateNormal(points[j]);
			while (nextStepPoints.size() > 0) {
				Plane* plane = new Plane();
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x + 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x - 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k != 2 || y < verticalCount - 1) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								if (neighbourPoints[k]->normal.length() == 0) {
									calculateNormal(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							auto temp = (normals[k] - normal).length();
							auto temp2 = (normals[(k + 1) % 4] - normal).length();
							if (normals[k].length()>0 && normals[(k + 1) % 4].length() > 0 && 
								(normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {								
								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, planeDistanceTreshold, nullptr,
									plane);
								break;
							}
						}
					}
				}
				if (plane->points.size() > 0)
				{
					planes.push_back(plane);
					currentPlaneId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		int originalSize = planes[i]->points.size();
		while (isThereBridge(planes[i]->points)) {}		
		if (originalSize != planes[i]->points.size() && planes[i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[i]->points[0], nullptr, nullptr }, planeDistanceTreshold, planes[i],
					plane);
				if (plane->points.size() < planes[i]->points.size()) {
					for (size_t j = 0; j < planes[i]->points.size(); j++) {
						if (planes[i]->points[j]->plane != planes[i]) {
							planes[i]->points.erase(planes[i]->points.begin() + j);
							j--;
						}
					}
					planes.push_back(plane);
					currentPlaneId++;
				}
				else 
				{
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[i];
					}
					delete plane;
					break;
				}
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		calculateBounds(planes[i]);	
	}/**/
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec" << std::endl;
}

void findNextPoint(Point* startPoint, size_t direction, bool isPreviousSpike, bool wasThereNonSpike, /*out*/ std::vector<std::pair<Point*, size_t>>& currentEdge,
	size_t dbgPlaneIndex, std::vector<Plane*> dbgPlanes)
{
	Point* currentPoint = nullptr;
	std::pair<Point*, size_t> previousSavedPoint = {nullptr, 0};
	bool isFirstPoint = true;
	while (currentPoint != startPoint) {
		if (!currentPoint)
			currentPoint = startPoint;
		Point* neighbourPoint = nullptr;
		size_t x = currentPoint->horizontalIndex;
		size_t y = currentPoint->verticalIndex;
		if (spikeType(currentPoint, isFirstPoint ? -1 : ((direction + 1) % 4)) == 0)
			isPreviousSpike = true;
		else {
			if (currentPoint->isMarked) {
				currentEdge.push_back({ currentPoint, direction });
				for (size_t i = 0; i < 4; i++) {
					switch (direction) {
					case 0: //to right
					neighbourPoint = points[getOffset(x + 1, y)];
					break;
					case 1: //to down
					neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
					break;
					case 2: //to left
					neighbourPoint = points[getOffset(x - 1, y)];
					break;
					case 3: //to up
					neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
					break;
					default:
					break;
					}
					if (neighbourPoint && neighbourPoint->plane != startPoint->plane && neighbourPoint->plane != nullptr) {
						currentPoint->neighbourPlaneNeighbours.push_back(neighbourPoint);
					}
					direction += direction == 3 ? -3 : 1;
				}
			}
		}
		currentPoint->isMarked = false;
		currentPoint->isMarked2 = false;
		isFirstPoint = false;
		if (currentPoint->horizontalIndex == 71 && currentPoint->verticalIndex == 5)
			std::cout << "asd";
		for (size_t i = 0; i < 4; i++) {
			switch (direction) {
			case 0: //to right
			neighbourPoint = points[getOffset(x + 1, y)];
			break;
			case 1: //to down
			neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
			break;
			case 2: //to left
			neighbourPoint = points[getOffset(x - 1, y)];
			break;
			case 3: //to up
			neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
			break;
			default:
			break;
			}
			if (neighbourPoint == startPoint) {
				currentPoint = startPoint;
				break;
			}
			if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr && neighbourPoint->isMarked 
				&& (isPreviousSpike || spikeType(neighbourPoint, -1) <= 1 || spikeType(currentPoint, direction) > 0))
			{
				if (!wasThereNonSpike && spikeType(currentPoint, -1) > 1) {
					startPoint = currentPoint;
					isPreviousSpike = false;
					wasThereNonSpike = true;
				}
				if (!isPreviousSpike)
					previousSavedPoint = { currentPoint, (direction + (4 - i)) % 4 };
				if (spikeType(neighbourPoint, -1) == 1 || (wasThereNonSpike && currentEdge.size() > 1 && isPreviousSpike && 
					spikeType(neighbourPoint, -1) == 2 && neighbourPoint->isMarked)) {
					auto savedPoint = spikeType(neighbourPoint, -1) == 1 ? neighbourPoint : currentPoint;
					currentEdge.push_back({ savedPoint, 0 });
					savedPoint->isMarked = false;
					savedPoint->isMarked2 = false;
					currentPoint = previousSavedPoint.first;
					direction = previousSavedPoint.second;
					isPreviousSpike = false;
					break;
				}
				else {
					currentPoint = neighbourPoint;
				}
				direction = (direction + 3) % 4;
				break;
			}
			direction += direction == 3 ? -3 : 1;
			if (i == 3) {
				if (!wasThereNonSpike) {
					currentEdge.clear();
					break;
				}
				currentPoint = currentEdge[currentEdge.size() - 2].first;
				direction = currentEdge[currentEdge.size() - 2].second;
				std::cout << "INVALID EDGE SEARCH" << std::endl;;
			}
		}
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

bool isStraightPoint(size_t pointIndex, std::vector<Point*>& edge, bool& isPreviousConcave)
{
	bool previousConcaveStore = isPreviousConcave;
	size_t neighbourCount = 0;
	Point* point = edge[pointIndex];
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Plane* plane = point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i < 3) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane && (neighbourPoints[i]->outlineId == 0 || neighbourPoints[i]->outlineId == point->outlineId)) {
			neighbourCount++;
			isNeighbour[i] = true;
		}
	}
	isPreviousConcave = neighbourCount == 4;
	if (neighbourCount == 3 || (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3])))) {
		return true;
	}
	if (neighbourCount == 4 && (pointIndex == 0 ? edge[edge.size() - 1] : edge[pointIndex - 1])->isCorner)
		return true;
	if (previousConcaveStore) {
		edge[pointIndex - 1] -> isCorner = false;
	}
	return false;
}

void findEdgePoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) if(isEdgePoint(planes[i]->points[j])) edgePointsInPlane.push_back(planes[i]->points[j]);
		while (edgePointsInPlane.size() > 0) 
		{
			std::vector<Point*> tempEdgePointsInPlane;
			Point* startPoint = edgePointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
				if ((edgePointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - edgePointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| edgePointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = edgePointsInPlane[j]->horizontalIndex;
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
				else if (edgePointsInPlane[j]->horizontalIndex == minHorizontalCoord && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
			}
			size_t direction = 0;
			if (spikeType(startPoint, -1) == 0) {
				size_t x = startPoint->horizontalIndex;
				size_t y = startPoint->verticalIndex;
				startPoint->plane = nullptr;
				startPoint->isMarked = false;
				for (size_t j = 0; j < planes[i]->points.size(); j++) {
					if (planes[i]->points[j] == startPoint) {
						planes[i]->points.erase(planes[i]->points.begin() + j);
						break;
					}
				}
				startPoint = points[getOffset(x + 1, y)];
				direction = 3;
			}
			else if(startPoint->verticalIndex > 0 && points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)] &&
				points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)]->isMarked2)
				direction = 1;
			std::vector<std::pair<Point*, size_t>> currentEdge;
			findNextPoint(startPoint, direction, false, false, currentEdge, i, planes);
			for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
				if (edgePointsInPlane[j]->isMarked) tempEdgePointsInPlane.push_back(edgePointsInPlane[j]);
			}
			edgePointsInPlane = tempEdgePointsInPlane;
			if (currentEdge.size() > 0) {
				if (direction == 1) {
					currentEdge.insert(currentEdge.begin(), currentEdge[currentEdge.size() - 1]);
					currentEdge.pop_back();
				}
				std::vector<Point*> savedEdge;
				for (size_t k = 0; k < currentEdge.size(); k++) {
					currentEdge[k].first->outlineId = currentOutlineId;
					savedEdge.push_back(currentEdge[k].first);
				}
				currentOutlineId++;
				planes[i]->edges.push_back(savedEdge);
			}
		}
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = false;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = false;
	}
}

bool arePlanesNeighbours(Plane p1, Plane p2, std::pair<int, int>& horizontalCommonBounds, std::pair<int, int>& verticalCommonBounds)
{
	if (p1.horizontalBounds.first > p1.horizontalBounds.second)
		p1.horizontalBounds.second += horizontalCount;
	double p1HorizontalSize = p1.horizontalBounds.second - p1.horizontalBounds.first;
	double p1VerticalSize = p1.verticalBounds.second - p1.verticalBounds.first;
	std::pair<int, int> p1Center = {(int)(p1.horizontalBounds.first + p1HorizontalSize / 2) % horizontalCount, 
		p1.verticalBounds.first + p1VerticalSize / 2 };

	if (p2.horizontalBounds.first > p2.horizontalBounds.second)
		p2.horizontalBounds.second += horizontalCount;
	double p2HorizontalSize = p2.horizontalBounds.second - p2.horizontalBounds.first;
	double p2VerticalSize = p2.verticalBounds.second - p2.verticalBounds.first;
	std::pair<int, int> p2Center = { (int)(p2.horizontalBounds.first + p2HorizontalSize / 2) % horizontalCount,
		p2.verticalBounds.first + p2VerticalSize / 2 };

	if (abs(p1Center.first - p2Center.first) > horizontalCount / 2) 
	{
		if (p1Center.first > p2Center.first)
			p2Center.first += horizontalCount;
		if (p1Center.first < p2Center.first)
			p1Center.first += horizontalCount;
	}
	if (abs(p1Center.first - p2Center.first) <= (p1HorizontalSize + p2HorizontalSize) / 2 + 1 &&
		abs(p1Center.second - p2Center.second) <= (p1VerticalSize + p2VerticalSize) / 2 + 1) 
	{
		horizontalCommonBounds = {abs(p1.horizontalBounds.first - p2.horizontalBounds.first) > horizontalCount / 2
			? std::min(p1.horizontalBounds.first, p2.horizontalBounds.first) : std::max(p1.horizontalBounds.first, p2.horizontalBounds.first),
			abs(p1.horizontalBounds.second - p2.horizontalBounds.second) > horizontalCount / 2
			? std::max(p1.horizontalBounds.second, p2.horizontalBounds.second) : std::min(p1.horizontalBounds.second, p2.horizontalBounds.second)};
		if (horizontalCommonBounds.second < horizontalCommonBounds.first)
			horizontalCommonBounds = { horizontalCommonBounds.second, horizontalCommonBounds.first };

		verticalCommonBounds = { std::max(p1.verticalBounds.first, p2.verticalBounds.first),
			std::min(p1.verticalBounds.second, p2.verticalBounds.second) };
		if (verticalCommonBounds.second < verticalCommonBounds.first)
			verticalCommonBounds = { verticalCommonBounds.second, verticalCommonBounds.first };
		return true;
	}
	return false;
}

const double newPointAcceptTreshold = 0.95;
const double inf = 1000000;

Point* createNewPoint(Vec3<double> newPointPos, Point* point, Point* neighbour, size_t addedCount)
{
	Point* newPoint = new Point(newPointPos, neighbour->horizontalIndex, verticalCount, point->plane);
	addedPoints.push_back(newPoint);
	newPoint->isCorner = true;
	newPoint->outlineId = point->outlineId;
	for (size_t j = 0; j < point->plane->edges.size(); j++) {
		if (point->plane->edges[j][0]->outlineId == point->outlineId) {
			for (size_t k = 0; k < point->plane->edges[j].size(); k++) {
				if (point->plane->edges[j][k] == point) {
					point->plane->edges[j].insert(point->plane->edges[j].begin() + k + 1 + addedCount, newPoint);
					break;
				}
			}
			break;
		}
	}
	newPoint->neighbourPlaneNeighbours.push_back(neighbour);
	return newPoint;
}

Point* addNewPoint(Point* point, Point*& neighbour, Plane* plane, size_t addedCount)
{
	if (neighbour->verticalIndex == verticalCount) { //created by other plane
		for (size_t j = 0; j < point->plane->edges.size(); j++) {
			if (point->plane->edges[j][0]->outlineId == point->outlineId) {
				for (size_t k = 0; k < point->plane->edges[j].size(); k++) {
					if (point->plane->edges[j][k] == point) {
						for (size_t l = 0; l < 4; l++) 
						{
							if (point->plane->edges[j][k + l + 1] == neighbour) {
								point->plane->edges[j].insert(point->plane->edges[j].begin() + k + 1 + addedCount, neighbour);
								point->plane->edges[j].erase(point->plane->edges[j].begin() + k + l + 2);
								break;
							}
						}
						break;
					}
				}
				break;
			}
		}
		return neighbour;
	}
	Vec3<double> dir = { 0,0,0 };
	switch (areNeighbours(point, neighbour)) {
	case 1:
	dir = point->plane->pointDirections.first;
	break;
	case 2:
	dir = point->plane->pointDirections.first * -1;
	break;
	case 3:
	dir = point->plane->pointDirections.second;
	break;
	case 4:
	dir = point->plane->pointDirections.second * -1;
	break;
	default:
	break;
	}
	Vec3<double> newPointPos = point->position - dir * Vec3<double>::dot_product(point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf || isnan(newPointPos.x) || isnan(newPointPos.y) || isnan(newPointPos.z))
		return nullptr;
	Vec3<double> dirToNew = newPointPos - point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold)
		return nullptr;
	Vec3<double> neighbourDir = { 0,0,0 };
	switch (areNeighbours(neighbour, point)) {
	case 1:
	neighbourDir = plane->pointDirections.first;
	break;
	case 2:
	neighbourDir = plane->pointDirections.first * -1;
	break;
	case 3:
	neighbourDir = plane->pointDirections.second;
	break;
	case 4:
	neighbourDir = plane->pointDirections.second * -1;
	break;
	default:
	break;
	}	
	Vec3<double> neighbourNewPointPos = neighbour->position - neighbourDir * Vec3<double>::dot_product(neighbour->position
		- point->plane->planePointPos, point->plane->normal) / Vec3<double>::dot_product(neighbourDir, point->plane->normal);
	if (abs(neighbourNewPointPos.x) > inf || abs(neighbourNewPointPos.y) > inf || abs(neighbourNewPointPos.z) > inf || isnan(neighbourNewPointPos.x) ||
		isnan(neighbourNewPointPos.y) || isnan(neighbourNewPointPos.z))
		return nullptr;
	Vec3<double> dirToNewNeighbour = neighbourNewPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(neighbourDir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold) 
		return nullptr;
	auto newPos = (newPointPos + neighbourNewPointPos) / 2;
	bool isNeighbourEdge = false;
	Point* newPoint;
	Point* newNeighbourPoint;
	newPoint = createNewPoint(newPos, point, neighbour, addedCount);
	for (size_t i = 0; i < neighbour->neighbourPlaneNeighbours.size(); i++) 
	{
		if (neighbour->neighbourPlaneNeighbours[i] == point) 		
		{
			isNeighbourEdge = true;
			newNeighbourPoint = createNewPoint(newPos, neighbour, point, 0);
			neighbour->neighbourPlaneNeighbours[i] = newNeighbourPoint;
			break;
		}
	}
	if (!isNeighbourEdge) {
		newNeighbourPoint = createNewPoint(newPos, neighbour, point, 0);
		neighbour->neighbourPlaneNeighbours.push_back(newNeighbourPoint);
	}
	newPoint->neighbourPlaneNeighbours.push_back(newNeighbourPoint);
	newNeighbourPoint->neighbourPlaneNeighbours.push_back(newPoint);
	return newPoint;
}

void connectPlanes()
{
	std::vector<std::vector<bool>> wasFirstGeneratedVec;
	std::vector<Point*> createdPoints;
	for (size_t i = 0; i < planes.size(); i++) {
		wasFirstGeneratedVec.push_back({});
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			wasFirstGeneratedVec[i].push_back({false});
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {
				if (planes[i]->edges[j][k]->verticalIndex == verticalCount) 
				{
					continue;
				}
				size_t addedCount = 0;
				if(planes[i]->edges[j][k]->isCorner && planes[i]->edges[j][k]->neighbourPlaneNeighbours.size() > 0)
				{
					for (auto neighbour : planes[i]->edges[j][k]->neighbourPlaneNeighbours) {
						auto newPoint = addNewPoint(planes[i]->edges[j][k], neighbour, neighbour->plane, addedCount);
						if (newPoint) {
							if (k == 0 && ((planes[i]->edges[j][k]->horizontalIndex - neighbour->horizontalIndex + horizontalCount) % horizontalCount
								== 1)) wasFirstGeneratedVec[i][wasFirstGeneratedVec[i].size() - 1] = true;
							planes[i]->edges[j][k]->isCorner = false;
							createdPoints.push_back(newPoint);
							addedCount++;
						}
						else {
							planes[i]->edges[j].insert(planes[i]->edges[j].begin() + k + 1 + addedCount, nullptr);
							addedCount++;
						}
					}
					k += addedCount;
				}
			}			
		}
	}
	std::vector<std::vector<std::vector<std::pair<size_t, size_t>>>> cornersToDelete;
	for (size_t i = 0; i < planes.size(); i++) {
		cornersToDelete.push_back({});
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			cornersToDelete[i].push_back({});
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {
				if (planes[i]->edges[j][k]) {
					if (planes[i]->edges[j][k]->verticalIndex == verticalCount)
						cornersToDelete[i][j].push_back({ k,  planes[i]->edges[j][k]->neighbourPlaneNeighbours[0]->plane->id });
				}
				else 
				{
					cornersToDelete[i][j].push_back({ 0, 0 });
					planes[i]->edges[j].erase(planes[i]->edges[j].begin() + k);
					k--;
				}
			}
			int newPointStartIndex = wasFirstGeneratedVec[i][j] ? 1 : 0;
			int newPointEndIndex = newPointStartIndex;
			while (newPointEndIndex < (int)cornersToDelete[i][j].size() - 1) {
				if (cornersToDelete[i][j][newPointStartIndex].second > 0) {
					while (newPointEndIndex + 1 < cornersToDelete[i][j].size()
						&& cornersToDelete[i][j][newPointStartIndex].second == cornersToDelete[i][j][newPointEndIndex + 1].second) {
						newPointEndIndex++;
					}
					for (size_t k = cornersToDelete[i][j][newPointStartIndex].first + 1; k < cornersToDelete[i][j][newPointEndIndex].first; k++) {
						planes[i]->edges[j][k]->isCorner = false;
					}
				}
				newPointStartIndex = newPointEndIndex + 1;
				newPointEndIndex = newPointStartIndex;
			}
		}
	}
	for (size_t i = 0; i < createdPoints.size(); i++) {
		if (createdPoints[i]->isCorner)
			createdPoints[i]->neighbourPlaneNeighbours[1]->isCorner = true;
	}
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k]->edges.size(); i++) {
			bool isPreviousConcave = false;
			for (size_t j = 0; j < planes[k]->edges[i].size(); j++) {
				if (!isStraightPoint(j, planes[k]->edges[i], isPreviousConcave)) {
					planes[k]->edges[i][j]->isCorner = true;
				}
			}
		}
	}
}

double distancePointFromLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line)
{
	Vec3<double> directionVector = line.first - line.second;
	return Vec3<double>::crossProduct(p - line.first, directionVector).length() / directionVector.length();
}

bool isOnLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line, double treshold)
{
	return distancePointFromLine(p, line) < treshold;
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i] = nullptr;
	}
}

void exportObjects()
{
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models/processed_obj_" 
				+ std::to_string(currentCornerId - 1) + ".obj");
			MyFile << "o Mesh" << std::endl;
			std::vector<Point*> corners;
			currentCornerIndex = 0;
			for (size_t k = 0; k < planes[i]->edges[j].size(); k++) {
				if (planes[i]->edges[j][k]->isCorner) {
					planes[i]->edges[j][k]->cornerId = currentCornerId;
					planes[i]->edges[j][k]->cornerIndex = currentCornerIndex;
					corners.push_back(planes[i]->edges[j][k]);
					currentCornerIndex++;
				}
			}
			for (size_t k = 0; k < corners.size(); k++) {
				MyFile << "v " << -corners[k]->position.x << " " << corners[k]->position.y << " " << corners[k]->position.z << std::endl;
			}
			MyFile << "f ";
			for (size_t j = 1; j < corners.size() + 1; j++) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (size_t j = corners.size(); j > 0; j--) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile.close();
			currentCornerId++;
		}
	}
}

void processData() {
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	//findEdgePoints();
	//findCorners();
	//connectPlanes2();
	//exportObjects();
	writeData();
}

int main()
{
    readData();
	processData();
    return 0;
}