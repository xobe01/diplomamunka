#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>
#include "kernel.h"
#include <chrono>

struct Point
{
	size_t horizontalIndex;
	size_t verticalIndex;
	Vec3<double> position;
	Plane* plane{ 0 };
	size_t cornerId{ 0 };
	size_t outlineId{ 0 };
	size_t cornerIndex{ 0 };
	Vec3<double> normal = { 0, 0, 0 };
	bool isMarked{ false };
	bool isMarked2{ false };
	std::vector<Point*> neighbourPlaneNeighbours;
	std::vector<Point*> createdNeighbourPoints;
	bool isCorner;
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, Plane* _plane) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), plane(_plane)
	{
		isCorner = false;
		neighbourPlaneNeighbours = { nullptr, nullptr, nullptr, nullptr };
		createdNeighbourPoints = { nullptr, nullptr, nullptr, nullptr };
	};
	std::string printCoordinates()
	{
		return std::to_string(horizontalIndex) + " " + std::to_string(verticalIndex) + '\n';
	}
};

struct Plane {
	std::vector<Point*> points;
	std::vector<std::pair<Point*, std::vector<std::pair<Point*, int>>>> edges;
	Vec3<double> planePointPos;
	Vec3<double> normal;
	std::pair<Vec3<double>, Vec3<double>> pointDirections;
	size_t id;
};

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Plane*> planes;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPlaneId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;
const double planeDistanceTreshold = 0.05;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void readData()
{
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, nullptr);
		}		
	}
    MyReadFile.close();
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i])
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
			';' << (points[i]->plane ? points[i]->plane->id : 0) << ';' << points[i]->outlineId << ';' << points[i]->cornerId
			<< ';' << points[i]->cornerIndex << std::endl;
	}
}

void writePlanes(std::ofstream& MyFile)
{
	MyFile << planes.size() << std::endl;
	for (size_t i = 0; i < planes.size(); i++) {
		MyFile << planes[i]->planePointPos.x << ';' << planes[i]->planePointPos.y << ';' << planes[i]->planePointPos.z << ';' 
			<< planes[i]->normal.x << ';' << planes[i]->normal.y << ';' << planes[i]->normal.z << std::endl;
	}
}

void writeData()
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	writePlanes(MyFile);
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	MyFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

int spikeType(Point* p, int arriveDirection, bool onlyMarkedNeighbours)
{
	//arriveDirection
	// 0 - from left
	// 1 - from up
	// 2 - from rigth
	// 3 - from down
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	size_t neighbourCount = 0;
	bool isNeighbour[4] = { false, false, false, false };
	bool diagIsNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };

	Point* diagNeighbourPoints[4] = { points[getOffset(x - 1, y - 1)], points[getOffset(x + 1, y - 1)], points[getOffset(x + 1, y + 1)], points[getOffset(x - 1, y + 1)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->plane == p->plane && 
			(!onlyMarkedNeighbours || neighbourPoints[j]->isMarked2)) {
			neighbourCount++;
			isNeighbour[j] = true;
		}
	}
	for (size_t j = 0; j < 4; j++) {
		if (diagNeighbourPoints[j] && (j > 1 || y > 0) && (j < 2 || y < verticalCount - 1) && diagNeighbourPoints[j]->plane == p->plane &&
			(!onlyMarkedNeighbours || diagNeighbourPoints[j]->isMarked2)) {
			diagIsNeighbour[j] = true;
		}
	}
	if (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[2] && !diagIsNeighbour[0]) || (isNeighbour[0] && isNeighbour[3] && !diagIsNeighbour[1]) 
		|| (isNeighbour[1] && isNeighbour[2] && !diagIsNeighbour[3]) || (isNeighbour[1] && isNeighbour[3] && !diagIsNeighbour[2])))
		return -1;
	if ((neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3]))) || (neighbourCount > 1 && 
		((arriveDirection == 0 && !isNeighbour[1]) || (arriveDirection == 1 && !isNeighbour[2]) || (arriveDirection == 2 && !isNeighbour[0])
			|| (arriveDirection == 3 && !isNeighbour[3]))))
		return 0;
	if (neighbourCount == 1)
		return 1;
	if (neighbourCount > 1)
		return 2;
	return 3;
}

bool checkIfBridge(Point* p)
{
	if (spikeType(p, -1, false) == -1)
		return true;
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;

	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
				points[getOffset(x + 1, y)] };
	for (size_t j = 0; j < 4; j++) {
		if (neighbourPoints[j] && neighbourPoints[j]->plane == p->plane && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) &&
			spikeType(neighbourPoints[j], -1, false) > 1)
			neighbourCount++;
	}
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || points[getOffset(x - 1, y - 1)]->plane != p->plane)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || points[getOffset(x + 1, y + 1)]->plane != p->plane))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || points[getOffset(x + 1, y - 1)]->plane != p->plane)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || points[getOffset(x - 1, y + 1)]->plane != p->plane))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& planePoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : planePoints)
		if (p->plane != nullptr && checkIfBridge(p)) {
			theresBridge = true;
			p->plane = nullptr;
		}
		else
			newPoints.push_back(p);
	planePoints = newPoints;
	return theresBridge;
}

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(p1->position - center->position, p2->position - center-> position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x + 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x - 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

void choosePoints(const Vec3<Point*> planePoints, Plane* basePlane, /*out*/ Plane* plane)
{
	if (basePlane) {
		plane->pointDirections = basePlane->pointDirections;
		plane->normal = basePlane->normal;
	}
	else 
	{
		auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.y->position -
			planePoints.x->position, planePoints.z->position - planePoints.x->position));
		Vec3<double> horizontalDirection = { 0,0,0 };
		Vec3<double> verticalDirection = { 0,0,0 };
		Point* neighbours[2] = { planePoints.y, planePoints.z };
		for each (auto neighbour in neighbours) {
			switch (areNeighbours(planePoints.x, neighbour)) {
			case 1:
			horizontalDirection = neighbour->position - planePoints.x->position;
			break;
			case 2:
			verticalDirection = neighbour->position - planePoints.x->position;
			break;
			case 3:
			horizontalDirection = planePoints.x->position - neighbour->position;
			break;
			case 4:
			verticalDirection = planePoints.x->position - neighbour->position;
			break;
			default:
			break;
			}
		}
		plane->pointDirections = { horizontalDirection, verticalDirection };
		plane->normal = normal;
	}
	plane->planePointPos = planePoints.x->position;
	plane->id = currentPlaneId;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->plane = plane;
	plane->points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(plane->normal, neighbourPoints[j]->position - plane->planePointPos));
					if (dist <= planeDistanceTreshold) {
						plane->points.push_back(neighbourPoints[j]);
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->plane = plane;
						tempNextStepPoints.push_back(neighbourPoints[j]);
						if ((neighbourPoints[j]->normal - plane->normal).length() < 0.05) {
							plane->normal = ((plane->normal * (plane->points.size() - 1) + neighbourPoints[j]->normal)) / plane->points.size();
						}
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void findPlanes()
{
	size_t minPointCount = 10;
	size_t counter = 1;
	double normalTreshold = 0.01;
	for (size_t i = 0; i < points.size(); i++)
		if (points[i]) {
			points[i]->isMarked = true;
			points[i]->isMarked2 = true;
			calculateNormal(points[i]);
		}
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			nextStepPoints.push_back(points[j]);
			calculateNormal(points[j]);
			while (nextStepPoints.size() > 0) {
				Plane* plane = new Plane();
				bool planeMerged;
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x + 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x - 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k != 2 || y < verticalCount - 1) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								if (neighbourPoints[k]->normal.length() == 0) {
									calculateNormal(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if (normals[k].length() > 0 && normals[(k + 1) % 4].length() > 0 &&
								(normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {
								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, nullptr,
									plane);
								break;
							}
						}
					}
				}
				if (plane->points.size() > 0)
				{
					planes.push_back(plane);
					currentPlaneId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		int originalSize = planes[i]->points.size();
		while (isThereBridge(planes[i]->points)) {}		
		if (originalSize != planes[i]->points.size() && planes[i]->points.size() > 0) //cutting plane		
		{
			for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
			while (true) {
				Plane* plane = new Plane();
				choosePoints({ planes[i]->points[0], nullptr, nullptr }, planes[i], plane);
				if (plane->points.size() < planes[i]->points.size()) {
					for (size_t j = 0; j < planes[i]->points.size(); j++) {
						if (planes[i]->points[j]->plane != planes[i]) {
							planes[i]->points.erase(planes[i]->points.begin() + j);
							j--;
						}
					}
					planes.push_back(plane);
					currentPlaneId++;
				}
				else 
				{
					for (size_t j = 0; j < plane->points.size(); j++) {
						plane->points[j]->plane = planes[i];
					}
					delete plane;
					break;
				}
			}
		}
	}
	/*for (size_t i = 0; i < planes.size(); i++) {
		calculateBounds(planes[i]);	
	}/**/	
}

void findNextPoint(Point*& startPoint, size_t direction, bool isPreviousSpike, bool wasThereNonSpike, /*out*/ 
	std::vector<std::pair<Point*, int>>& currentEdge, size_t dbgPlaneIndex, std::vector<Plane*> dbgPlanes)
{
	Point* currentPoint = nullptr;
	std::pair<Point*, size_t> previousSavedPoint = {nullptr, 0};
	bool isFirstPoint = true;
	bool comeFromDeadEnd = false;
	while (currentPoint != startPoint || comeFromDeadEnd)
	{
		if (!currentPoint)
			currentPoint = startPoint;
		Point* neighbourPoint = nullptr;
		size_t x = currentPoint->horizontalIndex;
		size_t y = currentPoint->verticalIndex;
		if (spikeType(currentPoint, isFirstPoint ? -1 : ((direction + 1) % 4), false) == 0)
			isPreviousSpike = true;
		else {
			if (currentPoint->isMarked) {
				currentEdge.push_back({ currentPoint, direction });				
			}
		}
		currentPoint->isMarked = false;
		currentPoint->isMarked2 = false;
		isFirstPoint = false;
		for (size_t i = 0; i < 4; i++) {
			switch (direction) {
			case 0: //to right
			neighbourPoint = points[getOffset(x + 1, y)];
			break;
			case 1: //to down
			neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
			break;
			case 2: //to left
			neighbourPoint = points[getOffset(x - 1, y)];
			break;
			case 3: //to up
			neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
			break;
			default:
			break;
			}
			if (neighbourPoint == startPoint) {
				currentPoint = startPoint;
				comeFromDeadEnd = false;
				break;
			}
			if (neighbourPoint && neighbourPoint->plane == startPoint->plane && neighbourPoint->plane != nullptr && neighbourPoint->isMarked 
				&& (isPreviousSpike || spikeType(neighbourPoint, -1, false) <= 1 || spikeType(currentPoint, direction, false) > 0))
			{
					if (!wasThereNonSpike && spikeType(currentPoint, -1, false) > 1) {
					startPoint = currentPoint;
					isPreviousSpike = false;
					wasThereNonSpike = true;
				}
				if (!isPreviousSpike)
					previousSavedPoint = { currentPoint, (direction + (4 - i)) % 4 };
				if (spikeType(neighbourPoint, -1, false) == 1 || (wasThereNonSpike && currentEdge.size() > 1 && isPreviousSpike &&
					spikeType(neighbourPoint, -1, false) == 2 && neighbourPoint->isMarked)) {
					auto savedPoint = spikeType(neighbourPoint, -1, false) == 1 ? neighbourPoint : currentPoint;
					currentEdge.push_back({ savedPoint, (direction + 3) % 4 });
					savedPoint->isMarked = false;
					savedPoint->isMarked2 = false;
					currentPoint = previousSavedPoint.first;
					direction = previousSavedPoint.second;
					isPreviousSpike = false;
					comeFromDeadEnd = true;
					break;
				}
				else {
					comeFromDeadEnd = false;
					currentPoint = neighbourPoint;
				}
				direction = (direction + 3) % 4;
				break;
			}
			direction += direction == 3 ? -3 : 1;
			if (i == 3) {
				if (!wasThereNonSpike) {
					currentEdge.clear();
					return;
				}
				if (currentPoint == startPoint)
					return;
				currentPoint = currentEdge[currentEdge.size() - 2].first;
				direction = currentEdge[currentEdge.size() - 2].second;
				std::cout << "INVALID EDGE SEARCH" << std::endl;;
			}
		}
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

void findEdgePoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = true;
		for (size_t j = 0; j < planes[i]->points.size(); j++) if(isEdgePoint(planes[i]->points[j])) edgePointsInPlane.push_back(planes[i]->points[j]);
		while (edgePointsInPlane.size() > 0) 
		{
			std::vector<Point*> tempEdgePointsInPlane;
			Point* startPoint = edgePointsInPlane[0];
			size_t minHorizontalCoord = startPoint->horizontalIndex;
			size_t minVerticalCoord = startPoint->verticalIndex;
			for (size_t j = 1; j < edgePointsInPlane.size(); j++) {
				if ((edgePointsInPlane[j]->horizontalIndex < minHorizontalCoord && minHorizontalCoord - edgePointsInPlane[j]->horizontalIndex < horizontalCount / 2)
					|| edgePointsInPlane[j]->horizontalIndex > minHorizontalCoord + horizontalCount / 2) {
					minHorizontalCoord = edgePointsInPlane[j]->horizontalIndex;
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
				else if (edgePointsInPlane[j]->horizontalIndex == minHorizontalCoord && edgePointsInPlane[j]->verticalIndex < minVerticalCoord) {
					minVerticalCoord = edgePointsInPlane[j]->verticalIndex;
					startPoint = edgePointsInPlane[j];
				}
			}
			size_t direction = 0;
			if (spikeType(startPoint, -1, true) == -1) {
				size_t x = startPoint->horizontalIndex;
				size_t y = startPoint->verticalIndex;
				startPoint->plane = nullptr;
				startPoint->isMarked = false;
				for (size_t j = 0; j < planes[i]->points.size(); j++) {
					if (planes[i]->points[j] == startPoint) {
						planes[i]->points.erase(planes[i]->points.begin() + j);
						break;
					}
				}
				startPoint = points[getOffset(x + 1, y)];
				direction = 3;
			}
			else if(startPoint->verticalIndex > 0 && points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)] &&
				points[getOffset(startPoint->horizontalIndex, startPoint->verticalIndex - 1)]->isMarked2)
				direction = 1;
			std::vector<std::pair<Point*, int>> currentEdge;
			findNextPoint(startPoint, direction, false, false, currentEdge, i, planes);
			for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
				if (edgePointsInPlane[j]->isMarked) tempEdgePointsInPlane.push_back(edgePointsInPlane[j]);
			}
			edgePointsInPlane = tempEdgePointsInPlane;
			if (currentEdge.size() > 0) {
				if (direction == 1) {
					currentEdge.insert(currentEdge.begin(), currentEdge[currentEdge.size() - 1]);
					currentEdge.pop_back();
				}
				for (size_t k = 0; k < currentEdge.size(); k++) {
					currentEdge[k].first->outlineId = currentOutlineId;
				}
				currentOutlineId++;
				planes[i]->edges.push_back({ startPoint, currentEdge });
			}
		}
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked = false;
		for (size_t j = 0; j < planes[i]->points.size(); j++) planes[i]->points[j]->isMarked2 = false;
	}
}

const double newPointAcceptTreshold = 0.95;
const double inf = 1000000;

bool isStraightPoint(size_t pointIndex, std::pair<Point*, std::vector<std::pair<Point*, int>>>& edge, size_t& previousNeighbourCount)
{
	Point* point = edge.second[pointIndex].first;
	if (point->isCorner) return false; // first point can modify last point
	size_t previousNeighbourCountStore = previousNeighbourCount;
	size_t neighbourCount = 0;
	size_t neighbourEdgeCount = 0;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Plane* plane = point->plane;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i != 1) && neighbourPoints[i] && neighbourPoints[i]->plane &&
			neighbourPoints[i]->plane == plane) {
			neighbourCount++;
			isNeighbour[i] = true;
			if (neighbourPoints[i]->outlineId > 0)
				neighbourEdgeCount++;
		}
	}
	previousNeighbourCount = neighbourCount;
	if (areNeighbours(pointIndex < (edge.second.size() - 1) ? edge.second[pointIndex + 1].first : edge.first, point) == 0 && neighbourCount < 3) {
		(pointIndex > 0 ? edge.second[pointIndex - 1].first : edge.first)->isCorner = false;
		(pointIndex > 0 ? edge.second[pointIndex - 2].first : edge.second[edge.second.size() - 1].first)->isCorner = true;
		(pointIndex < (edge.second.size() - 1) ? edge.second[pointIndex + 1].first : edge.first)->isCorner = true;
		return false;
	}
	if ((neighbourCount == 3 && (neighbourEdgeCount == 2 && (pointIndex == edge.second.size() - 1 || areNeighbours(point, edge.second[pointIndex + 1].first)
		> 0)))) {
		return true;
	}
	if (neighbourCount == 4 && (pointIndex == 0 ? edge.second[edge.second.size() - 1].first : edge.second[pointIndex - 1].first)->isCorner &&
		previousNeighbourCountStore == 2)
		return true;
	if (previousNeighbourCountStore == 4) {
		edge.second[pointIndex - 1].first->isCorner = false;
	}
	return false;
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k]->edges.size(); i++) {
			size_t previousNeighbourCount = 0;
			for (size_t j = 0; j < planes[k]->edges[i].second.size(); j++) {
				if (!isStraightPoint(j, planes[k]->edges[i], previousNeighbourCount)) {
					planes[k]->edges[i].second[j].first->isCorner = true;
				}
			}
		}
	}
}

Point* createNewPoint(Vec3<double> newPointPos, Point* point, std::vector<Point*> neighbours, size_t addedCount, bool createBeforePoint = false)
{
	Point* newPoint = new Point(newPointPos, neighbours[0]->horizontalIndex, verticalCount, point->plane);
	addedPoints.push_back(newPoint);
	newPoint->isCorner = true;
	newPoint->outlineId = point->outlineId;
	for (size_t j = 0; j < point->plane->edges.size(); j++) {
		if (point->plane->edges[j].second[0].first->outlineId == point->outlineId) {
			for (size_t k = 0; k < point->plane->edges[j].second.size(); k++) {
				if (point->plane->edges[j].second[k].first == point) {
					point->plane->edges[j].second.insert(point->plane->edges[j].second.begin() + k + (createBeforePoint ? 0 : 1) + addedCount, 
						{ newPoint, -1 });
					break;
				}
			}
			break;
		}
	}
	for (size_t i = 0; i < neighbours.size(); i++) {
		newPoint->neighbourPlaneNeighbours[i] = neighbours[i];
	}
	return newPoint;
}

const double twoPointDifferenceTreshold = 0.5;

Point* addNewPoint(Point* point, Point*& neighbour, Plane* plane, size_t addedCount, size_t neighbourIndex)
{
	if (point->createdNeighbourPoints[neighbourIndex] != nullptr) { //created by other plane
		auto createdNeighbour = point->createdNeighbourPoints[neighbourIndex];
		for (size_t j = 0; j < point->plane->edges.size(); j++) {
			if (point->plane->edges[j].second[0].first->outlineId == point->outlineId) {
				for (size_t k = 0; k < point->plane->edges[j].second.size(); k++) {
					if (point->plane->edges[j].second[k].first == point) {
						for (size_t l = 0; l < 4; l++) 
						{
							if (point->plane->edges[j].second[k + l + 1].first == createdNeighbour) {
								point->plane->edges[j].second.insert(point->plane->edges[j].second.begin() + k + 1 + addedCount, { createdNeighbour, -1 });
								point->plane->edges[j].second.erase(point->plane->edges[j].second.begin() + k + l + 2);
								break;
							}
						}
						break;
					}
				}
				break;
			}
		}
		return createdNeighbour;
	}
	Vec3<double> dir = { 0,0,0 };
	switch (areNeighbours(point, neighbour)) {
	case 1:
	dir = point->plane->pointDirections.first;
	break;
	case 2:
	dir = point->plane->pointDirections.second;
	break;
	case 3:
	dir = point->plane->pointDirections.first * -1;
	break;
	case 4:
	dir = point->plane->pointDirections.second * -1;
	break;
	default:
	break;
	}
	Vec3<double> newPointPos = point->position - dir * Vec3<double>::dot_product(point->position
		- plane->planePointPos, plane->normal) / Vec3<double>::dot_product(dir, plane->normal);
	if (abs(newPointPos.x) > inf || abs(newPointPos.y) > inf || abs(newPointPos.z) > inf || isnan(newPointPos.x) || isnan(newPointPos.y) || isnan(newPointPos.z))
		return nullptr;
	Vec3<double> dirToNew = newPointPos - point->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(dir), Vec3<double>::normalize(dirToNew)) < newPointAcceptTreshold && 
		(point->position - newPointPos).length() > planeDistanceTreshold)
		return nullptr;
	Vec3<double> neighbourDir = { 0,0,0 };
	switch (areNeighbours(neighbour, point)) {
	case 1:
	neighbourDir = plane->pointDirections.first;
	break;
	case 2:
	neighbourDir = plane->pointDirections.second;
	break;
	case 3:
	neighbourDir = plane->pointDirections.first * -1;
	break;
	case 4:
	neighbourDir = plane->pointDirections.second * -1;
	break;
	default:
	break;
	}	
	Vec3<double> neighbourNewPointPos = neighbour->position - neighbourDir * Vec3<double>::dot_product(neighbour->position
		- point->plane->planePointPos, point->plane->normal) / Vec3<double>::dot_product(neighbourDir, point->plane->normal);
	if (abs(neighbourNewPointPos.x) > inf || abs(neighbourNewPointPos.y) > inf || abs(neighbourNewPointPos.z) > inf || isnan(neighbourNewPointPos.x) ||
		isnan(neighbourNewPointPos.y) || isnan(neighbourNewPointPos.z))
		return nullptr;
	Vec3<double> dirToNewNeighbour = neighbourNewPointPos - neighbour->position;
	if (Vec3<double>::dot_product(Vec3<double>::normalize(neighbourDir), Vec3<double>::normalize(dirToNewNeighbour)) < newPointAcceptTreshold &&
		(neighbour->position - neighbourNewPointPos).length() > planeDistanceTreshold)
		return nullptr;
	if ((newPointPos - neighbourNewPointPos).length() > twoPointDifferenceTreshold)
		return nullptr;
	auto newPos = (newPointPos + neighbourNewPointPos) / 2;
	//bool isNeighbourEdge = false;
	Point* newPoint;
	Point* newNeighbourPoint;
	newPoint = createNewPoint(newPos, point, { neighbour }, addedCount);
	point->createdNeighbourPoints[neighbourIndex] = newPoint;
	for (size_t i = 0; i < neighbour->neighbourPlaneNeighbours.size(); i++) 
	{
		if (neighbour->neighbourPlaneNeighbours[i] == point) 		
		{
			//isNeighbourEdge = true;
			newNeighbourPoint = createNewPoint(newPos, neighbour, { point }, 0);
			neighbour->createdNeighbourPoints[i] = newNeighbourPoint;
			break;
		}
	}
	/*if (!isNeighbourEdge) {
		newNeighbourPoint = createNewPoint(newPos, neighbour, { point }, 0);
		neighbour->neighbourPlaneNeighbours.push_back(newNeighbourPoint);
	}*/
	newPoint->neighbourPlaneNeighbours[1] = newNeighbourPoint;
	newNeighbourPoint->neighbourPlaneNeighbours[1] = newPoint;
	return newPoint;
}

void findPlaneConnections()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		for (size_t j = 0; j < planes[i]->edges.size(); j++) 
		{
			for (size_t k = 0; k < planes[i]->edges[j].second.size(); k++)
			{
				Point* point = planes[i]->edges[j].second[k].first;
				int direction = planes[i]->edges[j].second[k].second;
				size_t x = point->horizontalIndex;
				size_t y = point->verticalIndex;
				Point* neighbourPoints[4] = { points[getOffset(x + 1, y)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
					points[getOffset(x, y - 1)] };
				for (size_t i = 0; i < 4; i++) {
					if ((y > 0 || direction != 3) && (y < verticalCount - 1 || direction != 1) && neighbourPoints[direction] && 
						neighbourPoints[direction]->outlineId > 0 && neighbourPoints[direction]->outlineId != point->outlineId && 
						neighbourPoints[direction]->plane != point->plane) {
						point->neighbourPlaneNeighbours[i] = neighbourPoints[direction];
					}
					else if((y == 0 && direction == 3) || (y == verticalCount - 1 && direction == 1) || !neighbourPoints[direction] ||
						neighbourPoints[direction]->plane == nullptr)
						point->neighbourPlaneNeighbours[i] = nullptr;
					direction += direction == 3 ? -3 : 1;
				}
			}
		}
	}
}

void createPlaneCorner(Point* point, std::vector<Point*> point1, std::vector<Point*> point2)
{
	Plane* p1 = point->plane;
	Plane* p2 = point1[2]->plane;
	Plane* p3 = point2[2]->plane;
	auto cross1 = Vec3<double>::crossProduct(p2->normal, p3->normal);
	auto cross2 = Vec3<double>::crossProduct(p3->normal, p1->normal);
	auto cross3 = Vec3<double>::crossProduct(p1->normal, p2->normal);

	auto denom = Vec3<double>::dot_product(p1->normal, cross1);

	auto planeDist1 = Vec3<double>::dot_product(p1->planePointPos, p1->normal);
	auto planeDist2 = Vec3<double>::dot_product(p2->planePointPos, p2->normal);
	auto planeDist3 = Vec3<double>::dot_product(p3->planePointPos, p3->normal);

	cross1 = cross1 * planeDist1;
	cross2 = cross2 * planeDist2;
	cross3 = cross3 * planeDist3;

	auto cornerPoint = (cross1 + cross2 + cross3) / denom;

	createNewPoint(cornerPoint, point1[0], {point1[2], point2[2]}, 0);
	createNewPoint(cornerPoint, point1[1], { point }, 0, true);
	createNewPoint(cornerPoint, point2[1], { point }, 0);
}

void connectPlanes()
{
	std::vector<std::vector<bool>> wasFirstGeneratedVec;
	std::vector<Point*> createdPoints;
	for (size_t i = 0; i < planes.size(); i++) {
		wasFirstGeneratedVec.push_back({});
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			wasFirstGeneratedVec[i].push_back({false});
			for (size_t k = 0; k < planes[i]->edges[j].second.size(); k++) {
				auto point = planes[i]->edges[j].second[k].first;
				if (point->verticalIndex == verticalCount)
				{
					continue;
				}
				size_t addedCount = 0;
				if(point->isCorner)
				{
					for (size_t l = 0; l < point->neighbourPlaneNeighbours.size(); l++) {
						Point* newPoint = nullptr;
						if (point->neighbourPlaneNeighbours[l])
						{
							newPoint = addNewPoint(point, point->neighbourPlaneNeighbours[l], point->neighbourPlaneNeighbours[l]->plane, addedCount, l);
							if (newPoint) {
								if (k == 0 && ((planes[i]->edges[j].second[k].first->horizontalIndex -
									point->neighbourPlaneNeighbours[l]->horizontalIndex + horizontalCount) %
									horizontalCount == 1)) wasFirstGeneratedVec[i][wasFirstGeneratedVec[i].size() - 1] = true;
								planes[i]->edges[j].second[k].first->isCorner = false;
								createdPoints.push_back(newPoint);
								addedCount++;
							}
							else {
								planes[i]->edges[j].second.insert(planes[i]->edges[j].second.begin() + k + 1 + addedCount, { nullptr, -1 });
								addedCount++;
							}
						}						
					}
					k += addedCount;
				}
			}			
		}
	}

	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			for (size_t k = 0; k < planes[i]->edges[j].second.size(); k++) {
				auto point = planes[i]->edges[j].second[k].first;
				if (point) {
					for (size_t l = 0; l < planes[i]->edges[j].second[k].first->createdNeighbourPoints.size(); l++) {
						auto createdPoint1 = planes[i]->edges[j].second[k].first->createdNeighbourPoints[l];
						auto createdPoint2 = planes[i]->edges[j].second[k].first->createdNeighbourPoints[l == 3 ? 0 : (l + 1)];
						if (createdPoint1 && createdPoint2 && createdPoint1->neighbourPlaneNeighbours[1]->plane !=
							createdPoint2->neighbourPlaneNeighbours[1]->plane) {
							createPlaneCorner(planes[i]->edges[j].second[k].first, { createdPoint1, createdPoint1->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l] }, { createdPoint2, createdPoint2->neighbourPlaneNeighbours[1],
								point->neighbourPlaneNeighbours[l == 3 ? 0 : (l + 1)] });
						}
					}
				}
			}
		}
	}

	std::vector<std::vector<std::vector<std::pair<size_t, size_t>>>> cornersToDelete;
	for (size_t i = 0; i < planes.size(); i++) {
		cornersToDelete.push_back({});
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			cornersToDelete[i].push_back({});
			for (size_t k = 0; k < planes[i]->edges[j].second.size(); k++) {
				if (planes[i]->edges[j].second[k].first) {
					if (planes[i]->edges[j].second[k].first->verticalIndex == verticalCount) {
						for (size_t l = 0; l < planes[i]->edges[j].second[k].first->neighbourPlaneNeighbours.size(); l++) {
							if (planes[i]->edges[j].second[k].first->neighbourPlaneNeighbours[l] &&
								planes[i]->edges[j].second[k].first->neighbourPlaneNeighbours[l]->verticalIndex < verticalCount)
								cornersToDelete[i][j].push_back({ k,  planes[i]->edges[j].second[k].first->neighbourPlaneNeighbours[l]->plane->id });
						}
					}						
				}
				else 
				{
					cornersToDelete[i][j].push_back({ 0, 0 });
					planes[i]->edges[j].second.erase(planes[i]->edges[j].second.begin() + k);
					k--;
				}
			}
			int newPointStartIndex = wasFirstGeneratedVec[i][j] ? 1 : 0;
			int newPointEndIndex = newPointStartIndex;
			while (newPointEndIndex < (int)cornersToDelete[i][j].size() - 1) {
				if (cornersToDelete[i][j][newPointStartIndex].second > 0) {
					while (newPointEndIndex + 1 < cornersToDelete[i][j].size()
						&& cornersToDelete[i][j][newPointStartIndex].second == cornersToDelete[i][j][newPointEndIndex + 1].second) {
						newPointEndIndex++;
					}
					for (size_t k = cornersToDelete[i][j][newPointStartIndex].first + 1; k < cornersToDelete[i][j][newPointEndIndex].first; k++) {
						planes[i]->edges[j].second[k].first->isCorner = false;
					}
					if(newPointEndIndex == cornersToDelete[i][j].size() - 1 && newPointStartIndex != newPointEndIndex && wasFirstGeneratedVec[i][j] 
						&& cornersToDelete[i][j][0].second == cornersToDelete[i][j][newPointStartIndex].second)
						planes[i]->edges[j].second[cornersToDelete[i][j][cornersToDelete[i][j].size() - 1].first].first->isCorner = false;
				}
				newPointStartIndex = newPointEndIndex + 1;
				newPointEndIndex = newPointStartIndex;
			}
		}
	}
	for (size_t i = 0; i < createdPoints.size(); i++) {
		if (createdPoints[i]->isCorner)
			createdPoints[i]->neighbourPlaneNeighbours[1]->isCorner = true;
	}
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i] = nullptr;
	}
}

void exportObjects()
{
	for (size_t i = 0; i < planes.size(); i++) {
		for (size_t j = 0; j < planes[i]->edges.size(); j++) {
			std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models/processed_obj_" 
				+ std::to_string(currentCornerId - 1) + ".obj");
			MyFile << "o Mesh" << std::endl;
			std::vector<Point*> corners;
			currentCornerIndex = 0;
			for (size_t k = 0; k < planes[i]->edges[j].second.size(); k++) {
				if (planes[i]->edges[j].second[k].first->isCorner) {
					planes[i]->edges[j].second[k].first->cornerId = currentCornerId;
					planes[i]->edges[j].second[k].first->cornerIndex = currentCornerIndex;
					corners.push_back(planes[i]->edges[j].second[k].first);
					currentCornerIndex++;
				}
			}
			for (size_t k = 0; k < corners.size(); k++) {
				MyFile << "v " << -corners[k]->position.x << " " << corners[k]->position.y << " " << corners[k]->position.z << std::endl;
			}
			MyFile << "f ";
			for (size_t j = 1; j < corners.size() + 1; j++) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile << "f ";
			for (size_t j = corners.size(); j > 0; j--) {
				MyFile << j << " ";
			}
			MyFile << std::endl;
			MyFile.close();
			currentCornerId++;
		}
	}
}

void processData() {
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	findEdgePoints();
	findCorners();
	findPlaneConnections();
	connectPlanes();
	exportObjects();
	writeData();
}

int main()
{
    readData();
	auto start = std::chrono::steady_clock::now();
	processData();
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec" << std::endl;
    return 0;
}