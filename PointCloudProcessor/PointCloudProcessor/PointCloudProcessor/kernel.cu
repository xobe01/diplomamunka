#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>

struct Point
{
	Vec3<double> position;
	size_t horizontalIndex;
	size_t verticalIndex;
	size_t id{ 0 };
	size_t cornerId{ 0 };
	size_t outlineId{ 0 };
	size_t cornerIndex{ 0 };
	Vec3<double> normal = {0, 0, 0};
	bool isMarked{ false };
	bool isMarked2{ false };
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, size_t _id) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), id(_id) {};
};

struct Plane {
	std::vector<Point*> points;
	std::pair<double, double> horizontalBounds;
	std::pair<double, double> verticalBounds;
	std::vector<std::vector<Point*>> edges;
	Vec3<double> planePointPos;
	Vec3<double> normal;
};

std::vector<Point*> points;
std::vector<Point*> addedPoints;
std::vector<Plane> planes;
std::vector<std::vector<Point*>> filteredCorners;
std::vector<int> verticalCounts;
size_t horizontalCount;
size_t verticalCount;
int currentCornerId = 1;
int currentSeparatedObjectId = 1;
int currentPointId = 1;
int currentOutlineId = 1;
int currentCornerIndex = 0;
const double objectPointDistance = 5;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void ReadData()
{
	verticalCounts.push_back(0);
    std::string myText;		  
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, id);
		}		
	}
    MyReadFile.close();
}

void writePoints(const std::vector<Point*> points, std::ofstream& MyFile)
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i])
			MyFile << points[i]->position.to_string() << ';' << points[i]->horizontalIndex << ';' << points[i]->verticalIndex <<
			';' << points[i]->id << ';' << points[i]->outlineId << ';' << points[i]->cornerId
			<< ';' << points[i]->cornerIndex << std::endl;
	}
}

void writeData()
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	writePoints(points, MyFile);
	writePoints(addedPoints, MyFile);
	MyFile.close();
}

void groundSegmentation() { //TODO point struktúra megvátozott
	double groundLevel = 100;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y < groundLevel) groundLevel = points[i]->position.y;
	}
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.y <= groundLevel) {
			points[i] = nullptr;
		}
	}
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

bool checkIfBridge(Point* p)
{
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;
	//fel-le
	bool isUpNotNeightbour = y == 0 || !points[getOffset(x, y - 1)] || points[getOffset(x, y - 1)]->id != p->id;
	bool isDownNotNeightbour = y == verticalCount - 1 || !points[getOffset(x, y + 1)] || points[getOffset(x, y + 1)]->id != p->id;
	if (!isUpNotNeightbour)
		neighbourCount++;
	if (!isDownNotNeightbour)
		neighbourCount++;
	//jobbra-balra
	bool isLeftNotNeightbour = !points[getOffset(x - 1, y)] || points[getOffset(x - 1, y)]->id != p->id;
	bool isRighttNotNeightbour = !points[getOffset(x + 1, y)] || points[getOffset(x + 1, y)]->id != p->id;
	if (!isLeftNotNeightbour)
		neighbourCount++;
	if (!isRighttNotNeightbour)
		neighbourCount++;
	if (((isLeftNotNeightbour && isRighttNotNeightbour) || (isUpNotNeightbour && isDownNotNeightbour))) {
		return true;
	}

	//átlósan
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || points[getOffset(x - 1, y - 1)]->id != p->id)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || points[getOffset(x + 1, y + 1)]->id != p->id))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || points[getOffset(x + 1, y - 1)]->id != p->id)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || points[getOffset(x - 1, y + 1)]->id != p->id))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(std::vector<Point*>& checkPoints)
{
	std::vector<Point*> newPoints;
	bool theresBridge = false;
	for (auto p : checkPoints)
		if (p->id > 0 && checkIfBridge(p)) {
			theresBridge = true;
			p->id = 0;
		}
		else
			newPoints.push_back(p);
	checkPoints = newPoints;
	return theresBridge;
}

void checkForGaps(/*out*/ std::vector<Point*>& chosenPoints)
{  
	for (auto p : chosenPoints) p->isMarked = true;
	while (isThereBridge(chosenPoints)) {}
	std::vector<Point*> nonProcessedPoints(chosenPoints);
	std::vector<Point*> nextStepPoints;
	std::vector<Point*> bestPartitionPoints;
	while (bestPartitionPoints.size() < nonProcessedPoints.size()) {
		std::vector<Point*> currentPartitionPoints;
		currentPartitionPoints.push_back(nonProcessedPoints[0]);
		nextStepPoints.push_back(nonProcessedPoints[0]);
		nonProcessedPoints[0]->isMarked = false;
		while (nextStepPoints.size() > 0) {
			std::vector<Point*> tempNextStepPoints;
			for (size_t i = 0; i < nextStepPoints.size(); i++) {
				size_t x = nextStepPoints[i]->horizontalIndex;
				size_t y = nextStepPoints[i]->verticalIndex;
				Point* neighbourPoint = points[getOffset(x, y - 1)];
				if (y > 0 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x, y + 1)];
				if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x - 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x + 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
			}
			nextStepPoints = tempNextStepPoints;
		}
		if (currentPartitionPoints.size() > bestPartitionPoints.size()) bestPartitionPoints = currentPartitionPoints;
		std::vector<Point*> tempNonProcessedPoints;
		for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
			if (nonProcessedPoints[i]->isMarked) tempNonProcessedPoints.push_back(nonProcessedPoints[i]);
		}
		nonProcessedPoints = tempNonProcessedPoints;
	}
	for (size_t i = 0; i < chosenPoints.size(); i++) {
		chosenPoints[i]->isMarked = false;
	}
	chosenPoints = bestPartitionPoints;
}

#include <chrono>

Vec3<double> getNormal(Point* center, Point* p1, Point* p2)
{
	return Vec3<double>::crossProduct(center->position - p1->position, center->position - p2->position);
}

void calculateNormal(Point* point)
{
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint1 = points[getOffset(x, y - 1)];
	Point* neighbourPoint2 = points[getOffset(x - 1, y)];
	Point* neighbourPoint3 = points[getOffset(x, y + 1)];
	Point* neighbourPoint4 = points[getOffset(x + 1, y)];
	if (y > 0 && neighbourPoint1 && neighbourPoint2) {
		point->normal = point->normal + getNormal(point, neighbourPoint1, neighbourPoint2);
	}
	if (y < verticalCount - 1 && neighbourPoint2 && neighbourPoint3) {
		point->normal = point->normal + getNormal(point, neighbourPoint2, neighbourPoint3);
	}
	if (y < verticalCount - 1 && neighbourPoint3 && neighbourPoint4) {
		point->normal = point->normal + getNormal(point, neighbourPoint3, neighbourPoint4);
	}
	if (y > 0 && neighbourPoint4 && neighbourPoint1) {
		point->normal = point->normal + getNormal(point, neighbourPoint4, neighbourPoint1);
	}
	point->normal = Vec3<double>::normalize(point->normal);
}

void choosePoints(const Vec3<Point*> planePoints, double acceptTreshold, /*out*/ Plane& plane)
{
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.x->position -
		planePoints.y->position, planePoints.z->position - planePoints.y->position));
	plane.normal = normal;
	plane.planePointPos = planePoints.x->position;
	std::vector<Point*>nextStepPoints;
	nextStepPoints.push_back(planePoints.x);
	planePoints.x->isMarked = false;
	planePoints.x->isMarked2 = false;
	planePoints.x->id = currentPointId;
	plane.points.push_back(planePoints.x);
	while (nextStepPoints.size() > 0) {
		std::vector<Point*> tempNextStepPoints;
		for (size_t i = 0; i < nextStepPoints.size(); i++) {
			size_t x = nextStepPoints[i]->horizontalIndex;
			size_t y = nextStepPoints[i]->verticalIndex;
			Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)], 
				points[getOffset(x + 1, y)] };
			for (size_t j = 0; j < 4; j++) {
				if (neighbourPoints[j] && (j > 0 || y > 0) && (j < 3 || y < verticalCount - 1) && neighbourPoints[j]->isMarked2) {
					double dist = abs(Vec3<double>::dot_product(normal, neighbourPoints[j]->position - planePoints.y->position));
					if (dist <= acceptTreshold) {
						plane.points.push_back(neighbourPoints[j]);						
						neighbourPoints[j]->isMarked = false;
						neighbourPoints[j]->isMarked2 = false;
						neighbourPoints[j]->id = currentPointId;
						tempNextStepPoints.push_back(neighbourPoints[j]);
					}
				}
			}
		}
		nextStepPoints = tempNextStepPoints;
	}
}

void calculateBounds(Plane& plane)
{
	plane.horizontalBounds = std::make_pair(plane.points[0]->horizontalIndex, plane.points[0]->horizontalIndex);
	plane.verticalBounds = std::make_pair(plane.points[0]->verticalIndex, plane.points[0]->verticalIndex);
	for (size_t i = 1; i < plane.points.size(); i++) 
	{
		if (plane.points[i]->horizontalIndex > plane.horizontalBounds.second)
			plane.horizontalBounds.second = plane.points[i]->horizontalIndex;
		else if (plane.points[i]->horizontalIndex < plane.horizontalBounds.first)
			plane.horizontalBounds.first = plane.points[i]->horizontalIndex;
		if (plane.points[i]->verticalIndex > plane.verticalBounds.second)
			plane.verticalBounds.second = plane.points[i]->verticalIndex;
		else if (plane.points[i]->verticalIndex < plane.verticalBounds.first)
			plane.verticalBounds.first = plane.points[i]->verticalIndex;
	}
}

void findPlanes()
{
	auto start = std::chrono::steady_clock::now();
	size_t minPointCount = 10;
	size_t counter = 1;
	double planeDistanceTreshold = 0.01;
	double normalTreshold = 0.1;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = true;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = true;
	std::vector<Point*> nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) {
		if (points[j] && points[j]->isMarked) {
			Plane plane;		
			nextStepPoints.push_back(points[j]);
			calculateNormal(points[j]);
			while (nextStepPoints.size() > 0) {
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					if (nextStepPoints[i]->isMarked2) {
						auto normal = nextStepPoints[i]->normal;
						Vec3<double> normals[4] = { {0,0,0}, {0,0,0}, {0,0,0}, {0,0,0} };
						size_t x = nextStepPoints[i]->horizontalIndex;
						size_t y = nextStepPoints[i]->verticalIndex;
						Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x - 1, y)], points[getOffset(x, y + 1)],
							points[getOffset(x + 1, y)] };
						for (size_t k = 0; k < 4; k++) {
							if (neighbourPoints[k] && (k > 0 || y > 0) && (k < verticalCount - 1 || y < 3) && neighbourPoints[k]->isMarked2) {
								if (neighbourPoints[k]->isMarked) {
									neighbourPoints[k]->isMarked = false;
									tempNextStepPoints.push_back(neighbourPoints[k]);
								}
								if (neighbourPoints[k]->normal.length() == 0) {
									calculateNormal(neighbourPoints[k]);
								}
								normals[k] = neighbourPoints[k]->normal;
							}
						}
						for (size_t k = 0; k < 4; k++) {
							if ((normals[k] - normal).length() < normalTreshold && (normals[(k + 1) % 4] - normal).length() < normalTreshold) {
								choosePoints({ nextStepPoints[i], neighbourPoints[k], neighbourPoints[(k + 1) % 4] }, planeDistanceTreshold,
									plane);
								break;
							}
						}
					}
				}
				if (plane.points.size() > 0)
				{
					planes.push_back(plane);
					plane = Plane();
					currentPointId++;
				}
				nextStepPoints = tempNextStepPoints;
			}
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked2 = false;
	for (size_t i = 0; i < planes.size(); i++) {
		while (isThereBridge(planes[i].points)) {}
	}
	for (size_t i = 0; i < planes.size(); i++) {
		calculateBounds(planes[i]);	
	}
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec" << std::endl;
}

void findNextEdgePoint(Point* startPoint, Point* currentPoint, std::vector<Point*>& edgePoints, size_t direction)
{
	edgePoints.push_back(currentPoint);
	size_t x = currentPoint->horizontalIndex;
	size_t y = currentPoint->verticalIndex;
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 4; i++) {
		switch (direction) {
		case 0: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && neighbourPoint->id == startPoint->id) {
			if (neighbourPoint == startPoint) {
			}
			else 
			{
				findNextEdgePoint(startPoint, neighbourPoint, edgePoints, (direction + 3) % 4);
			}
			break;
		}
		direction += direction == 3 ? -3 : 1;
	}
	if (neighbourPoint == nullptr) {
		std::cout << "PLANE WITHOUT RELEVANT OUTLINE!" << std::endl;
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked)
		return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

int areNeighbours(Point* p1, Point* p2)
{
	if (points[getOffset(p1->horizontalIndex + 1, p1->verticalIndex)] == p2)
		return 1;
	if (points[getOffset(p1->horizontalIndex - 1, p1->verticalIndex)] == p2)
		return 2;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex + 1)] == p2)
		return 3;
	if (points[getOffset(p1->horizontalIndex, p1->verticalIndex - 1)] == p2)
		return 4;
	return 0;
}

bool isStraightEdgePoint(Point* point, std::vector<Point*>& edge, bool& isPreviousConcave)
{
	bool previousConcaveStore = isPreviousConcave;
	size_t neighbourCount = 0;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	size_t planeId = point->id;
	bool isNeighbour[4] = { false, false, false, false };
	Point* neighbourPoints[4] = { points[getOffset(x, y - 1)], points[getOffset(x, y + 1)], points[getOffset(x - 1, y)],
		points[getOffset(x + 1, y)] };
	for (size_t i = 0; i < 4; i++) {
		if ((y > 0 || i > 0) && (y < verticalCount - 1 || i < 3) && neighbourPoints[i] && neighbourPoints[i]->id == planeId) {
			neighbourCount++;
			isNeighbour[i] = true;
		}
	}
	isPreviousConcave = neighbourCount == 4;
	if (neighbourCount == 3 || (neighbourCount == 2 && ((isNeighbour[0] && isNeighbour[1]) || (isNeighbour[2] && isNeighbour[3])))) {
		return true;
	}
	if (neighbourCount == 4 && edge.size() > 0 && areNeighbours(point, edge[edge.size() - 1]) > 0)
		return true;
	if (previousConcaveStore && edge.size() > 0 && areNeighbours(point, edge[edge.size() - 1]) > 0) {
		edge[edge.size() - 1] = point;
		return true;
	}
	return false;
}

void findEdgePoints()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i].points.size(); j++) {
			planes[i].points[j]->isMarked = true;
		}		
		for (size_t j = 0; j < planes[i].points.size(); j++) if(isEdgePoint(planes[i].points[j])) edgePointsInPlane.push_back(planes[i].points[j]);
		
		bool isFirstEdge = true;
		for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
			if (edgePointsInPlane[j]->isMarked) {
				std::vector<Point*> edgePoints;
				findNextEdgePoint(edgePointsInPlane[j], edgePointsInPlane[j], edgePoints, isFirstEdge ? 0 : 1);
				if (!isFirstEdge) {
					edgePoints.insert(edgePoints.begin(), edgePoints[edgePoints.size() - 1]);
					edgePoints.pop_back();
				}
				for (size_t k = 0; k < edgePoints.size(); k++) 
				{
					edgePoints[k]->isMarked = false;
					edgePoints[k]->outlineId = currentOutlineId;
				}
				currentOutlineId++;
				planes[i].edges.push_back(edgePoints);
				isFirstEdge = false;
			}
		}
	}
}

bool arePlanesNeighbours(Plane p1, Plane p2, std::pair<int, int>& horizontalCommonBounds, std::pair<int, int>& verticalCommonBounds)
{
	if (p1.horizontalBounds.first > p1.horizontalBounds.second)
		p1.horizontalBounds.second += horizontalCount;
	double p1HorizontalSize = p1.horizontalBounds.second - p1.horizontalBounds.first;
	double p1VerticalSize = p1.verticalBounds.second - p1.verticalBounds.first;
	std::pair<int, int> p1Center = {(int)(p1.horizontalBounds.first + p1HorizontalSize / 2) % horizontalCount, 
		p1.verticalBounds.first + p1VerticalSize / 2 };

	if (p2.horizontalBounds.first > p2.horizontalBounds.second)
		p2.horizontalBounds.second += horizontalCount;
	double p2HorizontalSize = p2.horizontalBounds.second - p2.horizontalBounds.first;
	double p2VerticalSize = p2.verticalBounds.second - p2.verticalBounds.first;
	std::pair<int, int> p2Center = { (int)(p2.horizontalBounds.first + p2HorizontalSize / 2) % horizontalCount,
		p2.verticalBounds.first + p2VerticalSize / 2 };

	if (abs(p1Center.first - p2Center.first) > horizontalCount / 2) 
	{
		if (p1Center.first > p2Center.first)
			p2Center.first += horizontalCount;
		if (p1Center.first < p2Center.first)
			p1Center.first += horizontalCount;
	}
	if (abs(p1Center.first - p2Center.first) <= (p1HorizontalSize + p2HorizontalSize) / 2 + 1 &&
		abs(p1Center.second - p2Center.second) <= (p1VerticalSize + p2VerticalSize) / 2 + 1) 
	{
		horizontalCommonBounds = {abs(p1.horizontalBounds.first - p2.horizontalBounds.first) > horizontalCount / 2
			? std::min(p1.horizontalBounds.first, p2.horizontalBounds.first) : std::max(p1.horizontalBounds.first, p2.horizontalBounds.first),
			abs(p1.horizontalBounds.second - p2.horizontalBounds.second) > horizontalCount / 2
			? std::max(p1.horizontalBounds.second, p2.horizontalBounds.second) : std::min(p1.horizontalBounds.second, p2.horizontalBounds.second)};
		if (horizontalCommonBounds.second < horizontalCommonBounds.first)
			horizontalCommonBounds = { horizontalCommonBounds.second, horizontalCommonBounds.first };

		verticalCommonBounds = { std::max(p1.verticalBounds.first, p2.verticalBounds.first),
			std::min(p1.verticalBounds.second, p2.verticalBounds.second) };
		if (verticalCommonBounds.second < verticalCommonBounds.first)
			verticalCommonBounds = { verticalCommonBounds.second, verticalCommonBounds.first };
		return true;
	}
	return false;
}

void connectPlanes()
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		for (size_t j = i + 1; j < planes.size(); j++) 
		{
			std::pair<int, int> horizontalCommonBounds;
			std::pair<int, int> verticalCommonBounds;
			if (arePlanesNeighbours(planes[i], planes[j], horizontalCommonBounds, verticalCommonBounds)) 				
			{
				for (size_t k = 0; k < planes[i].edges.size(); k++) 
				{
					for (size_t l = 0; l < planes[j].edges.size(); l++) 
					{
						for (size_t m = 0; m < planes[i].edges[k].size(); m++) 
						{
							for (size_t n = 0; n < planes[j].edges[l].size(); n++) 
							{
								int neighbourType = areNeighbours(planes[i].edges[k][m], planes[j].edges[l][n]);
								if (neighbourType > 0)
								{
									std::cout << neighbourType << std::endl;
									Vec3<double> dir = { 0,0,0 };
									size_t x = planes[i].edges[k][m]->horizontalIndex;
									size_t y = planes[i].edges[k][m]->verticalIndex;
									switch (neighbourType) {
									case 1:
									dir = planes[i].edges[k][m]->position - points[getOffset(x - 1, y)]->position;
									break;
									case 2:
									dir = planes[i].edges[k][m]->position - points[getOffset(x + 1, y)]->position;
									break;
									case 3:
									dir = planes[i].edges[k][m]->position - points[getOffset(x, y - 1)]->position;
									break;
									case 4:
									dir = planes[i].edges[k][m]->position - points[getOffset(x, y + 1)]->position;
									break;
									default:
									break;
									}
									Vec3<double> newPointPos = planes[i].edges[k][m]->position - dir * Vec3<double>::dot_product(planes[i].edges[k][m]->position
										- planes[j].planePointPos, planes[j].normal) / Vec3<double>::dot_product(dir, planes[j].normal);
									Point* newPoint = new Point(newPointPos, 0, 0, planes[i].edges[k][m]->id);
									addedPoints.push_back(newPoint);
								}
							}
						}
					}
				}
			}
		}
	}
}

void findCorners()
{
	for (size_t k = 0; k < planes.size(); k++) {
		for (size_t i = 0; i < planes[k].edges.size(); i++) {
			bool isPreviousConcave = false;
			std::vector<Point*> corners;
			for (size_t j = 0; j < planes[k].edges[i].size(); j++) {
				if (!isStraightEdgePoint(planes[k].edges[i][j], corners, isPreviousConcave)) {
					corners.push_back(planes[k].edges[i][j]);
				}
			}
			if (corners.size() >= 3) {
				filteredCorners.push_back(corners);
				for (size_t k = 0; k < corners.size(); k++) {
					corners[k]->cornerId = currentCornerId;
					corners[k]->cornerIndex = currentCornerIndex;
					currentCornerIndex++;
				}
				currentCornerIndex = 0;
				currentCornerId++;
			}
		}
	}
}

double distancePointFromLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line)
{
	Vec3<double> directionVector = line.first - line.second;
	return Vec3<double>::crossProduct(p - line.first, directionVector).length() / directionVector.length();
}

bool isOnLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line, double treshold)
{
	return distancePointFromLine(p, line) < treshold;
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i] = nullptr;
	}
}

void exportObjects()
{
	for (size_t i = 0; i < filteredCorners.size(); i++) {
		std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomamunka/Diplomamunka/Diplomamunka/Assets/Resources/Generated_Models/processed_obj_" + std::to_string(i) + ".obj");
		MyFile << "o Mesh" << std::endl;
		for (size_t j = 0; j < filteredCorners[i].size(); j++) {
			MyFile << "v " << -filteredCorners[i][j]->position.x << " " << filteredCorners[i][j]->position.y << " " <<
				filteredCorners[i][j]->position.z << std::endl;
		}
		MyFile << "f ";
		for (size_t j = 1; j < filteredCorners[i].size() + 1; j++) {
			MyFile << j << " ";
		}
		MyFile << std::endl;
		MyFile << "f ";
		for (size_t j = filteredCorners[i].size(); j > 0; j--) {
			MyFile << j << " ";
		}
		MyFile << std::endl;
		MyFile.close();
	}
}

void processData() {
	groundSegmentation();
	egoCarSegmentation();
	findPlanes();
	findEdgePoints();
	connectPlanes();
	findCorners();
	exportObjects();
	writeData();
}

int main()
{
    ReadData();
	processData();
    return 0;
}