#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include "Vec3.cpp"
#include <algorithm>
#include <sstream>
#include <vector>

struct Point
{
	Vec3<double> position;
	size_t horizontalIndex;
	size_t verticalIndex;
	size_t id{ 0 };
	bool deleted{ false };
	bool isMarked{ false };
	Point(Vec3<double> _position, size_t _horizontalIndex, size_t _verticalIndex, size_t _id) : position(_position), horizontalIndex(_horizontalIndex),
		verticalIndex(_verticalIndex), id(_id) {};
};

std::vector<Point*> points;
std::vector<std::vector<Point*>> separatedPoints;
std::vector<int> verticalCounts;
size_t pointCount;
size_t horizontalCount;
size_t verticalCount;
int currentId = 1;
const double objectPointDistance = 5;

size_t getOffset(int horizontalIndex, int verticalIndex)
{
	if (horizontalIndex < 0) horizontalIndex = horizontalCount - 1 + horizontalIndex;
	else if (horizontalIndex > horizontalCount - 1) horizontalIndex = horizontalIndex - horizontalCount;
	if (verticalIndex < 0) verticalIndex = verticalCount - 1 + verticalIndex;
	else if (verticalIndex > verticalCount - 1) verticalIndex = verticalIndex - verticalCount;
	return horizontalIndex * verticalCount + verticalIndex;
}

void ReadData()
{
	verticalCounts.push_back(0);
    std::string myText;
    std::ifstream MyReadFile("C:/Users/ungbo/Desktop/BME/_Diplomaterv/Diplomamunka/Diplomamunka/Assets/Resources/points_raw.txt");
	getline(MyReadFile, myText);
	pointCount = std::stoi(myText);
	getline(MyReadFile, myText);
	horizontalCount = std::stoi(myText);
	getline(MyReadFile, myText);
	verticalCount = std::stoi(myText);
	points.resize(verticalCount * horizontalCount);
    while (getline(MyReadFile, myText)) {
		std::replace(myText.begin(), myText.end(), ',', '.');
		std::stringstream ss(myText);
		std::string _x, _y, _z, _horizontalIndex, _verticalIndex, _id;
		std::getline(ss, _x, ';');
		if (_x == myText) {
			verticalCounts.push_back(std::stoi(myText));
		}
		else {
			std::getline(ss, _y, ';');
			std::getline(ss, _z, ';');
			std::getline(ss, _horizontalIndex, ';');
			std::getline(ss, _verticalIndex, ';');
			std::getline(ss, _id, ';');
			double x = std::stof(_x);
			double y = std::stof(_y);
			double z = std::stof(_z);
			int id = std::stoi(_id);
			int horizontalIndex = std::stoi(_horizontalIndex);
			int verticalIndex = std::stoi(_verticalIndex);
			points[getOffset(horizontalIndex, verticalIndex)] = new Point({ x, y, z }, horizontalIndex, verticalIndex, id);
		}		
	}
    MyReadFile.close();
}

void WriteData(std::vector<Point*> exportPoints)
{
	std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomaterv/Diplomamunka/Diplomamunka/Assets/Resources/points_processed.txt");
	for (size_t i = 0; i < exportPoints.size(); i++) {
		if (exportPoints[i] && !exportPoints[i]->deleted)
			MyFile << exportPoints[i]->position.to_string() << ';' << exportPoints[i]->horizontalIndex << ';' << exportPoints[i]->verticalIndex <<
			';' << exportPoints[i]->id << std::endl;
	}
	MyFile.close();
}

void GroundSegmentation() { //TODO point struktúra megvátozott
	/*double groundLevel = points[0].position.y;
	for (size_t i = 0; i < pointCount; i++) {
		if (points[i].position.y < groundLevel) groundLevel = points[i].position.y;
	}

	for (size_t i = 0; i < pointCount; i++) {
		if (points[i].position.y == groundLevel)
			points[i].deleted = true;
	}*/
}

void choosePoints(const Vec3<Point*> planePoints, const std::vector<Point*>& nonProcessedPoints, double acceptTreshold, 
	/*out*/ std::vector<Point*>& chosenPoints, /*out*/ double std)
{
	std = 0;
	std::vector<double> distances;
	double avg = 0;
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(planePoints.x->position - 
		planePoints.y->position, planePoints.z->position -planePoints.y->position));
	for (size_t i = 0; i < nonProcessedPoints.size(); i++)
	{
		double dist = abs(Vec3<double>::dot_product(normal, nonProcessedPoints[i]->position - planePoints.y->position));
		if (dist <= acceptTreshold) 
		{
			avg += dist;
			distances.push_back(dist);
			chosenPoints.push_back(nonProcessedPoints[i]);
		}
	}
	avg /= chosenPoints.size();
	for (size_t i = 0; i < chosenPoints.size(); i++)
	{
		std += pow(distances[i] - avg, 2);
	}
	std /= chosenPoints.size();
}

#include <random>

std::mt19937 gen(100);

Vec3<Point*> pick3Points(const std::vector<Point*>& nonProcessedPoints)
{
	std::uniform_int_distribution<int> distr(0, nonProcessedPoints.size() - 1);
	size_t index1 = distr(gen);
	size_t index2;
	size_t index3;
	do {
		index2 = distr(gen);
	} while (index1 == index2);

	do {
		index3 = distr(gen);
	} while (index1 == index3 || index2 == index3);

	Vec3<double> v1(1, 2, 3);
	Vec3<double> v2(3, 1, 1);
	Vec3<double> v3(1, 5, 2);
	auto normal = Vec3<double>::normalize(Vec3<double>::crossProduct(v1 -
		v2, v3 - v2));
	double dist = abs(Vec3<double>::dot_product(v3 - v2, normal));
	return Vec3<Point*>(nonProcessedPoints[index1], nonProcessedPoints[index2], nonProcessedPoints[index3]);
}

bool checkIfBridge(Point* p)
{
	if (!p->isMarked) return false;
	size_t x = p->horizontalIndex;
	size_t y = p->verticalIndex;
	int neighbourCount = 0;
	//fel-le
	bool isUpNotNeightbour = y == 0 || !points[getOffset(x, y - 1)] || !points[getOffset(x, y - 1)]->isMarked;
	bool isDownNotNeightbour = y == verticalCount - 1 || !points[getOffset(x, y + 1)] || !points[getOffset(x, y + 1)]->isMarked;
	if (isUpNotNeightbour && isDownNotNeightbour) return true;
	if (!isUpNotNeightbour)
		neighbourCount++;
	if (!isDownNotNeightbour)
		neighbourCount++;
	//jobbra-balra
	bool isLeftNotNeightbour = !points[getOffset(x - 1, y)] || !points[getOffset(x - 1, y)]->isMarked;
	bool isRighttNotNeightbour = !points[getOffset(x + 1, y)] || !points[getOffset(x + 1, y)]->isMarked;
	if (isLeftNotNeightbour && isRighttNotNeightbour) return true;
	if (!isLeftNotNeightbour)
		neighbourCount++;
	if (!isRighttNotNeightbour)
		neighbourCount++;
	//átlósan
	if ((((y > 0 && (!points[getOffset(x - 1, y - 1)] || !points[getOffset(x - 1, y - 1)]->isMarked)) &&
		(y < verticalCount - 1 && (!points[getOffset(x + 1, y + 1)] || !points[getOffset(x + 1, y + 1)]->isMarked))) ||
		((y > 0 && (!points[getOffset(x + 1, y - 1)] || !points[getOffset(x + 1, y - 1)]->isMarked)) &&
			(y < verticalCount - 1 && (!points[getOffset(x - 1, y + 1)] || !points[getOffset(x - 1, y + 1)]->isMarked))))
		&& neighbourCount > 2)
		return true;
	return false;
}

bool isThereBridge(const std::vector<Point*>& checkPoints)
{
	bool theresBridge = false;
	for (auto p : checkPoints) 
		if (checkIfBridge(p)) {
			theresBridge = true;
			p->isMarked = false;
		}
	return theresBridge;
}

void checkForGaps(/*out*/ std::vector<Point*>& chosenPoints)
{  
	for (auto p : chosenPoints) p->isMarked = true;
	while (isThereBridge(chosenPoints)) {}
	std::vector<Point*> nonProcessedPoints(chosenPoints);
	std::vector<Point*> nextStepPoints;
	std::vector<Point*> bestPartitionPoints;
	while (bestPartitionPoints.size() < nonProcessedPoints.size()) {
		std::vector<Point*> currentPartitionPoints;
		currentPartitionPoints.push_back(nonProcessedPoints[0]);
		nextStepPoints.push_back(nonProcessedPoints[0]);
		nonProcessedPoints[0]->isMarked = false;
		while (nextStepPoints.size() > 0) {
			std::vector<Point*> tempNextStepPoints;
			for (size_t i = 0; i < nextStepPoints.size(); i++) {
				size_t x = nextStepPoints[i]->horizontalIndex;
				size_t y = nextStepPoints[i]->verticalIndex;
				Point* neighbourPoint = points[getOffset(x, y - 1)];
				if (y > 0 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x, y + 1)];
				if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x - 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
				neighbourPoint = points[getOffset(x + 1, y)];
				if (neighbourPoint && neighbourPoint->isMarked) {
					neighbourPoint->isMarked = false;
					currentPartitionPoints.push_back(neighbourPoint);
					tempNextStepPoints.push_back(neighbourPoint);
				}
			}
			nextStepPoints = tempNextStepPoints;
		}
		if (currentPartitionPoints.size() > bestPartitionPoints.size()) bestPartitionPoints = currentPartitionPoints;
		std::vector<Point*> tempNonProcessedPoints;
		for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
			if (nonProcessedPoints[i]->isMarked) tempNonProcessedPoints.push_back(nonProcessedPoints[i]);
		}
		nonProcessedPoints = tempNonProcessedPoints;
	}
	for (size_t i = 0; i < chosenPoints.size(); i++) {
		chosenPoints[i]->isMarked = false;
	}
	chosenPoints = bestPartitionPoints;
}

void RANSAC(std::vector<Point*>& nonProcessedPoints)
{
	/*size_t maxPointInPLane = nonProcessedPoints.size()/10;
	double findChance = 0.9;
	double eps = 1 - (double)maxPointInPLane / nonProcessedPoints.size();
	size_t N = log(1 - findChance) / log(1 - pow(1 - eps, 3));*/
	size_t N = 100;
	double bestStd = 100000;
	double acceptTreshold = 0.1;
	std::vector<Point*> bestPoints;
	for (size_t i = 0; i < N; i++) 
	{
		std::vector<Point*> chosenPoints;
		std::vector<Point*> chosenPoints2;
		double std = 0;
		auto pickedPoints = pick3Points(nonProcessedPoints);
		choosePoints(pickedPoints, nonProcessedPoints, acceptTreshold, chosenPoints, std);
		bool temptt = currentId == 13 && chosenPoints.size() == 1748;
		checkForGaps(chosenPoints); //TODO std
		if (chosenPoints.size() >= 3) {
			choosePoints(pick3Points(chosenPoints), nonProcessedPoints, acceptTreshold, chosenPoints2, std);
			checkForGaps(chosenPoints2); //TODO std
			chosenPoints = chosenPoints2;
		}		
		if (chosenPoints.size() > bestPoints.size() || (chosenPoints.size() == bestPoints.size() && std < bestStd))
		{
			bestPoints = chosenPoints;
			bestStd = std;
		}
	}
	for (size_t i = 0; i < bestPoints.size(); i++) {
		bestPoints[i]->isMarked = true;
	}
}

void ObjectSegmentation()
{
	double treshold = 1;
	for (size_t i = 0; i < points.size(); i++) if(points[i]) points[i]->isMarked = true;
	std::vector<Point*>nextStepPoints;
	for (size_t j = 0; j < points.size(); j++) 
	{
		if (points[j] && points[j]->isMarked) {
			std::vector<Point*> objectPoints;
			objectPoints.push_back(points[j]);
			nextStepPoints.push_back(points[j]);
			points[j]->isMarked = false;
			while (nextStepPoints.size() > 0) {
				std::vector<Point*> tempNextStepPoints;
				for (size_t i = 0; i < nextStepPoints.size(); i++) {
					size_t x = nextStepPoints[i]->horizontalIndex;
					size_t y = nextStepPoints[i]->verticalIndex;
					Point* neighbourPoint = points[getOffset(x, y - 1)];
					if (y > 0 && neighbourPoint && neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position, neighbourPoint->position) < treshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
					}
					neighbourPoint = points[getOffset(x, y + 1)];
					if (y < verticalCount - 1 && neighbourPoint && neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position, neighbourPoint->position) < treshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
					}
					neighbourPoint = points[getOffset(x - 1, y)];
					if (neighbourPoint && neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position, neighbourPoint->position) < treshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
					}
					neighbourPoint = points[getOffset(x + 1, y)];
					if (neighbourPoint && neighbourPoint->isMarked && Vec3<double>::distance(nextStepPoints[i]->position, neighbourPoint->position) < treshold) {
						neighbourPoint->isMarked = false;
						tempNextStepPoints.push_back(neighbourPoint);
						objectPoints.push_back(neighbourPoint);
					}
				}
				nextStepPoints = tempNextStepPoints;
			}
			separatedPoints.push_back(objectPoints);
		}
	}
	for (size_t i = 0; i < points.size(); i++) if (points[i]) points[i]->isMarked = false;
}

#include <chrono>

void findPlanes(std::vector<std::vector<Point*>>& planes)
{
	auto start = std::chrono::steady_clock::now();
	std::vector<Point*> nonProcessedPoints;
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && !points[i]->deleted)
			nonProcessedPoints.push_back(points[i]);
	}

	size_t planeSearchIter = 100;
	for (size_t j = 0; j < separatedPoints.size(); j++) {
		nonProcessedPoints = separatedPoints[j];
	
		while (true) {
			std::vector<Point*> plane;
			if (nonProcessedPoints.size() >= 50)
				RANSAC(nonProcessedPoints);
			else break;
			std::vector<Point*> tempPoints;
			for (size_t i = 0; i < nonProcessedPoints.size(); i++) {
				if (nonProcessedPoints[i]->isMarked) {
					nonProcessedPoints[i]->isMarked = false;
					plane.push_back(nonProcessedPoints[i]);
				}
				else {
					tempPoints.push_back(nonProcessedPoints[i]);
				}
			}
			if (plane.size() < 50) break;
			planes.push_back(plane);
			for (size_t i = 0; i < plane.size(); i++) {
				plane[i]->id = currentId;
			}
			currentId++;
			nonProcessedPoints = tempPoints;
			std::cout << nonProcessedPoints.size() << std::endl;
		}
	}
	auto end = std::chrono::steady_clock::now();
	std::cout << "Elapsed time in seconds: "
		<< std::chrono::duration_cast<std::chrono::seconds>(end - start).count()
		<< " sec";
}

void findNextEdgePoint(Point* startPoint, Point* currentPoint, std::vector<Point*>& edgePoints, size_t direction)
{
	edgePoints.push_back(currentPoint);
	size_t x = currentPoint->horizontalIndex;
	size_t y = currentPoint->verticalIndex;
	Point* neighbourPoint = nullptr;
	for (size_t i = 0; i < 3; i++) {
		switch (direction) {
		case 0: //to right
		neighbourPoint = points[getOffset(x + 1, y)];
		break;
		case 1: //to down
		neighbourPoint = y == verticalCount - 1 ? nullptr : points[getOffset(x, y + 1)];
		break;
		case 2: //to left
		neighbourPoint = points[getOffset(x - 1, y)];
		break;
		case 3: //to up
		neighbourPoint = y == 0 ? nullptr : points[getOffset(x, y - 1)];
		break;
		default:
		break;
		}
		if (neighbourPoint && neighbourPoint->isMarked) {
			neighbourPoint->isMarked = false;
			if (neighbourPoint == startPoint) {
			}
			else 
			{
				findNextEdgePoint(startPoint, neighbourPoint, edgePoints, (direction + 3) % 4);
			}
			break;
		}
		direction += direction == 3 ? -3 : 1;
	}
	if (neighbourPoint == nullptr) {
		std::cout << "PLANE WITHOUT RELEVANT OUTLINE!" << std::endl;
	}
}

bool isEdgePoint(Point* point)
{
	if (!point->isMarked) return false;
	size_t x = point->horizontalIndex;
	size_t y = point->verticalIndex;
	Point* neighbourPoint = points[getOffset(x, y - 1)];
	if (y == 0 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x, y + 1)];
	if (y == verticalCount - 1 || !neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x - 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	neighbourPoint = points[getOffset(x + 1, y)];
	if (!neighbourPoint || !neighbourPoint->isMarked) {
		return true;
	}
	return false;
}

void findEdgePoints(std::vector<std::vector<Point*>>& planes, /*out*/ std::vector<std::vector<std::vector<Point*>>>& edges)
{
	for (size_t i = 0; i < planes.size(); i++) 
	{
		std::vector<std::vector<Point*>> edgesInPlane;
		std::vector<Point*> edgePointsInPlane;
		for (size_t j = 0; j < planes[i].size(); j++) planes[i][j]->isMarked = true;
		for (size_t j = 0; j < planes[i].size(); j++) if(isEdgePoint(planes[i][j])) edgePointsInPlane.push_back(planes[i][j]);
		bool isFirstEdge = true;
		for (size_t j = 0; j < edgePointsInPlane.size(); j++) {
			if (isEdgePoint(edgePointsInPlane[j])) {
				std::vector<Point*> edgePoints;
				findNextEdgePoint(edgePointsInPlane[j], edgePointsInPlane[j], edgePoints, isFirstEdge ? 0 : 1);
				if (!isFirstEdge) {
					edgePoints.insert(edgePoints.begin(), edgePoints[edgePoints.size() - 1]);
					edgePoints.pop_back();
				}
				edgesInPlane.push_back(edgePoints);
				for (auto p : edgePoints) {
					//if (p) p->id = currentId;
				}
				//currentId++;
				isFirstEdge = false;
				break; //TODO remove
			}
		}
		for (size_t j = 0; j < planes[i].size(); j++) planes[i][j]->isMarked = false;
		edges.push_back(edgesInPlane);
	}
}

double distancePointFromLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line)
{
	Vec3<double> directionVector = line.first - line.second;
	return Vec3<double>::length(Vec3<double>::crossProduct(p - line.first, directionVector)) / Vec3<double>::length(directionVector);
}

bool isOnLine(Vec3<double> p, std::pair<Vec3<double>, Vec3<double>> line, double treshold)
{
	return distancePointFromLine(p, line) < treshold;
}

void createEdgeSegments(const std::vector<std::vector<std::vector<Point*>>>& edges, /*out*/ std::vector < std::vector<Point*>>& corners)
{
	for (auto plane : edges) {
		for (auto edge : plane) {
			std::pair<Vec3<double>, Vec3<double>> linePoints;
			std::vector<Point*> currentCorners;
			bool newSegment = true;
			for (size_t i = 0; i < edge.size(); i++) {
				if (i == 0 || !isOnLine(edge[i]->position, linePoints, 0.1)) newSegment = true;
				if (newSegment) {
					newSegment = false;
					currentCorners.push_back(edge[i == 0 ? i : (i - 1)]);
					edge[i == 0 ? i : (i - 1)]->id = currentId;
					if (i == 0) {
						linePoints = std::make_pair(edge[i]->position, edge[i + 1]->position);
						i++;
					}
					else {
						linePoints = std::make_pair(edge[i-1]->position, edge[i]->position);
					}
				}
			}
			currentId++;
			corners.push_back(currentCorners);
		}
	}
}

void egoCarSegmentation()
{
	for (size_t i = 0; i < points.size(); i++) {
		if (points[i] && points[i]->position.x <= 1 && points[i]->position.x >= -1 &&
			points[i]->position.y <= 1 && points[i]->position.y >= -1 &&
			points[i]->position.z <= 2.5 && points[i]->position.z >= -2.5)
			points[i]->deleted = true;
	}
}

void filterCornerPoints(const std::vector<std::vector<Point*>>& corners, std::vector<std::vector<Point*>>& filteredCorners)
{
	for (auto currentEdgeCorners : corners) {
		std::vector<Point*> currentCorners;
		currentEdgeCorners.push_back(currentEdgeCorners[0]);
		currentCorners.push_back(currentEdgeCorners[0]);
		size_t startCornerIndex = 0;
		for (size_t i = startCornerIndex + 2; i < currentEdgeCorners.size(); i++) {
			std::pair<Vec3<double>, Vec3<double>> direction = std::make_pair(currentEdgeCorners[startCornerIndex]->position, currentEdgeCorners[i]->position);
			for (size_t j = startCornerIndex + 1; j < i; j++) {
				if (!isOnLine(currentEdgeCorners[j]->position, direction, 0.0)) {
					currentCorners.push_back(currentEdgeCorners[i - 1]);
					startCornerIndex = i - 1;
					i = startCornerIndex + 1;
					break;
				}
			}
		}
		filteredCorners.push_back(currentCorners);
	}
}

void exportObjects(std::vector<std::vector<Point*>> filteredCorners)
{
	for (size_t i = 0; i < filteredCorners.size(); i++) {
		std::ofstream MyFile("C:/Users/ungbo/Desktop/BME/_Diplomaterv/Diplomamunka/Diplomamunka/Assets/Resources/processed_obj_" + std::to_string(i) + ".obj");
		MyFile << "o Mesh" << std::endl;
		for (size_t j = 0; j < filteredCorners[i].size(); j++) {
			MyFile << "v " << -filteredCorners[i][j]->position.x << " " << filteredCorners[i][j]->position.y << " " <<
				filteredCorners[i][j]->position.z << std::endl;
		}
		MyFile << "f ";
		for (size_t j = 1; j < filteredCorners[i].size() + 1; j++) {
			MyFile << j << " ";
		}
		MyFile << std::endl;
		MyFile << "f ";
		for (size_t j = filteredCorners[i].size(); j > 0; j--) {
			MyFile << j << " ";
		}
		MyFile << std::endl;
		MyFile.close();
	}
}

void ProcessData() {
	egoCarSegmentation();
	//GroundSegmentation();
	ObjectSegmentation();
	//WriteData(points);

	std::vector<std::vector<Point*>> planes;
	findPlanes(planes);

	WriteData(points);

	/*std::vector<std::vector<std::vector<Point*>>> edges;
	findEdgePoints(planes, edges);
	std::vector<std::vector<Point*>> corners;
	createEdgeSegments(edges, corners);
	std::vector<std::vector<Point*>> filteredCorners;
	filterCornerPoints(corners, filteredCorners);
	exportObjects(filteredCorners);
	std::vector<Point*> writePoints;
	for (size_t i = 0; i < filteredCorners.size(); i++) {
		for (size_t j = 0; j < filteredCorners[i].size(); j++) {
			writePoints.push_back(filteredCorners[i][j]);
		}
	}
	WriteData(writePoints);*/
}



int main()
{
    ReadData();
	ProcessData();
	//WriteData();
    return 0;
}